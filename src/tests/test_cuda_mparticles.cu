#include "hip/hip_runtime.h"

#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define cudaCheck(ierr) do {						\
    if (ierr != hipSuccess)						\
      fprintf(stderr, "IERR = %d (%s)\n", ierr, hipGetErrorName(ierr)); \
    assert(ierr == hipSuccess);					\
  } while(0)

struct cuda_domain_info {
  int nr_patches;
  int mx[3]; // number of cells per patch
  int bs[3]; // size of each block / super-cell
  double dx[3]; // size of a single cell
};

void
cuda_domain_info_set_test_1(struct cuda_domain_info *info)
{
  info->nr_patches = 1;
  info->mx[0] = 1; info->mx[1] = 4; info->mx[2] = 2;
  info->bs[0] = 1; info->bs[1] = 1; info->bs[2] = 1;
  info->dx[0] = 1.; info->dx[1] = 10.; info->dx[2] = 10.;
  for (int d = 0; d < 3; d++) {
    assert(info->mx[d] % info->bs[d] == 0);
  }
};

struct cuda_mparticles {
  unsigned int nr_prts;     // total # of particles in all patches
  unsigned int nr_alloced;  // arrays are alloced for this # of particles
  unsigned int nr_patches;
  unsigned int nr_blocks;
  unsigned int *nr_prts_by_patch;
  int mx[3];      // number of cells per direction in each patch
  int b_mx[3];    // number of blocks per direction in each patch
  float dx[3];    // cell size (in actual length units)
  float b_dxi[3]; // inverse of block size (in actual length units)

  // per particle
  float4 *d_xi4, *d_pxi4;
  float4 *d_alt_xi4, *d_alt_pxi4;
  unsigned int *d_bidx;
  unsigned int *d_id;

  // per patch
  unsigned int *d_nr_prts_by_patch;

  // per block
  unsigned int *d_off;
};

void
cuda_mparticles_set_domain_info(struct cuda_mparticles *cuda_mprts,
				const struct cuda_domain_info *info)
{
  cuda_mprts->nr_patches = info->nr_patches;
  for (int d = 0; d < 3; d++) {
    cuda_mprts->mx[d] = info->mx[d];
    cuda_mprts->b_mx[d] = info->mx[d] / info->bs[d];
    cuda_mprts->dx[d] = info->dx[d];
    cuda_mprts->b_dxi[d] = 1.f / (info->bs[d] * info->dx[d]);
  }
  cuda_mprts->nr_blocks = info->nr_patches *
    cuda_mprts->b_mx[0] * cuda_mprts->b_mx[1] * cuda_mprts->b_mx[2];
}

void
cuda_mparticles_alloc(struct cuda_mparticles *cuda_mprts, int nr_prts)
{
  hipError_t ierr;
  unsigned int nr_alloced = nr_prts * 1.4;
  cuda_mprts->nr_prts = nr_prts;
  cuda_mprts->nr_alloced = nr_alloced;
  cuda_mprts->nr_prts_by_patch = new unsigned int[cuda_mprts->nr_patches];

  assert(cuda_mprts->nr_patches == 1);
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    cuda_mprts->nr_prts_by_patch[p] = nr_prts;
  }

  ierr = hipMalloc((void **) &cuda_mprts->d_xi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cuda_mprts->d_pxi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cuda_mprts->d_alt_xi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cuda_mprts->d_alt_pxi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cuda_mprts->d_bidx, nr_alloced * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cuda_mprts->d_id, nr_alloced * sizeof(unsigned int)); cudaCheck(ierr);

  ierr = hipMalloc((void **) &cuda_mprts->d_nr_prts_by_patch, cuda_mprts->nr_patches * sizeof(unsigned int)); cudaCheck(ierr);

  ierr = hipMalloc((void **) &cuda_mprts->d_off, (cuda_mprts->nr_blocks + 1) * sizeof(unsigned int)); cudaCheck(ierr);
}

void
cuda_mparticles_free(struct cuda_mparticles *cuda_mprts)
{
  hipError_t ierr;

  delete[] cuda_mprts->nr_prts_by_patch;
  
  ierr = hipFree(cuda_mprts->d_xi4); cudaCheck(ierr);
  ierr = hipFree(cuda_mprts->d_pxi4); cudaCheck(ierr);
  ierr = hipFree(cuda_mprts->d_alt_xi4); cudaCheck(ierr);
  ierr = hipFree(cuda_mprts->d_alt_pxi4); cudaCheck(ierr);
  ierr = hipFree(cuda_mprts->d_bidx); cudaCheck(ierr);
  ierr = hipFree(cuda_mprts->d_id); cudaCheck(ierr);

  ierr = hipFree(cuda_mprts->d_nr_prts_by_patch); cudaCheck(ierr);

  ierr = hipFree(cuda_mprts->d_off); cudaCheck(ierr);
}

void
cuda_mparticles_copy_to_dev(struct cuda_mparticles *cuda_mprts)
{
  hipError_t ierr;
  
  ierr = hipMemcpy(cuda_mprts->d_nr_prts_by_patch, cuda_mprts->nr_prts_by_patch,
		    cuda_mprts->nr_patches * sizeof(unsigned int),
		    hipMemcpyHostToDevice); cudaCheck(ierr);
}

void
cuda_mparticles_set_test_1(struct cuda_mparticles *cuda_mprts)
{
  int nr_prts = cuda_mprts->mx[0] * cuda_mprts->mx[1] * cuda_mprts->mx[2]
    * cuda_mprts->nr_patches;
  cuda_mparticles_alloc(cuda_mprts, nr_prts);
  
  thrust::device_ptr<float4> d_xi4(cuda_mprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cuda_mprts->d_pxi4);

  int *mx = cuda_mprts->mx;
  float *dx = cuda_mprts->dx;
  
  int n = 0;
  for (int i = 0; i < mx[0]; i++) {
    for (int j = 0; j < mx[1]; j++) {
      for (int k = 0; k < mx[2]; k++) {
	d_xi4[n] = (float4) { dx[0] * (i + .5f),
			      dx[1] * (j + .5f),
			      dx[1] * (k + .5f), 1. };
	d_pxi4[n] = (float4) { i, j, k, 2. };
	n++;
      }
    }
  }
}

void
cuda_mparticles_dump(struct cuda_mparticles *cuda_mprts)
{
  int nr_prts = cuda_mprts->nr_prts;
  
  thrust::device_ptr<float4> d_xi4(cuda_mprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cuda_mprts->d_pxi4);
  thrust::device_ptr<unsigned int> d_bidx(cuda_mprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cuda_mprts->d_id);
  thrust::device_ptr<unsigned int> d_off(cuda_mprts->d_off);

  printf("cuda_mparticles_dump: nr_prts = %d\n", nr_prts); 
  for (int n = 0; n < nr_prts; n++) {
    float4 xi4 = d_xi4[n], pxi4 = d_pxi4[n];
    unsigned int bidx = d_bidx[n], id = d_id[n];
    printf("cuda_mparticles_dump: [%d] %g %g %g // %g // %g %g %g // %g || bidx %d id %d\n",
	   n, xi4.x, xi4.y, xi4.z, xi4.w, pxi4.x, pxi4.y, pxi4.z, pxi4.w,
	   bidx, id);
  }

  for (int b = 0; b <= cuda_mprts->nr_blocks; b++) {
    unsigned int off = d_off[b];
    printf("cuda_mparticles_dump: off[%d] = %d\n", b, off);
  }
}

// ----------------------------------------------------------------------
// cuda_params

struct cuda_params {
  unsigned int b_mx[3];
  float b_dxi[3];
};

void
cuda_params_set(struct cuda_params *prm, const struct cuda_mparticles *cuda_mprts)
{
  for (int d = 0; d < 3; d++) {
    prm->b_mx[d]  = cuda_mprts->b_mx[d];
    prm->b_dxi[d] = cuda_mprts->b_dxi[d];
  }
}

void
cuda_params_free(struct cuda_params *prm)
{
}

// ----------------------------------------------------------------------
// cuda_mparticles_find_block_indices_ids_total

#define THREADS_PER_BLOCK 512

__global__ static void
mprts_find_block_indices_ids_total(struct cuda_params prm, float4 *d_xi4, unsigned int *d_nr_prts,
				   unsigned int *d_bidx, unsigned int *d_id, int nr_patches)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  unsigned int off = 0;
  for (int p = 0; p < nr_patches; p++) {
    if (n < d_nr_prts[p]) {
      float4 xi4 = d_xi4[n + off];
      unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
      unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);
      
      int block_idx;
      if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
	block_idx = -1; // not supposed to happen here!
      } else {
	block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
      }
      d_bidx[n + off] = block_idx;
      d_id[n + off] = n + off;
    }
    off += d_nr_prts[p];
  }
}

void
cuda_mparticles_find_block_indices_ids_total(struct cuda_mparticles *cuda_mprts)
{
  if (cuda_mprts->nr_patches == 0) {
    return;
  }

  int max_nr_prts = 0;
  int nr_prts = 0;
  //mprts_cuda->nr_prts_send = 0;
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    if (cuda_mprts->nr_prts_by_patch[p] > max_nr_prts) {
      max_nr_prts = cuda_mprts->nr_prts_by_patch[p];
    }
    nr_prts += cuda_mprts->nr_prts_by_patch[p];
    // struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    // struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    // mprts_cuda->nr_prts_send += cuda->bnd_n_send;
    // if (prts->n_part > max_n_part) {
    //   max_n_part = prts->n_part;
    // }
    // cuda->h_dev->n_part = prts->n_part;
    // nr_prts += prts->n_part;
  }
  //mprts_cuda->nr_prts = nr_prts;
  assert(cuda_mprts->nr_prts == nr_prts);
  cuda_mparticles_copy_to_dev(cuda_mprts);

  struct cuda_params prm;
  cuda_params_set(&prm, cuda_mprts);
    
  dim3 dimGrid((max_nr_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  dim3 dimBlock(THREADS_PER_BLOCK);

  mprts_find_block_indices_ids_total<<<dimGrid, dimBlock>>>(prm, cuda_mprts->d_xi4, 
							    cuda_mprts->d_nr_prts_by_patch,
							    cuda_mprts->d_bidx,
							    cuda_mprts->d_id,
							    cuda_mprts->nr_patches);
  cuda_params_free(&prm);
}

// ======================================================================
// cuda_mparticles_swap_alt

void
cuda_mparticles_swap_alt(struct cuda_mparticles *cuda_mprts)
{
  float4 *tmp_xi4 = cuda_mprts->d_alt_xi4;
  float4 *tmp_pxi4 = cuda_mprts->d_alt_pxi4;
  cuda_mprts->d_alt_xi4 = cuda_mprts->d_xi4;
  cuda_mprts->d_alt_pxi4 = cuda_mprts->d_pxi4;
  cuda_mprts->d_xi4 = tmp_xi4;
  cuda_mprts->d_pxi4 = tmp_pxi4;
}

// ======================================================================
// reorder_and_offsets

__global__ static void
mprts_reorder_and_offsets(int nr_prts, float4 *xi4, float4 *pxi4, float4 *alt_xi4, float4 *alt_pxi4,
			  unsigned int *d_bidx, unsigned int *d_ids, unsigned int *d_off, int last_block)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > nr_prts)
    return;

  int block, prev_block;
  if (i < nr_prts) {
    alt_xi4[i] = xi4[d_ids[i]];
    alt_pxi4[i] = pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else { // needed if there is no particle in the last block
    block = last_block;
  }

  // OPT: d_bidx[i-1] could use shmem
  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_off[b] = i;
  }
}

void
cuda_mparticles_reorder_and_offsets(struct cuda_mparticles *cuda_mprts)
{
  if (cuda_mprts->nr_patches == 0) {
    return;
  }

  dim3 dimGrid((cuda_mprts->nr_prts + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  dim3 dimBlock(THREADS_PER_BLOCK);

  mprts_reorder_and_offsets<<<dimGrid, dimBlock>>>(cuda_mprts->nr_prts, cuda_mprts->d_xi4, cuda_mprts->d_pxi4,
						   cuda_mprts->d_alt_xi4, cuda_mprts->d_alt_pxi4,
						   cuda_mprts->d_bidx, cuda_mprts->d_id,
						   cuda_mprts->d_off, cuda_mprts->nr_blocks);

  cuda_mparticles_swap_alt(cuda_mprts);
  //  psc_mparticles_cuda_copy_to_dev(mprts);
}



int
main(void)
{
  struct cuda_mparticles _cuda_mprts, *cuda_mprts = &_cuda_mprts;

  struct cuda_domain_info info;
  cuda_domain_info_set_test_1(&info);

  cuda_mparticles_set_domain_info(cuda_mprts, &info);
  cuda_mparticles_set_test_1(cuda_mprts);
  cuda_mparticles_dump(cuda_mprts);

  cuda_mparticles_find_block_indices_ids_total(cuda_mprts);
  cuda_mparticles_dump(cuda_mprts);

  thrust::device_ptr<unsigned int> d_bidx(cuda_mprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cuda_mprts->d_id);
  thrust::stable_sort_by_key(d_bidx, d_bidx + cuda_mprts->nr_prts, d_id);
  cuda_mparticles_dump(cuda_mprts);

  cuda_mparticles_reorder_and_offsets(cuda_mprts);
  cuda_mparticles_dump(cuda_mprts);
  
  cuda_mparticles_free(cuda_mprts);
}
