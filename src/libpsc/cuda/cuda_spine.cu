#include "hip/hip_runtime.h"

#undef _GLIBCXX_USE_INT128

#include "cuda_mparticles.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include "psc_cuda.h"
#include "particles_cuda.h"
#include <mrc_profile.h>

#include <b40c/radixsort_reduction_kernel.h>
#include <b40c/radixsort_scanscatter_kernel3.h>

using namespace b40c_thrust;

typedef unsigned int K;
typedef unsigned int V;

static const int RADIX_BITS = 4;

// layout of the spine
//     lt             self             rb        # from left-top .. self .. right-bottom 
//     0   1   2   3   4   5   6   7   8   NEW
// b0 |   |   |   |   |   |   |   |   |   |   |
// b1 |   |   |   |   |   |   |   |   |   |   |
// b2 |   |   |   |   |   |   |   |   |   |   |
// ...
// bn |   |   |   |   |   |   |   |   |   |   |

//    |   |   |   |   |   |   |   |   |   |   |   |   | ... |   | # oob
//     b0  b1  b2  b3                                        bn

// ======================================================================
// cuda_mprts_bidx_to_key

template<int NBLOCKS_X, int NBLOCKS_Y, int NBLOCKS_Z>
__global__ static void
mprts_bidx_to_key(int nr_total_blocks, unsigned int *d_off, unsigned int *d_bidx)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    unsigned int bidx = d_bidx[n];
    if (bidx == nr_total_blocks) {
      bidx = CUDA_BND_S_OOB;
    } else {
      int b_diff = bid - bidx + NBLOCKS_Y + 1;
      int d1 = b_diff % NBLOCKS_Y;
      int d2 = b_diff / NBLOCKS_Y;
      bidx = d2 * 3 + d1;
    }
    d_bidx[n] = bidx;
  }
}

void
cuda_mprts_bidx_to_key(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  unsigned int nr_total_blocks = mprts_cuda->nr_total_blocks;
  
  int *b_mx = mprts_cuda->b_mx;
  if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    mprts_bidx_to_key<1, 8, 8><<<nr_total_blocks, THREADS_PER_BLOCK>>>
      (mprts_cuda->nr_total_blocks, mprts_cuda->d_off, cmprts->d_bidx);
  } else {
    mprintf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
}

void
cuda_mprts_bidx_to_key_gold(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  unsigned int nr_total_blocks = mprts_cuda->nr_total_blocks;
  unsigned int n_blocks_per_patch = cmprts->n_blocks_per_patch;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_off(mprts_cuda->d_off);

  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + nr_total_blocks + 1);

  int *b_mx = mprts_cuda->b_mx;

  for (int bid = 0; bid < nr_total_blocks; bid++) {
    int p = bid / n_blocks_per_patch;
    for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
      assert((h_bidx[n] >= p * n_blocks_per_patch && h_bidx[n] < (p+1) * n_blocks_per_patch) ||
	     (h_bidx[n] == nr_total_blocks));
      int bidx;
      if (h_bidx[n] == mprts_cuda->nr_total_blocks) {
	bidx = CUDA_BND_S_OOB;
      } else {
	int b_diff = bid - h_bidx[n] + b_mx[1] + 1;
	int d1 = b_diff % b_mx[1];
	int d2 = b_diff / b_mx[1];
	bidx = d2 * 3 + d1;
      }
      
      h_bidx[n] = bidx;
    }
  }

  thrust::copy(h_bidx.begin(), h_bidx.end(), d_bidx);
}

// ======================================================================
// cuda_mprts_count_received

__global__ static void
mprts_count_received(int nr_total_blocks, unsigned int *d_alt_bidx, unsigned int *d_spine_cnts)
{
  int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (bid < nr_total_blocks) {
    d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = d_alt_bidx[bid];
  }
}

void
cuda_mprts_count_received(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  unsigned int nr_total_blocks = mprts_cuda->nr_total_blocks;
  
  mprts_count_received<<<nr_total_blocks, THREADS_PER_BLOCK>>>
    (mprts_cuda->nr_total_blocks, mprts_cuda->d_bnd_spine_cnts + 10 * nr_total_blocks,
     mprts_cuda->d_bnd_spine_cnts);
}

void
cuda_mprts_count_received_gold(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  int nr_total_blocks = mprts_cuda->nr_total_blocks;

  thrust::device_ptr<unsigned int> d_spine_cnts(mprts_cuda->d_bnd_spine_cnts);

  thrust::host_vector<unsigned int> h_spine_cnts(1 + nr_total_blocks * (10 + 1));

  thrust::copy(d_spine_cnts, d_spine_cnts + 1 + nr_total_blocks * (10 + 1), h_spine_cnts.begin());

  for (int bid = 0; bid < nr_total_blocks; bid++) {
    h_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = h_spine_cnts[10 * nr_total_blocks + bid];
  }

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts);
}

void
cuda_mprts_count_received_v1(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  int nr_total_blocks = mprts_cuda->nr_total_blocks;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_spine_cnts(mprts_cuda->d_bnd_spine_cnts);

  thrust::host_vector<unsigned int> h_bidx(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_spine_cnts(1 + nr_total_blocks * (10 + 1));

  thrust::copy(d_bidx, d_bidx + cmprts->n_prts, h_bidx.begin());
  thrust::copy(d_spine_cnts, d_spine_cnts + 1 + nr_total_blocks * (10 + 1), h_spine_cnts.begin());
  for (int n = cmprts->n_prts - mprts_cuda->nr_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < mprts_cuda->nr_total_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }
  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts);
}

// ======================================================================
// cuda_mprts_scan_scatter_received

static void __global__
mprts_scan_scatter_received(unsigned int nr_recv, unsigned int nr_prts_prev,
			    unsigned int *d_spine_sums, unsigned int *d_alt_bidx,
			    unsigned int *d_bidx, unsigned int *d_ids)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (n >= nr_recv) {
    return;
  }

  n += nr_prts_prev;

  int nn = d_spine_sums[d_bidx[n] * 10 + CUDA_BND_S_NEW] + d_alt_bidx[n];
  d_ids[nn] = n;
}

void
cuda_mprts_scan_scatter_received(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  int nr_recv = mprts_cuda->nr_prts_recv;
  int nr_prts_prev = cmprts->n_prts - nr_recv;

  int dimGrid = (nr_recv + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  mprts_scan_scatter_received<<<dimGrid, THREADS_PER_BLOCK>>>
    (nr_recv, nr_prts_prev, mprts_cuda->d_bnd_spine_sums, mprts_cuda->d_alt_bidx,
     cmprts->d_bidx, cmprts->d_id);
}

void
cuda_mprts_scan_scatter_received_gold(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  int nr_total_blocks = mprts_cuda->nr_total_blocks;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_alt_bidx(mprts_cuda->d_alt_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_spine_sums(mprts_cuda->d_bnd_spine_sums);

  thrust::host_vector<unsigned int> h_bidx(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_alt_bidx(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_id(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_spine_sums(1 + nr_total_blocks * (10 + 1));

  thrust::copy(d_spine_sums, d_spine_sums + nr_total_blocks * 11, h_spine_sums.begin());
  thrust::copy(d_bidx, d_bidx + cmprts->n_prts, h_bidx.begin());
  thrust::copy(d_alt_bidx, d_alt_bidx + cmprts->n_prts, h_alt_bidx.begin());
  for (int n = cmprts->n_prts - mprts_cuda->nr_prts_recv; n < cmprts->n_prts; n++) {
    int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW] + h_alt_bidx[n];
    h_id[nn] = n;
  }
  thrust::copy(h_id.begin(), h_id.end(), d_id);
}

// ======================================================================
// cuda_mprts_spine_reduce

void
cuda_mprts_spine_reduce(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  unsigned int nr_total_blocks = mprts_cuda->nr_total_blocks;
  int *b_mx = mprts_cuda->b_mx;

  thrust::device_ptr<unsigned int> d_spine_cnts(mprts_cuda->d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(mprts_cuda->d_bnd_spine_sums);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_off(mprts_cuda->d_off);

  // OPT?
  thrust::fill(d_spine_cnts, d_spine_cnts + 1 + nr_total_blocks * (CUDA_BND_STRIDE + 1), 0);

  const int threads = B40C_RADIXSORT_THREADS;
  if (b_mx[0] == 1 && b_mx[1] == 2 && b_mx[2] == 2) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 2, 2> <<<nr_total_blocks, threads>>>
      (mprts_cuda->d_bnd_spine_cnts, cmprts->d_bidx, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 4 && b_mx[2] == 4) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 4, 4> <<<nr_total_blocks, threads>>>
      (mprts_cuda->d_bnd_spine_cnts, cmprts->d_bidx, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 8, 8> <<<nr_total_blocks, threads>>>
      (mprts_cuda->d_bnd_spine_cnts, cmprts->d_bidx, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 16, 16> <<<nr_total_blocks, threads>>>
      (mprts_cuda->d_bnd_spine_cnts, cmprts->d_bidx, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 32, 32> <<<nr_total_blocks, threads>>>
      (mprts_cuda->d_bnd_spine_cnts, cmprts->d_bidx, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 64, 64> <<<nr_total_blocks, threads>>>
      (mprts_cuda->d_bnd_spine_cnts, cmprts->d_bidx, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
                      NopFunctor<K>, 128, 128> <<<nr_total_blocks, threads>>>
      (mprts_cuda->d_bnd_spine_cnts, cmprts->d_bidx, mprts_cuda->d_off, nr_total_blocks);
  } else {
    mprintf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();

  thrust::exclusive_scan(d_spine_cnts + nr_total_blocks * 10,
			 d_spine_cnts + nr_total_blocks * 10 + nr_total_blocks + 1,
			 d_spine_sums + nr_total_blocks * 10);
}

void
cuda_mprts_spine_reduce_gold(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  unsigned int nr_total_blocks = mprts_cuda->nr_total_blocks;
  unsigned int n_blocks_per_patch = cmprts->n_blocks_per_patch;
  int *b_mx = mprts_cuda->b_mx;

  thrust::device_ptr<unsigned int> d_spine_cnts(mprts_cuda->d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(mprts_cuda->d_bnd_spine_sums);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_off(mprts_cuda->d_off);

  thrust::fill(d_spine_cnts, d_spine_cnts + 1 + nr_total_blocks * (CUDA_BND_STRIDE + 1), 0);

  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + nr_total_blocks + 1);
  thrust::host_vector<unsigned int> h_spine_cnts(d_spine_cnts, d_spine_cnts + 1 + nr_total_blocks * (CUDA_BND_STRIDE + 1));

  
  for (int p = 0; p < mprts->nr_patches; p++) {
    for (int b = 0; b < n_blocks_per_patch; b++) {
      unsigned int bid = b + p * n_blocks_per_patch;
      for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
	unsigned int key = h_bidx[n];
	if (key < 9) {
	  int dy = key % 3;
	  int dz = key / 3;
	  int by = b % b_mx[1];
	  int bz = b / b_mx[1];
	  unsigned int bby = by + 1 - dy;
	  unsigned int bbz = bz + 1 - dz;
	  unsigned int bb = bbz * b_mx[1] + bby;
	  if (bby < b_mx[1] && bbz < b_mx[2]) {
	    h_spine_cnts[(bb + p * n_blocks_per_patch) * 10 + key]++;
	  } else {
	    assert(0);
	  }
	} else if (key == CUDA_BND_S_OOB) {
	  h_spine_cnts[b_mx[1]*b_mx[2]*mprts->nr_patches * 10 + bid]++;
	}
      }
    }
  }  

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts);
  thrust::exclusive_scan(d_spine_cnts + nr_total_blocks * 10,
			 d_spine_cnts + nr_total_blocks * 10 + nr_total_blocks + 1,
			 d_spine_sums + nr_total_blocks * 10);
}

// ======================================================================
// cuda_mprts_sort_pairs_device

void
cuda_mprts_sort_pairs_device(struct psc_mparticles *mprts)
{
  static int pr_A, pr_B, pr_C, pr_D;
  if (!pr_B) {
    pr_A = prof_register("xchg_cnt_recvd", 1., 0, 0);
    pr_B = prof_register("xchg_top_scan", 1., 0, 0);
    pr_C = prof_register("xchg_ss_recvd", 1., 0, 0);
    pr_D = prof_register("xchg_bottom_scan", 1., 0, 0);
  }

  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  int nr_total_blocks = mprts_cuda->nr_total_blocks;

  prof_start(pr_A);
  cuda_mprts_count_received(mprts);
  prof_stop(pr_A);

  prof_start(pr_B);
  // FIXME why isn't 10 + 0 enough?
  thrust::device_ptr<unsigned int> d_spine_cnts(mprts_cuda->d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(mprts_cuda->d_bnd_spine_sums);
  thrust::exclusive_scan(d_spine_cnts, d_spine_cnts + 1 + nr_total_blocks * (10 + 1), d_spine_sums);
  prof_stop(pr_B);

  prof_start(pr_C);
  cuda_mprts_scan_scatter_received(mprts);
  prof_stop(pr_C);

  prof_start(pr_D);
  int *b_mx = mprts_cuda->b_mx;
  if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			8, 8> 
      <<<nr_total_blocks, B40C_RADIXSORT_THREADS>>>
      (mprts_cuda->d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			16, 16> 
      <<<nr_total_blocks, B40C_RADIXSORT_THREADS>>>
      (mprts_cuda->d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			32, 32> 
      <<<nr_total_blocks, B40C_RADIXSORT_THREADS>>>
      (mprts_cuda->d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			64, 64> 
      <<<nr_total_blocks, B40C_RADIXSORT_THREADS>>>
      (mprts_cuda->d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, mprts_cuda->d_off, nr_total_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
                        NopFunctor<K>,
                        NopFunctor<K>,
                        128, 128>
      <<<nr_total_blocks, B40C_RADIXSORT_THREADS>>>
      (mprts_cuda->d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, mprts_cuda->d_off, nr_total_blocks);
  } else {
    mprintf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();
  prof_stop(pr_D);

  // d_ids now contains the indices to reorder by
}

void
cuda_mprts_sort_pairs_gold(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  unsigned int n_blocks_per_patch = cmprts->n_blocks_per_patch;
  int nr_total_blocks = mprts_cuda->nr_total_blocks;
  int *b_mx = mprts_cuda->b_mx;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_off(mprts_cuda->d_off);
  thrust::device_ptr<unsigned int> d_spine_cnts(mprts_cuda->d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(mprts_cuda->d_bnd_spine_sums);

  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_id(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + nr_total_blocks + 1);
  thrust::host_vector<unsigned int> h_spine_cnts(d_spine_cnts, d_spine_cnts + 1 + nr_total_blocks * (10 + 1));

  thrust::host_vector<unsigned int> h_spine_sums(1 + nr_total_blocks * (10 + 1));

  for (int n = cmprts->n_prts - mprts_cuda->nr_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < mprts_cuda->nr_total_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }

  thrust::exclusive_scan(h_spine_cnts.begin(), h_spine_cnts.end(), h_spine_sums.begin());
  thrust::copy(h_spine_sums.begin(), h_spine_sums.end(), d_spine_sums);

  for (int bid = 0; bid < nr_total_blocks; bid++) {
    int b = bid % n_blocks_per_patch;
    int p = bid / n_blocks_per_patch;
    for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
      unsigned int key = h_bidx[n];
      if (key < 9) {
	int dy = key % 3;
	int dz = key / 3;
	int by = b % b_mx[1];
	int bz = b / b_mx[1];
	unsigned int bby = by + 1 - dy;
	unsigned int bbz = bz + 1 - dz;
	assert(bby < b_mx[1] && bbz < b_mx[2]);
	unsigned int bb = bbz * b_mx[1] + bby;
	int nn = h_spine_sums[(bb + p * n_blocks_per_patch) * 10 + key]++;
	h_id[nn] = n;
      } else { // OOB
	assert(0);
      }
    }
  }
  for (int n = cmprts->n_prts - mprts_cuda->nr_prts_recv; n < cmprts->n_prts; n++) {
      int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
      h_id[nn] = n;
  }

  thrust::copy(h_id.begin(), h_id.end(), d_id);
  // d_ids now contains the indices to reorder by
}

// ======================================================================
// cuda_mprts_update_offsets

__global__ static void
mprts_update_offsets(int nr_total_blocks, unsigned int *d_off, unsigned int *d_spine_sums)
{
  int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  
  if (bid <= nr_total_blocks) {
    d_off[bid] = d_spine_sums[bid * CUDA_BND_STRIDE + 0];
  }
}

void
cuda_mprts_update_offsets(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  int nr_total_blocks = mprts_cuda->nr_total_blocks;
  int dimGrid = (nr_total_blocks + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  mprts_update_offsets<<<dimGrid, THREADS_PER_BLOCK>>>
    (mprts_cuda->nr_total_blocks, mprts_cuda->d_off, mprts_cuda->d_bnd_spine_sums);
}

void
cuda_mprts_update_offsets_gold(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  int nr_total_blocks = mprts_cuda->nr_total_blocks;

  thrust::device_ptr<unsigned int> d_spine_sums(mprts_cuda->d_bnd_spine_sums);
  thrust::device_ptr<unsigned int> d_off(mprts_cuda->d_off);

  thrust::host_vector<unsigned int> h_spine_sums(d_spine_sums, d_spine_sums + 1 + nr_total_blocks * (10 + 1));
  thrust::host_vector<unsigned int> h_off(nr_total_blocks + 1);

  for (int bid = 0; bid <= nr_total_blocks; bid++) {
    h_off[bid] = h_spine_sums[bid * 10];
  }

  thrust::copy(h_off.begin(), h_off.end(), d_off);
}

