#include "hip/hip_runtime.h"

#include "collision_cuda_impl.hxx"

#include "balance.hxx"
#include "cuda_collision.cuh"
#include "mparticles_cuda.hxx"
#include "../libpsc/psc_sort/psc_sort_impl.hxx"

// ======================================================================
// CollisionCuda

template <typename Mparticles, typename RngState>
CollisionCuda<Mparticles, RngState>::CollisionCuda(const Grid_t& grid,
                                                   int interval, double nu)
  : fwd_{new CudaCollision<typename Mparticles::CudaMparticles, RngState>{
    interval, nu, int(1. / grid.norm.cori + .5),
      grid.dt}}, // FIXME nicell hack
    c_(grid, interval, nu)
{}

template <typename Mparticles, typename RngState>
void CollisionCuda<Mparticles, RngState>::operator()(Mparticles& mprts)
{
#if 1
  fwd_->sort_by_cell(*mprts.cmprts());
  auto& h_mprts = mprts.template get_as<MparticlesSingle>();
  //SortCountsort2<MparticlesSingle> sort;
  //sort(h_mprts);
  c_(h_mprts);
  mprts.put_as(h_mprts);
#else
  (*fwd_)(*mprts.cmprts());
#endif
}

template <typename Mparticles, typename RngState>
int CollisionCuda<Mparticles, RngState>::interval() const
{
  return fwd_->interval();
}

template struct CollisionCuda<MparticlesCuda<BS144>>;
template struct CollisionCuda<MparticlesCuda<BS444>>;

template struct CollisionCuda<MparticlesCuda<BS144>, RngStateFake>;
