#include "hip/hip_runtime.h"

#include <psc_cuda.h>
#include "cuda_sort2.h"

#include <thrust/scan.h>
#include <thrust/device_vector.h>

#define PFX(x) xchg_##x
#include "constants.c"

// FIXME const mem for dims?
// FIXME probably should do our own loop rather than use blockIdx

__global__ static void
exchange_particles(int n_part, particles_cuda_dev_t d_part,
		   int ldimsx, int ldimsy, int ldimsz)
{
  int ldims[3] = { ldimsx, ldimsy, ldimsz };
  int xm[3];

  for (int d = 0; d < 3; d++) {
    xm[d] = ldims[d] / d_consts.dxi[d];
  }

  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      d_part.xi4[i].x * d_consts.dxi[0],
      d_part.xi4[i].y * d_consts.dxi[1],
      d_part.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = cuda_fint(xi[d]);
    }
    if (pos[1] < 0) {
      d_part.xi4[i].y += xm[1];
      if (d_part.xi4[i].y >= xm[1])
	d_part.xi4[i].y = 0.f;
    }
    if (pos[2] < 0) {
      d_part.xi4[i].z += xm[2];
      if (d_part.xi4[i].z >= xm[2])
	d_part.xi4[i].z = 0.f;
    }
    if (pos[1] >= ldims[1]) {
      d_part.xi4[i].y -= xm[1];
    }
    if (pos[2] >= ldims[2]) {
      d_part.xi4[i].z -= xm[2];
    }
  }
}

EXTERN_C void
cuda_exchange_particles(int p, struct psc_particles *prts)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_patch *patch = &ppsc->patch[p];

  xchg_set_constants(prts, NULL);

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     exchange_particles, (prts->n_part, cuda->d_part,
				  patch->ldims[0], patch->ldims[1], patch->ldims[2]));
}

EXTERN_C void
cuda_alloc_block_indices(struct psc_particles *prts, unsigned int **d_bidx)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  check(hipMalloc((void **) d_bidx, cuda->n_alloced * sizeof(**d_bidx)));
}

EXTERN_C void
cuda_free_block_indices(unsigned int *d_bidx)
{
  check(hipFree(d_bidx));
}

EXTERN_C void
cuda_copy_bidx_from_dev(struct psc_particles *prts, unsigned int *h_bidx, unsigned int *d_bidx)
{
  check(hipMemcpy(h_bidx, d_bidx, prts->n_part * sizeof(*h_bidx),
		   hipMemcpyDeviceToHost));
}

EXTERN_C void
cuda_copy_bidx_to_dev(struct psc_particles *prts, unsigned int *d_bidx, unsigned int *h_bidx)
{
  check(hipMemcpy(d_bidx, h_bidx, prts->n_part * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
}

// ======================================================================
// cuda_find_block_indices

__global__ static void
find_block_indices(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		   int dimy, float b_dyi, float b_dzi, int b_my, int b_mz)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    float4 xi4 = d_part.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx = block_pos_z * b_my + block_pos_y;
    d_bidx[i] = block_idx;
  }
}

EXTERN_C void
cuda_find_block_indices(struct psc_particles *prts, unsigned int *d_bidx)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices, (prts->n_part, cuda->d_part, d_bidx,
				  cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				  cuda->b_mx[1], cuda->b_mx[2]));
}

// ======================================================================
// cuda_find_block_indices_ids

__global__ static void
find_block_indices_ids(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		       unsigned int *d_ids, int dimy, float b_dyi, float b_dzi, int b_my, int b_mz)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    float4 xi4 = d_part.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx = block_pos_z * b_my + block_pos_y;
    d_bidx[i] = block_idx;
    d_ids[i] = i;
  }
}

EXTERN_C void
cuda_find_block_indices_ids(struct psc_particles *prts, unsigned int *d_bidx,
			    unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices_ids, (prts->n_part, cuda->d_part, d_bidx, d_ids,
				      cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				      cuda->b_mx[1], cuda->b_mx[2]));
}

// ======================================================================
// cuda_find_block_indices_2
//
// like cuda_find_block_indices, but handles out-of-bound
// particles

__global__ static void
find_block_indices_2(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		     int dimy, float b_dyi, float b_dzi,
		     int b_my, int b_mz, int start)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x + start;
  if (i < n_part) {
    float4 xi4 = d_part.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx;
    if (block_pos_y >= b_my || block_pos_z >= b_mz) {
      block_idx = b_my * b_mz;
    } else {
      block_idx = block_pos_z * b_my + block_pos_y;
    }
    d_bidx[i] = block_idx;
  }
}

EXTERN_C void
cuda_find_block_indices_2(struct psc_particles *prts, unsigned int *d_bidx,
			  int start)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { ((prts->n_part - start) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices_2, (prts->n_part, cuda->d_part, d_bidx,
				    cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				    cuda->b_mx[1], cuda->b_mx[2], start));
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_2

EXTERN_C void
cuda_mprts_find_block_indices_2(struct psc_mparticles *mprts)
{
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    if (psc_particles_ops(prts) != &psc_particles_cuda_ops) {
      continue;
    }
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    cuda->bnd_cnt = (unsigned int *) calloc(cuda->nr_blocks, sizeof(*cuda->bnd_cnt));
    cuda_find_block_indices_2(prts, cuda->d_part.bidx, 0);
  }
}

// ----------------------------------------------------------------------

EXTERN_C void
_cuda_find_block_indices_2(struct psc_particles *prts, unsigned int *d_bidx,
			   int start)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4 = new float4[prts->n_part];
  float4 *pxi4 = new float4[prts->n_part];
  unsigned int *bidx = new unsigned int[prts->n_part];
  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);

  float b_dyi = cuda->b_dxi[1], b_dzi = cuda->b_dxi[2];
  int b_my = cuda->b_mx[1], b_mz = cuda->b_mx[2];
  for (int i = start; i < prts->n_part; i++) {
    unsigned int block_pos_y = cuda_fint(xi4[i].y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4[i].z * b_dzi);

    int block_idx;
    if (block_pos_y >= b_my || block_pos_z >= b_mz) {
      block_idx = cuda->nr_blocks;
    } else {
      block_idx = block_pos_z * b_my + block_pos_y;
    }
    bidx[i] = block_idx;
  }
  
  cuda_copy_bidx_to_dev(prts, d_bidx, bidx);
  delete[] xi4;
  delete[] pxi4;
  delete[] bidx;
}

// ======================================================================
// cuda_find_block_indices_3

EXTERN_C void
cuda_find_block_indices_3(struct psc_particles *prts, unsigned int *d_bidx,
			  unsigned int *d_alt_bidx,
			  int start, unsigned int *bn_idx, unsigned int *bn_off)
{
  // for consistency, use same block indices that we counted earlier
  check(hipMemcpy(d_bidx + start, bn_idx, (prts->n_part - start) * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
  // abuse of alt_bidx!!! FIXME
  check(hipMemcpy(d_alt_bidx + start, bn_off, (prts->n_part - start) * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
}

// ======================================================================
// reorder_send_buf

__global__ static void
reorder_send_buf(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		 unsigned int *d_sums, unsigned int nr_blocks)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    if (d_bidx[i] == nr_blocks) {
      int j = d_sums[i] + n_part;
      d_part.xi4[j] = d_part.xi4[i];
      d_part.pxi4[j] = d_part.pxi4[i];
    }
  }
}

EXTERN_C void
cuda_reorder_send_buf(int p, struct psc_particles *prts, 
		      unsigned int *d_bidx, unsigned int *d_sums, int n_send)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  assert(prts->n_part + n_send <= cuda->n_alloced);

  // OPT: don't pass offset, get it in device code
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder_send_buf, (prts->n_part, cuda->d_part, d_bidx, d_sums, cuda->nr_blocks));
}

EXTERN_C void
_cuda_reorder_send_buf(int p, struct psc_particles *prts, 
		       unsigned int *d_bidx, unsigned int *d_sums, int n_send)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int n_part = prts->n_part;
  int n_total = n_part + n_send;
  assert(n_total <= cuda->n_alloced);
  float4 *xi4 = new float4[n_total];
  float4 *pxi4 = new float4[n_total];
  unsigned int *bidx = new unsigned int[n_total];
  unsigned int *sums = new unsigned int[n_total];
  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);
  cuda_copy_bidx_from_dev(prts, sums, d_sums);

  for (int i = 0; i < prts->n_part; i++) {
    if (bidx[i] == cuda->nr_blocks) {
      int j = sums[i] + prts->n_part;
      xi4[j] = xi4[i];
      pxi4[j] = pxi4[i];
    }
  }

  prts->n_part = n_total;
  __particles_cuda_to_device(prts, xi4, pxi4, NULL, NULL);
  prts->n_part = n_part;
  delete[] xi4;
  delete[] pxi4;
  delete[] bidx;
  delete[] sums;
}

// ======================================================================
// reorder_and_offsets

__global__ static void
reorder_and_offsets(int n_part, particles_cuda_dev_t d_part, float4 *xi4, float4 *pxi4,
		    unsigned int *d_bidx, unsigned int *d_ids, int nr_blocks)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > n_part)
    return;

  int block, prev_block;
  if (i < n_part) {
    xi4[i] = d_part.xi4[d_ids[i]];
    pxi4[i] = d_part.pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else if (i == n_part) { // needed if there is no particle in the last block
    block = nr_blocks;
  }

  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_part.offsets[b] = i;
  }
}

EXTERN_C void
cuda_reorder_and_offsets(struct psc_particles *prts, unsigned int *d_bidx,
			 unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *alt_xi4 = cuda->d_part.alt_xi4;
  float4 *alt_pxi4 = cuda->d_part.alt_pxi4;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder_and_offsets, (prts->n_part, cuda->d_part, alt_xi4, alt_pxi4,
				   d_bidx, d_ids, cuda->nr_blocks));

  cuda->d_part.alt_xi4 = cuda->d_part.xi4;
  cuda->d_part.alt_pxi4 = cuda->d_part.pxi4;
  cuda->d_part.xi4 = alt_xi4;
  cuda->d_part.pxi4 = alt_pxi4;
}

void
_cuda_reorder_and_offsets(struct psc_particles *prts, unsigned int *d_bidx,
			  unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4 = new float4[prts->n_part];
  float4 *pxi4 = new float4[prts->n_part];
  float4 *alt_xi4 = new float4[prts->n_part];
  float4 *alt_pxi4 = new float4[prts->n_part];
  unsigned int *bidx = new unsigned int[prts->n_part];
  unsigned int *ids = new unsigned int[prts->n_part];
  int *offsets = new int[cuda->nr_blocks + 2];

  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);
  cuda_copy_bidx_from_dev(prts, ids, d_ids);

  for (int i = 0; i < prts->n_part; i++) {
    alt_xi4[i] = xi4[ids[i]];
    alt_pxi4[i] = pxi4[ids[i]];

    int block = bidx[i];
    int prev_block = (i > 0) ? (int) bidx[i-1] : -1;
    for (int b = prev_block + 1; b <= block; b++) {
      offsets[b] = i;
    }
  }
  int block = cuda->nr_blocks + 1;
  int prev_block = bidx[prts->n_part - 1];
  for (int b = prev_block + 1; b <= block; b++) {
    offsets[b] = prts->n_part;
  }

  float4 *d_alt_xi4 = cuda->d_part.alt_xi4;
  float4 *d_alt_pxi4 = cuda->d_part.alt_pxi4;
  cuda->d_part.alt_xi4 = cuda->d_part.xi4;
  cuda->d_part.alt_pxi4 = cuda->d_part.pxi4;
  cuda->d_part.xi4 = d_alt_xi4;
  cuda->d_part.pxi4 = d_alt_pxi4;

  __particles_cuda_to_device(prts, alt_xi4, alt_pxi4, offsets, NULL);
  delete[] xi4;
  delete[] pxi4;
  delete[] alt_xi4;
  delete[] alt_pxi4;
  delete[] bidx;
  delete[] ids;
  delete[] offsets;
}

// ======================================================================
// cuda_reorder

__global__ static void
reorder(int n_part, particles_cuda_dev_t d_part, float4 *xi4, float4 *pxi4,
	unsigned int *d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < n_part) {
    xi4[i] = d_part.xi4[d_ids[i]];
    pxi4[i] = d_part.pxi4[d_ids[i]];
  }
}

EXTERN_C void
cuda_reorder(struct psc_particles *prts, unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *alt_xi4 = cuda->d_part.alt_xi4;
  float4 *alt_pxi4 = cuda->d_part.alt_pxi4;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder, (prts->n_part, cuda->d_part, alt_xi4, alt_pxi4, d_ids));

  cuda->d_part.alt_xi4 = cuda->d_part.xi4;
  cuda->d_part.alt_pxi4 = cuda->d_part.pxi4;
  cuda->d_part.xi4 = alt_xi4;
  cuda->d_part.pxi4 = alt_pxi4;
}

// ======================================================================
// cuda_exclusive_scan

EXTERN_C int
_cuda_exclusive_scan(int p, struct psc_particles *prts,
		    unsigned int *d_vals, unsigned int *d_sums)
{
  unsigned int *vals = new unsigned int[prts->n_part];
  unsigned int *sums = new unsigned int[prts->n_part];
  cuda_copy_bidx_from_dev(prts, vals, d_vals);

  unsigned int sum = 0;
  for (int i = 0; i < prts->n_part; i++) {
    sums[i] = sum;
    sum += vals[i];
  }

  cuda_copy_bidx_to_dev(prts, d_sums, sums);
  delete[] sums;
  delete[] vals;
  return sum;
}

EXTERN_C int
cuda_exclusive_scan(int p, struct psc_particles *prts, unsigned int *_d_vals, unsigned int *_d_sums)
{
  thrust::device_ptr<unsigned int> d_vals(_d_vals);
  thrust::device_ptr<unsigned int> d_sums(_d_sums);
  thrust::exclusive_scan(d_vals, d_vals + prts->n_part, d_sums);
  int sum = d_sums[prts->n_part - 1] + d_vals[prts->n_part - 1];
  return sum;
}

