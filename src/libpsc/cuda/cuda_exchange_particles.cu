#include "hip/hip_runtime.h"

#undef _GLIBCXX_USE_INT128

#include "cuda_mparticles.h"

#include "cuda_sort2.h"
#include "particles_cuda.h"
#include "psc_bnd_cuda.h"
#include "psc_particles_as_cuda.h"

#if 0
#define PFX(x) xchg_##x
#include "constants.c"

// FIXME const mem for dims?
// FIXME probably should do our own loop rather than use blockIdx

__global__ static void
exchange_particles(int n_part, particles_cuda_dev_t h_dev,
		   int ldimsx, int ldimsy, int ldimsz)
{
  int ldims[3] = { ldimsx, ldimsy, ldimsz };
  int xm[3];

  for (int d = 0; d < 3; d++) {
    xm[d] = ldims[d] / d_consts.dxi[d];
  }

  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      h_dev.xi4[i].x * d_consts.dxi[0],
      h_dev.xi4[i].y * d_consts.dxi[1],
      h_dev.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = __float2int_rd(xi[d]);
    }
    if (pos[1] < 0) {
      h_dev.xi4[i].y += xm[1];
      if (h_dev.xi4[i].y >= xm[1])
	h_dev.xi4[i].y = 0.f;
    }
    if (pos[2] < 0) {
      h_dev.xi4[i].z += xm[2];
      if (h_dev.xi4[i].z >= xm[2])
	h_dev.xi4[i].z = 0.f;
    }
    if (pos[1] >= ldims[1]) {
      h_dev.xi4[i].y -= xm[1];
    }
    if (pos[2] >= ldims[2]) {
      h_dev.xi4[i].z -= xm[2];
    }
  }
}

EXTERN_C void
cuda_exchange_particles(int p, struct psc_particles *prts)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_patch *patch = &ppsc->patch[p];

  xchg_set_constants(prts, NULL);

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     exchange_particles, (prts->n_part, *cuda->h_dev,
				  patch->ldims[0], patch->ldims[1], patch->ldims[2]));
}
#endif

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_2_total
//
// like cuda_find_block_indices, but handles out-of-bound
// particles

__global__ static void
mprts_find_block_indices_2_total(struct cuda_params prm, float4 *d_xi4,
				 unsigned int *d_off,
				 unsigned int *d_bidx, int nr_patches)
{
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % prm.b_mx[2];
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);
  int p = blockIdx.y / prm.b_mx[2];

  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  // FIXME/OPT, could be done better like reorder_send_buf
  int block_begin = d_off[bid + p * nr_blocks];
  int block_end   = d_off[bid + p * nr_blocks + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_xi4[n];
    unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = nr_blocks * nr_patches;
    } else {
      block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
    }
    d_bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_indices_2_total(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (mprts->nr_patches == 0) {
    return;
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, cmprts, NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_indices_2_total, (prm, cmprts->d_xi4, cmprts->d_off,
						cmprts->d_bidx, mprts->nr_patches));
  free_params(&prm);
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_keys

__global__ static void
mprts_find_block_keys(struct cuda_params prm, float4 *d_xi4,
		      unsigned int *d_off,
		      unsigned int *d_bidx, int nr_total_blocks)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];
  int p = bid / nr_blocks;

  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_xi4[n];
    unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = CUDA_BND_S_OOB;
    } else {
      int bidx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
      int b_diff = bid - bidx + prm.b_mx[1] + 1;
      int d1 = b_diff % prm.b_mx[1];
      int d2 = b_diff / prm.b_mx[1];
      block_idx = d2 * 3 + d1;
    }
    d_bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_keys(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (mprts->nr_patches == 0) {
    return;
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, cmprts, NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { cmprts->n_blocks, 1 };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_keys, (prm, cmprts->d_xi4, cmprts->d_off,
				     cmprts->d_bidx, cmprts->n_blocks));
  free_params(&prm);
}

