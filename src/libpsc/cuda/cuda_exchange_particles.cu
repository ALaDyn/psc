#include "hip/hip_runtime.h"

#include <psc_cuda.h>
#include "cuda_sort2.h"
#include "particles_cuda.h"
#include "psc_bnd_cuda.h"

#include <thrust/scan.h>
#include <thrust/device_vector.h>

#define PFX(x) xchg_##x
#include "constants.c"

// FIXME const mem for dims?
// FIXME probably should do our own loop rather than use blockIdx

__global__ static void
exchange_particles(int n_part, particles_cuda_dev_t h_dev,
		   int ldimsx, int ldimsy, int ldimsz)
{
  int ldims[3] = { ldimsx, ldimsy, ldimsz };
  int xm[3];

  for (int d = 0; d < 3; d++) {
    xm[d] = ldims[d] / d_consts.dxi[d];
  }

  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      h_dev.xi4[i].x * d_consts.dxi[0],
      h_dev.xi4[i].y * d_consts.dxi[1],
      h_dev.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = cuda_fint(xi[d]);
    }
    if (pos[1] < 0) {
      h_dev.xi4[i].y += xm[1];
      if (h_dev.xi4[i].y >= xm[1])
	h_dev.xi4[i].y = 0.f;
    }
    if (pos[2] < 0) {
      h_dev.xi4[i].z += xm[2];
      if (h_dev.xi4[i].z >= xm[2])
	h_dev.xi4[i].z = 0.f;
    }
    if (pos[1] >= ldims[1]) {
      h_dev.xi4[i].y -= xm[1];
    }
    if (pos[2] >= ldims[2]) {
      h_dev.xi4[i].z -= xm[2];
    }
  }
}

EXTERN_C void
cuda_exchange_particles(int p, struct psc_particles *prts)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_patch *patch = &ppsc->patch[p];

  xchg_set_constants(prts, NULL);

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     exchange_particles, (prts->n_part, *cuda->h_dev,
				  patch->ldims[0], patch->ldims[1], patch->ldims[2]));
}

// ======================================================================
// cuda_find_block_indices

__global__ static void
find_block_indices(int n_part, particles_cuda_dev_t h_dev, unsigned int *d_bidx,
		   int dimy, float b_dyi, float b_dzi, int b_my, int b_mz)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    float4 xi4 = h_dev.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx = block_pos_z * b_my + block_pos_y;
    d_bidx[i] = block_idx;
  }
}

EXTERN_C void
cuda_find_block_indices(struct psc_particles *prts, unsigned int *d_bidx)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices, (prts->n_part, *cuda->h_dev, d_bidx,
				  cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				  cuda->b_mx[1], cuda->b_mx[2]));
}

// ======================================================================
// cuda_find_block_indices_ids

__global__ static void
find_block_indices_ids(int n_part, particles_cuda_dev_t h_dev, unsigned int *d_bidx,
		       unsigned int *d_ids, int dimy, float b_dyi, float b_dzi, int b_my, int b_mz)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    float4 xi4 = h_dev.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx = block_pos_z * b_my + block_pos_y;
    d_bidx[i] = block_idx;
    d_ids[i] = i;
  }
}

EXTERN_C void
cuda_find_block_indices_ids(struct psc_particles *prts, unsigned int *d_bidx,
			    unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices_ids, (prts->n_part, *cuda->h_dev, d_bidx, d_ids,
				      cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				      cuda->b_mx[1], cuda->b_mx[2]));
}

// ======================================================================
// cuda_find_block_indices_2
//
// like cuda_find_block_indices, but handles out-of-bound
// particles

__global__ static void
find_block_indices_2(int n_part, particles_cuda_dev_t h_dev, unsigned int *d_bidx,
		     int dimy, float b_dyi, float b_dzi,
		     int b_my, int b_mz, int start)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x + start;
  if (i < n_part) {
    float4 xi4 = h_dev.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx;
    if (block_pos_y >= b_my || block_pos_z >= b_mz) {
      block_idx = b_my * b_mz;
    } else {
      block_idx = block_pos_z * b_my + block_pos_y;
    }
    d_bidx[i] = block_idx;
  }
}

EXTERN_C void
cuda_find_block_indices_2(struct psc_particles *prts, unsigned int *d_bidx,
			  int start)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { ((prts->n_part - start) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices_2, (prts->n_part, *cuda->h_dev, d_bidx,
				    cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				    cuda->b_mx[1], cuda->b_mx[2], start));
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_2

__global__ static void
mprts_find_block_indices_2(struct cuda_params prm, particles_cuda_dev_t *d_cp_prts,
			   unsigned int *d_bidx, int nr_patches)
{
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y;
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);

  unsigned int off = 0;
  for (int p = 0; p < nr_patches; p++) {
    int block_begin = d_cp_prts[p].offsets[bid];
    int block_end   = d_cp_prts[p].offsets[bid+1];

    for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
      float4 xi4 = d_cp_prts[p].xi4[n];
      unsigned int block_pos_y = cuda_fint(xi4.y * prm.b_dxi[1]);
      unsigned int block_pos_z = cuda_fint(xi4.z * prm.b_dxi[2]);
      
      int block_idx;
      if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
	block_idx = prm.b_mx[1] * prm.b_mx[2];
      } else {
	block_idx = block_pos_z * prm.b_mx[1] + block_pos_y;
      }
      d_bidx[off + n] = block_idx;
    }
    off += d_cp_prts[p].n_part;
  }
}

EXTERN_C void
cuda_mprts_find_block_indices_2(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }
  struct cuda_params prm;
  set_params(&prm, ppsc, psc_mparticles_get_patch(mprts, 0), NULL);
  
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { prm.b_mx[1], prm.b_mx[2] };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_indices_2, (prm, mprts_cuda->d_dev,
					  mprts_cuda->d_bidx, mprts->nr_patches));
  free_params(&prm);
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_2_total

__global__ static void
mprts_find_block_indices_2_total(struct cuda_params prm, particles_cuda_dev_t *d_cp_prts,
				 unsigned int *d_bidx, int nr_patches)
{
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % prm.b_mx[2];
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);
  int p = blockIdx.y / prm.b_mx[2];

  // FIXME/OPT, could be done better like reorder_send_buf
  int block_begin = d_cp_prts[p].d_off[bid];
  int block_end   = d_cp_prts[p].d_off[bid+1];

  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_cp_prts[0].xi4[n];
    unsigned int block_pos_y = cuda_fint(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = cuda_fint(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = nr_blocks * nr_patches;
    } else {
      block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
    }
    d_bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_indices_2_total(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, psc_mparticles_get_patch(mprts, 0), NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_indices_2_total, (prm, psc_mparticles_cuda(mprts)->d_dev,
						mprts_cuda->d_bidx, mprts->nr_patches));
  free_params(&prm);
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_ids_total

__global__ static void
mprts_find_block_indices_ids_total(struct cuda_params prm, particles_cuda_dev_t *d_cp_prts,
				   unsigned int *d_bidx, unsigned int *d_ids, int nr_patches)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  unsigned int off = 0;
  for (int p = 0; p < nr_patches; p++) {
    if (n < d_cp_prts[p].n_part) {
      float4 xi4 = d_cp_prts[0].xi4[n + off];
      unsigned int block_pos_y = cuda_fint(xi4.y * prm.b_dxi[1]);
      unsigned int block_pos_z = cuda_fint(xi4.z * prm.b_dxi[2]);
      
      int block_idx;
      if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
	block_idx = -1; // not supposed to happen here!
      } else {
	block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
      }
      d_bidx[n + off] = block_idx;
      d_ids[n + off] = n + off;
    }
    off += d_cp_prts[p].n_part;
  }
}

EXTERN_C void
cuda_mprts_find_block_indices_ids_total(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  int max_n_part = 0;
  mprts_cuda->nr_prts_send = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    mprts_cuda->nr_prts_send += cuda->bnd_n_send;
    if (prts->n_part > max_n_part) {
      max_n_part = prts->n_part;
    }
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, psc_mparticles_get_patch(mprts, 0), NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (max_n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };

  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_indices_ids_total, (prm, psc_mparticles_cuda(mprts)->d_dev,
						  mprts_cuda->d_bidx, mprts_cuda->d_ids,
						  mprts->nr_patches));
  free_params(&prm);
}

// ----------------------------------------------------------------------

EXTERN_C void
_cuda_find_block_indices_2(struct psc_particles *prts, unsigned int *d_bidx,
			   int start)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4 = new float4[prts->n_part];
  float4 *pxi4 = new float4[prts->n_part];
  unsigned int *bidx = new unsigned int[prts->n_part];
  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);

  float b_dyi = cuda->b_dxi[1], b_dzi = cuda->b_dxi[2];
  int b_my = cuda->b_mx[1], b_mz = cuda->b_mx[2];
  for (int i = start; i < prts->n_part; i++) {
    unsigned int block_pos_y = cuda_fint(xi4[i].y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4[i].z * b_dzi);

    int block_idx;
    if (block_pos_y >= b_my || block_pos_z >= b_mz) {
      block_idx = cuda->nr_blocks;
    } else {
      block_idx = block_pos_z * b_my + block_pos_y;
    }
    bidx[i] = block_idx;
  }
  
  cuda_copy_bidx_to_dev(prts, d_bidx, bidx);
  delete[] xi4;
  delete[] pxi4;
  delete[] bidx;
}

// ======================================================================
// cuda_find_block_indices_3

EXTERN_C void
cuda_find_block_indices_3(struct psc_particles *prts, unsigned int *d_bidx,
			  unsigned int *d_alt_bidx,
			  int start, unsigned int *bn_idx, unsigned int *bn_off)
{
  // for consistency, use same block indices that we counted earlier
  check(hipMemcpy(d_bidx + start, bn_idx, (prts->n_part - start) * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
  // abuse of alt_bidx!!! FIXME
  check(hipMemcpy(d_alt_bidx + start, bn_off, (prts->n_part - start) * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
}

// ======================================================================
// cuda_find_block_indices_3

EXTERN_C void
cuda_mprts_find_block_indices_3(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    unsigned int start = cuda->bnd_n_part_save;
    // for consistency, use same block indices that we counted earlier
    check(hipMemcpy(mprts_cuda->d_bidx + off + start, cuda->bnd_idx,
		     (prts->n_part - start) * sizeof(*mprts_cuda->d_bidx),
		     hipMemcpyHostToDevice));
    // abuse of alt_bidx!!! FIXME
    check(hipMemcpy(mprts_cuda->d_alt_bidx + off + start, cuda->bnd_off,
		     (prts->n_part - start) * sizeof(*mprts_cuda->d_alt_bidx),
		     hipMemcpyHostToDevice));
    off += cuda->n_alloced;
  }
}

// ======================================================================
// reorder_send_buf

__global__ static void
reorder_send_buf(int n_part, particles_cuda_dev_t h_dev, unsigned int *d_bidx,
		 unsigned int *d_sums, unsigned int nr_blocks)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    if (d_bidx[i] == nr_blocks) {
      int j = d_sums[i] + n_part;
      h_dev.xi4[j] = h_dev.xi4[i];
      h_dev.pxi4[j] = h_dev.pxi4[i];
    }
  }
}

EXTERN_C void
cuda_reorder_send_buf(int p, struct psc_particles *prts, 
		      unsigned int *d_bidx, unsigned int *d_sums, int n_send)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  assert(prts->n_part + n_send <= cuda->n_alloced);

  // OPT: don't pass offset, get it in device code
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder_send_buf, (prts->n_part, *cuda->h_dev, d_bidx, d_sums, cuda->nr_blocks));
}

__global__ static void
mprts_reorder_send_buf_total(int nr_prts, int nr_oob, unsigned int *d_bidx, unsigned int *d_sums,
			     float4 *d_xi4, float4 *d_pxi4,
			     float4 *d_xchg_xi4, float4 *d_xchg_pxi4)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i >= nr_prts)
    return;

  if (d_bidx[i] == nr_oob) {
    int j = d_sums[i];
    d_xchg_xi4[j]  = d_xi4[i];
    d_xchg_pxi4[j] = d_pxi4[i];
  }
}

EXTERN_C void
cuda_mprts_reorder_send_buf_total(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  if (mprts->nr_patches == 0)
    return;

  struct psc_particles_cuda *cuda = psc_particles_cuda(psc_mparticles_get_patch(mprts, 0));
  
  mprts_cuda->nr_prts_send = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    mprts_cuda->nr_prts_send += cuda->bnd_n_send;
  }

  float4 *xchg_xi4 = mprts_cuda->d_xi4 + mprts_cuda->nr_prts;
  float4 *xchg_pxi4 = mprts_cuda->d_pxi4 + mprts_cuda->nr_prts;
  assert(mprts_cuda->nr_prts + mprts_cuda->nr_prts_send < mprts_cuda->nr_alloced);
  int nr_oob = cuda->nr_blocks * mprts->nr_patches;
  
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (mprts_cuda->nr_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_reorder_send_buf_total, (mprts_cuda->nr_prts, nr_oob,
					    mprts_cuda->d_bidx, mprts_cuda->d_sums,
					    mprts_cuda->d_xi4, mprts_cuda->d_pxi4,
					    xchg_xi4, xchg_pxi4));
}

EXTERN_C void
_cuda_reorder_send_buf(int p, struct psc_particles *prts, 
		       unsigned int *d_bidx, unsigned int *d_sums, int n_send)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int n_part = prts->n_part;
  int n_total = n_part + n_send;
  assert(n_total <= cuda->n_alloced);
  float4 *xi4 = new float4[n_total];
  float4 *pxi4 = new float4[n_total];
  unsigned int *bidx = new unsigned int[n_total];
  unsigned int *sums = new unsigned int[n_total];
  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);
  cuda_copy_bidx_from_dev(prts, sums, d_sums);

  for (int i = 0; i < prts->n_part; i++) {
    if (bidx[i] == cuda->nr_blocks) {
      int j = sums[i] + prts->n_part;
      xi4[j] = xi4[i];
      pxi4[j] = pxi4[i];
    }
  }

  prts->n_part = n_total;
  __particles_cuda_to_device(prts, xi4, pxi4, NULL);
  prts->n_part = n_part;
  delete[] xi4;
  delete[] pxi4;
  delete[] bidx;
  delete[] sums;
}

// ======================================================================

static void
psc_particles_cuda_swap_alt(struct psc_particles *prts)
{
  // FIXME (eventually)
  // this function should not exist, since mprts needs to be swapped, too,
  // but isn't available here.
  // but due to sorting in copy_from/to, it's inevitable, so we fix up 
  // the mprts pointers elsewhere for nwo
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

  float4 *alt_xi4 = cuda->h_dev->alt_xi4;
  float4 *alt_pxi4 = cuda->h_dev->alt_pxi4;
  cuda->h_dev->alt_xi4 = cuda->h_dev->xi4;
  cuda->h_dev->alt_pxi4 = cuda->h_dev->pxi4;
  cuda->h_dev->xi4 = alt_xi4;
  cuda->h_dev->pxi4 = alt_pxi4;
}

static void
psc_mparticles_cuda_swap_alt(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    psc_particles_cuda_swap_alt(prts);
  }
  float4 *tmp_xi4 = mprts_cuda->d_alt_xi4;
  float4 *tmp_pxi4 = mprts_cuda->d_alt_pxi4;
  mprts_cuda->d_alt_xi4 = mprts_cuda->d_xi4;
  mprts_cuda->d_alt_pxi4 = mprts_cuda->d_pxi4;
  mprts_cuda->d_xi4 = tmp_xi4;
  mprts_cuda->d_pxi4 = tmp_pxi4;
}

// ======================================================================
// reorder_and_offsets

__global__ static void
reorder_and_offsets(int n_part, particles_cuda_dev_t h_dev, float4 *xi4, float4 *pxi4,
		    unsigned int *d_bidx, unsigned int *d_ids, int nr_blocks)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > n_part)
    return;

  int block, prev_block;
  if (i < n_part) {
    xi4[i] = h_dev.xi4[d_ids[i]];
    pxi4[i] = h_dev.pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else if (i == n_part) { // needed if there is no particle in the last block
    block = nr_blocks;
  }

  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    h_dev.offsets[b] = i;
  }
}

EXTERN_C void
cuda_reorder_and_offsets(struct psc_particles *prts, unsigned int *d_bidx,
			 unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *alt_xi4 = cuda->h_dev->alt_xi4;
  float4 *alt_pxi4 = cuda->h_dev->alt_pxi4;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder_and_offsets, (prts->n_part, *cuda->h_dev, alt_xi4, alt_pxi4,
				   d_bidx, d_ids, cuda->nr_blocks));

  psc_particles_cuda_swap_alt(prts);
}

void
_cuda_reorder_and_offsets(struct psc_particles *prts, unsigned int *d_bidx,
			  unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4 = new float4[prts->n_part];
  float4 *pxi4 = new float4[prts->n_part];
  float4 *alt_xi4 = new float4[prts->n_part];
  float4 *alt_pxi4 = new float4[prts->n_part];
  unsigned int *bidx = new unsigned int[prts->n_part];
  unsigned int *ids = new unsigned int[prts->n_part];
  int *offsets = new int[cuda->nr_blocks + 2];

  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);
  cuda_copy_bidx_from_dev(prts, ids, d_ids);

  for (int i = 0; i < prts->n_part; i++) {
    alt_xi4[i] = xi4[ids[i]];
    alt_pxi4[i] = pxi4[ids[i]];

    int block = bidx[i];
    int prev_block = (i > 0) ? (int) bidx[i-1] : -1;
    for (int b = prev_block + 1; b <= block; b++) {
      offsets[b] = i;
    }
  }
  int block = cuda->nr_blocks + 1;
  int prev_block = bidx[prts->n_part - 1];
  for (int b = prev_block + 1; b <= block; b++) {
    offsets[b] = prts->n_part;
  }

  psc_particles_cuda_swap_alt(prts);

  __particles_cuda_to_device(prts, alt_xi4, alt_pxi4, offsets);
  delete[] xi4;
  delete[] pxi4;
  delete[] alt_xi4;
  delete[] alt_pxi4;
  delete[] bidx;
  delete[] ids;
  delete[] offsets;
}

__global__ static void
mprts_reorder_and_offsets(int nr_prts, float4 *xi4, float4 *pxi4, float4 *alt_xi4, float4 *alt_pxi4,
			  unsigned int *d_bidx, unsigned int *d_ids, unsigned int *d_off, int last_block)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > nr_prts)
    return;

  int block, prev_block;
  if (i < nr_prts) {
    alt_xi4[i] = xi4[d_ids[i]];
    alt_pxi4[i] = pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else { // needed if there is no particle in the last block
    block = last_block;
  }

  // OPT: d_bidx[i-1] could use shmem
  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_off[b] = i;
  }
}

void
cuda_mprts_reorder_and_offsets(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }
  int nr_blocks = psc_particles_cuda(psc_mparticles_get_patch(mprts, 0))->nr_blocks;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (mprts_cuda->nr_prts + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_reorder_and_offsets, (mprts_cuda->nr_prts, mprts_cuda->d_xi4, mprts_cuda->d_pxi4,
					 mprts_cuda->d_alt_xi4, mprts_cuda->d_alt_pxi4,
					 mprts_cuda->d_bidx, mprts_cuda->d_ids,
					 mprts_cuda->d_off, mprts->nr_patches * nr_blocks));

  psc_mparticles_cuda_swap_alt(mprts);
  psc_mparticles_cuda_copy_to_dev(mprts);
}

// ======================================================================
// cuda_reorder

__global__ static void
reorder(int n_part, particles_cuda_dev_t h_dev, float4 *xi4, float4 *pxi4,
	unsigned int *d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < n_part) {
    xi4[i] = h_dev.xi4[d_ids[i]];
    pxi4[i] = h_dev.pxi4[d_ids[i]];
  }
}

EXTERN_C void
cuda_reorder(struct psc_particles *prts, unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *alt_xi4 = cuda->h_dev->alt_xi4;
  float4 *alt_pxi4 = cuda->h_dev->alt_pxi4;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder, (prts->n_part, *cuda->h_dev, alt_xi4, alt_pxi4, d_ids));

  cuda->h_dev->alt_xi4 = cuda->h_dev->xi4;
  cuda->h_dev->alt_pxi4 = cuda->h_dev->pxi4;
  cuda->h_dev->xi4 = alt_xi4;
  cuda->h_dev->pxi4 = alt_pxi4;
}

// ======================================================================
// cuda_exclusive_scan

EXTERN_C int
_cuda_exclusive_scan(int p, struct psc_particles *prts,
		    unsigned int *d_vals, unsigned int *d_sums)
{
  unsigned int *vals = new unsigned int[prts->n_part];
  unsigned int *sums = new unsigned int[prts->n_part];
  cuda_copy_bidx_from_dev(prts, vals, d_vals);

  unsigned int sum = 0;
  for (int i = 0; i < prts->n_part; i++) {
    sums[i] = sum;
    sum += vals[i];
  }

  cuda_copy_bidx_to_dev(prts, d_sums, sums);
  delete[] sums;
  delete[] vals;
  return sum;
}

EXTERN_C int
cuda_exclusive_scan(int p, struct psc_particles *prts, unsigned int *_d_vals, unsigned int *_d_sums)
{
  thrust::device_ptr<unsigned int> d_vals(_d_vals);
  thrust::device_ptr<unsigned int> d_sums(_d_sums);
  thrust::exclusive_scan(d_vals, d_vals + prts->n_part, d_sums);
  int sum = d_sums[prts->n_part - 1] + d_vals[prts->n_part - 1];
  return sum;
}

// ======================================================================
// cuda_mprts_copy_from_dev

void
cuda_mprts_copy_from_dev(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  mprts_cuda->h_bnd_xi4 = new float4[mprts_cuda->nr_prts_send];
  mprts_cuda->h_bnd_pxi4 = new float4[mprts_cuda->nr_prts_send];

  check(hipMemcpy(mprts_cuda->h_bnd_xi4, mprts_cuda->d_xi4 + mprts_cuda->nr_prts,
		   mprts_cuda->nr_prts_send * sizeof(float4), hipMemcpyDeviceToHost));
  check(hipMemcpy(mprts_cuda->h_bnd_pxi4, mprts_cuda->d_pxi4 + mprts_cuda->nr_prts,
		   mprts_cuda->nr_prts_send * sizeof(float4), hipMemcpyDeviceToHost));
}

// ----------------------------------------------------------------------
// cuda_mprts_convert_from_cuda

void
cuda_mprts_convert_from_cuda(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  float4 *bnd_xi4 = mprts_cuda->h_bnd_xi4;
  float4 *bnd_pxi4 = mprts_cuda->h_bnd_pxi4;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    cuda->bnd_prts = new particle_single_t[cuda->bnd_n_send];
    for (int n = 0; n < cuda->bnd_n_send; n++) {
      particle_single_t *prt = &cuda->bnd_prts[n];
      prt->xi      = bnd_xi4[n].x;
      prt->yi      = bnd_xi4[n].y;
      prt->zi      = bnd_xi4[n].z;
      prt->kind    = cuda_float_as_int(bnd_xi4[n].w);
      prt->pxi     = bnd_pxi4[n].x;
      prt->pyi     = bnd_pxi4[n].y;
      prt->pzi     = bnd_pxi4[n].z;
      prt->qni_wni = bnd_pxi4[n].w;
    }
    bnd_xi4 += cuda->bnd_n_send;
    bnd_pxi4 += cuda->bnd_n_send;
  }
  delete[] mprts_cuda->h_bnd_xi4;
  delete[] mprts_cuda->h_bnd_pxi4;
}

// ======================================================================
// cuda_mprts_copy_to_dev

void
cuda_mprts_copy_to_dev(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  float4 *d_alt_xi4 = mprts_cuda->d_alt_xi4;
  float4 *d_alt_pxi4 = mprts_cuda->d_alt_pxi4;
  float4 *d_xi4 = mprts_cuda->d_xi4;
  float4 *d_pxi4 = mprts_cuda->d_pxi4;
  unsigned int *d_bidx = mprts_cuda->d_bidx;
  unsigned int *d_alt_bidx = mprts_cuda->d_alt_bidx;
  unsigned int *d_ids = mprts_cuda->d_ids;
  unsigned int *d_alt_ids = mprts_cuda->d_alt_ids;
  unsigned int *d_sums = mprts_cuda->d_sums;

 for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    assert(d_alt_xi4 + prts->n_part <= mprts_cuda->d_alt_xi4 + mprts_cuda->nr_alloced);
    check(hipMemcpy(d_alt_xi4, cuda->h_dev->xi4,
		     cuda->bnd_n_part_save * sizeof(*cuda->h_dev->alt_xi4),
		     hipMemcpyDeviceToDevice));
    check(hipMemcpy(d_alt_xi4 + cuda->bnd_n_part_save, cuda->bnd_xi4,
		     (prts->n_part - cuda->bnd_n_part_save) * sizeof(*cuda->bnd_xi4),
		     hipMemcpyHostToDevice));
    check(hipMemcpy(d_alt_pxi4, cuda->h_dev->pxi4,
		     cuda->bnd_n_part_save * sizeof(*cuda->h_dev->alt_xi4),
		     hipMemcpyDeviceToDevice));
    check(hipMemcpy(d_alt_pxi4 + cuda->bnd_n_part_save, cuda->bnd_pxi4,
		     (prts->n_part - cuda->bnd_n_part_save) * sizeof(*cuda->bnd_pxi4),
		     hipMemcpyHostToDevice));
    cuda->n_alloced = prts->n_part;
    cuda->h_dev->alt_xi4 = d_alt_xi4;
    cuda->h_dev->alt_pxi4 = d_alt_pxi4;
    cuda->h_dev->xi4 = d_xi4;
    cuda->h_dev->pxi4 = d_pxi4;
    d_alt_xi4 += cuda->n_alloced;
    d_alt_pxi4 += cuda->n_alloced;
    d_xi4 += cuda->n_alloced;
    d_pxi4 += cuda->n_alloced;
    d_bidx += cuda->n_alloced;
    d_alt_bidx += cuda->n_alloced;
    d_ids += cuda->n_alloced;
    d_alt_ids += cuda->n_alloced;
    d_sums += cuda->n_alloced;
  }
  psc_mparticles_cuda_swap_alt(mprts);
  psc_mparticles_cuda_copy_to_dev(mprts);
}

void
cuda_mprts_copy_to_dev_v1(struct psc_mparticles *mprts)
{
  hipStream_t stream[mprts->nr_patches];
  for (int p = 0; p < mprts->nr_patches; p++) {
    hipStreamCreate(&stream[p]);
  }
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    check(hipMemcpyAsync(cuda->h_dev->xi4 + cuda->bnd_n_part_save, cuda->bnd_xi4,
			  (prts->n_part - cuda->bnd_n_part_save) * sizeof(*cuda->bnd_xi4),
			  hipMemcpyHostToDevice, stream[p]));
    check(hipMemcpyAsync(cuda->h_dev->pxi4 + cuda->bnd_n_part_save, cuda->bnd_pxi4,
			  (prts->n_part - cuda->bnd_n_part_save) * sizeof(*cuda->bnd_pxi4),
			  hipMemcpyHostToDevice, stream[p]));
  }

  for (int p = 0; p < mprts->nr_patches; p++) {
    hipStreamSynchronize(stream[p]);
    hipStreamDestroy(stream[p]);
  }
}

// ======================================================================
// cuda_mprts_sort

void
cuda_mprts_sort(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  unsigned int *d_bidx = mprts_cuda->d_bidx;
  unsigned int *h_bidx = new unsigned int[mprts_cuda->nr_alloced];
  unsigned int *h_bidx_save = h_bidx;
  check(hipMemcpy(h_bidx, d_bidx, mprts_cuda->nr_alloced * sizeof(float),
		   hipMemcpyDeviceToHost));
  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    check(hipMemcpy(d_bidx, h_bidx, cuda->n_alloced * sizeof(float),
		     hipMemcpyHostToDevice));
    // OPT: when calculating bidx, do preprocess then
    void *sp = sort_pairs_3_create(cuda->b_mx);
    sort_pairs_3_device(sp, d_bidx, mprts_cuda->d_alt_bidx + off, mprts_cuda->d_alt_ids + off,
			prts->n_part, cuda->h_dev->offsets,
			cuda->bnd_n_part_save, cuda->bnd_cnt);
    sort_pairs_3_destroy(sp);
    //    cuda->h_dev->bidx = d_bidx;
    d_bidx += 122880;
    h_bidx += cuda->n_alloced;
    off += cuda->n_alloced;
  }
  delete[] h_bidx_save;

  unsigned int *h_alt_ids = new unsigned int[mprts_cuda->nr_alloced];
  check(hipMemcpy(h_alt_ids, mprts_cuda->d_alt_ids, mprts_cuda->nr_alloced * sizeof(unsigned int),
		   hipMemcpyDeviceToHost));

  off = 0;
  unsigned int off2 = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    assert(off == cuda->h_dev->xi4 - mprts_cuda->d_xi4);
    for (int n = 0; n < prts->n_part - cuda->bnd_n_send; n++) {
      h_alt_ids[off2 + n] = h_alt_ids[off + n] + off;
    }
    off += cuda->n_alloced;
    off2 += prts->n_part - cuda->bnd_n_send;
  }
  check(hipMemcpy(mprts_cuda->d_alt_ids, h_alt_ids,
		   mprts_cuda->nr_alloced * sizeof(unsigned int),
		   hipMemcpyHostToDevice));

  delete[] h_alt_ids;
  psc_mparticles_cuda_copy_to_dev(mprts);
}

// ======================================================================
// cuda_mprts_reorder

__global__ static void
mprts_reorder(int nr_prts, unsigned int *alt_ids,
	      float4 *xi4, float4 *pxi4,
	      float4 *alt_xi4, float4 *alt_pxi4)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < nr_prts) {
    int j = alt_ids[i];
    alt_xi4[i] = xi4[j];
    alt_pxi4[i] = pxi4[j];
  }
}

void
cuda_mprts_reorder(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    prts->n_part -= cuda->bnd_n_send;
    cuda->h_dev->xi4 = mprts_cuda->d_xi4 + off;
    cuda->h_dev->pxi4 = mprts_cuda->d_pxi4 + off;
    cuda->h_dev->alt_xi4 = mprts_cuda->d_alt_xi4 + off;
    cuda->h_dev->alt_pxi4 = mprts_cuda->d_alt_pxi4 + off;
    cuda->n_alloced = prts->n_part;
    off += prts->n_part;
  }
  mprts_cuda->nr_prts = off;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (mprts_cuda->nr_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_reorder, (mprts_cuda->nr_prts, mprts_cuda->d_alt_ids,
			     mprts_cuda->d_xi4, mprts_cuda->d_pxi4,
			     mprts_cuda->d_alt_xi4, mprts_cuda->d_alt_pxi4));
  
  psc_mparticles_cuda_swap_alt(mprts);
  cuda_mprts_find_off(mprts);
}

// ======================================================================
// cuda_mprts_check_ordered

void
cuda_mprts_check_ordered(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  psc_mparticles_cuda_copy_to_dev(mprts); // update n_part, particle pointers
  cuda_mprts_find_block_indices_2(mprts);

  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    unsigned int *bidx = new unsigned int[prts->n_part];
    cuda_copy_bidx_from_dev(prts, bidx, mprts_cuda->d_bidx + off);
    
    float4 *xi4 = new float4[prts->n_part];
    float4 *pxi4 = new float4[prts->n_part];
    __particles_cuda_from_device(prts, xi4, pxi4);

    unsigned int last = 0;
    for (int n = 0; n < prts->n_part; n++) {
      unsigned int block_pos_y = cuda_fint(xi4[n].y * cuda->b_dxi[1]);
      unsigned int block_pos_z = cuda_fint(xi4[n].z * cuda->b_dxi[2]);
      
      int block_idx;
      if (block_pos_y >= cuda->b_mx[1] || block_pos_z >= cuda->b_mx[2]) {
	block_idx = cuda->nr_blocks;
      } else {
	block_idx = block_pos_z * cuda->b_mx[1] + block_pos_y;
      }
      if (block_idx != bidx[n]) {
	mprintf("n = %d bidx = %d block_idx = %d bp [%d:%d] real [%g:%g]\n",
		n, bidx[n], block_idx, block_pos_y, block_pos_z,
		xi4[n].y * cuda->b_dxi[1], xi4[n].z * cuda->b_dxi[2]);
	static int error_cnt;
	assert(error_cnt++ < 10);
      }
      if (!(bidx[n] >= last && bidx[n] < cuda->nr_blocks)) {
	mprintf("n = %d bidx = %d last = %d\n", n, bidx[n], last);
	static int error_cnt;
	assert(error_cnt++ < 10);
      }
      last = block_idx;
    }

    delete[] bidx;
    delete[] xi4;
    delete[] pxi4;

    off += prts->n_part;
  }
}

// ======================================================================
// cuda_mprts_check_ordered_offsets

void
cuda_mprts_check_ordered_offsets(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  psc_mparticles_cuda_copy_to_dev(mprts); // update n_part, particle pointers
  cuda_mprts_find_block_indices_2(mprts);

  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    unsigned int *bidx = new unsigned int[prts->n_part];
    cuda_copy_bidx_from_dev(prts, bidx, mprts_cuda->d_bidx + off);
    
    float4 *xi4 = new float4[prts->n_part];
    float4 *pxi4 = new float4[prts->n_part];
    __particles_cuda_from_device(prts, xi4, pxi4);

    unsigned int *offsets = new unsigned int[cuda->nr_blocks+1];
    cuda_copy_offsets_from_dev(prts, offsets);
    assert(offsets[0] == 0);
    assert(offsets[cuda->nr_blocks] == prts->n_part);

    unsigned int last = 0;
    for (int b = 0; b < cuda->nr_blocks; b++) {
      for (int n = offsets[b]; n < offsets[b+1]; n++) {
	unsigned int block_pos_y = cuda_fint(xi4[n].y * cuda->b_dxi[1]);
	unsigned int block_pos_z = cuda_fint(xi4[n].z * cuda->b_dxi[2]);
	
	int block_idx;
	if (block_pos_y >= cuda->b_mx[1] || block_pos_z >= cuda->b_mx[2]) {
	  block_idx = cuda->nr_blocks;
	} else {
	  block_idx = block_pos_z * cuda->b_mx[1] + block_pos_y;
	}
	if (block_idx != bidx[n]) {
	  mprintf("n = %d bidx = %d block_idx = %d bp [%d:%d] real [%g:%g]\n",
		  n, bidx[n], block_idx, block_pos_y, block_pos_z,
		  xi4[n].y * cuda->b_dxi[1], xi4[n].z * cuda->b_dxi[2]);
	  static int error_cnt;
	  assert(error_cnt++ < 10);
	}
	if (!(bidx[n] >= last && bidx[n] < cuda->nr_blocks)) {
	  mprintf("n = %d bidx = %d last = %d\n", n, bidx[n], last);
	  static int error_cnt;
	  assert(error_cnt++ < 10);
	}
	if (bidx[n] != b) {
	  mprintf("n = %d bidx = %d block_idx = %d b = %d\n",
		  n, bidx[n], block_idx, b);
	  static int error_cnt;
	  assert(error_cnt++ < 10);
	}
	last = block_idx;
      }
    }
    delete[] offsets;
    delete[] bidx;
    delete[] xi4;
    delete[] pxi4;

    off += prts->n_part;
  }
}

// ======================================================================
// cuda_mprts_free

void
cuda_mprts_free(struct psc_mparticles *mprts)
{
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    free(cuda->bnd_idx);
    free(cuda->bnd_off);
    free(cuda->bnd_cnt);
    free(cuda->bnd_prts);
    free(cuda->bnd_xi4);
    free(cuda->bnd_pxi4);
  }
}

// ======================================================================
// cuda_mprts_check_ordered_total

void
cuda_mprts_check_ordered_total(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  psc_mparticles_cuda_copy_to_dev(mprts); // update n_part, particle pointers
  cuda_mprts_find_block_indices_2_total(mprts);

  unsigned int last = 0;
  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    unsigned int *bidx = new unsigned int[prts->n_part];
    cuda_copy_bidx_from_dev(prts, bidx, mprts_cuda->d_bidx + off);
    
    for (int n = 0; n < prts->n_part; n++) {
      if (!(bidx[n] >= last && bidx[n] < mprts->nr_patches * cuda->nr_blocks)) {
	mprintf("p = %d, n = %d bidx = %d last = %d\n", p, n, bidx[n], last);
	assert(0);
      }
      last = bidx[n];
    }

    delete[] bidx;

    off += prts->n_part;
  }
}

// ======================================================================
// cuda_mprts_compact

void
cuda_mprts_compact(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  float4 *d_alt_xi4 = mprts_cuda->d_alt_xi4;
  float4 *d_alt_pxi4 = mprts_cuda->d_alt_pxi4;
  float4 *d_xi4 = mprts_cuda->d_xi4;
  float4 *d_pxi4 = mprts_cuda->d_pxi4;
  unsigned int *d_bidx = mprts_cuda->d_bidx;
  unsigned int *d_alt_bidx = mprts_cuda->d_alt_bidx;
  unsigned int *d_sums = mprts_cuda->d_sums;
  unsigned int *d_ids = mprts_cuda->d_ids;
  unsigned int *d_alt_ids = mprts_cuda->d_alt_ids;

  int nr_prts = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    assert(d_alt_xi4 + prts->n_part <= mprts_cuda->d_alt_xi4 + mprts_cuda->nr_alloced);
    check(hipMemcpy(d_alt_xi4, cuda->h_dev->xi4,
		     prts->n_part * sizeof(*cuda->h_dev->alt_xi4),
		     hipMemcpyDeviceToDevice));
    check(hipMemcpy(d_alt_pxi4, cuda->h_dev->pxi4,
		     prts->n_part * sizeof(*cuda->h_dev->alt_pxi4),
		     hipMemcpyDeviceToDevice));
    nr_prts += prts->n_part;
    cuda->n_alloced = prts->n_part;
    cuda->h_dev->alt_xi4 = d_alt_xi4;
    cuda->h_dev->alt_pxi4 = d_alt_pxi4;
    cuda->h_dev->xi4 = d_xi4;
    cuda->h_dev->pxi4 = d_pxi4;
    d_alt_xi4 += cuda->n_alloced;
    d_alt_pxi4 += cuda->n_alloced;
    d_xi4 += cuda->n_alloced;
    d_pxi4 += cuda->n_alloced;
    d_bidx += cuda->n_alloced;
    d_alt_bidx += cuda->n_alloced;
    d_ids += cuda->n_alloced;
    d_alt_ids += cuda->n_alloced;
    d_sums += cuda->n_alloced;
  }
  mprts_cuda->nr_prts = nr_prts;
  psc_mparticles_cuda_swap_alt(mprts);
  psc_mparticles_cuda_copy_to_dev(mprts);
}

void
cuda_mprts_find_off(struct psc_mparticles *mprts)
{
  unsigned int off = 0;
  unsigned int last = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    int *offsets = new int[cuda->nr_blocks + 1];
    check(hipMemcpy(offsets, cuda->h_dev->offsets,
		     (cuda->nr_blocks + 1) * sizeof(*offsets),
		     hipMemcpyDeviceToHost));
    assert(offsets[cuda->nr_blocks] == prts->n_part);
    for (int n = 0; n <= cuda->nr_blocks; n++) {
      offsets[n] += off;
    }
    assert(offsets[0] == last);
    last = offsets[cuda->nr_blocks];
    check(hipMemcpy(cuda->h_dev->d_off, offsets,
		     (cuda->nr_blocks + 1) * sizeof(*offsets),
		     hipMemcpyHostToDevice));
    delete[] offsets;

    off += prts->n_part;
  }
}

void
cuda_mprts_find_offsets(struct psc_mparticles *mprts)
{
  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    int *d_off = new int[cuda->nr_blocks + 1];
    check(hipMemcpy(d_off, cuda->h_dev->d_off,
		     (cuda->nr_blocks + 1) * sizeof(*d_off),
		     hipMemcpyDeviceToHost));
    assert(d_off[cuda->nr_blocks] == prts->n_part + off);
    for (int n = 0; n <= cuda->nr_blocks; n++) {
      d_off[n] -= off;
    }
    assert(d_off[0] == 0);
    check(hipMemcpy(cuda->h_dev->offsets, d_off,
		     (cuda->nr_blocks + 1) * sizeof(*d_off),
		     hipMemcpyHostToDevice));
    delete[] d_off;

    off += prts->n_part;
  }
}
