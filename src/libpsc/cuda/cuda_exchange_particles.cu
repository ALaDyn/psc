#include "hip/hip_runtime.h"

#undef _GLIBCXX_USE_INT128

#include "cuda_mparticles.h"

#include "cuda_sort2.h"
#include "particles_cuda.h"
#include "psc_bnd_cuda.h"
#include "psc_particles_as_cuda.h"

#define PFX(x) xchg_##x
#include "constants.c"

#if 0
// FIXME const mem for dims?
// FIXME probably should do our own loop rather than use blockIdx

__global__ static void
exchange_particles(int n_part, particles_cuda_dev_t h_dev,
		   int ldimsx, int ldimsy, int ldimsz)
{
  int ldims[3] = { ldimsx, ldimsy, ldimsz };
  int xm[3];

  for (int d = 0; d < 3; d++) {
    xm[d] = ldims[d] / d_consts.dxi[d];
  }

  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      h_dev.xi4[i].x * d_consts.dxi[0],
      h_dev.xi4[i].y * d_consts.dxi[1],
      h_dev.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = __float2int_rd(xi[d]);
    }
    if (pos[1] < 0) {
      h_dev.xi4[i].y += xm[1];
      if (h_dev.xi4[i].y >= xm[1])
	h_dev.xi4[i].y = 0.f;
    }
    if (pos[2] < 0) {
      h_dev.xi4[i].z += xm[2];
      if (h_dev.xi4[i].z >= xm[2])
	h_dev.xi4[i].z = 0.f;
    }
    if (pos[1] >= ldims[1]) {
      h_dev.xi4[i].y -= xm[1];
    }
    if (pos[2] >= ldims[2]) {
      h_dev.xi4[i].z -= xm[2];
    }
  }
}

EXTERN_C void
cuda_exchange_particles(int p, struct psc_particles *prts)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_patch *patch = &ppsc->patch[p];

  xchg_set_constants(prts, NULL);

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     exchange_particles, (prts->n_part, *cuda->h_dev,
				  patch->ldims[0], patch->ldims[1], patch->ldims[2]));
}
#endif

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_2_total
//
// like cuda_find_block_indices, but handles out-of-bound
// particles

__global__ static void
mprts_find_block_indices_2_total(struct cuda_params prm, float4 *d_xi4,
				 unsigned int *d_off,
				 unsigned int *d_bidx, int nr_patches)
{
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % prm.b_mx[2];
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);
  int p = blockIdx.y / prm.b_mx[2];

  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  // FIXME/OPT, could be done better like reorder_send_buf
  int block_begin = d_off[bid + p * nr_blocks];
  int block_end   = d_off[bid + p * nr_blocks + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_xi4[n];
    unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = nr_blocks * nr_patches;
    } else {
      block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
    }
    d_bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_indices_2_total(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (mprts->nr_patches == 0) {
    return;
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, cmprts, NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_indices_2_total, (prm, cmprts->d_xi4, cmprts->d_off,
						cmprts->d_bidx, mprts->nr_patches));
  free_params(&prm);
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_keys

__global__ static void
mprts_find_block_keys(struct cuda_params prm, float4 *d_xi4,
		      unsigned int *d_off,
		      unsigned int *d_bidx, int nr_total_blocks)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];
  int p = bid / nr_blocks;

  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_xi4[n];
    unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = CUDA_BND_S_OOB;
    } else {
      int bidx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
      int b_diff = bid - bidx + prm.b_mx[1] + 1;
      int d1 = b_diff % prm.b_mx[1];
      int d2 = b_diff / prm.b_mx[1];
      block_idx = d2 * 3 + d1;
    }
    d_bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_keys(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (mprts->nr_patches == 0) {
    return;
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, cmprts, NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { cmprts->n_blocks, 1 };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_keys, (prm, cmprts->d_xi4, cmprts->d_off,
				     cmprts->d_bidx, cmprts->n_blocks));
  free_params(&prm);
}

// ======================================================================
// cuda_mprts_find_block_indices_3

EXTERN_C void
cuda_mprts_find_block_indices_3(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  unsigned int nr_recv = cmprts->bnd.n_prts_recv;
  unsigned int nr_prts_prev = cmprts->n_prts - nr_recv;

  // for consistency, use same block indices that we counted earlier
  // OPT unneeded?
  check(hipMemcpy(cmprts->d_bidx + nr_prts_prev, cmprts->bnd.h_bnd_idx,
		   nr_recv * sizeof(*cmprts->d_bidx),
		   hipMemcpyHostToDevice));
  // slight abuse of the now unused last part of spine_cnts
  check(hipMemcpy(cmprts->bnd.d_bnd_spine_cnts + 10 * cmprts->n_blocks,
		   cmprts->bnd.h_bnd_cnt,
		   cmprts->n_blocks * sizeof(*cmprts->bnd.d_bnd_spine_cnts),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(cmprts->bnd.d_alt_bidx + nr_prts_prev, cmprts->bnd.h_bnd_off,
		   nr_recv * sizeof(*cmprts->bnd.d_alt_bidx),
		   hipMemcpyHostToDevice));

  free(cmprts->bnd.h_bnd_idx);
  free(cmprts->bnd.h_bnd_off);
}

// ======================================================================
// mprts_reorder_send_buf_total

__global__ static void
mprts_reorder_send_buf_total(int nr_prts, int nr_total_blocks,
			     unsigned int *d_bidx, unsigned int *d_sums,
			     float4 *d_xi4, float4 *d_pxi4,
			     float4 *d_xchg_xi4, float4 *d_xchg_pxi4)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i >= nr_prts)
    return;

  if (d_bidx[i] == CUDA_BND_S_OOB) {
    int j = d_sums[i];
    d_xchg_xi4[j]  = d_xi4[i];
    d_xchg_pxi4[j] = d_pxi4[i];
  }
}

EXTERN_C void
cuda_mprts_reorder_send_buf_total(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (mprts->nr_patches == 0)
    return;

  float4 *xchg_xi4 = cmprts->d_xi4 + cmprts->n_prts;
  float4 *xchg_pxi4 = cmprts->d_pxi4 + cmprts->n_prts;
  assert(cmprts->n_prts + cmprts->bnd.n_prts_send < cmprts->n_alloced);
  
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (cmprts->n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_reorder_send_buf_total, (cmprts->n_prts, cmprts->n_blocks,
					    cmprts->d_bidx, cmprts->bnd.d_sums,
					    cmprts->d_xi4, cmprts->d_pxi4,
					    xchg_xi4, xchg_pxi4));
}

// ======================================================================
// cuda_mprts_copy_from_dev

void
cuda_mprts_copy_from_dev(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (mprts->nr_patches == 0) {
    return;
  }

  cmprts->bnd.h_bnd_xi4 = new float4[cmprts->bnd.n_prts_send];
  cmprts->bnd.h_bnd_pxi4 = new float4[cmprts->bnd.n_prts_send];

  assert(cmprts->n_prts + cmprts->bnd.n_prts_send < cmprts->n_alloced);

  check(hipMemcpy(cmprts->bnd.h_bnd_xi4, cmprts->d_xi4 + cmprts->n_prts,
		   cmprts->bnd.n_prts_send * sizeof(float4), hipMemcpyDeviceToHost));
  check(hipMemcpy(cmprts->bnd.h_bnd_pxi4, cmprts->d_pxi4 + cmprts->n_prts,
		   cmprts->bnd.n_prts_send * sizeof(float4), hipMemcpyDeviceToHost));
}

//======================================================================
// cuda_mprts_convert_from_cuda

void
cuda_mprts_convert_from_cuda(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (mprts->nr_patches == 0) {
    return;
  }

  float4 *bnd_xi4 = cmprts->bnd.h_bnd_xi4;
  float4 *bnd_pxi4 = cmprts->bnd.h_bnd_pxi4;
  for (int p = 0; p < mprts->nr_patches; p++) {
    cmprts->bnd.bpatch[p].prts = new particle_t[cmprts->bnd.bpatch[p].n_send];
    for (int n = 0; n < cmprts->bnd.bpatch[p].n_send; n++) {
      particle_t *prt = &cmprts->bnd.bpatch[p].prts[n];
      prt->xi      = bnd_xi4[n].x;
      prt->yi      = bnd_xi4[n].y;
      prt->zi      = bnd_xi4[n].z;
      prt->kind    = cuda_float_as_int(bnd_xi4[n].w);
      prt->pxi     = bnd_pxi4[n].x;
      prt->pyi     = bnd_pxi4[n].y;
      prt->pzi     = bnd_pxi4[n].z;
      prt->qni_wni = bnd_pxi4[n].w;
    }
    bnd_xi4 += cmprts->bnd.bpatch[p].n_send;
    bnd_pxi4 += cmprts->bnd.bpatch[p].n_send;
  }
  delete[] cmprts->bnd.h_bnd_xi4;
  delete[] cmprts->bnd.h_bnd_pxi4;
}

// ======================================================================
// cuda_mprts_copy_to_dev

void
cuda_mprts_copy_to_dev(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  float4 *d_xi4 = cmprts->d_xi4;
  float4 *d_pxi4 = cmprts->d_pxi4;

  unsigned int nr_recv = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    nr_recv += cmprts->bnd.bpatch[p].n_recv;
  }
  assert(cmprts->n_prts + nr_recv <= cmprts->n_alloced);

  check(hipMemcpy(d_xi4 + cmprts->n_prts, cmprts->bnd.h_bnd_xi4,
		   nr_recv * sizeof(*d_xi4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_pxi4 + cmprts->n_prts, cmprts->bnd.h_bnd_pxi4,
		   nr_recv * sizeof(*d_pxi4),
		   hipMemcpyHostToDevice));

  free(cmprts->bnd.h_bnd_xi4);
  free(cmprts->bnd.h_bnd_pxi4);

  cmprts->bnd.n_prts_recv = nr_recv;
  cmprts->n_prts += nr_recv;
}

// ======================================================================
// cuda_mprts_sort

void
cuda_mprts_sort(struct psc_mparticles *mprts, int *n_prts_by_patch)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  cuda_mprts_sort_pairs_device(mprts);

  for (int p = 0; p < mprts->nr_patches; p++) {
    n_prts_by_patch[p] += cmprts->bnd.bpatch[p].n_recv - cmprts->bnd.bpatch[p].n_send;
  }
  cmprts->n_prts -= cmprts->bnd.n_prts_send;
}

