#include "hip/hip_runtime.h"

#include <psc_cuda.h>
#include "cuda_sort2.h"
#include "particles_cuda.h"

#include <thrust/scan.h>
#include <thrust/device_vector.h>

#define PFX(x) xchg_##x
#include "constants.c"

// FIXME const mem for dims?
// FIXME probably should do our own loop rather than use blockIdx

__global__ static void
exchange_particles(int n_part, particles_cuda_dev_t d_part,
		   int ldimsx, int ldimsy, int ldimsz)
{
  int ldims[3] = { ldimsx, ldimsy, ldimsz };
  int xm[3];

  for (int d = 0; d < 3; d++) {
    xm[d] = ldims[d] / d_consts.dxi[d];
  }

  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      d_part.xi4[i].x * d_consts.dxi[0],
      d_part.xi4[i].y * d_consts.dxi[1],
      d_part.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = cuda_fint(xi[d]);
    }
    if (pos[1] < 0) {
      d_part.xi4[i].y += xm[1];
      if (d_part.xi4[i].y >= xm[1])
	d_part.xi4[i].y = 0.f;
    }
    if (pos[2] < 0) {
      d_part.xi4[i].z += xm[2];
      if (d_part.xi4[i].z >= xm[2])
	d_part.xi4[i].z = 0.f;
    }
    if (pos[1] >= ldims[1]) {
      d_part.xi4[i].y -= xm[1];
    }
    if (pos[2] >= ldims[2]) {
      d_part.xi4[i].z -= xm[2];
    }
  }
}

EXTERN_C void
cuda_exchange_particles(int p, struct psc_particles *prts)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_patch *patch = &ppsc->patch[p];

  xchg_set_constants(prts, NULL);

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     exchange_particles, (prts->n_part, cuda->d_part,
				  patch->ldims[0], patch->ldims[1], patch->ldims[2]));
}

EXTERN_C void
cuda_alloc_block_indices(struct psc_particles *prts, unsigned int **d_bidx)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  check(hipMalloc((void **) d_bidx, cuda->n_alloced * sizeof(**d_bidx)));
}

EXTERN_C void
cuda_free_block_indices(unsigned int *d_bidx)
{
  check(hipFree(d_bidx));
}

EXTERN_C void
cuda_copy_bidx_from_dev(struct psc_particles *prts, unsigned int *h_bidx, unsigned int *d_bidx)
{
  check(hipMemcpy(h_bidx, d_bidx, prts->n_part * sizeof(*h_bidx),
		   hipMemcpyDeviceToHost));
}

EXTERN_C void
cuda_copy_bidx_to_dev(struct psc_particles *prts, unsigned int *d_bidx, unsigned int *h_bidx)
{
  check(hipMemcpy(d_bidx, h_bidx, prts->n_part * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
}

// ======================================================================
// cuda_find_block_indices

__global__ static void
find_block_indices(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		   int dimy, float b_dyi, float b_dzi, int b_my, int b_mz)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    float4 xi4 = d_part.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx = block_pos_z * b_my + block_pos_y;
    d_bidx[i] = block_idx;
  }
}

EXTERN_C void
cuda_find_block_indices(struct psc_particles *prts, unsigned int *d_bidx)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices, (prts->n_part, cuda->d_part, d_bidx,
				  cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				  cuda->b_mx[1], cuda->b_mx[2]));
}

// ======================================================================
// cuda_find_block_indices_ids

__global__ static void
find_block_indices_ids(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		       unsigned int *d_ids, int dimy, float b_dyi, float b_dzi, int b_my, int b_mz)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    float4 xi4 = d_part.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx = block_pos_z * b_my + block_pos_y;
    d_bidx[i] = block_idx;
    d_ids[i] = i;
  }
}

EXTERN_C void
cuda_find_block_indices_ids(struct psc_particles *prts, unsigned int *d_bidx,
			    unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices_ids, (prts->n_part, cuda->d_part, d_bidx, d_ids,
				      cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				      cuda->b_mx[1], cuda->b_mx[2]));
}

// ======================================================================
// cuda_find_block_indices_2
//
// like cuda_find_block_indices, but handles out-of-bound
// particles

__global__ static void
find_block_indices_2(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		     int dimy, float b_dyi, float b_dzi,
		     int b_my, int b_mz, int start)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x + start;
  if (i < n_part) {
    float4 xi4 = d_part.xi4[i];
    unsigned int block_pos_y = cuda_fint(xi4.y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4.z * b_dzi);

    int block_idx;
    if (block_pos_y >= b_my || block_pos_z >= b_mz) {
      block_idx = b_my * b_mz;
    } else {
      block_idx = block_pos_z * b_my + block_pos_y;
    }
    d_bidx[i] = block_idx;
  }
}

EXTERN_C void
cuda_find_block_indices_2(struct psc_particles *prts, unsigned int *d_bidx,
			  int start)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { ((prts->n_part - start) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_block_indices_2, (prts->n_part, cuda->d_part, d_bidx,
				    cuda->map.dims[1], cuda->b_dxi[1], cuda->b_dxi[2],
				    cuda->b_mx[1], cuda->b_mx[2], start));
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_2

__global__ static void
mprts_find_block_indices_2(struct cuda_params prm, struct cuda_patch_prts *d_cp_prts)
{
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % prm.b_mx[2];
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);
  int p = blockIdx.y / prm.b_mx[2];

  int block_begin = d_cp_prts[p].d_part.offsets[bid];
  int block_end   = d_cp_prts[p].d_part.offsets[bid+1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_cp_prts[p].d_part.xi4[n];
    unsigned int block_pos_y = cuda_fint(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = cuda_fint(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = prm.b_mx[1] * prm.b_mx[2];
    } else {
      block_idx = block_pos_z * prm.b_mx[1] + block_pos_y;
    }
    d_cp_prts[p].d_part.bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_indices_2(struct cuda_mprts *cuda_mprts)
{
  if (cuda_mprts->nr_patches > 0) {
    struct cuda_params prm;
    set_params(&prm, ppsc, cuda_mprts->mprts_cuda[0], NULL);
    
    int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
    int dimGrid[2]  = { prm.b_mx[1], prm.b_mx[2] * cuda_mprts->nr_patches };

    RUN_KERNEL(dimGrid, dimBlock,
	       mprts_find_block_indices_2, (prm, cuda_mprts->d_cp_prts));
    free_params(&prm);
  }
}

// ----------------------------------------------------------------------

EXTERN_C void
_cuda_find_block_indices_2(struct psc_particles *prts, unsigned int *d_bidx,
			   int start)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4 = new float4[prts->n_part];
  float4 *pxi4 = new float4[prts->n_part];
  unsigned int *bidx = new unsigned int[prts->n_part];
  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);

  float b_dyi = cuda->b_dxi[1], b_dzi = cuda->b_dxi[2];
  int b_my = cuda->b_mx[1], b_mz = cuda->b_mx[2];
  for (int i = start; i < prts->n_part; i++) {
    unsigned int block_pos_y = cuda_fint(xi4[i].y * b_dyi);
    unsigned int block_pos_z = cuda_fint(xi4[i].z * b_dzi);

    int block_idx;
    if (block_pos_y >= b_my || block_pos_z >= b_mz) {
      block_idx = cuda->nr_blocks;
    } else {
      block_idx = block_pos_z * b_my + block_pos_y;
    }
    bidx[i] = block_idx;
  }
  
  cuda_copy_bidx_to_dev(prts, d_bidx, bidx);
  delete[] xi4;
  delete[] pxi4;
  delete[] bidx;
}

// ======================================================================
// cuda_find_block_indices_3

EXTERN_C void
cuda_find_block_indices_3(struct psc_particles *prts, unsigned int *d_bidx,
			  unsigned int *d_alt_bidx,
			  int start, unsigned int *bn_idx, unsigned int *bn_off)
{
  // for consistency, use same block indices that we counted earlier
  check(hipMemcpy(d_bidx + start, bn_idx, (prts->n_part - start) * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
  // abuse of alt_bidx!!! FIXME
  check(hipMemcpy(d_alt_bidx + start, bn_off, (prts->n_part - start) * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
}

// ======================================================================
// reorder_send_buf

__global__ static void
reorder_send_buf(int n_part, particles_cuda_dev_t d_part, unsigned int *d_bidx,
		 unsigned int *d_sums, unsigned int nr_blocks)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    if (d_bidx[i] == nr_blocks) {
      int j = d_sums[i] + n_part;
      d_part.xi4[j] = d_part.xi4[i];
      d_part.pxi4[j] = d_part.pxi4[i];
    }
  }
}

EXTERN_C void
cuda_reorder_send_buf(int p, struct psc_particles *prts, 
		      unsigned int *d_bidx, unsigned int *d_sums, int n_send)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  assert(prts->n_part + n_send <= cuda->n_alloced);

  // OPT: don't pass offset, get it in device code
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder_send_buf, (prts->n_part, cuda->d_part, d_bidx, d_sums, cuda->nr_blocks));
}

__global__ static void
mprts_reorder_send_buf(struct cuda_params prm, struct cuda_patch_prts *d_cp_prts, int nr_patches)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  for (int p = 0; p < nr_patches; p++) {
#if 0
    __shared__ struct cuda_patch_prts cp_prts;

    __syncthreads();
    if (threadIdx.x < sizeof(cp_prts) / sizeof(int)) {
      ((int *) &cp_prts)[threadIdx.x] = ((int *) &d_cp_prts[p])[threadIdx.x];
    }
    __syncthreads();
#else
    struct cuda_patch_prts cp_prts = d_cp_prts[p];
#endif
    
    if (i < cp_prts.n_part) {
      if (cp_prts.d_part.bidx[i] == nr_blocks) {
	int j = cp_prts.d_part.sums[i] + cp_prts.n_part;
	cp_prts.d_part.xi4[j]  = cp_prts.d_part.xi4[i];
	cp_prts.d_part.pxi4[j] = cp_prts.d_part.pxi4[i];
      }
    }
  }
}

EXTERN_C void
cuda_mprts_reorder_send_buf(struct cuda_mprts *cuda_mprts)
{
  if (cuda_mprts->nr_patches > 0) {
    struct cuda_params prm;
    set_params(&prm, ppsc, cuda_mprts->mprts_cuda[0], NULL);

    int max_n_part = 0;
    for (int p = 0; p < cuda_mprts->nr_patches; p++) {
      struct psc_particles *prts = cuda_mprts->mprts_cuda[p];
      struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
      assert(prts->n_part + cuda->bnd_n_send <= cuda->n_alloced);
      if (prts->n_part > max_n_part) {
	max_n_part = prts->n_part;
      }
    }

    int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
    int dimGrid[2]  = { (max_n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };

    RUN_KERNEL(dimGrid, dimBlock,
		 mprts_reorder_send_buf, (prm, cuda_mprts->d_cp_prts, cuda_mprts->nr_patches));

    free_params(&prm);
  }
}

EXTERN_C void
_cuda_reorder_send_buf(int p, struct psc_particles *prts, 
		       unsigned int *d_bidx, unsigned int *d_sums, int n_send)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int n_part = prts->n_part;
  int n_total = n_part + n_send;
  assert(n_total <= cuda->n_alloced);
  float4 *xi4 = new float4[n_total];
  float4 *pxi4 = new float4[n_total];
  unsigned int *bidx = new unsigned int[n_total];
  unsigned int *sums = new unsigned int[n_total];
  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);
  cuda_copy_bidx_from_dev(prts, sums, d_sums);

  for (int i = 0; i < prts->n_part; i++) {
    if (bidx[i] == cuda->nr_blocks) {
      int j = sums[i] + prts->n_part;
      xi4[j] = xi4[i];
      pxi4[j] = pxi4[i];
    }
  }

  prts->n_part = n_total;
  __particles_cuda_to_device(prts, xi4, pxi4, NULL, NULL);
  prts->n_part = n_part;
  delete[] xi4;
  delete[] pxi4;
  delete[] bidx;
  delete[] sums;
}

// ======================================================================
// reorder_and_offsets

__global__ static void
reorder_and_offsets(int n_part, particles_cuda_dev_t d_part, float4 *xi4, float4 *pxi4,
		    unsigned int *d_bidx, unsigned int *d_ids, int nr_blocks)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > n_part)
    return;

  int block, prev_block;
  if (i < n_part) {
    xi4[i] = d_part.xi4[d_ids[i]];
    pxi4[i] = d_part.pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else if (i == n_part) { // needed if there is no particle in the last block
    block = nr_blocks;
  }

  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_part.offsets[b] = i;
  }
}

EXTERN_C void
cuda_reorder_and_offsets(struct psc_particles *prts, unsigned int *d_bidx,
			 unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *alt_xi4 = cuda->d_part.alt_xi4;
  float4 *alt_pxi4 = cuda->d_part.alt_pxi4;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder_and_offsets, (prts->n_part, cuda->d_part, alt_xi4, alt_pxi4,
				   d_bidx, d_ids, cuda->nr_blocks));

  cuda->d_part.alt_xi4 = cuda->d_part.xi4;
  cuda->d_part.alt_pxi4 = cuda->d_part.pxi4;
  cuda->d_part.xi4 = alt_xi4;
  cuda->d_part.pxi4 = alt_pxi4;
}

void
_cuda_reorder_and_offsets(struct psc_particles *prts, unsigned int *d_bidx,
			  unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4 = new float4[prts->n_part];
  float4 *pxi4 = new float4[prts->n_part];
  float4 *alt_xi4 = new float4[prts->n_part];
  float4 *alt_pxi4 = new float4[prts->n_part];
  unsigned int *bidx = new unsigned int[prts->n_part];
  unsigned int *ids = new unsigned int[prts->n_part];
  int *offsets = new int[cuda->nr_blocks + 2];

  __particles_cuda_from_device(prts, xi4, pxi4);
  cuda_copy_bidx_from_dev(prts, bidx, d_bidx);
  cuda_copy_bidx_from_dev(prts, ids, d_ids);

  for (int i = 0; i < prts->n_part; i++) {
    alt_xi4[i] = xi4[ids[i]];
    alt_pxi4[i] = pxi4[ids[i]];

    int block = bidx[i];
    int prev_block = (i > 0) ? (int) bidx[i-1] : -1;
    for (int b = prev_block + 1; b <= block; b++) {
      offsets[b] = i;
    }
  }
  int block = cuda->nr_blocks + 1;
  int prev_block = bidx[prts->n_part - 1];
  for (int b = prev_block + 1; b <= block; b++) {
    offsets[b] = prts->n_part;
  }

  float4 *d_alt_xi4 = cuda->d_part.alt_xi4;
  float4 *d_alt_pxi4 = cuda->d_part.alt_pxi4;
  cuda->d_part.alt_xi4 = cuda->d_part.xi4;
  cuda->d_part.alt_pxi4 = cuda->d_part.pxi4;
  cuda->d_part.xi4 = d_alt_xi4;
  cuda->d_part.pxi4 = d_alt_pxi4;

  __particles_cuda_to_device(prts, alt_xi4, alt_pxi4, offsets, NULL);
  delete[] xi4;
  delete[] pxi4;
  delete[] alt_xi4;
  delete[] alt_pxi4;
  delete[] bidx;
  delete[] ids;
  delete[] offsets;
}

// ======================================================================
// cuda_reorder

__global__ static void
reorder(int n_part, particles_cuda_dev_t d_part, float4 *xi4, float4 *pxi4,
	unsigned int *d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < n_part) {
    xi4[i] = d_part.xi4[d_ids[i]];
    pxi4[i] = d_part.pxi4[d_ids[i]];
  }
}

EXTERN_C void
cuda_reorder(struct psc_particles *prts, unsigned int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *alt_xi4 = cuda->d_part.alt_xi4;
  float4 *alt_pxi4 = cuda->d_part.alt_pxi4;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     reorder, (prts->n_part, cuda->d_part, alt_xi4, alt_pxi4, d_ids));

  cuda->d_part.alt_xi4 = cuda->d_part.xi4;
  cuda->d_part.alt_pxi4 = cuda->d_part.pxi4;
  cuda->d_part.xi4 = alt_xi4;
  cuda->d_part.pxi4 = alt_pxi4;
}

// ======================================================================
// cuda_exclusive_scan

EXTERN_C int
_cuda_exclusive_scan(int p, struct psc_particles *prts,
		    unsigned int *d_vals, unsigned int *d_sums)
{
  unsigned int *vals = new unsigned int[prts->n_part];
  unsigned int *sums = new unsigned int[prts->n_part];
  cuda_copy_bidx_from_dev(prts, vals, d_vals);

  unsigned int sum = 0;
  for (int i = 0; i < prts->n_part; i++) {
    sums[i] = sum;
    sum += vals[i];
  }

  cuda_copy_bidx_to_dev(prts, d_sums, sums);
  delete[] sums;
  delete[] vals;
  return sum;
}

EXTERN_C int
cuda_exclusive_scan(int p, struct psc_particles *prts, unsigned int *_d_vals, unsigned int *_d_sums)
{
  thrust::device_ptr<unsigned int> d_vals(_d_vals);
  thrust::device_ptr<unsigned int> d_sums(_d_sums);
  thrust::exclusive_scan(d_vals, d_vals + prts->n_part, d_sums);
  int sum = d_sums[prts->n_part - 1] + d_vals[prts->n_part - 1];
  return sum;
}

// ======================================================================
// cuda_mprts_copy_from_dev

void
cuda_mprts_copy_from_dev(struct cuda_mprts *cuda_mprts)
{
  hipStream_t stream[cuda_mprts->nr_patches];
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    struct psc_particles *prts = cuda_mprts->mprts_cuda[p];
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    cuda->bnd_n_part = 0;
    cuda->bnd_prts = NULL;
    
    int n_send = cuda->bnd_n_send;
    cuda->bnd_xi4  = new float4[n_send];
    cuda->bnd_pxi4 = new float4[n_send];
    cuda->bnd_idx  = new unsigned int[n_send];
    cuda->bnd_off  = new unsigned int[n_send];

    hipStreamCreate(&stream[p]);
  }    
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    struct psc_particles *prts = cuda_mprts->mprts_cuda[p];
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    int n_send = cuda->bnd_n_send;
    check(hipMemcpyAsync(cuda->bnd_xi4, cuda->d_part.xi4 + cuda->bnd_n_part_save,
			  n_send * sizeof(*cuda->bnd_xi4), hipMemcpyDeviceToHost, stream[p]));
    check(hipMemcpyAsync(cuda->bnd_pxi4, cuda->d_part.pxi4 + cuda->bnd_n_part_save,
			  n_send * sizeof(*cuda->bnd_pxi4), hipMemcpyDeviceToHost, stream[p]));
  }
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    hipStreamSynchronize(stream[p]);
    hipStreamDestroy(stream[p]);
  }
}

// ======================================================================
// cuda_mprts_copy_to_dev

void
cuda_mprts_copy_to_dev(struct cuda_mprts *cuda_mprts)
{
  hipStream_t stream[cuda_mprts->nr_patches];
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    hipStreamCreate(&stream[p]);
  }
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    struct psc_particles *prts = cuda_mprts->mprts_cuda[p];
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    check(hipMemcpy(cuda->d_part.xi4 + cuda->bnd_n_part_save, cuda->bnd_xi4,
		     (prts->n_part - cuda->bnd_n_part_save) * sizeof(*cuda->bnd_xi4),
		     hipMemcpyHostToDevice));
    check(hipMemcpy(cuda->d_part.pxi4 + cuda->bnd_n_part_save, cuda->bnd_pxi4,
		     (prts->n_part - cuda->bnd_n_part_save) * sizeof(*cuda->bnd_pxi4),
		     hipMemcpyHostToDevice));
  }

  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    struct psc_particles *prts = cuda_mprts->mprts_cuda[p];
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    hipStreamSynchronize(stream[p]);
    hipStreamDestroy(stream[p]);
    free(cuda->bnd_prts);
    free(cuda->bnd_xi4);
    free(cuda->bnd_pxi4);
  }
}

