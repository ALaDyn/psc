#include "hip/hip_runtime.h"

#undef _GLIBCXX_USE_INT128

#include "cuda_mparticles.h"

#include "cuda_sort2.h"
#include "particles_cuda.h"
#include "psc_bnd_cuda.h"

#define PFX(x) xchg_##x
#include "constants.c"

#if 0
// FIXME const mem for dims?
// FIXME probably should do our own loop rather than use blockIdx

__global__ static void
exchange_particles(int n_part, particles_cuda_dev_t h_dev,
		   int ldimsx, int ldimsy, int ldimsz)
{
  int ldims[3] = { ldimsx, ldimsy, ldimsz };
  int xm[3];

  for (int d = 0; d < 3; d++) {
    xm[d] = ldims[d] / d_consts.dxi[d];
  }

  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      h_dev.xi4[i].x * d_consts.dxi[0],
      h_dev.xi4[i].y * d_consts.dxi[1],
      h_dev.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = __float2int_rd(xi[d]);
    }
    if (pos[1] < 0) {
      h_dev.xi4[i].y += xm[1];
      if (h_dev.xi4[i].y >= xm[1])
	h_dev.xi4[i].y = 0.f;
    }
    if (pos[2] < 0) {
      h_dev.xi4[i].z += xm[2];
      if (h_dev.xi4[i].z >= xm[2])
	h_dev.xi4[i].z = 0.f;
    }
    if (pos[1] >= ldims[1]) {
      h_dev.xi4[i].y -= xm[1];
    }
    if (pos[2] >= ldims[2]) {
      h_dev.xi4[i].z -= xm[2];
    }
  }
}

EXTERN_C void
cuda_exchange_particles(int p, struct psc_particles *prts)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_patch *patch = &ppsc->patch[p];

  xchg_set_constants(prts, NULL);

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     exchange_particles, (prts->n_part, *cuda->h_dev,
				  patch->ldims[0], patch->ldims[1], patch->ldims[2]));
}
#endif

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_2_total
//
// like cuda_find_block_indices, but handles out-of-bound
// particles

__global__ static void
mprts_find_block_indices_2_total(struct cuda_params prm, float4 *d_xi4,
				 unsigned int *d_off,
				 unsigned int *d_bidx, int nr_patches)
{
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % prm.b_mx[2];
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);
  int p = blockIdx.y / prm.b_mx[2];

  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  // FIXME/OPT, could be done better like reorder_send_buf
  int block_begin = d_off[bid + p * nr_blocks];
  int block_end   = d_off[bid + p * nr_blocks + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_xi4[n];
    unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = nr_blocks * nr_patches;
    } else {
      block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
    }
    d_bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_indices_2_total(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_indices_2_total, (prm, cmprts->d_xi4, mprts_cuda->d_off,
						cmprts->d_bidx, mprts->nr_patches));
  free_params(&prm);
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_keys

__global__ static void
mprts_find_block_keys(struct cuda_params prm, float4 *d_xi4,
		      unsigned int *d_off,
		      unsigned int *d_bidx, int nr_total_blocks)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];
  int p = bid / nr_blocks;

  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    float4 xi4 = d_xi4[n];
    unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
    unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);

    int block_idx;
    if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
      block_idx = CUDA_BND_S_OOB;
    } else {
      int bidx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
      int b_diff = bid - bidx + prm.b_mx[1] + 1;
      int d1 = b_diff % prm.b_mx[1];
      int d2 = b_diff / prm.b_mx[1];
      block_idx = d2 * 3 + d1;
    }
    d_bidx[n] = block_idx;
  }
}

EXTERN_C void
cuda_mprts_find_block_keys(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, NULL);
    
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { mprts_cuda->nr_total_blocks, 1 };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_find_block_keys, (prm, cmprts->d_xi4, mprts_cuda->d_off,
				     cmprts->d_bidx, mprts_cuda->nr_total_blocks));
  free_params(&prm);
}

// ======================================================================
// cuda_mprts_find_block_indices_3

EXTERN_C void
cuda_mprts_find_block_indices_3(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  unsigned int nr_recv = mprts_cuda->nr_prts_recv;
  unsigned int nr_prts_prev = cmprts->n_prts - nr_recv;

  // for consistency, use same block indices that we counted earlier
  // OPT unneeded?
  check(hipMemcpy(cmprts->d_bidx + nr_prts_prev, mprts_cuda->h_bnd_idx,
		   nr_recv * sizeof(*cmprts->d_bidx),
		   hipMemcpyHostToDevice));
  // slight abuse of the now unused last part of spine_cnts
  check(hipMemcpy(mprts_cuda->d_bnd_spine_cnts + 10 * mprts_cuda->nr_total_blocks,
		   mprts_cuda->h_bnd_cnt,
		   mprts_cuda->nr_total_blocks * sizeof(*mprts_cuda->d_bnd_spine_cnts),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(mprts_cuda->d_alt_bidx + nr_prts_prev, mprts_cuda->h_bnd_off,
		   nr_recv * sizeof(*mprts_cuda->d_alt_bidx),
		   hipMemcpyHostToDevice));

  free(mprts_cuda->h_bnd_idx);
  free(mprts_cuda->h_bnd_off);
}

// ======================================================================
// mprts_reorder_send_buf_total

__global__ static void
mprts_reorder_send_buf_total(int nr_prts, int nr_total_blocks,
			     unsigned int *d_bidx, unsigned int *d_sums,
			     float4 *d_xi4, float4 *d_pxi4,
			     float4 *d_xchg_xi4, float4 *d_xchg_pxi4)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i >= nr_prts)
    return;

  if (d_bidx[i] == CUDA_BND_S_OOB) {
    int j = d_sums[i];
    d_xchg_xi4[j]  = d_xi4[i];
    d_xchg_pxi4[j] = d_pxi4[i];
  }
}

EXTERN_C void
cuda_mprts_reorder_send_buf_total(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);

  if (mprts->nr_patches == 0)
    return;

  float4 *xchg_xi4 = cmprts->d_xi4 + cmprts->n_prts;
  float4 *xchg_pxi4 = cmprts->d_pxi4 + cmprts->n_prts;
  assert(cmprts->n_prts + mprts_cuda->nr_prts_send < cmprts->n_alloced);
  
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (cmprts->n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_reorder_send_buf_total, (cmprts->n_prts, mprts_cuda->nr_total_blocks,
					    cmprts->d_bidx, mprts_cuda->d_sums,
					    cmprts->d_xi4, cmprts->d_pxi4,
					    xchg_xi4, xchg_pxi4));
}

// ======================================================================
// cuda_mprts_reorder

__global__ static void
mprts_reorder(int nr_prts, unsigned int *d_ids,
	      float4 *xi4, float4 *pxi4,
	      float4 *alt_xi4, float4 *alt_pxi4)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < nr_prts) {
    int j = d_ids[i];
    alt_xi4[i] = xi4[j];
    alt_pxi4[i] = pxi4[j];
  }
}

void
cuda_mprts_reorder(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (cmprts->n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_reorder, (cmprts->n_prts, cmprts->d_id,
			     cmprts->d_xi4, cmprts->d_pxi4,
			     cmprts->d_alt_xi4, cmprts->d_alt_pxi4));
  
  cuda_mparticles_swap_alt(cmprts);
}

// ======================================================================
// reorder_and_offsets

__global__ static void
mprts_reorder_and_offsets(int nr_prts, float4 *xi4, float4 *pxi4, float4 *alt_xi4, float4 *alt_pxi4,
			  unsigned int *d_bidx, unsigned int *d_ids, unsigned int *d_off, int last_block)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > nr_prts)
    return;

  int block, prev_block;
  if (i < nr_prts) {
    alt_xi4[i] = xi4[d_ids[i]];
    alt_pxi4[i] = pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else { // needed if there is no particle in the last block
    block = last_block;
  }

  // OPT: d_bidx[i-1] could use shmem
  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_off[b] = i;
  }
}

void
cuda_mprts_reorder_and_offsets(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);

  if (mprts->nr_patches == 0) {
    return;
  }
  int nr_blocks = psc_particles_cuda(psc_mparticles_get_patch(mprts, 0))->nr_blocks;

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (cmprts->n_prts + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     mprts_reorder_and_offsets, (cmprts->n_prts, cmprts->d_xi4, cmprts->d_pxi4,
					 cmprts->d_alt_xi4, cmprts->d_alt_pxi4,
					 cmprts->d_bidx, cmprts->d_id,
					 mprts_cuda->d_off, mprts->nr_patches * nr_blocks));

  cuda_mparticles_swap_alt(cmprts);
  psc_mparticles_cuda_copy_to_dev(mprts);
}

// ======================================================================
// cuda_mprts_copy_from_dev

void
cuda_mprts_copy_from_dev(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  mprts_cuda->h_bnd_xi4 = new float4[mprts_cuda->nr_prts_send];
  mprts_cuda->h_bnd_pxi4 = new float4[mprts_cuda->nr_prts_send];

  check(hipMemcpy(mprts_cuda->h_bnd_xi4, cmprts->d_xi4 + cmprts->n_prts,
		   mprts_cuda->nr_prts_send * sizeof(float4), hipMemcpyDeviceToHost));
  check(hipMemcpy(mprts_cuda->h_bnd_pxi4, cmprts->d_pxi4 + cmprts->n_prts,
		   mprts_cuda->nr_prts_send * sizeof(float4), hipMemcpyDeviceToHost));
}

//======================================================================
// cuda_mprts_convert_from_cuda

void
cuda_mprts_convert_from_cuda(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }

  float4 *bnd_xi4 = mprts_cuda->h_bnd_xi4;
  float4 *bnd_pxi4 = mprts_cuda->h_bnd_pxi4;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    cuda->bnd_prts = new particle_single_t[cuda->bnd_n_send];
    for (int n = 0; n < cuda->bnd_n_send; n++) {
      particle_single_t *prt = &cuda->bnd_prts[n];
      prt->xi      = bnd_xi4[n].x;
      prt->yi      = bnd_xi4[n].y;
      prt->zi      = bnd_xi4[n].z;
      prt->kind    = cuda_float_as_int(bnd_xi4[n].w);
      prt->pxi     = bnd_pxi4[n].x;
      prt->pyi     = bnd_pxi4[n].y;
      prt->pzi     = bnd_pxi4[n].z;
      prt->qni_wni = bnd_pxi4[n].w;
    }
    bnd_xi4 += cuda->bnd_n_send;
    bnd_pxi4 += cuda->bnd_n_send;
  }
  delete[] mprts_cuda->h_bnd_xi4;
  delete[] mprts_cuda->h_bnd_pxi4;
}

// ======================================================================
// cuda_mprts_copy_to_dev

void
cuda_mprts_copy_to_dev(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);

  float4 *d_xi4 = cmprts->d_xi4;
  float4 *d_pxi4 = cmprts->d_pxi4;

  unsigned int nr_recv = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    nr_recv += cuda->bnd_n_recv;
  }
  assert(cmprts->n_prts + nr_recv <= cmprts->n_alloced);

  check(hipMemcpy(d_xi4 + cmprts->n_prts, mprts_cuda->h_bnd_xi4,
		   nr_recv * sizeof(*d_xi4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_pxi4 + cmprts->n_prts, mprts_cuda->h_bnd_pxi4,
		   nr_recv * sizeof(*d_pxi4),
		   hipMemcpyHostToDevice));

  free(mprts_cuda->h_bnd_xi4);
  free(mprts_cuda->h_bnd_pxi4);

  mprts_cuda->nr_prts_recv = nr_recv;
  cmprts->n_prts += nr_recv;
}

// ======================================================================
// cuda_mprts_sort

void
cuda_mprts_sort(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  cuda_mprts_sort_pairs_device(mprts);

  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    prts->n_part += cuda->bnd_n_recv - cuda->bnd_n_send;
    mprts_cuda->h_n_prts[p] = prts->n_part;
  }
  cmprts->n_prts -= mprts_cuda->nr_prts_send;
  psc_mparticles_cuda_copy_to_dev(mprts);
}

// ======================================================================
// cuda_mprts_check_ordered_total

void
cuda_mprts_check_ordered_total(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;

  cuda_mprts_find_block_indices_2_total(mprts);

  unsigned int last = 0;
  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);

    unsigned int *bidx = new unsigned int[prts->n_part];
    cuda_copy_bidx_from_dev(prts, bidx, cmprts->d_bidx + off);
    
    for (int n = 0; n < prts->n_part; n++) {
      if (!(bidx[n] >= last && bidx[n] < mprts->nr_patches * cuda->nr_blocks)) {
	mprintf("p = %d, n = %d bidx = %d last = %d\n", p, n, bidx[n], last);
	assert(0);
      }
      last = bidx[n];
    }

    delete[] bidx;

    off += prts->n_part;
  }
}

