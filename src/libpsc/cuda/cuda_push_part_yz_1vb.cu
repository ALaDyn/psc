#include "hip/hip_runtime.h"

#include "psc_cuda.h"

#define NO_CHECKERBOARD
//#define DEBUG

#define SW (2)

#include "cuda_common.h"

#define MAX_KINDS (4)

struct cuda_params {
  real dt;
  real dxi[3];
  real dqs;
  real fnqs;
  real fnqys, fnqzs;
  int mx[3];
  int ilg[3];
  int b_mx[3];
  int *d_error_count;
  real dq[MAX_KINDS];
};

struct cuda_patch {
  particles_cuda_dev_t d_part;
  real *d_flds;
};

__device__ int *__d_error_count;

static void
set_params(struct cuda_params *prm, struct psc *psc,
	   struct psc_particles *prts, struct psc_fields *pf, int *d_error_count)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  prm->dt = psc->dt;
  for (int d = 0; d < 3; d++) {
    prm->dxi[d] = 1.f / ppsc->dx[d];
    prm->mx[d] = pf->im[d];
    prm->ilg[d] = pf->ib[d];
    prm->b_mx[d] = cuda->b_mx[d];
  }
  prm->dqs    = .5f * psc->coeff.eta * psc->dt;
  prm->fnqs   = sqr(psc->coeff.alpha) * psc->coeff.cori / psc->coeff.eta;
  prm->fnqys  = psc->dx[1] * prm->fnqs / psc->dt;
  prm->fnqzs  = psc->dx[2] * prm->fnqs / psc->dt;
  prm->d_error_count = d_error_count;
  assert(psc->nr_kinds <= MAX_KINDS);
  for (int k = 0; k < psc->nr_kinds; k++) {
    prm->dq[k] = prm->dqs * psc->kinds[k].q / psc->kinds[k].m;
  }
}

// ======================================================================
// field caching

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
class F3cache {
  real *fld_cache;

public:
  __device__ F3cache(real *_fld_cache, real *d_flds, int l[3],
		     struct cuda_params prm) :
    fld_cache(_fld_cache)
  {
    int ti = threadIdx.x;
    int n = BLOCKSIZE_X * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4);
    while (ti < n) {
      int tmp = ti;
      int jx = tmp % BLOCKSIZE_X;
      tmp /= BLOCKSIZE_X;
      int jy = tmp % (BLOCKSIZE_Y + 4) - 2;
      tmp /= BLOCKSIZE_Y + 4;
      int jz = tmp % (BLOCKSIZE_Z + 4) - 2;
      //    tmp /= BLOCKSIZE_Z + 4;
      //    int m = tmp + EX;
      //    printf("n %d ti %d m %d, jx %d,%d,%d\n", n, ti, m, jx, jy, jz);
      // currently it seems faster to do the loop rather than do m by threadidx
      for (int m = EX; m <= HZ; m++) {
	(*this)(m, jx,jy,jz) = F3_DEV_YZ(m, jy+l[1],jz+l[2]);
      }
      ti += blockDim.x;
    }
    __syncthreads();
  }

  __host__ __device__ real operator()(int fldnr, int jx, int jy, int jz) const
  {
    int off = ((((fldnr-EX)
		 *(BLOCKSIZE_Z + 4) + ((jz)-(-2)))
		*(BLOCKSIZE_Y + 4) + ((jy)-(-2)))
	       *1 + ((jx)));
    return fld_cache[off];
  }
  __host__ __device__ real& operator()(int fldnr, int jx, int jy, int jz)
  {
    int off = ((((fldnr-EX)
		 *(BLOCKSIZE_Z + 4) + ((jz)-(-2)))
		*(BLOCKSIZE_Y + 4) + ((jy)-(-2)))
	       *1 + ((jx)));
    return fld_cache[off];
  }
};

// ----------------------------------------------------------------------
// push_xi
//
// advance position using velocity

__device__ static void
push_xi(struct d_particle *p, const real vxi[3], real dt)
{
  int d;
  for (d = 1; d < 3; d++) {
    p->xi[d] += dt * vxi[d];
  }
}

// ----------------------------------------------------------------------
// calc_vxi
//
// calculate velocity from momentum

__device__ static void
calc_vxi(real vxi[3], struct d_particle p)
{
  real root = rsqrtr(real(1.) + sqr(p.pxi[0]) + sqr(p.pxi[1]) + sqr(p.pxi[2]));

  int d;
  for (d = 0; d < 3; d++) {
    vxi[d] = p.pxi[d] * root;
  }
}

// ----------------------------------------------------------------------
// push_pxi_dt
//
// advance moments according to EM fields

__device__ static void
push_pxi_dt(struct d_particle *p,
	    real exq, real eyq, real ezq, real hxq, real hyq, real hzq,
	    struct cuda_params prm)
{
  int kind = __float_as_int(p->kind_as_float);
  real dq = prm.dq[kind];
  real pxm = p->pxi[0] + dq*exq;
  real pym = p->pxi[1] + dq*eyq;
  real pzm = p->pxi[2] + dq*ezq;
  
  real root = dq * rsqrtr(real(1.) + sqr(pxm) + sqr(pym) + sqr(pzm));
  real taux = hxq * root, tauy = hyq * root, tauz = hzq * root;
  
  real tau = real(1.) / (real(1.) + sqr(taux) + sqr(tauy) + sqr(tauz));
  real pxp = ( (real(1.) + sqr(taux) - sqr(tauy) - sqr(tauz)) * pxm
	       +(real(2.)*taux*tauy + real(2.)*tauz)*pym
	       +(real(2.)*taux*tauz - real(2.)*tauy)*pzm)*tau;
  real pyp = ( (real(2.)*taux*tauy - real(2.)*tauz)*pxm
	       +(real(1.) - sqr(taux) + sqr(tauy) - sqr(tauz)) * pym
	       +(real(2.)*tauy*tauz + real(2.)*taux)*pzm)*tau;
  real pzp = ( (real(2.)*taux*tauz + real(2.)*tauy)*pxm
	       +(real(2.)*tauy*tauz - real(2.)*taux)*pym
	       +(real(1.) - sqr(taux) - sqr(tauy) + sqr(tauz))*pzm)*tau;
  
  p->pxi[0] = pxp + dq * exq;
  p->pxi[1] = pyp + dq * eyq;
  p->pxi[2] = pzp + dq * ezq;
}

#define OFF(g, d) o##g[d]
  
__device__ static real
ip1_to_grid_0(real h)
{
  return real(1.) - h;
}

__device__ static real
ip1_to_grid_p(real h)
{
  return h;
}

#define INTERP_FIELD_1ST(cache, exq, fldnr, g1, g2)			\
  do {									\
    int ddy = l##g1[1]-l0[1], ddz = l##g2[2]-l0[2];			\
    /* printf("C %g [%d,%d,%d]\n", F3C(fldnr, 0, ddy, ddz), 0, ddy, ddz); */ \
    exq =								\
      ip1_to_grid_0(OFF(g1, 1)) * ip1_to_grid_0(OFF(g2, 2)) *		\
      cache(fldnr, 0, ddy+0, ddz+0) +					\
      ip1_to_grid_p(OFF(g1, 1)) * ip1_to_grid_0(OFF(g2, 2)) *		\
      cache(fldnr, 0, ddy+1, ddz+0) +					\
      ip1_to_grid_0(OFF(g1, 1)) * ip1_to_grid_p(OFF(g2, 2)) *		\
      cache(fldnr, 0, ddy+0, ddz+1) +					\
      ip1_to_grid_p(OFF(g1, 1)) * ip1_to_grid_p(OFF(g2, 2)) *		\
      cache(fldnr, 0, ddy+1, ddz+1);					\
  } while(0)

// ----------------------------------------------------------------------
// push_part_one
//
// push one particle

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
push_part_one(int n, particles_cuda_dev_t d_particles,
	      const F3cache<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &cached_flds, int l0[3],
	      struct cuda_params prm)
{
  struct d_particle p;
  LOAD_PARTICLE(p, d_particles, n);

  // here we have x^{n+.5}, p^n
  
  // field interpolation

  int lh[3], lg[3];
  real oh[3], og[3];
  find_idx_off_1st(p.xi, lh, oh, real(-.5), prm.dxi);
  find_idx_off_1st(p.xi, lg, og, real(0.), prm.dxi);

  real exq, eyq, ezq, hxq, hyq, hzq;
  INTERP_FIELD_1ST(cached_flds, exq, EX, g, g);
  INTERP_FIELD_1ST(cached_flds, eyq, EY, h, g);
  INTERP_FIELD_1ST(cached_flds, ezq, EZ, g, h);
  INTERP_FIELD_1ST(cached_flds, hxq, HX, h, h);
  INTERP_FIELD_1ST(cached_flds, hyq, HY, g, h);
  INTERP_FIELD_1ST(cached_flds, hzq, HZ, h, g);

  // x^(n+0.5), p^n -> x^(n+0.5), p^(n+1.0) 
  
  push_pxi_dt(&p, exq, eyq, ezq, hxq, hyq, hzq, prm);

  STORE_PARTICLE_MOM(p, d_particles, n);
}

// ----------------------------------------------------------------------
// push_part_p1
//
// push particles

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
push_part_p1(particles_cuda_dev_t d_part, real *d_flds, struct cuda_params prm)
{
  __d_error_count = prm.d_error_count;
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y;

  int ci[3];
  ci[0] = 0;
  ci[1] = block_pos[1] * BLOCKSIZE_Y;
  ci[2] = block_pos[2] * BLOCKSIZE_Z;
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);

  int block_begin = d_part.offsets[bid];
  int block_end   = d_part.offsets[bid+1];

  extern __shared__ real fld_cache[];

  F3cache<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> cached_flds(fld_cache, d_flds, ci, prm);
  
  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    push_part_one(n, d_part, cached_flds, ci, prm);
  }
}

// ----------------------------------------------------------------------
// push_mprts_p1
//
// same as push_part_p1, different calling sequence

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
push_mprts_p1(struct cuda_params prm, struct cuda_patch *d_cpatch)
{
  __d_error_count = prm.d_error_count;
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % prm.b_mx[2];
  int p = blockIdx.y / prm.b_mx[2];

  int ci[3];
  ci[0] = 0;
  ci[1] = block_pos[1] * BLOCKSIZE_Y;
  ci[2] = block_pos[2] * BLOCKSIZE_Z;
  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);

  int block_begin = d_cpatch[p].d_part.offsets[bid];
  int block_end   = d_cpatch[p].d_part.offsets[bid+1];

  extern __shared__ real fld_cache[];

  F3cache<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> cached_flds(fld_cache, d_cpatch[p].d_flds, ci, prm);
  
  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    push_part_one(n, d_cpatch[p].d_part, cached_flds, ci, prm);
  }
}

// ----------------------------------------------------------------------
// cuda_push_part_p2

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_part_p2(struct psc_particles *prts, struct psc_fields *pf)
{
  struct psc_fields_cuda *pfc = psc_fields_cuda(pf);
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int *d_error_count;
  check(hipMalloc(&d_error_count, 1 * sizeof(int)));
  check(hipMemset(d_error_count, 0, 1 * sizeof(int)));

  struct cuda_params prm;
  set_params(&prm, ppsc, prts, pf, d_error_count);

  assert(cuda->nr_blocks == cuda->b_mx[1] * cuda->b_mx[2]);
  unsigned int shared_size = 6 * 1 * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4) * sizeof(real);

  dim3 dimGrid(cuda->b_mx[1], cuda->b_mx[2]);

  push_part_p1<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    <<<dimGrid, THREADS_PER_BLOCK, shared_size>>>
    (cuda->d_part, pfc->d_flds, prm);
  cuda_sync_if_enabled();

  int h_error_count[1];
  check(hipMemcpy(h_error_count, d_error_count, 1 * sizeof(int),
		   hipMemcpyDeviceToHost));
  check(hipFree(d_error_count));
  if (h_error_count[0] != 0) {
    printf("err cnt %d\n", h_error_count[0]);
  }
  assert(h_error_count[0] == 0);
}

// ----------------------------------------------------------------------
// cuda_push_mprts_a

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_a(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_particles **mprts_cuda = new struct psc_particles *[mprts->nr_patches];
  struct psc_fields **mflds_cuda = new struct psc_fields *[mflds->nr_patches];
  int nr_patches = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
    if (psc_particles_ops(prts) == &psc_particles_cuda_ops &&
	psc_fields_ops(flds) == &psc_fields_cuda_ops) {
      mprts_cuda[nr_patches] = prts;
      mflds_cuda[nr_patches] = flds;
      nr_patches++;
    }
  }

  if (nr_patches == 0) {
    delete[] mprts_cuda;
    delete[] mflds_cuda;
    return;
  }

  int *d_error_count;
  check(hipMalloc(&d_error_count, 1 * sizeof(int)));
  check(hipMemset(d_error_count, 0, 1 * sizeof(int)));

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts_cuda[0], mflds_cuda[0], d_error_count);

  struct cuda_patch *cpatch = new struct cuda_patch[nr_patches];
  for (int p = 0; p < nr_patches; p++) {
    struct psc_particles *prts = mprts_cuda[p];
    struct psc_fields *flds = mflds_cuda[p];

    cpatch[p].d_part = psc_particles_cuda(prts)->d_part;
    cpatch[p].d_flds = psc_fields_cuda(flds)->d_flds;
  }

  struct cuda_patch *d_cpatch;
  check(hipMalloc(&d_cpatch, nr_patches * sizeof(*d_cpatch)));
  check(hipMemcpy(d_cpatch, cpatch, nr_patches * sizeof(*d_cpatch),
		   hipMemcpyHostToDevice));
  
  // FIXME, why is this dynamic?
  unsigned int shared_size = 6 * 1 * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4) * sizeof(real);
    
  dim3 dimGrid(prm.b_mx[1], prm.b_mx[2] * nr_patches);
    
  push_mprts_p1<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    <<<dimGrid, THREADS_PER_BLOCK, shared_size>>>
    (prm, d_cpatch);
  cuda_sync_if_enabled();

  check(hipFree(d_cpatch));
  delete[] cpatch;

  delete[] mprts_cuda;
  delete[] mflds_cuda;

  int h_error_count[1];
  check(hipMemcpy(h_error_count, d_error_count, 1 * sizeof(int),
		   hipMemcpyDeviceToHost));
  check(hipFree(d_error_count));
  if (h_error_count[0] != 0) {
    printf("err cnt %d\n", h_error_count[0]);
  }
  assert(h_error_count[0] == 0);
}

// ======================================================================

// FIXME -> common.c

__device__ static void
find_idx_off_pos_1st(const real xi[3], int j[3], real h[3], real pos[3], real shift,
		     struct cuda_params prm)
{
  int d;
  for (d = 0; d < 3; d++) {
    pos[d] = xi[d] * prm.dxi[d] + shift;
    j[d] = cuda_fint(pos[d]);
    h[d] = pos[d] - j[d];
  }
}

__shared__ volatile bool do_read;
__shared__ volatile bool do_write;
__shared__ volatile bool do_reduce;
__shared__ volatile bool do_calc_j;

// OPT: take i < cell_end condition out of load
// OPT: reduce two at a time
// OPT: try splitting current calc / measuring by itself
// OPT: get rid of block_stride

__shared__ int ci0[3]; // cell index of lower-left cell in block

#define WARPS_PER_BLOCK (THREADS_PER_BLOCK / 32)

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
class SCurr {
  real *scurr;

public:
  __device__ SCurr(real *_scurr) :
    scurr(_scurr)
  {
  }

  __device__ void zero()
  {
    const int blockstride = ((((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32);
    int i = threadIdx.x;
    int N = blockstride * WARPS_PER_BLOCK;
    while (i < N) {
      scurr[i] = real(0.);
      i += THREADS_PER_BLOCK;
    }
  }

  __device__ void add_to_fld(real *d_flds, int m, struct cuda_params prm)
  {
    int i = threadIdx.x;
    int stride = (BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW);
    while (i < stride) {
      int rem = i;
      int jz = rem / (BLOCKSIZE_Y + 2*SW);
      rem -= jz * (BLOCKSIZE_Y + 2*SW);
      int jy = rem;
      jz -= SW;
      jy -= SW;
      real val = real(0.);
      // FIXME, opt
      for (int wid = 0; wid < WARPS_PER_BLOCK; wid++) {
	val += (*this)(wid, jy, jz);
      }
      F3_DEV_YZ(JXI+m, jy+ci0[1],jz+ci0[2]) += val;
      i += THREADS_PER_BLOCK;
    }
  }

  __device__ real operator()(int wid, int jy, int jz) const
  {
    const int blockstride = ((((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32);
    unsigned int off = (jz + SW) * (BLOCKSIZE_Y + 2*SW) + jy + SW + wid * blockstride;
#ifdef DEBUG
    if (off >= WARPS_PER_BLOCK * blockstride) {
      *__d_error_count++;
      off = 0;
    }
#endif

    return scurr[off];
  }
  __device__ real& operator()(int wid, int jy, int jz)
  {
    const int blockstride = ((((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32);
    unsigned int off = (jz + SW) * (BLOCKSIZE_Y + 2*SW) + jy + SW + wid * blockstride;
#ifdef DEBUG
    if (off >= WARPS_PER_BLOCK * blockstride) {
      *__d_error_count++;
      off = 0;
    }
#endif

    return scurr[off];
  }
  __device__ real operator()(int jy, int jz) const
  {
    return (*this)(threadIdx.x >> 5, jy, jz);
  }
  __device__ real& operator()(int jy, int jz)
  {
    return (*this)(threadIdx.x >> 5, jy, jz);
  }
};

// ======================================================================

// ----------------------------------------------------------------------
// current_add

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
current_add(SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr, int jy, int jz, real val)
{
  float *addr = &scurr(jy, jz);
  if (!do_write)
    return;

  if (do_reduce) {
#if __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
    atomicAdd(addr, val);
#else
#if 0
    while ((val = atomicExch(addr, atomicExch(addr, 0.0f)+val))!=0.0f);
#else
    int lid = threadIdx.x & 31;
    for (int i = 0; i < 32; i++) {
      if (lid == i) {
	*addr += val;
      }
    }
#endif
#endif
  } else {
    *addr += val;
  }
}

// ----------------------------------------------------------------------
// yz_calc_jx

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
yz_calc_jx(int i, particles_cuda_dev_t d_particles,
	   SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr,
	   struct cuda_params prm)
{
  struct d_particle p;
  if (do_read) {
    LOAD_PARTICLE(p, d_particles, i);
  }

  real vxi[3];
  calc_vxi(vxi, p);
  push_xi(&p, vxi, .5f * prm.dt);

  if (do_calc_j) {
    real fnqx = vxi[0] * p.qni_wni * prm.fnqs;
    
    int lf[3];
    real of[3];
    find_idx_off_1st(p.xi, lf, of, real(0.), prm.dxi);
    lf[1] -= ci0[1];
    lf[2] -= ci0[2];
    current_add(scurr, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnqx);
    current_add(scurr, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnqx);
    current_add(scurr, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnqx);
    current_add(scurr, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnqx);
  }
}

// ----------------------------------------------------------------------
// yz_calc_jy

__device__ static void
calc_dx1(real dx1[2], real x[2], real dx[2], int off[2])
{
  if (off[1] == 0) {
    dx1[0] = .5f * off[0] - x[0];
    if (dx[0] != 0.f)
      dx1[1] = dx[1] / dx[0] * dx1[0];
    else
      dx1[1] = 0.f;
  } else {
    dx1[1] = .5f * off[1] - x[1];
    if (dx[1] != 0.f)
      dx1[0] = dx[0] / dx[1] * dx1[1];
    else
      dx1[0] = 0.f;
  }
}

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
curr_2d_vb_cell(int i[2], real x[2], real dx[2], real qni_wni,
		real dxt[2], int off[2],
		SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_y,
		SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_z,
		struct cuda_params prm)
{
  real fnqy = qni_wni * prm.fnqys;
  real fnqz = qni_wni * prm.fnqzs;
  current_add(scurr_y, i[0],i[1]  , fnqy * dx[0] * (.5f - x[1] - .5f * dx[1]));
  current_add(scurr_y, i[0],i[1]+1, fnqy * dx[0] * (.5f + x[1] + .5f * dx[1]));
  current_add(scurr_z, i[0],i[1]  , fnqz * dx[1] * (.5f - x[0] - .5f * dx[0]));
  current_add(scurr_z, i[0]+1,i[1], fnqz * dx[1] * (.5f + x[0] + .5f * dx[0]));
  if (dxt) {
    dxt[0] -= dx[0];
    dxt[1] -= dx[1];
    x[0] += dx[0] - off[0];
    x[1] += dx[1] - off[1];
    i[0] += off[0];
    i[1] += off[1];
  }
}

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
yz_calc_jyjz(int i, particles_cuda_dev_t d_particles,
	     SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_y,
	     SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_z,
	     struct cuda_params prm)
{
  struct d_particle p;

  // OPT/FIXME, is it really better to reload the particle?
  if (do_read) {
    LOAD_PARTICLE(p, d_particles, i);
  }

  if (do_calc_j) {
    real vxi[3];
    real h0[3], h1[3];
    real xm[3], xp[3];
    
    int j[3], k[3];
    calc_vxi(vxi, p);
    
    find_idx_off_pos_1st(p.xi, j, h0, xm, real(0.), prm);

    // x^(n+0.5), p^(n+1.0) -> x^(n+1.5), p^(n+1.0) 
    push_xi(&p, vxi, prm.dt);
    STORE_PARTICLE_POS(p, d_particles, i);

    find_idx_off_pos_1st(p.xi, k, h1, xp, real(0.), prm);
    
    int idiff[2] = { k[1] - j[1], k[2] - j[2] };
    real dx[2] = { xp[1] - xm[1], xp[2] - xm[2] };
    real x[2] = { xm[1] - (j[1] + real(.5)), xm[2] - (j[2] + real(.5)) };
    int i[2] = { j[1] - ci0[1], j[2] - ci0[2] };
  
    int off[2];
    int first_dir, second_dir = -1;
    // FIXME, make sure we never div-by-zero?
    if (idiff[0] == 0 && idiff[1] == 0) {
      first_dir = -1;
    } else if (idiff[0] == 0) {
      first_dir = 1;
    } else if (idiff[1] == 0) {
      first_dir = 0;
    } else {
      real dx1[2];
      dx1[0] = .5f * idiff[0] - x[0];
      dx1[1] = dx[1] / dx[0] * dx1[0];
      if (fabsf(x[1] + dx1[1]) > .5f) {
	first_dir = 1;
      } else {
	first_dir = 0;
      }
      second_dir = 1 - first_dir;
    }
    
    if (first_dir >= 0) {
      real dx1[2];
      off[1-first_dir] = 0;
      off[first_dir] = idiff[first_dir];
      calc_dx1(dx1, x, dx, off);
      curr_2d_vb_cell(i, x, dx1, p.qni_wni, dx, off, scurr_y, scurr_z, prm);
    }
    
    if (second_dir >= 0) {
      real dx1[2];
      off[first_dir] = 0;
      off[second_dir] = idiff[second_dir];
      calc_dx1(dx1, x, dx, off);
      curr_2d_vb_cell(i, x, dx1, p.qni_wni, dx, off, scurr_y, scurr_z, prm);
    }
    
    curr_2d_vb_cell(i, x, dx, p.qni_wni, NULL, NULL, scurr_y, scurr_z, prm);
  }
}

// ======================================================================

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
push_part_p3(particles_cuda_dev_t d_particles, real *d_flds, int block_start, struct cuda_params prm)
{
  __d_error_count = prm.d_error_count;
  do_read = true;
  do_reduce = true;
  do_write = true;
  do_calc_j = true;

  __shared__ extern real _scurr[];

  const int block_stride = (((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32;

  SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> scurr_x(_scurr + 0 * WARPS_PER_BLOCK * block_stride);
  SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> scurr_y(_scurr + 1 * WARPS_PER_BLOCK * block_stride);
  SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> scurr_z(_scurr + 2 * WARPS_PER_BLOCK * block_stride);

  if (do_write) {
    scurr_x.zero();
    scurr_y.zero();
    scurr_z.zero();
  }

  int tid = threadIdx.x;
  int block_pos[3];
  block_pos[1] = blockIdx.x * 2;
  block_pos[2] = blockIdx.y * 2;
  block_pos[1] += block_start & 1;
  block_pos[2] += block_start >> 1;
  if (block_pos[1] >= prm.b_mx[1] ||
      block_pos[2] >= prm.b_mx[2])
    return;

  int bid = block_pos_to_block_idx(block_pos, prm.b_mx);
  __shared__ int s_block_end;
  if (tid == 0) {
    ci0[0] = 0;
    ci0[1] = block_pos[1] * BLOCKSIZE_Y;
    ci0[2] = block_pos[2] * BLOCKSIZE_Z;
    s_block_end = d_particles.offsets[bid + 1];
  }
  __syncthreads();

  int block_begin = d_particles.offsets[bid];

  for (int i = block_begin + tid; i < s_block_end; i += THREADS_PER_BLOCK) {
    yz_calc_jx(i, d_particles, scurr_x, prm);
    yz_calc_jyjz(i, d_particles, scurr_y, scurr_z, prm);
  }
  
  if (do_write) {
    __syncthreads();
    scurr_x.add_to_fld(d_flds, 0, prm);
    scurr_y.add_to_fld(d_flds, 1, prm);
    scurr_z.add_to_fld(d_flds, 2, prm);
  }
}

// ----------------------------------------------------------------------
// cuda_push_part_p3

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_part_p3(struct psc_particles *prts, struct psc_fields *pf)
{
  struct psc_fields_cuda *pfc = psc_fields_cuda(pf);
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int *d_error_count;
  check(hipMalloc(&d_error_count, 1 * sizeof(int)));
  check(hipMemset(d_error_count, 0, 1 * sizeof(int)));

  struct cuda_params prm;
  set_params(&prm, ppsc, prts, pf, d_error_count);

  unsigned int size = pf->im[0] * pf->im[1] * pf->im[2];
  check(hipMemset(pfc->d_flds + JXI * size, 0, 3 * size * sizeof(*pfc->d_flds)));

  assert(cuda->nr_blocks % 4 == 0); // FIXME, really?
  assert(cuda->nr_blocks == cuda->b_mx[1] * cuda->b_mx[2]);

  dim3 dimGrid((cuda->b_mx[1] + 1) / 2, (cuda->b_mx[2] + 1) / 2);
  const int block_stride = (((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32;
  unsigned int shared_size = 3 * WARPS_PER_BLOCK * block_stride * sizeof(real);

  for (int block_start = 0; block_start < 4; block_start++) {
    push_part_p3<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
      <<<dimGrid, THREADS_PER_BLOCK, shared_size>>>
      (cuda->d_part, pfc->d_flds, block_start, prm);
    cuda_sync_if_enabled();
  }

  int h_error_count[1];
  check(hipMemcpy(h_error_count, d_error_count, 1 * sizeof(int),
		   hipMemcpyDeviceToHost));
  check(hipFree(d_error_count));
  if (h_error_count[0] != 0) {
    printf("err cnt %d\n", h_error_count[0]);
  }
  assert(h_error_count[0] == 0);
}

// ----------------------------------------------------------------------
// cuda_push_mprts_b

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_b(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_particles **mprts_cuda = new struct psc_particles *[mprts->nr_patches];
  struct psc_fields **mflds_cuda = new struct psc_fields *[mflds->nr_patches];
  int nr_patches = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
    if (psc_particles_ops(prts) == &psc_particles_cuda_ops &&
	psc_fields_ops(flds) == &psc_fields_cuda_ops) {
      mprts_cuda[nr_patches] = prts;
      mflds_cuda[nr_patches] = flds;
      nr_patches++;
    }
  }

  if (nr_patches == 0) {
    delete[] mprts_cuda;
    delete[] mflds_cuda;
    return;
  }

  int *d_error_count;
  check(hipMalloc(&d_error_count, 1 * sizeof(int)));
  check(hipMemset(d_error_count, 0, 1 * sizeof(int)));

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts_cuda[0], mflds_cuda[0], d_error_count);

  struct cuda_patch *cpatch = new struct cuda_patch[nr_patches];
  for (int p = 0; p < nr_patches; p++) {
    struct psc_particles *prts = mprts_cuda[p];
    struct psc_fields *flds = mflds_cuda[p];

    cpatch[p].d_part = psc_particles_cuda(prts)->d_part;
    cpatch[p].d_flds = psc_fields_cuda(flds)->d_flds;
    assert(psc_particles_cuda(prts)->nr_blocks % 4 == 0); // FIXME, really?
  }
  
  struct cuda_patch *d_cpatch;
  check(hipMalloc(&d_cpatch, nr_patches * sizeof(*d_cpatch)));
  check(hipMemcpy(d_cpatch, cpatch, nr_patches * sizeof(*d_cpatch),
		   hipMemcpyHostToDevice));
  
  for (int p = 0; p < nr_patches; p++) {
    unsigned int size = prm.mx[0] * prm.mx[1] * prm.mx[2];
    check(hipMemset(cpatch[p].d_flds + JXI * size, 0, 3 * size * sizeof(*cpatch[p].d_flds)));
  }
	 
  const int block_stride = (((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32;
  dim3 dimGrid((prm.b_mx[1] + 1) / 2, (prm.b_mx[2] + 1) / 2);
  unsigned int shared_size = 3 * WARPS_PER_BLOCK * block_stride * sizeof(real);

  for (int block_start = 0; block_start < 4; block_start++) {
    for (int p = 0; p < nr_patches; p++) {
      push_part_p3<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
	<<<dimGrid, THREADS_PER_BLOCK, shared_size>>>
	(cpatch[p].d_part, cpatch[p].d_flds, block_start, prm);
      cuda_sync_if_enabled();
    }
  }
    
  check(hipFree(d_cpatch));
  delete[] cpatch;

  delete[] mprts_cuda;
  delete[] mflds_cuda;

  int h_error_count[1];
  check(hipMemcpy(h_error_count, d_error_count, 1 * sizeof(int),
		   hipMemcpyDeviceToHost));
  check(hipFree(d_error_count));
  if (h_error_count[0] != 0) {
    printf("err cnt %d\n", h_error_count[0]);
  }
  assert(h_error_count[0] == 0);
}

// ======================================================================

EXTERN_C void
yz2x2_1vb_cuda_push_part_p2(struct psc_particles *prts, struct psc_fields *pf)
{
  assert(0);
  //  cuda_push_part_p2<1, 2, 2>(prts, pf);
}

EXTERN_C void
yz2x2_1vb_cuda_push_part_p3(struct psc_particles *prts, struct psc_fields *pf, real *dummy,
			    int block_stride)
{
  assert(0);
  //  cuda_push_part_p3<1, 2, 2>(prts, pf);
}

EXTERN_C void
yz4x4_1vb_cuda_push_part_p2(struct psc_particles *prts, struct psc_fields *pf)
{
  cuda_push_part_p2<1, 4, 4>(prts, pf);
}

EXTERN_C void
yz4x4_1vb_cuda_push_part_p3(struct psc_particles *prts, struct psc_fields *pf, real *dummy,
			    int block_stride)
{
  cuda_push_part_p3<1, 4, 4>(prts, pf);
}

EXTERN_C void
yz8x8_1vb_cuda_push_part_p2(struct psc_particles *prts, struct psc_fields *pf)
{
  assert(0);
  //  cuda_push_part_p2<1, 8, 8>(prts, pf);
}

EXTERN_C void
yz8x8_1vb_cuda_push_part_p3(struct psc_particles *prts, struct psc_fields *pf, real *dummy,
			    int block_stride)
{
  assert(0);
  //  cuda_push_part_p3<1, 8, 8>(prts, pf);
}






EXTERN_C void
yz4x4_1vb_cuda_push_mprts_a(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  cuda_push_mprts_a<1, 4, 4>(mprts, mflds);
}

EXTERN_C void
yz4x4_1vb_cuda_push_mprts_b(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  cuda_push_mprts_b<1, 4, 4>(mprts, mflds);
}
