#include "hip/hip_runtime.h"

#include "psc_cuda.h"
#include "particles_cuda.h"

#include <mrc_profile.h>

// OPT: precalc offsets into fld_cache (including ci[])
// OPT: use more shmem?

#define LOAD_PARTICLE_(pp, d_xi4, d_pxi4, n) do {			\
    float4 xi4 = d_xi4[n];						\
    (pp).xi[0]         = xi4.x;						\
    (pp).xi[1]         = xi4.y;						\
    (pp).xi[2]         = xi4.z;						\
    (pp).kind_as_float = xi4.w;						\
    float4 pxi4 = d_pxi4[n];						\
    (pp).pxi[0]        = pxi4.x;					\
    (pp).pxi[1]        = pxi4.y;					\
    (pp).pxi[2]        = pxi4.z;					\
    (pp).qni_wni       = pxi4.w;					\
} while (0)

#define LOAD_PARTICLE_POS_(pp, d_xi4, n) do {				\
    float4 xi4 = d_xi4[n];						\
    (pp).xi[0]         = xi4.x;						\
    (pp).xi[1]         = xi4.y;						\
    (pp).xi[2]         = xi4.z;						\
    (pp).kind_as_float = xi4.w;						\
} while (0)

#define LOAD_PARTICLE_MOM_(pp, d_pxi4, n) do {				\
    float4 pxi4 = d_pxi4[n];						\
    (pp).pxi[0]        = pxi4.x;					\
    (pp).pxi[1]        = pxi4.y;					\
    (pp).pxi[2]        = pxi4.z;					\
    (pp).qni_wni       = pxi4.w;					\
} while (0)

#define STORE_PARTICLE_POS_(pp, d_xi4, n) do {				\
    d_xi4[n].x = (pp).xi[0];						\
    d_xi4[n].y = (pp).xi[1];						\
    d_xi4[n].z = (pp).xi[2];						\
    d_xi4[n].w = (pp).kind_as_float;					\
} while (0)

#define STORE_PARTICLE_MOM_(pp, d_pxi4, n) do {				\
    float4 pxi4 = { (pp).pxi[0], (pp).pxi[1], (pp).pxi[2], (pp).qni_wni }; \
    d_pxi4[n] = pxi4;							\
} while (0)

#define NO_CHECKERBOARD
//#define DEBUG

#define SW (2)

#include "cuda_common.h"

__device__ int *__d_error_count;

static __constant__ __device__ float c_dqs[4];
static __constant__ __device__ float c_dxi[3];
static __constant__ __device__ int c_mx[3];
static __constant__ __device__ int c_ilg[3];

static void
set_consts(struct cuda_params *prm)
{
  check(hipMemcpyToSymbol(HIP_SYMBOL(c_dqs), prm->dq, sizeof(c_dqs)));
  check(hipMemcpyToSymbol(HIP_SYMBOL(c_dxi), prm->dxi, sizeof(c_dxi)));
  check(hipMemcpyToSymbol(HIP_SYMBOL(c_mx), prm->mx, sizeof(c_mx)));
  check(hipMemcpyToSymbol(HIP_SYMBOL(c_ilg), prm->ilg, sizeof(c_ilg)));
}

void
set_params(struct cuda_params *prm, struct psc *psc,
	   struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  prm->dt = psc->dt;
  for (int d = 0; d < 3; d++) {
    prm->dxi[d] = 1.f / ppsc->patch[0].dx[d];
  }

  prm->dqs    = .5f * psc->coeff.eta * psc->dt;
  prm->fnqs   = sqr(psc->coeff.alpha) * psc->coeff.cori / psc->coeff.eta;
  prm->fnqys  = psc->patch[0].dx[1] * prm->fnqs / psc->dt;
  prm->fnqzs  = psc->patch[0].dx[2] * prm->fnqs / psc->dt;
  assert(psc->nr_kinds <= MAX_KINDS);
  for (int k = 0; k < psc->nr_kinds; k++) {
    prm->dq[k] = prm->dqs * psc->kinds[k].q / psc->kinds[k].m;
  }

  if (mprts && mprts->nr_patches > 0) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, 0);
    struct psc_particles_cuda *prts_cuda = psc_particles_cuda(prts);
    for (int d = 0; d < 3; d++) {
      prm->b_mx[d] = prts_cuda->b_mx[d];
      prm->b_dxi[d] = prts_cuda->b_dxi[d];
    }
  }

  if (mflds) {
    struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
    for (int d = 0; d < 3; d++) {
      prm->mx[d] = mflds_cuda->im[d];
      prm->ilg[d] = mflds_cuda->ib[d];
      assert(mflds_cuda->ib[d] == -3);
    }
    assert(mflds_cuda->im[0] == 7);
  }

  //  check(hipMalloc(&prm->d_error_count, 1 * sizeof(int)));
  //  check(hipMemset(prm->d_error_count, 0, 1 * sizeof(int)));
}

void
free_params(struct cuda_params *prm)
{
  //  int h_error_count[1];
  //  check(hipMemcpy(h_error_count, prm->d_error_count, 1 * sizeof(int),
  //		   hipMemcpyDeviceToHost));
  //  check(hipFree(prm->d_error_count));
  //  if (h_error_count[0] != 0) {
  //    printf("err cnt %d\n", h_error_count[0]);
  //  }
  //  assert(h_error_count[0] == 0);
}

// ======================================================================

void
psc_mparticles_cuda_copy_to_dev(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  check(hipMemcpy(mprts_cuda->d_dev, mprts_cuda->h_dev,
		   mprts->nr_patches * sizeof(*mprts_cuda->d_dev),
		   hipMemcpyHostToDevice));
}

// ======================================================================
// field caching

#if 0
template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
class F3cache {
  real *fld_cache;

public:
  __device__ F3cache(real *_fld_cache, real *d_flds, int l[3],
		     struct cuda_params prm) :
    fld_cache(_fld_cache)
  {
    int ti = threadIdx.x;
    int n = BLOCKSIZE_X * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4);
    while (ti < n) {
      int tmp = ti;
      int jx = tmp % BLOCKSIZE_X;
      tmp /= BLOCKSIZE_X;
      int jy = tmp % (BLOCKSIZE_Y + 4) - 2;
      tmp /= BLOCKSIZE_Y + 4;
      int jz = tmp % (BLOCKSIZE_Z + 4) - 2;
      //    tmp /= BLOCKSIZE_Z + 4;
      //    int m = tmp + EX;
      //    printf("n %d ti %d m %d, jx %d,%d,%d\n", n, ti, m, jx, jy, jz);
      // currently it seems faster to do the loop rather than do m by threadidx
      for (int m = EX; m <= HZ; m++) {
	(*this)(m, jx,jy,jz) = F3_DEV_YZ(m, jy+l[1],jz+l[2]);
      }
      ti += blockDim.x;
    }
    __syncthreads();
  }

  __host__ __device__ real operator()(int fldnr, int jx, int jy, int jz) const
  {
    int off = ((((fldnr-EX)
		 *(BLOCKSIZE_Z + 4) + ((jz)-(-2)))
		*(BLOCKSIZE_Y + 4) + ((jy)-(-2)))
	       *1 + ((jx)));
    return fld_cache[off];
  }
  __host__ __device__ real& operator()(int fldnr, int jx, int jy, int jz)
  {
    int off = ((((fldnr-EX)
		 *(BLOCKSIZE_Z + 4) + ((jz)-(-2)))
		*(BLOCKSIZE_Y + 4) + ((jy)-(-2)))
	       *1 + ((jx)));
    return fld_cache[off];
  }
};
#endif

#define F3_CACHE(fld_cache, m, jy, jz)					\
  ((fld_cache)[(((m-EX)							\
		 *(BLOCKSIZE_Z + 4) + ((jz)-(-2)))			\
		*(BLOCKSIZE_Y + 4) + ((jy)-(-2)))])

// ----------------------------------------------------------------------
// push_xi
//
// advance position using velocity

__device__ static void
push_xi(struct d_particle *p, const real vxi[3], real dt)
{
  int d;
  for (d = 1; d < 3; d++) {
    p->xi[d] += dt * vxi[d];
  }
}

// ----------------------------------------------------------------------
// calc_vxi
//
// calculate velocity from momentum

__device__ static void
calc_vxi(real vxi[3], struct d_particle p)
{
  real root = rsqrtr(real(1.) + sqr(p.pxi[0]) + sqr(p.pxi[1]) + sqr(p.pxi[2]));

  int d;
  for (d = 0; d < 3; d++) {
    vxi[d] = p.pxi[d] * root;
  }
}

// ----------------------------------------------------------------------
// push_pxi_dt
//
// advance moments according to EM fields

__device__ static void
push_pxi_dt(struct d_particle *p,
	     real exq, real eyq, real ezq, real hxq, real hyq, real hzq)
{
  int kind = __float_as_int(p->kind_as_float);
  real dq = c_dqs[kind];
  real pxm = p->pxi[0] + dq*exq;
  real pym = p->pxi[1] + dq*eyq;
  real pzm = p->pxi[2] + dq*ezq;
  
  real root = dq * rsqrtr(real(1.) + sqr(pxm) + sqr(pym) + sqr(pzm));
  real taux = hxq * root, tauy = hyq * root, tauz = hzq * root;
  
  real tau = real(1.) / (real(1.) + sqr(taux) + sqr(tauy) + sqr(tauz));
  real pxp = ( (real(1.) + sqr(taux) - sqr(tauy) - sqr(tauz)) * pxm
	       +(real(2.)*taux*tauy + real(2.)*tauz)*pym
	       +(real(2.)*taux*tauz - real(2.)*tauy)*pzm)*tau;
  real pyp = ( (real(2.)*taux*tauy - real(2.)*tauz)*pxm
	       +(real(1.) - sqr(taux) + sqr(tauy) - sqr(tauz)) * pym
	       +(real(2.)*tauy*tauz + real(2.)*taux)*pzm)*tau;
  real pzp = ( (real(2.)*taux*tauz + real(2.)*tauy)*pxm
	       +(real(2.)*tauy*tauz - real(2.)*taux)*pym
	       +(real(1.) - sqr(taux) - sqr(tauy) + sqr(tauz))*pzm)*tau;
  
  p->pxi[0] = pxp + dq * exq;
  p->pxi[1] = pyp + dq * eyq;
  p->pxi[2] = pzp + dq * ezq;
}

#define OFF(g, d) o##g[d]
  
__device__ static real
ip1_to_grid_0(real h)
{
  return real(1.) - h;
}

__device__ static real
ip1_to_grid_p(real h)
{
  return h;
}

#define INTERP_FIELD_1ST(cache, exq, fldnr, g1, g2)			\
  do {									\
    int ddy = l##g1[1]-l0[1], ddz = l##g2[2]-l0[2];			\
    /* printf("C %g [%d,%d,%d]\n", F3C(fldnr, 0, ddy, ddz), 0, ddy, ddz); */ \
    exq =								\
      ip1_to_grid_0(OFF(g1, 1)) * ip1_to_grid_0(OFF(g2, 2)) *		\
      F3_CACHE(fld_cache, fldnr, ddy+0, ddz+0) +			\
      ip1_to_grid_p(OFF(g1, 1)) * ip1_to_grid_0(OFF(g2, 2)) *		\
      F3_CACHE(fld_cache, fldnr, ddy+1, ddz+0) +			\
      ip1_to_grid_0(OFF(g1, 1)) * ip1_to_grid_p(OFF(g2, 2)) *		\
      F3_CACHE(fld_cache, fldnr, ddy+0, ddz+1) +			\
      ip1_to_grid_p(OFF(g1, 1)) * ip1_to_grid_p(OFF(g2, 2)) *		\
      F3_CACHE(fld_cache, fldnr, ddy+1, ddz+1);				\
  } while(0)

// ----------------------------------------------------------------------
// push_part_one
//
// push one particle

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
push_part_one(float4 *d_xi4, float4 *d_pxi4, real *fld_cache, int l0[3])
{
  struct d_particle p;
  LOAD_PARTICLE_POS_(p, d_xi4, 0);

  // here we have x^{n+.5}, p^n
  
  // field interpolation

  int lh[3], lg[3];
  real oh[3], og[3];
  find_idx_off_1st(p.xi, lh, oh, real(-.5), c_dxi);
  find_idx_off_1st(p.xi, lg, og, real(0.), c_dxi);

  real exq, eyq, ezq, hxq, hyq, hzq;
  INTERP_FIELD_1ST(cached_flds, exq, EX, g, g);
  INTERP_FIELD_1ST(cached_flds, eyq, EY, h, g);
  INTERP_FIELD_1ST(cached_flds, ezq, EZ, g, h);
  INTERP_FIELD_1ST(cached_flds, hxq, HX, h, h);
  INTERP_FIELD_1ST(cached_flds, hyq, HY, g, h);
  INTERP_FIELD_1ST(cached_flds, hzq, HZ, h, g);

  // x^(n+0.5), p^n -> x^(n+0.5), p^(n+1.0) 
  
  LOAD_PARTICLE_MOM_(p, d_pxi4, 0);
  push_pxi_dt(&p, exq, eyq, ezq, hxq, hyq, hzq);
  STORE_PARTICLE_MOM_(p, d_pxi4, 0);
}

// ----------------------------------------------------------------------
// push_part_one_reorder
//
// push one particle

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
push_part_one_reorder(int n, unsigned int *d_ids, float4 *d_xi4, float4 *d_pxi4,
		      float4 *d_alt_xi4, float4 *d_alt_pxi4,
		      real *fld_cache, int l0[3])
{
  struct d_particle p;
  LOAD_PARTICLE_POS_(p, d_xi4, d_ids[n]);
  STORE_PARTICLE_POS_(p, d_alt_xi4, n);

  // here we have x^{n+.5}, p^n
  
  // field interpolation

  int lh[3], lg[3];
  real oh[3], og[3];
  find_idx_off_1st(p.xi, lh, oh, real(-.5), c_dxi);
  find_idx_off_1st(p.xi, lg, og, real(0.), c_dxi);

  real exq, eyq, ezq, hxq, hyq, hzq;
  INTERP_FIELD_1ST(cached_flds, exq, EX, g, g);
  INTERP_FIELD_1ST(cached_flds, eyq, EY, h, g);
  INTERP_FIELD_1ST(cached_flds, ezq, EZ, g, h);
  INTERP_FIELD_1ST(cached_flds, hxq, HX, h, h);
  INTERP_FIELD_1ST(cached_flds, hyq, HY, g, h);
  INTERP_FIELD_1ST(cached_flds, hzq, HZ, h, g);

  // x^(n+0.5), p^n -> x^(n+0.5), p^(n+1.0) 
  
  LOAD_PARTICLE_MOM_(p, d_pxi4, d_ids[n]);
  push_pxi_dt(&p, exq, eyq, ezq, hxq, hyq, hzq);
  STORE_PARTICLE_MOM_(p, d_alt_pxi4, n);
}

// ----------------------------------------------------------------------
// push_mprts_p1
//
// push particles

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
push_mprts_p1(float4 *d_xi4, float4 *d_pxi4,
	      unsigned int *d_off, float *d_flds0, unsigned int size,
	      unsigned int b_my, unsigned int b_mz)
{
#if 0
  __d_error_count = prm.d_error_count;
#endif
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % b_mz;
  int p = blockIdx.y / b_mz;

  int ci[3];
  ci[0] = 0;
  ci[1] = block_pos[1] * BLOCKSIZE_Y;
  ci[2] = block_pos[2] * BLOCKSIZE_Z;

  __shared__ real fld_cache[6 * 1 * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4)];

  {
    real *d_flds = d_flds0 + p * size;

    int ti = threadIdx.x;
    int n = BLOCKSIZE_X * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4);
    while (ti < n) {
      int tmp = ti;
      int jy = tmp % (BLOCKSIZE_Y + 4) - 2;
      tmp /= BLOCKSIZE_Y + 4;
      int jz = tmp % (BLOCKSIZE_Z + 4) - 2;
      // OPT? currently it seems faster to do the loop rather than do m by threadidx
      for (int m = EX; m <= HZ; m++) {
	F3_CACHE(fld_cache, m, jy, jz) = F3_DEV_YZ_(m, jy+ci[1],jz+ci[2]);
      }
      ti += THREADS_PER_BLOCK;
    }
    __syncthreads();
  }


  int bid = blockIdx.y * b_my + blockIdx.x;
  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  float4 *xi4 = d_xi4 + block_begin + tid;
  float4 *pxi4 = d_pxi4 + block_begin + tid;
  float4 *xi4_end = d_xi4 + block_end;

  for (; xi4 < xi4_end; xi4 += THREADS_PER_BLOCK, pxi4 += THREADS_PER_BLOCK) {
    push_part_one<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>(xi4, pxi4, fld_cache, ci);
  }
}

// ----------------------------------------------------------------------
// push_mprts_p1_reorder
//
// push particles

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
push_mprts_p1_reorder(unsigned int *d_ids, float4 *d_xi4, float4 *d_pxi4,
		      float4 *d_alt_xi4, float4 *d_alt_pxi4,
		      unsigned int *d_off, float *d_flds0, unsigned int size,
		      unsigned int b_my, unsigned int b_mz)
{
#if 0
  __d_error_count = prm.d_error_count;
#endif
  int tid = threadIdx.x;

  int block_pos[3];
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % b_mz;
  int p = blockIdx.y / b_mz;

  int ci[3];
  ci[0] = 0;
  ci[1] = block_pos[1] * BLOCKSIZE_Y;
  ci[2] = block_pos[2] * BLOCKSIZE_Z;

  __shared__ real fld_cache[6 * 1 * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4)];

  {
    real *d_flds = d_flds0 + p * size;

    int ti = threadIdx.x;
    int n = BLOCKSIZE_X * (BLOCKSIZE_Y + 4) * (BLOCKSIZE_Z + 4);
    while (ti < n) {
      int tmp = ti;
      int jy = tmp % (BLOCKSIZE_Y + 4) - 2;
      tmp /= BLOCKSIZE_Y + 4;
      int jz = tmp % (BLOCKSIZE_Z + 4) - 2;
      // OPT? currently it seems faster to do the loop rather than do m by threadidx
      for (int m = EX; m <= HZ; m++) {
	F3_CACHE(fld_cache, m, jy, jz) = F3_DEV_YZ_(m, jy+ci[1],jz+ci[2]);
      }
      ti += THREADS_PER_BLOCK;
    }
    __syncthreads();
  }
  
  int bid = blockIdx.y * b_my + blockIdx.x;
  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  for (int n = (block_begin & ~31) + tid; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    push_part_one_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
      (n, d_ids, d_xi4, d_pxi4, d_alt_xi4, d_alt_pxi4, fld_cache, ci);
  }
}

// ----------------------------------------------------------------------
// cuda_push_mprts_a

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_a(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, mflds);
  set_consts(&prm);

  unsigned int size = mflds->nr_fields *
    mflds_cuda->im[0] * mflds_cuda->im[1] * mflds_cuda->im[2];
  
  dim3 dimGrid(prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches);
  
  push_mprts_p1<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (mprts_cuda->d_xi4, mprts_cuda->d_pxi4, mprts_cuda->d_off,
     mflds_cuda->d_flds, size, prm.b_mx[1], prm.b_mx[2]);
  cuda_sync_if_enabled();
  
  free_params(&prm);
}

// ----------------------------------------------------------------------
// psc_mparticles_cuda_swap_alt
// FIXME, duplicated

static void
psc_mparticles_cuda_swap_alt(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  float4 *tmp_xi4 = mprts_cuda->d_alt_xi4;
  float4 *tmp_pxi4 = mprts_cuda->d_alt_pxi4;
  mprts_cuda->d_alt_xi4 = mprts_cuda->d_xi4;
  mprts_cuda->d_alt_pxi4 = mprts_cuda->d_pxi4;
  mprts_cuda->d_xi4 = tmp_xi4;
  mprts_cuda->d_pxi4 = tmp_pxi4;
}

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 6)
mprts_reorder(struct cuda_params prm, unsigned int *d_off,
	      unsigned int *d_ids,
	      float4 *d_xi4, float4 *d_pxi4,
	      float4 *d_alt_xi4, float4 *d_alt_pxi4)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * prm.b_mx[1] + blockIdx.x;

  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  for (int n = (block_begin & ~31) + tid; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }

    int j = d_ids[n];
    d_alt_xi4[n] = d_xi4[j];
    d_alt_pxi4[n] = d_pxi4[j];
    d_ids[n] = n;//!!!
  }
}

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
mprts_reorder_x2(struct cuda_params prm, unsigned int *d_off,
		 unsigned int *d_ids,
		 float4 *d_xi4, float4 *d_pxi4,
		 float4 *d_alt_xi4, float4 *d_alt_pxi4)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * prm.b_mx[1] + blockIdx.x;

  int block_begin = d_off[bid];
  int block_end   = d_off[bid + 1];

  for (int n = block_begin + tid; n < block_end; n += THREADS_PER_BLOCK) {
    int j = n;
    d_alt_xi4[n] = d_xi4[j];
//    d_alt_pxi4[n] = d_pxi4[j];
    //    d_ids[n] = n;//!!!
  }
}

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
mprts_reorder_x3(struct cuda_params prm, unsigned int *d_off,
		 unsigned int *d_ids,
		 float4 *d_xi4, float4 *d_pxi4,
		 float4 *d_alt_xi4, float4 *d_alt_pxi4, int n_part)
{
  int tid = threadIdx.x;
  int bid = blockIdx.y * prm.b_mx[1] + blockIdx.x;

#if 0
  int block_begin = bid * 9760;
  int block_end = (bid+1) * 9760;
#else
  int block_begin = d_off[bid];
  int block_end = d_off[bid+1];
#endif

  for (int n = (block_begin & ~31) + tid; n < block_end; n += THREADS_PER_BLOCK) {
    if (n >= block_begin) {
      d_alt_xi4[n] = d_xi4[n];
      d_alt_pxi4[n] = d_pxi4[n];
    }
  }
}

// ----------------------------------------------------------------------
// cuda_push_mprts_a_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_a_reorder(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, mflds);
  set_consts(&prm);

  psc_mparticles_cuda_copy_to_dev(mprts);

  unsigned int size = mflds->nr_fields *
    mflds_cuda->im[0] * mflds_cuda->im[1] * mflds_cuda->im[2];
  
  dim3 dimGrid(prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches);

  push_mprts_p1_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (mprts_cuda->d_ids, mprts_cuda->d_xi4, mprts_cuda->d_pxi4,
     mprts_cuda->d_alt_xi4, mprts_cuda->d_alt_pxi4, mprts_cuda->d_off,
     mflds_cuda->d_flds, size, prm.b_mx[1], prm.b_mx[2]);
  cuda_sync_if_enabled();
  
  psc_mparticles_cuda_swap_alt(mprts);
    
  free_params(&prm);
}

// ----------------------------------------------------------------------
// cuda_push_mprts_a1_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_a1_reorder(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, mflds);
  set_consts(&prm);

  dim3 dimGrid(prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches);

  mprts_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (prm, mprts_cuda->d_off, mprts_cuda->d_ids,
     mprts_cuda->d_xi4, mprts_cuda->d_pxi4,
     mprts_cuda->d_alt_xi4, mprts_cuda->d_alt_pxi4);
  cuda_sync_if_enabled();

  free_params(&prm);
}
  
template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_a1_reorder_x2(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, mflds);
  set_consts(&prm);

  dim3 dimGrid(prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches);

  mprts_reorder_x2<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (prm, mprts_cuda->d_off, mprts_cuda->d_ids,
     mprts_cuda->d_xi4, mprts_cuda->d_pxi4,
     mprts_cuda->d_alt_xi4, mprts_cuda->d_alt_pxi4);
  cuda_sync_if_enabled();
  mprintf("n_p %d\n", psc_mparticles_get_patch(mprts, 0)->n_part);
  check(hipMemcpy(mprts_cuda->d_xi4, mprts_cuda->d_alt_xi4, psc_mparticles_get_patch(mprts, 0)->n_part * 16,
		   hipMemcpyDeviceToDevice));

  free_params(&prm);
}
  
template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_a1_reorder_x3(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, mflds);
  set_consts(&prm);

  int n_part = psc_mparticles_get_patch(mprts, 0)->n_part;
  int nr_blocks = prm.b_mx[1] * prm.b_mx[2] * mprts->nr_patches;
  mprintf("nr_blocks %d\n", nr_blocks);
  assert(n_part % nr_blocks == 0);
  mprintf("np/block %d\n", n_part / nr_blocks);

  unsigned  int *d_off;
  check(hipMalloc(&d_off, (nr_blocks + 1) * sizeof(int)));
  int *h_off = new int[nr_blocks+1];
  for (int b = 0; b <= nr_blocks; b++) {
    h_off[b] = b * (n_part / nr_blocks);
  }
  check(hipMemcpy(d_off, h_off, (nr_blocks + 1) * sizeof(int),
		   hipMemcpyHostToDevice));
  delete[] h_off;

  dim3 dimGrid(prm.b_mx[1], prm.b_mx[2] * mprts->nr_patches);

  mprts_reorder_x3<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (prm, mprts_cuda->d_off, mprts_cuda->d_ids,
     mprts_cuda->d_xi4, mprts_cuda->d_pxi4,
     mprts_cuda->d_alt_xi4, mprts_cuda->d_alt_pxi4, n_part);
  cuda_sync_if_enabled();

  check(hipFree(d_off));

  check(hipMemcpy(mprts_cuda->d_xi4, mprts_cuda->d_alt_xi4, n_part * 16,
		   hipMemcpyDeviceToDevice));
  check(hipMemcpy(mprts_cuda->d_pxi4, mprts_cuda->d_alt_pxi4, n_part * 16,
		   hipMemcpyDeviceToDevice));

  free_params(&prm);
}
  
// ======================================================================

// FIXME -> common.c

__device__ static void
find_idx_off_pos_1st(const real xi[3], int j[3], real h[3], real pos[3], real shift,
		     struct cuda_params prm)
{
  int d;
  for (d = 0; d < 3; d++) {
    pos[d] = xi[d] * prm.dxi[d] + shift;
    j[d] = __float2int_rd(pos[d]);
    h[d] = pos[d] - j[d];
  }
}

__shared__ volatile bool do_read;
__shared__ volatile bool do_write;
__shared__ volatile bool do_reduce;
__shared__ volatile bool do_calc_j;

// OPT: take i < cell_end condition out of load
// OPT: reduce two at a time
// OPT: try splitting current calc / measuring by itself
// OPT: get rid of block_stride

__shared__ int ci0[3]; // cell index of lower-left cell in block

#define WARPS_PER_BLOCK (THREADS_PER_BLOCK / 32)

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
class SCurr {
  real *scurr;

public:
  __device__ SCurr(real *_scurr) :
    scurr(_scurr)
  {
  }

  __device__ void zero()
  {
    const int blockstride = ((((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32);
    int i = threadIdx.x;
    int N = blockstride * WARPS_PER_BLOCK;
    while (i < N) {
      scurr[i] = real(0.);
      i += THREADS_PER_BLOCK;
    }
  }

  __device__ void add_to_fld(real *d_flds, int m, struct cuda_params prm)
  {
    int i = threadIdx.x;
    int stride = (BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW);
    while (i < stride) {
      int rem = i;
      int jz = rem / (BLOCKSIZE_Y + 2*SW);
      rem -= jz * (BLOCKSIZE_Y + 2*SW);
      int jy = rem;
      jz -= SW;
      jy -= SW;
      real val = real(0.);
      // FIXME, opt
      for (int wid = 0; wid < WARPS_PER_BLOCK; wid++) {
	val += (*this)(wid, jy, jz);
      }
      F3_DEV_YZ(JXI+m, jy+ci0[1],jz+ci0[2]) += val;
      i += THREADS_PER_BLOCK;
    }
  }

  __device__ real operator()(int wid, int jy, int jz) const
  {
    const int blockstride = ((((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32);
    unsigned int off = (jz + SW) * (BLOCKSIZE_Y + 2*SW) + jy + SW + wid * blockstride;
#ifdef DEBUG
    if (off >= WARPS_PER_BLOCK * blockstride) {
      (*__d_error_count)++;
      off = 0;
    }
#endif

    return scurr[off];
  }
  __device__ real& operator()(int wid, int jy, int jz)
  {
    const int blockstride = ((((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32);
    unsigned int off = (jz + SW) * (BLOCKSIZE_Y + 2*SW) + jy + SW + wid * blockstride;
#ifdef DEBUG
    if (off >= WARPS_PER_BLOCK * blockstride) {
      (*__d_error_count)++;
      off = 0;
    }
#endif

    return scurr[off];
  }
  __device__ real operator()(int jy, int jz) const
  {
    return (*this)(threadIdx.x >> 5, jy, jz);
  }
  __device__ real& operator()(int jy, int jz)
  {
    return (*this)(threadIdx.x >> 5, jy, jz);
  }
};

// ======================================================================

// ----------------------------------------------------------------------
// current_add

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
current_add(SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr, int jy, int jz, real val)
{
  float *addr = &scurr(jy, jz);
  if (!do_write)
    return;

  if (do_reduce) {
#if __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
    atomicAdd(addr, val);
#else
#if 0
    while ((val = atomicExch(addr, atomicExch(addr, 0.0f)+val))!=0.0f);
#else
    int lid = threadIdx.x & 31;
    for (int i = 0; i < 32; i++) {
      if (lid == i) {
	*addr += val;
      }
    }
#endif
#endif
  } else {
    *addr += val;
  }
}

// ----------------------------------------------------------------------
// yz_calc_jx

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
yz_calc_jx(int i, float4 *d_xi4, float4 *d_pxi4,
	   SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr,
	   struct cuda_params prm)
{
  struct d_particle p;
  if (do_read) {
    LOAD_PARTICLE_(p, d_xi4, d_pxi4, i);
  }

  real vxi[3];
  calc_vxi(vxi, p);
  push_xi(&p, vxi, .5f * prm.dt);

  if (do_calc_j) {
    real fnqx = vxi[0] * p.qni_wni * prm.fnqs;
    
    int lf[3];
    real of[3];
    find_idx_off_1st(p.xi, lf, of, real(0.), prm.dxi);
    lf[1] -= ci0[1];
    lf[2] -= ci0[2];
    current_add(scurr, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnqx);
    current_add(scurr, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnqx);
    current_add(scurr, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnqx);
    current_add(scurr, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnqx);
  }
}

// ----------------------------------------------------------------------
// yz_calc_jy

__device__ static void
calc_dx1(real dx1[2], real x[2], real dx[2], int off[2])
{
  real o0, x0, dx_0, dx_1, v0, v1;
  if (off[0] == 0) {
    o0 = off[1];
    x0 = x[1];
    dx_0 = dx[1];
    dx_1 = dx[0];
  } else {
    o0 = off[0];
    x0 = x[0];
    dx_0 = dx[0];
    dx_1 = dx[1];
  }
  if ((off[0] == 0 && off[1] == 0) || dx_0 == 0.f) {
    v0 = 0.f;
    v1 = 0.f;
  } else {
    v0 = .5f * o0 - x0;
    v1 = dx_1 / dx_0 * v0;
  }
  if (off[0] == 0) {
    dx1[0] = v1;
    dx1[1] = v0;
  } else {
    dx1[0] = v0;
    dx1[1] = v1;
  }
}

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
curr_2d_vb_cell(int i[2], real x[2], real dx[2], real qni_wni,
		SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_y,
		SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_z,
		struct cuda_params prm)
{
  if (dx[0] != 0.f) {
    real fnqy = qni_wni * prm.fnqys;
    current_add(scurr_y, i[0],i[1]  , fnqy * dx[0] * (.5f - x[1] - .5f * dx[1]));
    current_add(scurr_y, i[0],i[1]+1, fnqy * dx[0] * (.5f + x[1] + .5f * dx[1]));
  }
  if (dx[1] != 0.f) {
    real fnqz = qni_wni * prm.fnqzs;
    current_add(scurr_z, i[0],i[1]  , fnqz * dx[1] * (.5f - x[0] - .5f * dx[0]));
    current_add(scurr_z, i[0]+1,i[1], fnqz * dx[1] * (.5f + x[0] + .5f * dx[0]));
  }
}

__device__ static void
curr_2d_vb_cell_upd(int i[2], real x[2], real dx1[2], real dx[2], int off[2])
{
  dx[0] -= dx1[0];
  dx[1] -= dx1[1];
  x[0] += dx1[0] - off[0];
  x[1] += dx1[1] - off[1];
  i[0] += off[0];
  i[1] += off[1];
}

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static void
yz_calc_jyjz(int i, float4 *d_xi4, float4 *d_pxi4,
	     SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_y,
	     SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> &scurr_z,
	     struct cuda_params prm, int nr_total_blocks, int p_nr,
	     unsigned int *d_bidx, int bid)
{
  struct d_particle p;

  // OPT/FIXME, is it really better to reload the particle?
  if (do_read) {
    LOAD_PARTICLE_(p, d_xi4, d_pxi4, i);
  }

  if (do_calc_j) {
    real vxi[3];
    real h0[3], h1[3];
    real xm[3], xp[3];
    
    int j[3], k[3];
    calc_vxi(vxi, p);
    
    find_idx_off_pos_1st(p.xi, j, h0, xm, real(0.), prm);

    // x^(n+0.5), p^(n+1.0) -> x^(n+1.5), p^(n+1.0) 
    push_xi(&p, vxi, prm.dt);
    STORE_PARTICLE_POS_(p, d_xi4, i);
#if 1
    {
      unsigned int block_pos_y = __float2int_rd(p.xi[1] * prm.b_dxi[1]);
      unsigned int block_pos_z = __float2int_rd(p.xi[2] * prm.b_dxi[2]);
      int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

      int block_idx;
      if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
	block_idx = CUDA_BND_S_OOB;
      } else {
	int bidx = block_pos_z * prm.b_mx[1] + block_pos_y + p_nr * nr_blocks;
	int b_diff = bid - bidx + prm.b_mx[1] + 1;
	int d1 = b_diff % prm.b_mx[1];
	int d2 = b_diff / prm.b_mx[1];
	block_idx = d2 * 3 + d1;
      }
      d_bidx[i] = block_idx;
    }
#endif
    find_idx_off_pos_1st(p.xi, k, h1, xp, real(0.), prm);
    
    int idiff[2] = { k[1] - j[1], k[2] - j[2] };
    real dx[2] = { xp[1] - xm[1], xp[2] - xm[2] };
    real x[2] = { xm[1] - j[1] - real(.5), xm[2] - j[2] - real(.5) };
    int i[2] = { j[1] - ci0[1], j[2] - ci0[2] };

    real x0 = x[0] * idiff[0];
    real x1 = x[1] * idiff[1];
    int d_first = (abs(dx[1]) * (.5f - x0) >= abs(dx[0]) * (.5f - x1));

    int off[2];
    if (d_first == 0) {
      off[0] = idiff[0];
      off[1] = 0;
    } else {
      off[0] = 0;
      off[1] = idiff[1];
    }
    real dx1[2];
    calc_dx1(dx1, x, dx, off);
    curr_2d_vb_cell(i, x, dx1, p.qni_wni, scurr_y, scurr_z, prm);
    curr_2d_vb_cell_upd(i, x, dx1, dx, off);
    
    off[0] = idiff[0] - off[0];
    off[1] = idiff[1] - off[1];
    calc_dx1(dx1, x, dx, off);
    curr_2d_vb_cell(i, x, dx1, p.qni_wni, scurr_y, scurr_z, prm);
    curr_2d_vb_cell_upd(i, x, dx1, dx, off);
    
    curr_2d_vb_cell(i, x, dx, p.qni_wni, scurr_y, scurr_z, prm);
  }
}

// ======================================================================

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__global__ static void
push_mprts_p3(int block_start, struct cuda_params prm, float4 *d_xi4, float4 *d_pxi4,
	      unsigned int *d_off, int nr_total_blocks, unsigned int *d_bidx,
	      float *d_flds0, unsigned int size)
{
  __d_error_count = prm.d_error_count;
  do_read = true;
  do_reduce = true;
  do_write = true;
  do_calc_j = true;

  const int block_stride = (((BLOCKSIZE_Y + 2*SW) * (BLOCKSIZE_Z + 2*SW) + 31) / 32) * 32;
  __shared__ real _scurrx[WARPS_PER_BLOCK * block_stride];
  __shared__ real _scurry[WARPS_PER_BLOCK * block_stride];
  __shared__ real _scurrz[WARPS_PER_BLOCK * block_stride];

  SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> scurr_x(_scurrx);
  SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> scurr_y(_scurry);
  SCurr<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z> scurr_z(_scurrz);

  if (do_write) {
    scurr_x.zero();
    scurr_y.zero();
    scurr_z.zero();
  }

  int grid_dim_y = (prm.b_mx[2] + 1) / 2;
  int tid = threadIdx.x;
  int block_pos[3];
  block_pos[1] = blockIdx.x * 2;
  block_pos[2] = (blockIdx.y % grid_dim_y) * 2;
  block_pos[1] += block_start & 1;
  block_pos[2] += block_start >> 1;
  if (block_pos[1] >= prm.b_mx[1] ||
      block_pos[2] >= prm.b_mx[2])
    return;

  int p = blockIdx.y / grid_dim_y;

  int bid = block_pos_to_block_idx(block_pos, prm.b_mx) + p * prm.b_mx[1] * prm.b_mx[2];
  __shared__ int s_block_end;
  if (tid == 0) {
    ci0[0] = 0;
    ci0[1] = block_pos[1] * BLOCKSIZE_Y;
    ci0[2] = block_pos[2] * BLOCKSIZE_Z;
    s_block_end = d_off[bid + 1];
  }
  __syncthreads();

  int block_begin = d_off[bid];

  for (int i = block_begin + tid; i < s_block_end; i += THREADS_PER_BLOCK) {
    yz_calc_jx(i, d_xi4, d_pxi4, scurr_x, prm);
    yz_calc_jyjz(i, d_xi4, d_pxi4, scurr_y, scurr_z, prm, nr_total_blocks, p, d_bidx, bid);
  }
  
  if (do_write) {
    __syncthreads();
    real *d_flds = d_flds0 + p * size;
    scurr_x.add_to_fld(d_flds, 0, prm);
    scurr_y.add_to_fld(d_flds, 1, prm);
    scurr_z.add_to_fld(d_flds, 2, prm);
  }
}

// ----------------------------------------------------------------------
// cuda_push_mprts_b

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
cuda_push_mprts_b(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);

  if (mprts->nr_patches == 0)
    return;

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, mflds);

  unsigned int size;
  for (int p = 0; p < mflds->nr_patches; p++) {
    struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
    struct psc_fields_cuda *flds_cuda = psc_fields_cuda(flds);
    size = flds->im[0] * flds->im[1] * flds->im[2];
    check(hipMemset(flds_cuda->d_flds + JXI * size, 0,
		     3 * size * sizeof(*flds_cuda->d_flds)));
  }
  
  dim3 dimGrid((prm.b_mx[1] + 1) / 2, ((prm.b_mx[2] + 1) / 2) * mprts->nr_patches);
  
  for (int block_start = 0; block_start < 4; block_start++) {
    push_mprts_p3<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
      <<<dimGrid, THREADS_PER_BLOCK>>>
      (block_start, prm, mprts_cuda->d_xi4, mprts_cuda->d_pxi4, mprts_cuda->d_off,
       mprts_cuda->nr_total_blocks, mprts_cuda->d_bidx,
       mflds_cuda->d_flds, size * mflds->nr_fields);
    cuda_sync_if_enabled();
    }
  
  free_params(&prm);
}

// ======================================================================

EXTERN_C void
yz2x2_1vb_cuda_push_part_p2(struct psc_particles *prts, struct psc_fields *pf)
{
  assert(0);
  //  cuda_push_part_p2<1, 2, 2>(prts, pf);
}

EXTERN_C void
yz2x2_1vb_cuda_push_part_p3(struct psc_particles *prts, struct psc_fields *pf, real *dummy,
			    int block_stride)
{
  assert(0);
  //  cuda_push_part_p3<1, 2, 2>(prts, pf);
}

EXTERN_C void
yz8x8_1vb_cuda_push_part_p2(struct psc_particles *prts, struct psc_fields *pf)
{
  assert(0);
  //  cuda_push_part_p2<1, 8, 8>(prts, pf);
}

EXTERN_C void
yz8x8_1vb_cuda_push_part_p3(struct psc_particles *prts, struct psc_fields *pf, real *dummy,
			    int block_stride)
{
  assert(0);
  //  cuda_push_part_p3<1, 8, 8>(prts, pf);
}






EXTERN_C void
yz4x4_1vb_cuda_push_mprts_a(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  if (mprts->nr_patches == 0) {
    return;
  }

  static int pr_A, pr_B, pr_B1, pr_B2, pr_B3;
  if (!pr_A) {
    pr_A  = prof_register("a", 1., 0, 0);
    pr_B1 = prof_register("a1_reorder", 1., 0, 0);
    pr_B2 = prof_register("a2_reorder", 1., 0, 0);
    pr_B3 = prof_register("a3_reorder", 1., 0, 0);
    pr_B  = prof_register("a_reorder", 1., 0, 0);
  }

  psc_mparticles_cuda_copy_to_dev(mprts);
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (!mprts_cuda->need_reorder) {
    MHERE;
    prof_start(pr_A);
    cuda_push_mprts_a<1, 4, 4>(mprts, mflds);
    prof_stop(pr_A);
  } else {
    prof_start(pr_B1);
#if 0
    cuda_push_mprts_a1_reorder<1, 4, 4>(mprts, mflds);
    psc_mparticles_cuda_swap_alt(mprts);
#endif
    prof_stop(pr_B1);

#if 0
    prof_start(pr_B2);
    cuda_push_mprts_a1_reorder<1, 4, 4>(mprts, mflds);
    psc_mparticles_cuda_swap_alt(mprts);
    prof_stop(pr_B2);

    prof_start(pr_B3);
    cuda_push_mprts_a1_reorder_x3<1, 4, 4>(mprts, mflds);
    prof_stop(pr_B3);
#endif

    prof_start(pr_B);
    cuda_push_mprts_a_reorder<1, 4, 4>(mprts, mflds);
    mprts_cuda->need_reorder = false;
    prof_stop(pr_B);
  }
}

EXTERN_C void
yz4x4_1vb_cuda_push_mprts_b(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  cuda_push_mprts_b<1, 4, 4>(mprts, mflds);
}
