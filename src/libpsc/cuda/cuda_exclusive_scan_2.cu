
#include "psc_cuda.h"
#include "particles_cuda.h"

#include <thrust/functional.h>
#include <thrust/transform_scan.h>

struct count_if_equal : public thrust::unary_function<unsigned int, unsigned int> {
  const unsigned int value;

  __device__ __host__ count_if_equal(unsigned int _value) : value(_value) { }

  __device__ __host__ unsigned int operator()(unsigned int value_in) {
    return value_in == value;
  }
};

EXTERN_C int
cuda_exclusive_scan_2(struct psc_particles *prts, unsigned int *_d_vals,
		      unsigned int *_d_sums)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  thrust::device_ptr<unsigned int> d_vals(_d_vals);
  thrust::device_ptr<unsigned int> d_sums(_d_sums);

  count_if_equal unary_op(cuda->nr_blocks);
  thrust::transform_exclusive_scan(d_vals, d_vals + prts->n_part, d_sums, unary_op,
				   0, thrust::plus<unsigned int>());

  // OPT, don't mv to host
  int sum = d_sums[prts->n_part - 1] + (d_vals[prts->n_part - 1] == cuda->nr_blocks);
  return sum;
}

EXTERN_C int
_cuda_exclusive_scan_2(struct psc_particles *prts, unsigned int *d_bidx,
		       unsigned int *d_sums)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  unsigned int *bidx = new unsigned int[prts->n_part];
  unsigned int *sums = new unsigned int[prts->n_part];
  check(hipMemcpy(bidx, d_bidx, prts->n_part * sizeof(*bidx),
		   hipMemcpyDeviceToHost));

  unsigned int sum = 0;
  for (int i = 0; i < prts->n_part; i++) {
    sums[i] = sum;
    sum += (bidx[i] == cuda->nr_blocks ? 1 : 0);
  }

  check(hipMemcpy(d_sums, sums, prts->n_part * sizeof(*d_sums),
		   hipMemcpyHostToDevice));
  delete[] sums;
  delete[] bidx;
  return sum;
}

void
cuda_mprts_scan_send_buf(struct cuda_mprts *cuda_mprts)
{
  for (int p = 0; p < cuda_mprts->nr_patches; p++) {
    struct psc_particles *prts = cuda_mprts->mprts_cuda[p];
    struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
    cuda->bnd_n_send = cuda_exclusive_scan_2(prts, cuda->d_part.bidx, cuda->d_part.sums);
    cuda->bnd_n_part_save = prts->n_part;
  }
}
