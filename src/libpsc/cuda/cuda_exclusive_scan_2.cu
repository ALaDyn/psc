#include "hip/hip_runtime.h"

#undef _GLIBCXX_USE_INT128

#include "cuda_mparticles.h"

#include <thrust/functional.h>
#include <thrust/transform_scan.h>
#include <thrust/count.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "psc_cuda.h"
#include "particles_cuda.h"

#include <b40c/radixsort_scanscatter_kernel4.h>

using namespace b40c_thrust;

typedef unsigned int K;
typedef unsigned int V;

static const int RADIX_BITS = 4;

struct count_if_equal : public thrust::unary_function<unsigned int, unsigned int> {
  const unsigned int value;

  __device__ __host__ count_if_equal(unsigned int _value) : value(_value) { }

  __device__ __host__ unsigned int operator()(unsigned int value_in) {
    return value_in == value;
  }
};

#if 0

EXTERN_C int
cuda_exclusive_scan_2(struct psc_particles *prts, unsigned int *_d_vals,
		      unsigned int *_d_sums, int n_prts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(prts->mprts)->cmprts;
  thrust::device_ptr<unsigned int> d_vals(_d_vals);
  thrust::device_ptr<unsigned int> d_sums(_d_sums);
  
  count_if_equal unary_op(cmprts->n_blocks_per_patch);
  thrust::transform_exclusive_scan(d_vals, d_vals + n_prts, d_sums, unary_op,
				   0, thrust::plus<unsigned int>());

  // OPT, don't mv to host
  int sum = d_sums[n_prts - 1] + (d_vals[n_prts - 1] == cmprts->n_blocks_per_patch);
  return sum;
}

EXTERN_C int
_cuda_exclusive_scan_2(struct psc_particles *prts, unsigned int *d_bidx,
		       unsigned int *d_sums, int n_prts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(prts->mprts)->cmprts;
  unsigned int *bidx = new unsigned int[n_prts];
  unsigned int *sums = new unsigned int[n_prts];
  check(hipMemcpy(bidx, d_bidx, n_prts * sizeof(*bidx),
		   hipMemcpyDeviceToHost));

  unsigned int sum = 0;
  for (int i = 0; i < n_prts; i++) {
    sums[i] = sum;
    sum += (bidx[i] == cmprts->n_blocks_per_patch ? 1 : 0);
  }

  check(hipMemcpy(d_sums, sums, n_prts * sizeof(*d_sums),
		   hipMemcpyHostToDevice));
  delete[] sums;
  delete[] bidx;
  return sum;
}

#endif

void
cuda_mprts_find_n_send(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  unsigned int n_blocks = cmprts->n_blocks;

  thrust::device_ptr<unsigned int> d_spine_sums(cmprts->bnd.d_bnd_spine_sums);
  thrust::host_vector<unsigned int> h_spine_sums(n_blocks + 1);

  thrust::copy(d_spine_sums + n_blocks * 10,
	       d_spine_sums + n_blocks * 11 + 1,
	       h_spine_sums.begin());

  unsigned int off = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    unsigned int n_send = h_spine_sums[(p + 1) * cmprts->n_blocks_per_patch];
    cmprts->bnd.bpatch[p].n_send = n_send - off;
    off = n_send;
  }
  cmprts->bnd.n_prts_send = off;
}

// ======================================================================
// cuda_mprts_reorder_send_by_id

static void __global__
mprts_reorder_send_by_id(unsigned int nr_prts_send, unsigned int *d_xchg_ids,
			 float4 *d_xi4, float4 *d_pxi4,
			 float4 *d_xchg_xi4, float4 *d_xchg_pxi4)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (n >= nr_prts_send) {
    return;
  }

  unsigned int id = d_xchg_ids[n];
  d_xchg_xi4[n]  = d_xi4[id];
  d_xchg_pxi4[n] = d_pxi4[id];
}


void
cuda_mprts_reorder_send_by_id(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  if (cmprts->bnd.n_prts_send == 0) {
    return;
  }

  int dimGrid = (cmprts->bnd.n_prts_send + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  mprts_reorder_send_by_id<<<dimGrid, THREADS_PER_BLOCK>>>
    (cmprts->bnd.n_prts_send, cmprts->d_id + cmprts->n_prts - cmprts->bnd.n_prts_send,
     cmprts->d_xi4, cmprts->d_pxi4,
     cmprts->d_xi4 + cmprts->n_prts, cmprts->d_pxi4 + cmprts->n_prts);
  cuda_sync_if_enabled();
}

void
cuda_mprts_reorder_send_by_id_gold(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;
  
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::host_vector<unsigned int> h_id(d_id, d_id + cmprts->n_prts);
  thrust::host_vector<float4> h_xi4(d_xi4, d_xi4 + cmprts->n_prts + cmprts->bnd.n_prts_send);
  thrust::host_vector<float4> h_pxi4(d_pxi4, d_pxi4 + cmprts->n_prts + cmprts->bnd.n_prts_send);
  
  for (int n = 0; n < cmprts->bnd.n_prts_send; n++) {
    unsigned int id = h_id[cmprts->n_prts - cmprts->bnd.n_prts_send + n];
    h_xi4[cmprts->n_prts + n]  = h_xi4[id];
    h_pxi4[cmprts->n_prts + n] = h_pxi4[id];
  }

  thrust::copy(h_xi4.begin(), h_xi4.end(), d_xi4);
  thrust::copy(h_pxi4.begin(), h_pxi4.end(), d_pxi4);
}

// ======================================================================
// cuda_mprts_scan_send_buf_total

void
cuda_mprts_scan_send_buf_total_gold(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  unsigned int n_blocks = cmprts->n_blocks;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_sums(cmprts->bnd.d_sums);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_spine_sums(cmprts->bnd.d_bnd_spine_sums);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + n_blocks + 1);
  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_sums(d_sums, d_sums + cmprts->n_prts);
  
  for (unsigned int bid = 0; bid < n_blocks; bid++) {
    unsigned int sum = d_spine_sums[n_blocks * 10 + bid];
    for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
      if (h_bidx[n] == CUDA_BND_S_OOB) {
	h_sums[n] = sum;
	sum++;
      }
    }
  }

  thrust::copy(h_sums.begin(), h_sums.end(), d_sums);

  cuda_mprts_reorder_send_buf_total(mprts);
}

void
cuda_mprts_scan_send_buf_total(struct psc_mparticles *mprts)
{
  struct cuda_mparticles *cmprts = psc_mparticles_cuda(mprts)->cmprts;

  unsigned int n_blocks = cmprts->n_blocks;
  int *b_mx = cmprts->b_mx;

  // OPT, we could do this from the beginning and adapt find_n_send()
  thrust::device_ptr<unsigned int> d_spine_cnts(cmprts->bnd.d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(cmprts->bnd.d_bnd_spine_sums);
  thrust::exclusive_scan(d_spine_cnts + n_blocks * 10,
			 d_spine_cnts + n_blocks * 11 + 1,
			 d_spine_sums + n_blocks * 10,
			 cmprts->n_prts - cmprts->bnd.n_prts_send);
  // OPT, we could somehow not fill in ids for not oob at all
  // this should make sure at least those within bounds don't screw anything up
  thrust::fill(d_spine_sums, d_spine_sums + n_blocks * 10, 0);

  if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    ScanScatterDigits4<K, V, 0, RADIX_BITS, 0,
		       NopFunctor<K>,
		       NopFunctor<K>,
		       8, 8> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (cmprts->bnd.d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    ScanScatterDigits4<K, V, 0, RADIX_BITS, 0,
		       NopFunctor<K>,
		       NopFunctor<K>,
		       16, 16> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (cmprts->bnd.d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    ScanScatterDigits4<K, V, 0, RADIX_BITS, 0,
		       NopFunctor<K>,
		       NopFunctor<K>,
		       32, 32> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (cmprts->bnd.d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    ScanScatterDigits4<K, V, 0, RADIX_BITS, 0,
		       NopFunctor<K>,
		       NopFunctor<K>,
		       64, 64> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (cmprts->bnd.d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    ScanScatterDigits4<K, V, 0, RADIX_BITS, 0,
                       NopFunctor<K>,
                       NopFunctor<K>,
                       128, 128>
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (cmprts->bnd.d_bnd_spine_sums, cmprts->d_bidx,
       cmprts->d_id, cmprts->d_off, n_blocks);
  } else {
    mprintf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();

  cuda_mprts_reorder_send_by_id(mprts);
}

