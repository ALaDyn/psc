#include "hip/hip_runtime.h"

#include "psc_cuda.h"

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 4
#define BLOCKSIZE_Z 4

#define PFX(x) cuda_bnd_##x
#include "constants.c"

#define SW (3)

__global__ static void
fill_ghosts(real *d_flds, int mb, int me)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_mx[1] && iz < d_mx[2]))
    return;

  bool inside = true;
  int jy = iy, jz = iz;
  if (jy < SW           ) { jy += d_mx[1] - 2*SW; inside = false; }
  if (jy >= d_mx[1] - SW) { jy -= d_mx[1] - 2*SW; inside = false; }
  if (jz < SW           ) { jz += d_mx[2] - 2*SW; inside = false; }
  if (jz >= d_mx[2] - SW) { jz -= d_mx[2] - 2*SW; inside = false; }

  if (inside)
    return;

  for (int m = mb; m < me; m++) {
    F3_DEV(m, 0,iy-SW,iz-SW) = F3_DEV(m, 0,jy-SW,jz-SW);
  }
}

EXTERN_C void
cuda_fill_ghosts(int p, fields_cuda_t *pf, int mb, int me)
{
  particles_cuda_t pp;
  cuda_bnd_set_constants(&pp, pf);

  struct psc_patch *patch = &ppsc->patch[p];
  int dimBlock[2] = { BLOCKSIZE_Y, BLOCKSIZE_Z };
  int dimGrid[2]  = { (patch->ldims[1] + 2*SW + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		      (patch->ldims[2] + 2*SW + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  RUN_KERNEL(dimGrid, dimBlock,
	     fill_ghosts, (pf->d_flds, mb, me));
}

__global__ static void
add_ghosts(real *d_flds, int mb, int me)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_mx[1] - 2*SW && iz < d_mx[2] - 2*SW))
    return;

  if (iy < SW) {
    int jy = iy + (d_mx[1] - 2*SW);
    int jz = iz;
    for (int m = mb; m < me; m++) {
      F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
    }
    if (iz < SW) {
      jz = iz + (d_mx[2] - 2*SW);
      for (int m = mb; m < me; m++) {
	F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
      }
    }
    if (iz >= d_mx[2] - 3*SW) {
      jz = iz - (d_mx[2] - 2*SW);
      for (int m = mb; m < me; m++) {
	F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
      }
    }
  }
  if (iy >= d_mx[1] - 3*SW) {
    int jy = iy - (d_mx[1] - 2*SW);
    int jz = iz;
    for (int m = mb; m < me; m++) {
      F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
    }
    if (iz < SW) {
      jz = iz + (d_mx[2] - 2*SW);
      for (int m = mb; m < me; m++) {
	F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
      }
    }
    if (iz >= d_mx[2] - 3*SW) {
      jz = iz - (d_mx[2] - 2*SW);
      for (int m = mb; m < me; m++) {
	F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
      }
    }
  }
  if (iz < SW) {
    int jy = iy, jz = iz + (d_mx[2] - 2*SW);
    for (int m = mb; m < me; m++) {
      F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
    }
  }
  if (iz >= d_mx[2] - 3*SW) {
    int jy = iy, jz = iz - (d_mx[2] - 2*SW);
    for (int m = mb; m < me; m++) {
      F3_DEV(m, 0,iy,iz) += F3_DEV(m, 0,jy,jz);
    }
  }
}

EXTERN_C void
cuda_add_ghosts(int p, fields_cuda_t *pf, int mb, int me)
{
  particles_cuda_t pp;
  cuda_bnd_set_constants(&pp, pf);

  struct psc_patch *patch = &ppsc->patch[p];
  int dimBlock[2] = { BLOCKSIZE_Y, BLOCKSIZE_Z };
  int dimGrid[2]  = { (patch->ldims[1] + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		      (patch->ldims[2] + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  RUN_KERNEL(dimGrid, dimBlock,
	     add_ghosts, (pf->d_flds, mb, me));
}

