#include "hip/hip_runtime.h"

#include "grid.hxx"
#include "fields.hxx"
#include "cuda_mfields.h"
#include "cuda_mparticles.h"

#include "../vpic/PscRng.h"

#include "gtest/gtest.h"

// Rng hackiness

using Rng = PscRng;
using RngPool = PscRngPool<Rng>;

// enum hackiness

enum IP { // FIXME, dup
  IP_STD, // standard interpolation
  IP_EC,  // energy-conserving interpolation
};

enum DEPOSIT { // FIXME, dup
  DEPOSIT_VB_2D,
  DEPOSIT_VB_3D,
};

enum CURRMEM { // FIXME, dup
  CURRMEM_SHARED,
  CURRMEM_GLOBAL,
};

// profile hackiness

#include "mrc_profile.h"

struct prof_globals prof_globals; // FIXME

int
prof_register(const char *name, float simd, int flops, int bytes)
{
  return 0;
}

// ======================================================================
// class PushMprtsTest

struct PushMprtsTestParam
{
  double q;
  double m;
};

// ======================================================================
// class TestAccel

class TestAccel
{
  enum { // FIXME, duplicated
    JXI, JYI, JZI,
    EX , EY , EZ ,
    HX , HY , HZ ,
    N_FIELDS,
  };

public:
  TestAccel()
    : grid_({ 1, 1, 1 }, { L, L, L })
  {
  }

  ~TestAccel()
  {
    delete cmflds_;
  }

  void init_cmflds()
  {
    cmflds_ = new cuda_mfields(grid_, N_FIELDS, { 0, 2, 2 });
    fields_single_t flds = cmflds_->get_host_fields();
    Fields3d<fields_single_t> F(flds);

    F(EX, 0,0,0) = 1;
    F(EX, 0,1,0) = 1;
    F(EX, 0,0,1) = 1;
    F(EX, 0,1,1) = 1;
    
    F(EY, 0,0,0) = 2;
    F(EY, 0,0,1) = 2;
    //    F(EY, 1,0,0) = 2;
    //    F(EY, 1,0,1) = 2;
    
    F(EZ, 0,0,0) = 3;
    //    F(EZ, 1,0,0) = 3;
    F(EZ, 0,1,0) = 3;
    //    F(EZ, 1,1,0) = 3;

    cmflds_->copy_to_device(0, flds, 0, N_FIELDS);
    cmflds_->dump("accel.fld.json");
    flds.dtor();
  };

  double L = 1e10;
  unsigned int n_prts = 131;
  int n_steps = 10;
  cuda_mparticles::real_t eps = 1e-5;
  Int3 bs_ = { 1, 1, 1 };
  
  Grid_t grid_;
  cuda_mfields* cmflds_;
};

// ======================================================================
// class TestCyclo

class TestCyclo
{
  enum { // FIXME, duplicated
    JXI, JYI, JZI,
    EX , EY , EZ ,
    HX , HY , HZ ,
    N_FIELDS,
  };

public:
  TestCyclo()
    : grid_({ 1, 1, 1 }, { L, L, L })
  {
  }

  ~TestCyclo()
  {
    delete cmflds_;
  }

  void init_cmflds()
  {
    cmflds_ = new cuda_mfields(grid_, N_FIELDS, { 0, 2, 2 });
    fields_single_t flds = cmflds_->get_host_fields();
    Fields3d<fields_single_t> F(flds);

    F(HZ, 0,0,0) = 2. * M_PI / n_steps;
    F(HZ, 0,0,1) = 2. * M_PI / n_steps;

    cmflds_->copy_to_device(0, flds, 0, N_FIELDS);
    cmflds_->dump("cyclo.fld.json");
    flds.dtor();
  };

  double L = 1e10;
  unsigned int n_prts = 131;
  int n_steps = 64;
  cuda_mparticles::real_t eps = 1e-2;
  Int3 bs_ = { 1, 1, 1 };
  
  Grid_t grid_;
  cuda_mfields* cmflds_;
};

struct PushMprtsAccelTest : ::testing::TestWithParam<PushMprtsTestParam>
{
  TestAccel test;
  cuda_mparticles* cmprts_;

  void SetUp()
  {
    PushMprtsTestParam prm = GetParam();
  
    test.grid_.kinds.push_back(Grid_t::Kind(prm.q, prm.m, "test_species"));
    test.init_cmflds();

    cmprts_ = new cuda_mparticles(test.grid_, test.bs_);
    test.init_cmprts(cmprts_);
  }
};

struct PushMprtsCycloTest : ::testing::TestWithParam<PushMprtsTestParam>
{
  TestCyclo test;
  cuda_mparticles *cmprts_;
  
  void SetUp()
  {
    PushMprtsTestParam prm = GetParam();
  
    test.grid_.kinds.push_back(Grid_t::Kind(prm.q, prm.m, "test_species"));
    test.init_cmflds();

    cmprts_ = new cuda_mparticles(test.grid_, test.bs_);
    test.init_cmprts(cmprts_);
  }
};

TEST_P(PushMprtsAccelTest, Accel)
{
  // init particles
  RngPool rngpool;
  Rng *rng = rngpool[0];
  
  uint n_prts_by_patch[1] = { test.n_prts };
  
  cmprts_->reserve_all(n_prts_by_patch);
  
  std::vector<cuda_mparticles_prt> prts;
  prts.reserve(test.n_prts);
  
  for (int i = 0; i < test.n_prts; i++) {
    cuda_mparticles_prt prt = {};
    prt.xi[0] = rng->uniform(0, test.L);
    prt.xi[1] = rng->uniform(0, test.L);
    prt.xi[2] = rng->uniform(0, test.L);
    prt.qni_wni = 1.;
    
    prts.push_back(prt);
  }
  cmprts_->inject(prts.data(), n_prts_by_patch);
  //cmprts_->dump();
  
  int n_failed = 0;
  int n_steps = test.n_steps;
  
  for (int n = 0; n < n_steps; n++) {
    cuda_push_mprts_yz(cmprts_, test.cmflds_, test.bs_, IP_EC, DEPOSIT_VB_3D, CURRMEM_GLOBAL);
    cmprts_->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	if (std::abs(prt.pxi[0] - 1*(n+1)) > test.eps ||
	    std::abs(prt.pxi[1] - 2*(n+1)) > test.eps ||
	    std::abs(prt.pxi[2] - 3*(n+1)) > test.eps) {
	  printf("FAIL: n %d i %d px %g %g %g // exp %g %g %g\n", n, i,
		 prt.pxi[0], prt.pxi[1], prt.pxi[2],
		 1.*(n+1), 2.*(n+1), 3.*(n+1));
	  n_failed++;
	}
      });
    
    //cmprts_->dump();
  }
  EXPECT_EQ(n_failed, 0);
}

TEST_P(PushMprtsCycloTest, Cyclo)
{
  // init particles
  RngPool rngpool;
  Rng *rng = rngpool[0];
  
  uint n_prts_by_patch[1] = { test.n_prts };
  
  cmprts_->reserve_all(n_prts_by_patch);
  
  std::vector<cuda_mparticles_prt> prts;
  prts.reserve(test.n_prts);
  
  for (int i = 0; i < test.n_prts; i++) {
    cuda_mparticles_prt prt = {};
    prt.xi[0] = rng->uniform(0, test.L);
    prt.xi[1] = rng->uniform(0, test.L);
    prt.xi[2] = rng->uniform(0, test.L);
    prt.pxi[0] = 1.; // gamma = 2
    prt.pxi[1] = 1.;
    prt.pxi[2] = 1.;
    prt.qni_wni = rng->uniform(0, 1.);;
    
    prts.push_back(prt);
  }
  cmprts_->inject(prts.data(), n_prts_by_patch);
  //cmprts_->dump();
  
  int n_failed = 0;
  int n_steps = test.n_steps;
  
  for (int n = 0; n < n_steps; n++) {
    cuda_push_mprts_yz(cmprts_, test.cmflds_, test.bs_, IP_EC, DEPOSIT_VB_3D, CURRMEM_GLOBAL);
    double ux = (cos(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		 cos(2*M_PI*(0.125*n_steps)      /(double)n_steps));
    double uy = (sin(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		 sin(2*M_PI*(0.125*n_steps)      /(double)n_steps));
    double uz = 1.;
    cmprts_->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	if (std::abs(prt.pxi[0] - ux) > test.eps ||
	    std::abs(prt.pxi[1] - uy) > test.eps ||
	    std::abs(prt.pxi[2] - uz) > test.eps) {
	  printf("FAIL: n %d i %d px %g %g %g // exp %g %g %g\n", n, i,
		 prt.pxi[0], prt.pxi[1], prt.pxi[2], ux, uy, uz);
	  n_failed++;
	}
      });
    
    //cmprts_->dump();
  }
  EXPECT_EQ(n_failed, 0);
}

INSTANTIATE_TEST_CASE_P(T1,
                        PushMprtsAccelTest,
                        ::testing::Values(PushMprtsTestParam{1., 1.}));
INSTANTIATE_TEST_CASE_P(T1,
                        PushMprtsCycloTest,
                        ::testing::Values(PushMprtsTestParam{2., 1.}));
