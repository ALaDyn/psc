#include "hip/hip_runtime.h"

#include "grid.hxx"
#include "fields.hxx"
#include "cuda_mfields.h"
#include "cuda_mparticles.h"

#include "../vpic/PscRng.h"

using Rng = PscRng;
using RngPool = PscRngPool<Rng>;

#include "mrc_profile.h"

enum IP { // FIXME, dup
  IP_STD, // standard interpolation
  IP_EC,  // energy-conserving interpolation
};

enum DEPOSIT { // FIXME, dup
  DEPOSIT_VB_2D,
  DEPOSIT_VB_3D,
};

enum CURRMEM { // FIXME, dup
  CURRMEM_SHARED,
  CURRMEM_GLOBAL,
};

struct prof_globals prof_globals; // FIXME

int
prof_register(const char *name, float simd, int flops, int bytes)
{
  return 0;
}

// ======================================================================
// class TestAccel

class TestAccel
{
  enum { // FIXME, duplicated
    JXI, JYI, JZI,
    EX , EY , EZ ,
    HX , HY , HZ ,
    N_FIELDS,
  };

public:
  TestAccel()
    : grid_({ 1, 1, 1 }, { L, L, L })
  {
    bs_ = { 1, 1, 1 };
    init_grid();
    init_cmflds();
    init_cmprts();
  }

  ~TestAccel()
  {
    delete cmflds_;
    delete cmprts_;
  }

  void init_grid()
  {
    grid_.kinds.push_back(Grid_t::Kind(1.,  1., "test_species"));
  }
  
  void init_cmflds()
  {
    cmflds_ = new cuda_mfields(grid_, N_FIELDS, { 0, 2, 2 });
    fields_single_t flds = cmflds_->get_host_fields();
    Fields3d<fields_single_t> F(flds);

    F(EX, 0,0,0) = 1;
    F(EX, 0,1,0) = 1;
    F(EX, 0,0,1) = 1;
    F(EX, 0,1,1) = 1;
    
    F(EY, 0,0,0) = 2;
    F(EY, 0,0,1) = 2;
    //    F(EY, 1,0,0) = 2;
    //    F(EY, 1,0,1) = 2;
    
    F(EZ, 0,0,0) = 3;
    //    F(EZ, 1,0,0) = 3;
    F(EZ, 0,1,0) = 3;
    //    F(EZ, 1,1,0) = 3;

    cmflds_->copy_to_device(0, flds, 0, N_FIELDS);
    cmflds_->dump("accel.fld.json");
    flds.dtor();
  };

  void init_cmprts()
  {
    RngPool rngpool;
    Rng *rng = rngpool[0];

    uint n_prts_by_patch[1] = { n_prts };
    
    cmprts_ = new cuda_mparticles(grid_, bs_);
    cmprts_->reserve_all(n_prts_by_patch);
    
    std::vector<cuda_mparticles_prt> prts;
    prts.reserve(n_prts);
    
    for (int i = 0; i < n_prts; i++) {
      cuda_mparticles_prt prt = {};
      prt.xi[0] = rng->uniform(0, L);
      prt.xi[1] = rng->uniform(0, L);
      prt.xi[2] = rng->uniform(0, L);
      prt.qni_wni = 1.;

      prts.push_back(prt);
    }
    cmprts_->inject(prts.data(), n_prts_by_patch);

    //cmprts_->dump();
  }

  void run()
  {
    int n_failed = 0;

    for (int n = 0; n < n_steps; n++) {
      printf("advancing step %d\n", n);
      cuda_push_mprts_yz(cmprts_, cmflds_, bs_, IP_EC, DEPOSIT_VB_3D, CURRMEM_GLOBAL);
      cmprts_->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	  if (std::abs(prt.pxi[0] - 1*(n+1)) > eps ||
	      std::abs(prt.pxi[1] - 2*(n+1)) > eps ||
	      std::abs(prt.pxi[2] - 3*(n+1)) > eps) {
	    printf("FAIL: n %d i %d px %g %g %g // exp %g %g %g\n", n, i,
		   prt.pxi[0], prt.pxi[1], prt.pxi[2],
		   1.*(n+1), 2.*(n+1), 3.*(n+1));
	    n_failed++;
	  }
	});
      
      //cmprts_->dump();
    }
    assert(n_failed == 0);
  }

private:
  double L = 1e10;
  unsigned int n_prts = 131;
  int n_steps = 10;
  cuda_mparticles::real_t eps = 1e-5;
  Int3 bs_;
  
  Grid_t grid_;
  cuda_mfields* cmflds_;
  cuda_mparticles* cmprts_;
};

// ======================================================================
// class TestCyclo

class TestCyclo
{
  enum { // FIXME, duplicated
    JXI, JYI, JZI,
    EX , EY , EZ ,
    HX , HY , HZ ,
    N_FIELDS,
  };

public:
  TestCyclo()
    : grid_({ 1, 1, 1 }, { L, L, L })
  {
    bs_ = { 1, 1, 1 };
    init_grid();
    init_cmflds();
    init_cmprts();
  }

  ~TestCyclo()
  {
    delete cmflds_;
    delete cmprts_;
  }

  void init_grid()
  {
    grid_.kinds.push_back(Grid_t::Kind(2.,  1., "test_species"));
  }
  
  void init_cmflds()
  {
    cmflds_ = new cuda_mfields(grid_, N_FIELDS, { 0, 2, 2 });
    fields_single_t flds = cmflds_->get_host_fields();
    Fields3d<fields_single_t> F(flds);

    F(HZ, 0,0,0) = 2. * M_PI / n_steps;
    F(HZ, 0,0,1) = 2. * M_PI / n_steps;

    cmflds_->copy_to_device(0, flds, 0, N_FIELDS);
    cmflds_->dump("cyclo.fld.json");
    flds.dtor();
  };

  void init_cmprts()
  {
    RngPool rngpool;
    Rng *rng = rngpool[0];

    uint n_prts_by_patch[1] = { n_prts };
    
    cmprts_ = new cuda_mparticles(grid_, bs_);
    cmprts_->reserve_all(n_prts_by_patch);
    
    std::vector<cuda_mparticles_prt> prts;
    prts.reserve(n_prts);
    
    for (int i = 0; i < n_prts; i++) {
      cuda_mparticles_prt prt = {};
      prt.xi[0] = rng->uniform(0, L);
      prt.xi[1] = rng->uniform(0, L);
      prt.xi[2] = rng->uniform(0, L);
      prt.pxi[0] = 1.; // gamma = 2
      prt.pxi[1] = 1.;
      prt.pxi[2] = 1.;
      prt.qni_wni = rng->uniform(0, 1.);;

      prts.push_back(prt);
    }
    cmprts_->inject(prts.data(), n_prts_by_patch);

    //cmprts_->dump();
  }

  void run()
  {
    int n_failed = 0;

    for (int n = 0; n < n_steps; n++) {
      printf("advancing step %d\n", n);
      cuda_push_mprts_yz(cmprts_, cmflds_, bs_, IP_EC, DEPOSIT_VB_3D, CURRMEM_GLOBAL);
      double ux = (cos(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		   cos(2*M_PI*(0.125*n_steps)      /(double)n_steps));
      double uy = (sin(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		   sin(2*M_PI*(0.125*n_steps)      /(double)n_steps));
      double uz = 1.;
      cmprts_->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	  if (std::abs(prt.pxi[0] - ux) > eps ||
	      std::abs(prt.pxi[1] - uy) > eps ||
	      std::abs(prt.pxi[2] - uz) > eps) {
	    printf("FAIL: n %d i %d px %g %g %g // exp %g %g %g\n", n, i,
		   prt.pxi[0], prt.pxi[1], prt.pxi[2], ux, uy, uz);
	    n_failed++;
	  }
	});
  
      //cmprts_->dump();
    }
    assert(n_failed == 0);
  }

private:
  double L = 1e10;
  unsigned int n_prts = 131;
  int n_steps = 64;
  cuda_mparticles::real_t eps = 1e-2;
  Int3 bs_;
  
  Grid_t grid_;
  cuda_mfields* cmflds_;
  cuda_mparticles* cmprts_;
};

// ----------------------------------------------------------------------
// main

int
main(void)
{
  printf("=== Test: accel\n");
  TestAccel test_accel;
  test_accel.run();

  printf("=== Test: cyclo\n");
  TestCyclo test_cyclo;
  test_cyclo.run();
}
