#include "hip/hip_runtime.h"

#include "grid.hxx"
#include "fields.hxx"
#include "cuda_mfields.h"
#include "cuda_mparticles.h"

#include "../vpic/PscRng.h"

#include "gtest/gtest.h"

// Rng hackiness

using Rng = PscRng;
using RngPool = PscRngPool<Rng>;

// enum hackiness

enum IP { // FIXME, dup
  IP_STD, // standard interpolation
  IP_EC,  // energy-conserving interpolation
};

enum DEPOSIT { // FIXME, dup
  DEPOSIT_VB_2D,
  DEPOSIT_VB_3D,
};

enum CURRMEM { // FIXME, dup
  CURRMEM_SHARED,
  CURRMEM_GLOBAL,
};

enum { // FIXME, duplicated
  JXI, JYI, JZI,
  EX , EY , EZ ,
  HX , HY , HZ ,
  N_FIELDS,
};

// profile hackiness

#include "mrc_profile.h"

struct prof_globals prof_globals; // FIXME

int
prof_register(const char *name, float simd, int flops, int bytes)
{
  return 0;
}

// ======================================================================
// class PushMprtsTest

struct PushMprtsTestParam
{
  double q;
  double m;
  unsigned int n_prts;
  unsigned int n_steps;
  cuda_mparticles::real_t eps;
};

struct PushMprtsAccelTest : ::testing::TestWithParam<PushMprtsTestParam>
{
  Grid_t* grid_;
  cuda_mparticles* cmprts_;
  cuda_mfields* cmflds_;

  const double L = 1e10;
  const Int3 bs_ = { 1, 1, 1 };
  
  void SetUp()
  {
    PushMprtsTestParam prm = GetParam();
  
    grid_ = new Grid_t({ 1, 1, 1 }, { L, L, L });
    grid_->kinds.push_back(Grid_t::Kind(prm.q, prm.m, "test_species"));

    cmflds_ = new cuda_mfields(*grid_, N_FIELDS, { 0, 2, 2 });
    cmprts_ = new cuda_mparticles(*grid_, bs_);
  }
};

struct PushMprtsCycloTest : ::testing::TestWithParam<PushMprtsTestParam>
{
  Grid_t *grid_;
  cuda_mparticles *cmprts_;
  cuda_mfields *cmflds_;

  const double L = 1e10;
  const Int3 bs_ = { 1, 1, 1 };
  
  void SetUp()
  {
    PushMprtsTestParam prm = GetParam();
  
    grid_ = new Grid_t({ 1, 1, 1 }, { L, L, L });
    grid_->kinds.push_back(Grid_t::Kind(prm.q, prm.m, "test_species"));

    cmflds_ = new cuda_mfields(*grid_, N_FIELDS, { 0, 2, 2 });
    cmprts_ = new cuda_mparticles(*grid_, bs_);
  }
};

TEST_P(PushMprtsAccelTest, Accel)
{
  PushMprtsTestParam prm = GetParam();

  // init fields
  fields_single_t flds = cmflds_->get_host_fields();
  Fields3d<fields_single_t> F(flds);
  
  F(EX, 0,0,0) = 1;
  F(EX, 0,1,0) = 1;
  F(EX, 0,0,1) = 1;
  F(EX, 0,1,1) = 1;
  
  F(EY, 0,0,0) = 2;
  F(EY, 0,0,1) = 2;
  //    F(EY, 1,0,0) = 2;
  //    F(EY, 1,0,1) = 2;
  
  F(EZ, 0,0,0) = 3;
  //    F(EZ, 1,0,0) = 3;
  F(EZ, 0,1,0) = 3;
  //    F(EZ, 1,1,0) = 3;
  
  cmflds_->copy_to_device(0, flds, 0, N_FIELDS);
  cmflds_->dump("accel.fld.json");
  flds.dtor();
  
  // init particles
  RngPool rngpool;
  Rng *rng = rngpool[0];
  
  uint n_prts_by_patch[1] = { prm.n_prts };
  
  cmprts_->reserve_all(n_prts_by_patch);
  
  std::vector<cuda_mparticles_prt> prts;
  prts.reserve(prm.n_prts);
  
  for (int i = 0; i < prm.n_prts; i++) {
    cuda_mparticles_prt prt = {};
    prt.xi[0] = rng->uniform(0, L);
    prt.xi[1] = rng->uniform(0, L);
    prt.xi[2] = rng->uniform(0, L);
    prt.qni_wni = 1.;
    
    prts.push_back(prt);
  }
  cmprts_->inject(prts.data(), n_prts_by_patch);
  //cmprts_->dump();
  
  int n_failed = 0;
  for (int n = 0; n < prm.n_steps; n++) {
    cuda_push_mprts_yz(cmprts_, cmflds_, bs_, IP_EC, DEPOSIT_VB_3D, CURRMEM_GLOBAL);
    cmprts_->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	if (std::abs(prt.pxi[0] - 1*(n+1)) > prm.eps ||
	    std::abs(prt.pxi[1] - 2*(n+1)) > prm.eps ||
	    std::abs(prt.pxi[2] - 3*(n+1)) > prm.eps) {
	  printf("FAIL: n %d i %d px %g %g %g // exp %g %g %g\n", n, i,
		 prt.pxi[0], prt.pxi[1], prt.pxi[2],
		 1.*(n+1), 2.*(n+1), 3.*(n+1));
	  n_failed++;
	}
      });
    
    //cmprts_->dump();
  }
  EXPECT_EQ(n_failed, 0);
}

TEST_P(PushMprtsCycloTest, Cyclo)
{
  PushMprtsTestParam prm = GetParam();
  int n_steps = prm.n_steps;

  // init fields
  fields_single_t flds = cmflds_->get_host_fields();
  Fields3d<fields_single_t> F(flds);
  
  F(HZ, 0,0,0) = 2. * M_PI / n_steps;
  F(HZ, 0,0,1) = 2. * M_PI / n_steps;
  
  cmflds_->copy_to_device(0, flds, 0, N_FIELDS);
  cmflds_->dump("cyclo.fld.json");
  flds.dtor();

  // init particles
  RngPool rngpool;
  Rng *rng = rngpool[0];
  
  uint n_prts_by_patch[1] = { prm.n_prts };
  
  cmprts_->reserve_all(n_prts_by_patch);
  
  std::vector<cuda_mparticles_prt> prts;
  prts.reserve(prm.n_prts);
  
  for (int i = 0; i < prm.n_prts; i++) {
    cuda_mparticles_prt prt = {};
    prt.xi[0] = rng->uniform(0, L);
    prt.xi[1] = rng->uniform(0, L);
    prt.xi[2] = rng->uniform(0, L);
    prt.pxi[0] = 1.; // gamma = 2
    prt.pxi[1] = 1.;
    prt.pxi[2] = 1.;
    prt.qni_wni = rng->uniform(0, 1.);;
    
    prts.push_back(prt);
  }
  cmprts_->inject(prts.data(), n_prts_by_patch);
  //cmprts_->dump();
  
  int n_failed = 0;
  
  for (int n = 0; n < n_steps; n++) {
    cuda_push_mprts_yz(cmprts_, cmflds_, bs_, IP_EC, DEPOSIT_VB_3D, CURRMEM_GLOBAL);
    double ux = (cos(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		 cos(2*M_PI*(0.125*n_steps)      /(double)n_steps));
    double uy = (sin(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		 sin(2*M_PI*(0.125*n_steps)      /(double)n_steps));
    double uz = 1.;
    cmprts_->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	if (std::abs(prt.pxi[0] - ux) > prm.eps ||
	    std::abs(prt.pxi[1] - uy) > prm.eps ||
	    std::abs(prt.pxi[2] - uz) > prm.eps) {
	  printf("FAIL: n %d i %d px %g %g %g // exp %g %g %g\n", n, i,
		 prt.pxi[0], prt.pxi[1], prt.pxi[2], ux, uy, uz);
	  n_failed++;
	}
      });
    
    //cmprts_->dump();
  }
  EXPECT_EQ(n_failed, 0);
}

INSTANTIATE_TEST_CASE_P(T1,
                        PushMprtsAccelTest,
                        ::testing::Values(PushMprtsTestParam{ 1., 1., 131, 10, 1e-5 }));
INSTANTIATE_TEST_CASE_P(T1,
                        PushMprtsCycloTest,
                        ::testing::Values(PushMprtsTestParam{ 2., 1., 131, 64, 1e-2 }));
