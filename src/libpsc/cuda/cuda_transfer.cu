#include "hip/hip_runtime.h"

#include "psc_cuda.h"

EXTERN_C void
__particles_cuda_alloc(particles_cuda_t *pp, bool need_block_offsets,
		       bool need_cell_offsets)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *d_part = &pp->d_part;

  const int cells_per_block = BLOCKSIZE_X * BLOCKSIZE_Y * BLOCKSIZE_Z;

  check(hipMalloc((void **) &d_part->xi4, n_part * sizeof(float4)));
  check(hipMalloc((void **) &d_part->pxi4, n_part * sizeof(float4)));

  if (need_block_offsets) {
    check(hipMalloc((void **) &d_part->offsets, 
		     (pp->nr_blocks + 1) * sizeof(int)));
    check(hipMemcpy(&d_part->offsets[pp->nr_blocks], &n_part, sizeof(int),
		     hipMemcpyHostToDevice));
  }

  if (need_cell_offsets) {
    check(hipMalloc((void **) &d_part->c_offsets, 
		     (pp->nr_blocks * cells_per_block + 1) * sizeof(int)));
  }

  check(hipMalloc((void **) &d_part->c_pos, 
		   (pp->nr_blocks * cells_per_block * 3) * sizeof(int)));
}

EXTERN_C void
__particles_cuda_to_device(particles_cuda_t *pp, float4 *xi4, float4 *pxi4,
			   int *offsets, int *c_offsets, int *c_pos)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *d_part = &pp->d_part;

  const int cells_per_block = BLOCKSIZE_X * BLOCKSIZE_Y * BLOCKSIZE_Z;

  check(hipMemcpy(d_part->xi4, xi4, n_part * sizeof(*xi4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_part->pxi4, pxi4, n_part * sizeof(*pxi4),
		   hipMemcpyHostToDevice));
  if (offsets) {
    check(hipMemcpy(d_part->offsets, offsets,
		     pp->nr_blocks * sizeof(int), hipMemcpyHostToDevice));
  }
  if (c_offsets) {
    check(hipMemcpy(d_part->c_offsets,c_offsets,
		     (pp->nr_blocks * cells_per_block + 1) * sizeof(int),
		     hipMemcpyHostToDevice));
  }
  check(hipMemcpy(d_part->c_pos, c_pos,
		   (pp->nr_blocks * cells_per_block * 3) * sizeof(int),
		   hipMemcpyHostToDevice));
}

EXTERN_C void
__particles_cuda_from_device(particles_cuda_t *pp, float4 *xi4, float4 *pxi4)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *d_part = &pp->d_part;

  check(hipMemcpy(xi4, d_part->xi4, n_part * sizeof(*xi4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(pxi4, d_part->pxi4, n_part * sizeof(*pxi4),
		   hipMemcpyDeviceToHost));
}

EXTERN_C void
__particles_cuda_free(particles_cuda_t *pp)
{
  particles_cuda_dev_t *d_part = &pp->d_part;

  check(hipFree(d_part->xi4));
  check(hipFree(d_part->pxi4));
  check(hipFree(d_part->offsets));
  check(hipFree(d_part->c_offsets));
  check(hipFree(d_part->c_pos));
}

EXTERN_C void
__fields_cuda_to_device(fields_cuda_t *pf, real *h_flds, int mb, int me)
{
  assert(!ppsc->domain.use_pml);

  unsigned int size = pf->im[0] * pf->im[1] * pf->im[2];
  check(hipMalloc((void **) &pf->d_flds, pf->nr_comp * size * sizeof(float)));
  check(hipMemcpy(pf->d_flds + mb * size,
		   h_flds + mb * size,
		   (me - mb) * size * sizeof(float),
		   hipMemcpyHostToDevice));
}

EXTERN_C void
__fields_cuda_from_device(fields_cuda_t *pf, real *h_flds, int mb, int me)
{
  unsigned int size = pf->im[0] * pf->im[1] * pf->im[2];
  check(hipMemcpy(h_flds + mb * size,
		   pf->d_flds + mb * size,
		   (me - mb) * size * sizeof(float),
		   hipMemcpyDeviceToHost));
  check(hipFree(pf->d_flds));
}
