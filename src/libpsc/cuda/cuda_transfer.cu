#include "hip/hip_runtime.h"

#include "psc_cuda.h"
#include "particles_cuda.h"

#include <mrc_profile.h>

// FIXME, hardcoding is bad, needs to be consistent, etc...
#define BND  (3)
#define MAX_BND_COMPONENTS (3)

EXTERN_C void
__particles_cuda_to_device(struct psc_particles *prts, float4 *xi4, float4 *pxi4)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_mparticles *mprts = cuda->mprts;
  assert(mprts);
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  unsigned int off = 0;
  for (int p = 0; p < prts->p; p++) {
    off += psc_mparticles_get_patch(mprts, p)->n_part;
  }
  int n_part = prts->n_part;

  check(hipMemcpy(mprts_cuda->d_xi4 + off, xi4, n_part * sizeof(*xi4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(mprts_cuda->d_pxi4 + off, pxi4, n_part * sizeof(*pxi4),
		   hipMemcpyHostToDevice));
}

EXTERN_C void
__particles_cuda_from_device(struct psc_particles *prts, float4 *xi4, float4 *pxi4)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  struct psc_mparticles *mprts = cuda->mprts;
  assert(mprts);
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  unsigned int off = 0;
  for (int p = 0; p < prts->p; p++) {
    off += psc_mparticles_get_patch(mprts, p)->n_part;
  }
  int n_part = prts->n_part;

  check(hipMemcpy(xi4, mprts_cuda->d_xi4 + off, n_part * sizeof(*xi4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(pxi4, mprts_cuda->d_pxi4 + off, n_part * sizeof(*pxi4),
		   hipMemcpyDeviceToHost));
}

EXTERN_C void
cuda_copy_bidx_from_dev(struct psc_particles *prts, unsigned int *h_bidx, unsigned int *d_bidx)
{
  check(hipMemcpy(h_bidx, d_bidx, prts->n_part * sizeof(*h_bidx),
		   hipMemcpyDeviceToHost));
}

EXTERN_C void
cuda_copy_bidx_to_dev(struct psc_particles *prts, unsigned int *d_bidx, unsigned int *h_bidx)
{
  check(hipMemcpy(d_bidx, h_bidx, prts->n_part * sizeof(*d_bidx),
		   hipMemcpyHostToDevice));
}

void
__psc_mparticles_cuda_setup(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  if (mprts->nr_patches == 0) {
    return;
  }
  
  // FIXME we assume that every patch will have those same dims
  int *ldims = ppsc->patch[0].ldims;

  if (!mprts->flags) {
    // FIXME, they get set too late, so auto-dispatch "1vb" doesn't work
    mprts->flags = MP_NEED_BLOCK_OFFSETS | MP_BLOCKSIZE_4X4X4 | MP_NO_CHECKERBOARD;
  }

  int bs[3];
  for (int d = 0; d < 3; d++) {
    switch (mprts->flags & MP_BLOCKSIZE_MASK) {
    case MP_BLOCKSIZE_1X1X1: bs[d] = 1; break;
    case MP_BLOCKSIZE_2X2X2: bs[d] = 2; break;
    case MP_BLOCKSIZE_4X4X4: bs[d] = 4; break;
    case MP_BLOCKSIZE_8X8X8: bs[d] = 8; break;
    default: assert(0);
    }
    if (ppsc->domain.gdims[d] == 1) {
      bs[d] = 1;
    }
    mprts_cuda->blocksize[d] = bs[d];
    assert(ldims[d] % bs[d] == 0); // FIXME not sure what breaks if not
    mprts_cuda->b_mx[d] = (ldims[d] + bs[d] - 1) / bs[d];
    // assumes no AMR
    mprts_cuda->b_dxi[d] = 1.f / (mprts_cuda->blocksize[d] * ppsc->patch[0].dx[d]);
  }
  mprts_cuda->nr_blocks = mprts_cuda->b_mx[0] * mprts_cuda->b_mx[1] * mprts_cuda->b_mx[2];
  mprts_cuda->nr_total_blocks = mprts->nr_patches * mprts_cuda->nr_blocks;

  mprts_cuda->h_dev = new particles_cuda_dev_t[mprts->nr_patches];
  check(hipMalloc(&mprts_cuda->d_dev,
		   mprts->nr_patches * sizeof(*mprts_cuda->d_dev)));

  mprts_cuda->nr_prts = 0;
  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *prts_cuda = psc_particles_cuda(prts);
    mprts_cuda->nr_prts += prts->n_part;
    prts_cuda->mprts = mprts;
  }
  mprts_cuda->h_bnd_cnt = new unsigned int[mprts_cuda->nr_total_blocks];
  unsigned int nr_alloced = mprts_cuda->nr_prts * 1.2;
  mprts_cuda->nr_alloced = nr_alloced;

  check(hipMalloc((void **) &mprts_cuda->d_xi4, nr_alloced * sizeof(float4)));
  check(hipMalloc((void **) &mprts_cuda->d_pxi4, nr_alloced * sizeof(float4)));
  check(hipMalloc((void **) &mprts_cuda->d_alt_xi4, nr_alloced * sizeof(float4)));
  check(hipMalloc((void **) &mprts_cuda->d_alt_pxi4, nr_alloced * sizeof(float4)));
  check(hipMalloc((void **) &mprts_cuda->d_bidx, nr_alloced * sizeof(unsigned int)));
  check(hipMalloc((void **) &mprts_cuda->d_alt_bidx, nr_alloced * sizeof(unsigned int)));
  check(hipMalloc((void **) &mprts_cuda->d_ids, nr_alloced * sizeof(unsigned int)));
  check(hipMalloc((void **) &mprts_cuda->d_sums, nr_alloced * sizeof(unsigned int)));

  check(hipMalloc((void **) &mprts_cuda->d_off, 
		   (mprts_cuda->nr_total_blocks + 1) * sizeof(*mprts_cuda->d_off)));
  check(hipMalloc((void **) &mprts_cuda->d_bnd_spine_cnts,
		   (1 + mprts_cuda->nr_total_blocks * (CUDA_BND_STRIDE + 1)) * sizeof(unsigned int)));
  check(hipMalloc((void **) &mprts_cuda->d_bnd_spine_sums,
		   (1 + mprts_cuda->nr_total_blocks * (CUDA_BND_STRIDE + 1)) * sizeof(unsigned int)));

  for (int p = 0; p < mprts->nr_patches; p++) {
    struct psc_particles *prts = psc_mparticles_get_patch(mprts, p);
    struct psc_particles_cuda *prts_cuda = psc_particles_cuda(prts);

    prts_cuda->h_dev = &mprts_cuda->h_dev[p];
    prts_cuda->d_dev = &mprts_cuda->d_dev[p];
  }
}

void
__psc_mparticles_cuda_free(struct psc_mparticles *mprts)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);

  delete[] mprts_cuda->h_dev;
  delete[] mprts_cuda->h_bnd_cnt;

  check(hipFree(mprts_cuda->d_xi4));
  check(hipFree(mprts_cuda->d_pxi4));
  check(hipFree(mprts_cuda->d_alt_xi4));
  check(hipFree(mprts_cuda->d_alt_pxi4));
  check(hipFree(mprts_cuda->d_bidx));
  check(hipFree(mprts_cuda->d_alt_bidx));
  check(hipFree(mprts_cuda->d_ids));
  check(hipFree(mprts_cuda->d_sums));
  check(hipFree(mprts_cuda->d_bnd_spine_cnts));
  check(hipFree(mprts_cuda->d_bnd_spine_sums));

  check(hipFree(mprts_cuda->d_dev));
}

// ======================================================================
// ======================================================================
// fields

void
__psc_mfields_cuda_setup(struct psc_mfields *mflds)
{
  assert(!ppsc->domain.use_pml);
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);

  unsigned int total_size = 0;
  unsigned int buf_size = 0;
  for (int p = 0; p < mflds->nr_patches; p++) {
    struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
    if (p == 0) {
      for (int d = 0; d < 3; d++) {
	mflds_cuda->im[d] = flds->im[d];
	mflds_cuda->ib[d] = flds->ib[d];
      }
    } else {
      for (int d = 0; d < 3; d++) {
	assert(mflds_cuda->im[d] == flds->im[d]);
	assert(mflds_cuda->ib[d] == flds->ib[d]);
      }
    }
    unsigned int size = flds->im[0] * flds->im[1] * flds->im[2];
    total_size += size;
    if (flds->im[0] == 1 + 2*BND) {
      int B = 2*BND;
      buf_size = 2*B * (flds->im[1] + flds->im[2] - 2*B);
    }
  }

  check(hipMalloc((void **) &mflds_cuda->d_flds,
		   mflds->nr_fields * total_size * sizeof(float)));
  check(hipMalloc((void **) &mflds_cuda->d_bnd_buf,
		   MAX_BND_COMPONENTS * buf_size * mflds->nr_patches * sizeof(float)));
  mflds_cuda->h_bnd_buf = new float[MAX_BND_COMPONENTS * mflds->nr_patches * buf_size];
  float *d_flds = mflds_cuda->d_flds;

  for (int p = 0; p < mflds->nr_patches; p++) {
    struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
    assert(psc_fields_ops(flds) == &psc_fields_cuda_ops);
    struct psc_fields_cuda *flds_cuda = psc_fields_cuda(flds);

    unsigned int size = flds->im[0] * flds->im[1] * flds->im[2];
    flds_cuda->d_flds = d_flds;
    assert(d_flds == mflds_cuda->d_flds + p * flds->nr_comp * size);
    d_flds += flds->nr_comp * size;
    
    struct psc_fields_cuda_bnd *cf = &flds_cuda->bnd;
    int sz = 1;
    for (int d = 0; d < 3; d++) {
      if (flds->im[d] == 1 - 2 * flds->ib[d]) { // only 1 non-ghost point
	cf->im[d] = 1;
	cf->ib[d] = 0;
      } else {
	cf->im[d] = flds->im[d];
	cf->ib[d] = flds->ib[d];
      }
      sz *= cf->im[d];
    }
    cf->arr = new float [MAX_BND_COMPONENTS * sz * sizeof(*cf->arr)];
    cf->arr_off = cf->arr 
      - ((cf->ib[2] * cf->im[1] + cf->ib[1]) * cf->im[0] + cf->ib[0]);
  }
}

void
__psc_mfields_cuda_destroy(struct psc_mfields *mflds)
{
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);

  check(hipFree(mflds_cuda->d_flds));
  check(hipFree(mflds_cuda->d_bnd_buf));
  delete[] mflds_cuda->h_bnd_buf;

  for (int p = 0; p < mflds->nr_patches; p++) {
    struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
    struct psc_fields_cuda *flds_cuda = psc_fields_cuda(flds);
    struct psc_fields_cuda_bnd *cf = &flds_cuda->bnd;
    free(cf->arr);
  }
}

EXTERN_C void
__fields_cuda_to_device(struct psc_fields *pf, real *h_flds, int mb, int me)
{
  struct psc_fields_cuda *pfc = psc_fields_cuda(pf);
  unsigned int size = pf->im[0] * pf->im[1] * pf->im[2];
  check(hipMemcpy(pfc->d_flds + mb * size,
		   h_flds + mb * size,
		   (me - mb) * size * sizeof(float),
		   hipMemcpyHostToDevice));
}

EXTERN_C void
__fields_cuda_from_device(struct psc_fields *pf, real *h_flds, int mb, int me)
{
  struct psc_fields_cuda *pfc = psc_fields_cuda(pf);
  unsigned int size = pf->im[0] * pf->im[1] * pf->im[2];
  check(hipMemcpy(h_flds + mb * size,
		   pfc->d_flds + mb * size,
		   (me - mb) * size * sizeof(float),
		   hipMemcpyDeviceToHost));
}

// ======================================================================

enum {
  PACK,
  UNPACK,
};

// ======================================================================
// fields_device_pack

// FIXME/OPT: can probably be accelerated by making component the fast index

template<int B, int WHAT, int NR_COMPONENTS>
__global__ static void
k_fields_device_pack_yz(real *d_buf, real *d_flds, int gmy, int gmz,
			int nr_patches, int nr_fields)
{
  unsigned int buf_size = 2*B * (gmy + gmz - 2*B);
  int gmx = 2*BND + 1;
  int jx = BND;
  int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  int n_threads = NR_COMPONENTS * buf_size;
  int p = tid / n_threads;
  if (p >= nr_patches)
    return;

  int n = tid - p * n_threads;
  int m = n / buf_size; n -= m * buf_size;
  int jz, jy;
  if (n < B * gmy) {
    jz = n / gmy; n -= jz * gmy;
    jy = n;
  } else if (n < B * gmy + (gmz - 2*B) * 2*B) {
    n -= B * gmy;
    jz = n / (2*B); n -= jz * 2*B;
    if (n < B) {
      jy = n;
    } else {
      jy = n + gmy - 2*B;
    }
    jz += B;
  } else {
    n -= B * gmy + (gmz - 2*B) * 2*B;
    jz = n / gmy; n -= jz * gmy;
    jy = n;
    jz += gmz - B;
  }
  
  // FIXME, should use F3_DEV_YZ
  if (WHAT == PACK) {
    d_buf[tid] = d_flds[(((p * nr_fields + m) * gmz + jz) * gmy + jy) * gmx + jx];
  } else if (WHAT == UNPACK) {
    d_flds[(((p * nr_fields + m) * gmz + jz) * gmy + jy) * gmx + jx] = d_buf[tid]; 
  }
}

template<int B, bool pack>
static void
fields_device_pack_yz(struct psc_mfields *mflds, int mb, int me)
{
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
  unsigned int size = mflds_cuda->im[0] * mflds_cuda->im[1] * mflds_cuda->im[2];
  int gmy = mflds_cuda->im[1], gmz = mflds_cuda->im[2];
  unsigned int buf_size = 2*B * (gmy + gmz - 2*B);
  int n_threads = buf_size * (me - mb) * mflds->nr_patches;

  dim3 dimGrid((n_threads + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);
  dim3 dimBlock(THREADS_PER_BLOCK);
    
  const int NR_COMPONENTS = 3;
  assert(me - mb == NR_COMPONENTS);
  float *d_bnd_buf = mflds_cuda->d_bnd_buf;
  float *d_flds = mflds_cuda->d_flds + mb * size;
  k_fields_device_pack_yz<B, pack, NR_COMPONENTS> <<<dimGrid, dimBlock>>>
    (d_bnd_buf, d_flds, gmy, gmz, mflds->nr_patches,
     mflds->nr_fields);
  cuda_sync_if_enabled();
}

// ======================================================================
// fields_host_pack

#define WHAT do {					\
    if (what == PACK) {					\
      h_buf[tid++] = F3_CF_0(cf, m, 0,jy,jz);		\
    } else if (what == UNPACK) {			\
      F3_CF_0(cf, m, 0,jy,jz) = h_buf[tid++];		\
    }							\
  } while(0)

template<int B, int what>
static void
fields_host_pack_yz(struct psc_mfields *mflds, int mb, int me)
{
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
  int gmy = mflds_cuda->im[1], gmz = mflds_cuda->im[2];
  unsigned int buf_size = 2*B * (gmy + gmz - 2*B);

  for (int p = 0; p < mflds->nr_patches; p++) {
    struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
    struct psc_fields_cuda *flds_cuda = psc_fields_cuda(flds);
    struct psc_fields_cuda_bnd *cf = &flds_cuda->bnd;
    real *h_buf = mflds_cuda->h_bnd_buf + p * buf_size * MAX_BND_COMPONENTS;
    
    int gmy = cf->im[1], gmz = cf->im[2];
    int tid = 0;
    for (int m = 0; m < me - mb; m++) {
      for (int jz = 0; jz < B; jz++) {
	for (int jy = 0; jy < gmy; jy++) {
	  WHAT;
	}
      }
      for (int jz = B; jz < gmz - B; jz++) {
	for (int jy = 0; jy < B; jy++) {
	  WHAT;
	}
	for (int jy = gmy - B; jy < gmy; jy++) {
	  WHAT;
	}
      }
      for (int jz = gmz - B; jz < gmz; jz++) {
	for (int jy = 0; jy < gmy; jy++) {
	  WHAT;
	}
      }
    }
  }
}

template<int B>
static void
__fields_cuda_from_device_yz(struct psc_mfields *mflds, int mb, int me)
{
  static int pr1, pr2, pr3;
  if (!pr1) {
    pr1 = prof_register("field_device_pack", 1., 0, 0);
    pr2 = prof_register("cuda_memcpy", 1., 0, 0);
    pr3 = prof_register("field_host_unpack", 1., 0, 0);
  }
    
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
  int gmy = mflds_cuda->im[1], gmz = mflds_cuda->im[2];
  unsigned int buf_size = 2*B * (gmy + gmz - 2*B);
  assert(me - mb <= MAX_BND_COMPONENTS);
  assert(mflds_cuda->ib[1] == -BND);
  assert(mflds_cuda->im[1] >= 2 * B);
  assert(mflds_cuda->im[2] >= 2 * B);

  prof_start(pr1);
  fields_device_pack_yz<B, PACK>(mflds, mb, me);
  prof_stop(pr1);
  
  prof_start(pr2);
  check(hipMemcpy(mflds_cuda->h_bnd_buf, mflds_cuda->d_bnd_buf,
		   MAX_BND_COMPONENTS * buf_size * mflds->nr_patches *
		   sizeof(*mflds_cuda->h_bnd_buf),
		   hipMemcpyDeviceToHost));
  prof_stop(pr2);

  prof_start(pr3);
  fields_host_pack_yz<B, UNPACK>(mflds, mb, me);
  prof_stop(pr3);
}

template<int B>
static void
__fields_cuda_to_device_yz(struct psc_mfields *mflds, int mb, int me)
{
  static int pr1, pr2, pr3;
  if (!pr1) {
    pr1 = prof_register("field_host_pack", 1., 0, 0);
    pr2 = prof_register("cuda_memcpy", 1., 0, 0);
    pr3 = prof_register("field_device_unpack", 1., 0, 0);
  }
  
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
  int gmy = mflds_cuda->im[1], gmz = mflds_cuda->im[2];
  unsigned int buf_size = 2*B * (gmy + gmz - 2*B);
  assert(me - mb <= MAX_BND_COMPONENTS);
  assert(mflds_cuda->ib[1] == -BND);
  assert(mflds_cuda->im[1] >= 2 * B);
  assert(mflds_cuda->im[2] >= 2 * B);

  prof_start(pr1);
  fields_host_pack_yz<B, PACK>(mflds, mb, me);
  prof_stop(pr1);

  prof_start(pr2);
  check(hipMemcpy(mflds_cuda->d_bnd_buf, mflds_cuda->h_bnd_buf,
		   MAX_BND_COMPONENTS * buf_size * mflds->nr_patches *
		   sizeof(*mflds_cuda->d_bnd_buf),
		   hipMemcpyHostToDevice));
  prof_stop(pr2);

  prof_start(pr3);
  fields_device_pack_yz<B, UNPACK>(mflds, mb, me);
  prof_stop(pr3);
}

// ======================================================================

EXTERN_C void
__fields_cuda_from_device_inside(struct psc_mfields *mflds, int mb, int me)
{
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
  if (mflds_cuda->im[0] == 2 * -mflds_cuda->ib[0] + 1) {
    __fields_cuda_from_device_yz<2*BND>(mflds, mb, me);
  } else {
    for (int p = 0; p < mflds->nr_patches; p++) {
      struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
      struct psc_fields_cuda *flds_cuda = psc_fields_cuda(flds);
      unsigned int size = flds->im[0] * flds->im[1] * flds->im[2];
      check(hipMemcpy(flds_cuda->bnd.arr,
		       flds_cuda->d_flds + mb * size,
		       (me - mb) * size * sizeof(float),
		       hipMemcpyDeviceToHost));
    }
  }
}

EXTERN_C void
__fields_cuda_to_device_outside(struct psc_mfields *mflds, int mb, int me)
{
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
  if (mflds_cuda->im[0] == 2 * -mflds_cuda->ib[0] + 1) {
    __fields_cuda_to_device_yz<BND>(mflds, mb, me);
  } else {
    for (int p = 0; p < mflds->nr_patches; p++) {
      struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
      struct psc_fields_cuda *flds_cuda = psc_fields_cuda(flds);
      unsigned int size = flds->im[0] * flds->im[1] * flds->im[2];
      check(hipMemcpy(flds_cuda->d_flds + mb * size,
		       flds_cuda->bnd.arr,
		       (me - mb) * size * sizeof(float),
		       hipMemcpyHostToDevice));
    }
  }
}

EXTERN_C void
__fields_cuda_to_device_inside(struct psc_mfields *mflds, int mb, int me)
{
  struct psc_mfields_cuda *mflds_cuda = psc_mfields_cuda(mflds);
  if (mflds_cuda->im[0] == 2 * -mflds_cuda->ib[0] + 1) {
    __fields_cuda_to_device_yz<2*BND>(mflds, mb, me);
  } else {
    for (int p = 0; p < mflds->nr_patches; p++) {
      struct psc_fields *flds = psc_mfields_get_patch(mflds, p);
      struct psc_fields_cuda *flds_cuda = psc_fields_cuda(flds);
      unsigned int size = flds->im[0] * flds->im[1] * flds->im[2];
      check(hipMemcpy(flds_cuda->d_flds + mb * size,
		       flds_cuda->bnd.arr,
		       (me - mb) * size * sizeof(float),
		       hipMemcpyHostToDevice));
    }
  }
}
