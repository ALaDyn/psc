#include "hip/hip_runtime.h"

#include "cuda_iface.h"
#include "cuda_mparticles.h"
#include "cuda_bits.h"

#include "psc_particles_cuda.h"
#include "psc_particles_single.h"
#include "psc_particles_double.h"

#if 1
#define dprintf(...) mprintf(__VA_ARGS__)
#else
#define dprintf(...) do {} while (0)
#endif

psc_mparticles_cuda::psc_mparticles_cuda(Grid_t& grid, const Int3& bs)
{
  dprintf("CMPRTS: ctor\n");
  cmprts_ = new cuda_mparticles(grid, bs);
}

psc_mparticles_cuda::~psc_mparticles_cuda()
{
  dprintf("CMPRTS: dtor\n");
  delete cmprts_;
}

uint psc_mparticles_cuda::n_patches()
{
  dprintf("CMPRTS: n_patches\n");
  return cmprts_->n_patches;
}

void psc_mparticles_cuda::reserve_all(const uint *n_prts_by_patch)
{
  dprintf("CMPRTS: reserve_all\n");
  for (int p = 0; p < cmprts_->n_patches; p++) {
    dprintf("  p %d: %d\n", p, n_prts_by_patch[p]);
  }
  cmprts_->reserve_all(n_prts_by_patch);
}

void psc_mparticles_cuda::get_size_all(uint *n_prts_by_patch)
{
  dprintf("CMPRTS: get_size_all\n");
  cmprts_->get_size_all(n_prts_by_patch);
  for (int p = 0; p < cmprts_->n_patches; p++) {
    dprintf("  p %d: %d\n", p, n_prts_by_patch[p]);
  }
}

void psc_mparticles_cuda::resize_all(const uint *n_prts_by_patch)
{
  dprintf("CMPRTS: resize_all\n");
  cmprts_->resize_all(n_prts_by_patch);
}

uint psc_mparticles_cuda::get_n_prts()
{
  dprintf("CMPRTS: get_n_prts\n");
  return cmprts_->get_n_prts();
}

void psc_mparticles_cuda::set_particles(uint n_prts, uint off,
					void (*get_particle)(cuda_mparticles_prt *prt, int n, void *ctx),
					void *ctx)
{
  dprintf("CMPRTS: set_particles\n");
  cmprts_->set_particles(n_prts, off, get_particle, ctx);
}

void psc_mparticles_cuda::get_particles(uint n_prts, uint off,
					void (*put_particle)(cuda_mparticles_prt *, int, void *),
					void *ctx)
{
  dprintf("CMPRTS: get_particles\n");
  cmprts_->get_particles(n_prts, off, put_particle, ctx);
}

void psc_mparticles_cuda::to_device(float4 *xi4, float4 *pxi4, uint n_prts, uint off)
{
  dprintf("CMPRTS: to_device\n");
  cmprts_->to_device(xi4, pxi4, n_prts, off);
}

void psc_mparticles_cuda::from_device(float4 *xi4, float4 *pxi4, uint n_prts, uint off)
{
  dprintf("CMPRTS: from_device\n");
  cmprts_->from_device(xi4, pxi4, n_prts, off);
}

void psc_mparticles_cuda::setup_internals()
{
  dprintf("CMPRTS: setup_internals\n");
  cmprts_->setup_internals();
}

void psc_mparticles_cuda::inject(cuda_mparticles_prt *buf, uint *buf_n_by_patch)
{
  dprintf("CMPRTS: inject\n");
  cmprts_->inject(buf, buf_n_by_patch);
}

const particle_cuda_real_t* psc_mparticles_cuda::patch_get_b_dxi(int p)
{
  return cmprts_->patch_get_b_dxi(p);
}

const int* psc_mparticles_cuda::patch_get_b_mx(int p)
{
  return cmprts_->patch_get_b_mx(p);
}

psc_particle_cuda_buf_t* psc_mparticles_cuda::bnd_get_buffer(int p)
{
  return cmprts_->bnd_get_buffer(p);
}

void psc_mparticles_cuda::bnd_prep()
{
  cmprts_->bnd_prep();
}

void psc_mparticles_cuda::bnd_post()
{
  cmprts_->bnd_post();
}


// ======================================================================
// conversion

template<typename F>
void cuda_mparticles_base::set_particles(uint n_prts, uint off, F getter)
{
  float4 *xi4  = new float4[n_prts];
  float4 *pxi4 = new float4[n_prts];
  
  for (int n = 0; n < n_prts; n++) {
    struct cuda_mparticles_prt prt = getter(n);

    for (int d = 0; d < 3; d++) {
      int bi = fint(prt.xi[d] * b_dxi[d]);
      if (bi < 0 || bi >= b_mx[d]) {
	printf("XXX xi %g %g %g\n", prt.xi[0], prt.xi[1], prt.xi[2]);
	printf("XXX n %d d %d xi4[n] %g biy %d // %d\n",
	       n, d, prt.xi[d], bi, b_mx[d]);
	if (bi < 0) {
	  prt.xi[d] = 0.f;
	} else {
	  prt.xi[d] *= (1. - 1e-6);
	}
      }
      bi = floorf(prt.xi[d] * b_dxi[d]);
      assert(bi >= 0 && bi < b_mx[d]);
    }

    xi4[n].x  = prt.xi[0];
    xi4[n].y  = prt.xi[1];
    xi4[n].z  = prt.xi[2];
    xi4[n].w  = cuda_int_as_float(prt.kind);
    pxi4[n].x = prt.pxi[0];
    pxi4[n].y = prt.pxi[1];
    pxi4[n].z = prt.pxi[2];
    pxi4[n].w = prt.qni_wni;
  }

  to_device(xi4, pxi4, n_prts, off);
  
  delete[] xi4;
  delete[] pxi4;
}

// ----------------------------------------------------------------------
// get_particles

template<typename F>
void cuda_mparticles_base::get_particles(uint n_prts, uint off, F setter)
{
  float4 *xi4  = new float4[n_prts];
  float4 *pxi4 = new float4[n_prts];

  cuda_mparticles_reorder(static_cast<cuda_mparticles*>(this)); // FIXME
  from_device(xi4, pxi4, n_prts, off);
  
  for (int n = 0; n < n_prts; n++) {
    struct cuda_mparticles_prt prt;
    prt.xi[0]   = xi4[n].x;
    prt.xi[1]   = xi4[n].y;
    prt.xi[2]   = xi4[n].z;
    prt.kind    = cuda_float_as_int(xi4[n].w);
    prt.pxi[0]  = pxi4[n].x;
    prt.pxi[1]  = pxi4[n].y;
    prt.pxi[2]  = pxi4[n].z;
    prt.qni_wni = pxi4[n].w;

    setter(n, prt);

#if 0
    for (int d = 0; d < 3; d++) {
      int bi = fint(prt.xi[d] * b_dxi[d]);
      if (bi < 0 || bi >= b_mx[d]) {
	MHERE;
	mprintf("XXX xi %.10g %.10g %.10g\n", prt.xi[0], prt.xi[1], prt.xi[2]);
	mprintf("XXX n %d d %d xi %.10g b_dxi %.10g bi %d // %d\n",
		n, d, prt.xi[d] * b_dxi[d], b_dxi[d], bi, b_mx[d]);
      }
    }
#endif
  }

  delete[] (xi4);
  delete[] (pxi4);
}

template<typename MP>
struct ParticleGetter
{
  using particle_t = typename MP::particle_t;

  ParticleGetter(MP& mprts_other, int p)
    : mprts_other_(mprts_other), p_(p)
  {
  }

  cuda_mparticles_prt operator()(int n)
  {
    const particle_t& prt_other = mprts_other_[p_][n];

    cuda_mparticles_prt prt;
    prt.xi[0]   = prt_other.xi;
    prt.xi[1]   = prt_other.yi;
    prt.xi[2]   = prt_other.zi;
    prt.pxi[0]  = prt_other.pxi;
    prt.pxi[1]  = prt_other.pyi;
    prt.pxi[2]  = prt_other.pzi;
    prt.kind    = prt_other.kind_;
    prt.qni_wni = prt_other.qni_wni;

    return prt;
  }

private:
  MP& mprts_other_;
  int p_;
};

template<typename MP>
struct ParticleSetter
{
  using particle_t = typename MP::particle_t;

  ParticleSetter(MP& mprts_other, int p)
    : mprts_other_(mprts_other), p_(p)
  {
  }

  void operator()(int n, const cuda_mparticles_prt &prt)
  {
    particle_t& prt_other = mprts_other_[p_][n];

    prt_other.xi      = prt.xi[0];
    prt_other.yi      = prt.xi[1];
    prt_other.zi      = prt.xi[2];
    prt_other.kind_   = prt.kind;
    prt_other.pxi     = prt.pxi[0];
    prt_other.pyi     = prt.pxi[1];
    prt_other.pzi     = prt.pxi[2];
    prt_other.qni_wni = prt.qni_wni;
  }

private:
  MP& mprts_other_;
  int p_;
};

template<typename MP>
static void copy_from(mparticles_cuda_t mprts, MP mprts_other)
{
  uint n_prts_by_patch[mprts->n_patches()];
  mprts->get_size_all(n_prts_by_patch);
  
  uint off = 0;
  for (int p = 0; p < mprts.n_patches(); p++) {
    int n_prts = n_prts_by_patch[p];
    ParticleGetter<MP> getter(mprts_other, p);
    mprts.sub_->cmprts()->set_particles(n_prts, off, getter);

    off += n_prts;
  }
}

template<typename MP>
static void copy_to(mparticles_cuda_t mprts, MP mprts_other)
{
  uint n_prts_by_patch[mprts->n_patches()];
  mprts->get_size_all(n_prts_by_patch);

  uint off = 0;
  for (int p = 0; p < mprts.n_patches(); p++) {
    int n_prts = n_prts_by_patch[p];
    ParticleSetter<MP> setter(mprts_other, p);
    mprts.sub_->cmprts()->get_particles(n_prts, off, setter);

    off += n_prts;
  }
}

// ======================================================================
// conversion to "single"

void psc_mparticles_cuda::copy_from_single(struct psc_mparticles *mprts_cuda,
					   struct psc_mparticles *mprts, uint flags)
{
  copy_from(mparticles_cuda_t(mprts_cuda), mparticles_single_t(mprts));
}

void psc_mparticles_cuda::copy_to_single(struct psc_mparticles *mprts_cuda,
					 struct psc_mparticles *mprts, uint flags)
{
  copy_to(mparticles_cuda_t(mprts_cuda), mparticles_single_t(mprts));
}

// ======================================================================
// conversion to "double"

void psc_mparticles_cuda::copy_from_double(struct psc_mparticles *mprts_cuda,
					   struct psc_mparticles *mprts, uint flags)
{
  copy_from(mparticles_cuda_t(mprts_cuda), mparticles_double_t(mprts));
}

void psc_mparticles_cuda::copy_to_double(struct psc_mparticles *mprts_cuda,
					 struct psc_mparticles *mprts, uint flags)
{
  copy_to(mparticles_cuda_t(mprts_cuda), mparticles_double_t(mprts));
}

