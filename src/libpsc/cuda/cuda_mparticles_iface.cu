#include "hip/hip_runtime.h"

#include "cuda_iface.h"
#include "cuda_mparticles.h"

#include "psc_particles_cuda.h"
#include "psc_particles_single.h"
#include "psc_particles_double.h"

#if 1
#define dprintf(...) mprintf(__VA_ARGS__)
#else
#define dprintf(...) do {} while (0)
#endif

psc_mparticles_cuda::psc_mparticles_cuda(Grid_t& grid, const Int3& bs)
{
  dprintf("CMPRTS: ctor\n");
  cmprts_ = new cuda_mparticles(grid, bs);
}

psc_mparticles_cuda::~psc_mparticles_cuda()
{
  dprintf("CMPRTS: dtor\n");
  delete cmprts_;
}

uint psc_mparticles_cuda::n_patches()
{
  dprintf("CMPRTS: n_patches\n");
  return cmprts_->n_patches;
}

void psc_mparticles_cuda::reserve_all(const uint *n_prts_by_patch)
{
  dprintf("CMPRTS: reserve_all\n");
  for (int p = 0; p < cmprts_->n_patches; p++) {
    dprintf("  p %d: %d\n", p, n_prts_by_patch[p]);
  }
  cmprts_->reserve_all(n_prts_by_patch);
}

void psc_mparticles_cuda::get_size_all(uint *n_prts_by_patch)
{
  dprintf("CMPRTS: get_size_all\n");
  cmprts_->get_size_all(n_prts_by_patch);
  for (int p = 0; p < cmprts_->n_patches; p++) {
    dprintf("  p %d: %d\n", p, n_prts_by_patch[p]);
  }
}

void psc_mparticles_cuda::resize_all(const uint *n_prts_by_patch)
{
  dprintf("CMPRTS: resize_all\n");
  cmprts_->resize_all(n_prts_by_patch);
}

uint psc_mparticles_cuda::get_n_prts()
{
  dprintf("CMPRTS: get_n_prts\n");
  return cmprts_->get_n_prts();
}

void psc_mparticles_cuda::set_particles(uint n_prts, uint off,
					void (*get_particle)(cuda_mparticles_prt *prt, int n, void *ctx),
					void *ctx)
{
  dprintf("CMPRTS: set_particles\n");
  cmprts_->set_particles(n_prts, off, get_particle, ctx);
}

void psc_mparticles_cuda::get_particles(uint n_prts, uint off,
					void (*put_particle)(cuda_mparticles_prt *, int, void *),
					void *ctx)
{
  dprintf("CMPRTS: get_particles\n");
  cmprts_->get_particles(n_prts, off, put_particle, ctx);
}

void psc_mparticles_cuda::to_device(float4 *xi4, float4 *pxi4, uint n_prts, uint off)
{
  dprintf("CMPRTS: to_device\n");
  cmprts_->to_device(xi4, pxi4, n_prts, off);
}

void psc_mparticles_cuda::from_device(float4 *xi4, float4 *pxi4, uint n_prts, uint off)
{
  dprintf("CMPRTS: from_device\n");
  cmprts_->from_device(xi4, pxi4, n_prts, off);
}

void psc_mparticles_cuda::setup_internals()
{
  dprintf("CMPRTS: setup_internals\n");
  cmprts_->setup_internals();
}

void psc_mparticles_cuda::inject(cuda_mparticles_prt *buf, uint *buf_n_by_patch)
{
  dprintf("CMPRTS: inject\n");
  cmprts_->inject(buf, buf_n_by_patch);
}

const particle_cuda_real_t* psc_mparticles_cuda::patch_get_b_dxi(int p)
{
  return cmprts_->patch_get_b_dxi(p);
}

const int* psc_mparticles_cuda::patch_get_b_mx(int p)
{
  return cmprts_->patch_get_b_mx(p);
}

psc_particle_cuda_buf_t* psc_mparticles_cuda::bnd_get_buffer(int p)
{
  return cmprts_->bnd_get_buffer(p);
}

void psc_mparticles_cuda::bnd_prep()
{
  cmprts_->bnd_prep();
}

void psc_mparticles_cuda::bnd_post()
{
  cmprts_->bnd_post();
}


// ======================================================================
// conversion

template<typename MP>
struct copy_ctx
{
  copy_ctx(MP& _mprts_other, int _p)
    : mprts_other(_mprts_other), p(_p)
  {
  }
  
  MP& mprts_other;
  int p;
};

template<typename MP>
static void copy_from(mparticles_cuda_t mprts, MP mprts_other,
		      void (*get_particle)(struct cuda_mparticles_prt *prt, int n, void *ctx))
{
  uint n_prts_by_patch[mprts->n_patches()];
  mprts->get_size_all(n_prts_by_patch);
  
  uint off = 0;
  for (int p = 0; p < mprts.n_patches(); p++) {
    int n_prts = n_prts_by_patch[p];
    copy_ctx<MP> ctx(mprts_other, p);
    mprts->set_particles(n_prts, off, get_particle, &ctx);

    off += n_prts;
  }
}

template<typename MP>
static void copy_to(mparticles_cuda_t mprts, MP mprts_other,
		    void (*put_particle)(struct cuda_mparticles_prt *prt, int n, void *ctx))
{
  uint n_prts_by_patch[mprts->n_patches()];
  mprts->get_size_all(n_prts_by_patch);
  
  uint off = 0;
  for (int p = 0; p < mprts.n_patches(); p++) {
    int n_prts = n_prts_by_patch[p];
    copy_ctx<MP> ctx(mprts_other, p);
    mprts->get_particles(n_prts, off, put_particle, &ctx);

    off += n_prts;
  }
}

template<typename MP>
static void get_particle(struct cuda_mparticles_prt *prt, int n, void *_ctx)
{
  using particle_t = typename MP::particle_t;
  struct copy_ctx<MP> *ctx = (struct copy_ctx<MP> *) _ctx;
  particle_t *part = &ctx->mprts_other[ctx->p][n];

  prt->xi[0]   = part->xi;
  prt->xi[1]   = part->yi;
  prt->xi[2]   = part->zi;
  prt->pxi[0]  = part->pxi;
  prt->pxi[1]  = part->pyi;
  prt->pxi[2]  = part->pzi;
  prt->kind    = part->kind_;
  prt->qni_wni = part->qni_wni;
}

template<typename MP>
static void put_particle(struct cuda_mparticles_prt *prt, int n, void *_ctx)
{
  using particle_t = typename MP::particle_t;
  struct copy_ctx<MP> *ctx = (struct copy_ctx<MP> *) _ctx;
  particle_t *part = &ctx->mprts_other[ctx->p][n];
  
  part->xi      = prt->xi[0];
  part->yi      = prt->xi[1];
  part->zi      = prt->xi[2];
  part->kind_   = prt->kind;
  part->pxi     = prt->pxi[0];
  part->pyi     = prt->pxi[1];
  part->pzi     = prt->pxi[2];
  part->qni_wni = prt->qni_wni;
}

// ======================================================================
// conversion to "single"

void psc_mparticles_cuda::copy_from_single(struct psc_mparticles *mprts_cuda,
					   struct psc_mparticles *mprts, uint flags)
{
  copy_from(mparticles_cuda_t(mprts_cuda), mparticles_single_t(mprts), get_particle<mparticles_single_t>);
}

void psc_mparticles_cuda::copy_to_single(struct psc_mparticles *mprts_cuda,
					 struct psc_mparticles *mprts, uint flags)
{
  copy_to(mparticles_cuda_t(mprts_cuda), mparticles_single_t(mprts), put_particle<mparticles_single_t>);
}

// ======================================================================
// conversion to "double"

void psc_mparticles_cuda::copy_from_double(struct psc_mparticles *mprts_cuda,
					   struct psc_mparticles *mprts, uint flags)
{
  copy_from(mparticles_cuda_t(mprts_cuda), mparticles_double_t(mprts), get_particle<mparticles_double_t>);
}

void psc_mparticles_cuda::copy_to_double(struct psc_mparticles *mprts_cuda,
					 struct psc_mparticles *mprts, uint flags)
{
  copy_to(mparticles_cuda_t(mprts_cuda), mparticles_double_t(mprts), put_particle<mparticles_double_t>);
}

