#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"

#include <cstdio>
#include <cassert>

#define cudaCheck(ierr) do {						\
    if (ierr != hipSuccess)						\
      fprintf(stderr, "IERR = %d (%s)\n", ierr, hipGetErrorName(ierr)); \
    assert(ierr == hipSuccess);					\
  } while(0)

// ----------------------------------------------------------------------
// cuda_mparticles_create

struct cuda_mparticles *
cuda_mparticles_create()
{
  struct cuda_mparticles *cmprts = 
    (struct cuda_mparticles *) calloc(1, sizeof(*cmprts));

  return cmprts;
}

// ----------------------------------------------------------------------
// cuda_mparticles_destroy

void
cuda_mparticles_destroy(struct cuda_mparticles *cmprts)
{
  free(cmprts);
}

// ----------------------------------------------------------------------
// cuda_mparticles_alloc

void
cuda_mparticles_alloc(struct cuda_mparticles *cmprts, int nr_alloced)
{
  hipError_t ierr;

  ierr = hipMalloc((void **) &cmprts->d_xi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_pxi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_xi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_pxi4, nr_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_bidx, nr_alloced * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_id, nr_alloced * sizeof(unsigned int)); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_dealloc

void
cuda_mparticles_dealloc(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  ierr = hipFree(cmprts->d_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_bidx); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_id); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_swap_alt

void
cuda_mparticles_swap_alt(struct cuda_mparticles *cmprts)
{
  float4 *tmp_xi4 = cmprts->d_alt_xi4;
  float4 *tmp_pxi4 = cmprts->d_alt_pxi4;
  cmprts->d_alt_xi4 = cmprts->d_xi4;
  cmprts->d_alt_pxi4 = cmprts->d_pxi4;
  cmprts->d_xi4 = tmp_xi4;
  cmprts->d_pxi4 = tmp_pxi4;
}

