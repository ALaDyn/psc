#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_bits.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <cstdio>
#include <cassert>

// ----------------------------------------------------------------------
// cuda_mparticles_create

struct cuda_mparticles *
cuda_mparticles_create()
{
  struct cuda_mparticles *cmprts = new cuda_mparticles;
  std::memset(cmprts, 0, sizeof(*cmprts));

  return cmprts;
}

// ----------------------------------------------------------------------
// cuda_mparticles_set_domain_info

void
cuda_mparticles_set_domain_info(struct cuda_mparticles *cmprts,
				const struct cuda_domain_info *info)
{
  cmprts->n_patches = info->n_patches;
  cmprts->xb_by_patch = new float_3[cmprts->n_patches];
  for (int d = 0; d < 3; d++) {
    cmprts->ldims[d] = info->ldims[d];
    cmprts->bs[d] = info->bs[d];
    assert(info->ldims[d] % info->bs[d] == 0);
    cmprts->b_mx[d] = info->ldims[d] / info->bs[d];
    cmprts->dx[d] = info->dx[d];
    cmprts->b_dxi[d] = 1.f / (info->bs[d] * info->dx[d]);

    assert(info->xb_by_patch);
    for (int p = 0; p < cmprts->n_patches; p++) {
      cmprts->xb_by_patch[p][d] = info->xb_by_patch[p][d];
    }
  }
  cmprts->n_blocks_per_patch = cmprts->b_mx[0] * cmprts->b_mx[1] * cmprts->b_mx[2];
  cmprts->n_blocks = cmprts->n_patches * cmprts->n_blocks_per_patch;
}

// ----------------------------------------------------------------------
// cuda_mparticles_setup

void
cuda_mparticles_setup(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  ierr = hipMalloc(&cmprts->d_n_prts_by_patch, cmprts->n_patches * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc(&cmprts->d_off, (cmprts->n_blocks + 1) * sizeof(*cmprts->d_off)); cudaCheck(ierr);
  ierr = hipMemset(cmprts->d_off, 0, (cmprts->n_blocks + 1) * sizeof(*cmprts->d_off)); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_free_particle_mem

static void
cuda_mparticles_free_particle_mem(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  ierr = hipFree(cmprts->d_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_bidx); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_id); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_destroy

void
cuda_mparticles_destroy(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  ierr = hipFree(cmprts->d_n_prts_by_patch); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_off); cudaCheck(ierr);

  cuda_mparticles_free_particle_mem(cmprts);

  delete[] cmprts->xb_by_patch;
  delete cmprts;
}

// ----------------------------------------------------------------------
// cuda_mparticles_reserve

void
cuda_mparticles_reserve(struct cuda_mparticles *cmprts, unsigned int *n_prts_by_patch)
{
  hipError_t ierr;

  unsigned int size = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    size += n_prts_by_patch[p];
  }

  if (size <= cmprts->n_alloced) {
    return;
  }

  unsigned int n_alloced = max(size, 2 * cmprts->n_alloced);

  if (cmprts->n_alloced > 0) {
    cuda_mparticles_free_particle_mem(cmprts);
  }
  cmprts->n_alloced = n_alloced;

  ierr = hipMalloc((void **) &cmprts->d_xi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_pxi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_xi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_pxi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_bidx, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_id, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_to_device

void
cuda_mparticles_to_device(struct cuda_mparticles *cmprts, float4 *xi4, float4 *pxi4,
			  unsigned int n_prts, unsigned int off)
{
  hipError_t ierr;

  assert(off + n_prts <= cmprts->n_alloced);
  ierr = hipMemcpy(cmprts->d_xi4 + off, xi4, n_prts * sizeof(*xi4),
		    hipMemcpyHostToDevice); cudaCheck(ierr);
  ierr = hipMemcpy(cmprts->d_pxi4 + off, pxi4, n_prts * sizeof(*pxi4),
		    hipMemcpyHostToDevice); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_from_device

void
cuda_mparticles_from_device(struct cuda_mparticles *cmprts, float4 *xi4, float4 *pxi4,
			    unsigned int n_prts, unsigned int off)
{
  hipError_t ierr;

  assert(off + n_prts <= cmprts->n_alloced);
  ierr = hipMemcpy(xi4, cmprts->d_xi4 + off, n_prts * sizeof(*xi4),
		    hipMemcpyDeviceToHost); cudaCheck(ierr);
  ierr = hipMemcpy(pxi4, cmprts->d_pxi4 + off, n_prts * sizeof(*pxi4),
		    hipMemcpyDeviceToHost); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_dump_by_patch

void
cuda_mparticles_dump_by_patch(struct cuda_mparticles *cmprts, unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  printf("cuda_mparticles_dump_by_patch: n_prts = %d\n", cmprts->n_prts);
  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    float *xb = &cmprts->xb_by_patch[p][0];
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      float4 xi4 = d_xi4[n + off], pxi4 = d_pxi4[n + off];
      unsigned int bidx = d_bidx[n + off], id = d_id[n + off];
      printf("cuda_mparticles_dump_by_patch: [%d/%d] %g %g %g // %d // %g %g %g // %g b_idx %d id %d\n",
	     p, n, xi4.x + xb[0], xi4.y + xb[1], xi4.z + xb[2],
	     cuda_float_as_int(xi4.w),
	     pxi4.x, pxi4.y, pxi4.z, pxi4.w,
	     bidx, id);
    }
    off += n_prts_by_patch[p];
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_dump

void
cuda_mparticles_dump(struct cuda_mparticles *cmprts)
{
  int n_prts = cmprts->n_prts;
  
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);

  printf("cuda_mparticles_dump: n_prts = %d\n", n_prts);
  unsigned int off = 0;
  for (int b = 0; b < cmprts->n_blocks; b++) {
    unsigned int off_b = d_off[b], off_e = d_off[b+1];
    int p = b / cmprts->n_blocks_per_patch;
    printf("cuda_mparticles_dump: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    assert(d_off[b] == off);
    for (int n = d_off[b]; n < d_off[b+1]; n++) {
      float4 xi4 = d_xi4[n], pxi4 = d_pxi4[n];
      unsigned int bidx = d_bidx[n], id = d_id[n];
      printf("cuda_mparticles_dump: [%d] %g %g %g // %d // %g %g %g // %g || bidx %d id %d\n",
	     n, xi4.x, xi4.y, xi4.z, cuda_float_as_int(xi4.w), pxi4.x, pxi4.y, pxi4.z, pxi4.w,
	     bidx, id);
      assert(b == bidx);
    }
    off += off_e - off_b;
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_swap_alt

void
cuda_mparticles_swap_alt(struct cuda_mparticles *cmprts)
{
  float4 *tmp_xi4 = cmprts->d_alt_xi4;
  float4 *tmp_pxi4 = cmprts->d_alt_pxi4;
  cmprts->d_alt_xi4 = cmprts->d_xi4;
  cmprts->d_alt_pxi4 = cmprts->d_pxi4;
  cmprts->d_xi4 = tmp_xi4;
  cmprts->d_pxi4 = tmp_pxi4;
}

// ----------------------------------------------------------------------
// cuda_params2

struct cuda_params2 {
  unsigned int b_mx[3];
  float b_dxi[3];
};

static void
cuda_params2_set(struct cuda_params2 *prm, const struct cuda_mparticles *cuda_mprts)
{
  for (int d = 0; d < 3; d++) {
    prm->b_mx[d]  = cuda_mprts->b_mx[d];
    prm->b_dxi[d] = cuda_mprts->b_dxi[d];
  }
}

static void
cuda_params2_free(struct cuda_params2 *prm)
{
}

#define THREADS_PER_BLOCK 256

// ----------------------------------------------------------------------
// get_block_idx

static int
get_block_idx(struct cuda_mparticles *cmprts, float4 xi4, int p)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  float *b_dxi = cmprts->b_dxi;
  int *b_mx = cmprts->b_mx;
  
  unsigned int block_pos_y = (int) floorf(xi4.y * b_dxi[1]);
  unsigned int block_pos_z = (int) floorf(xi4.z * b_dxi[2]);

  int bidx;
  if (block_pos_y >= b_mx[1] || block_pos_z >= b_mx[2]) {
    bidx = -1;
  } else {
    bidx = (p * b_mx[2] + block_pos_z) * b_mx[1] + block_pos_y;
  }

  return bidx;
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_ids

__global__ static void
mprts_find_block_indices_ids(struct cuda_params2 prm, float4 *d_xi4,
			     int *d_n_prts_by_patch, unsigned int *d_bidx,
			     unsigned int *d_ids, int nr_patches)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  unsigned int off = 0;
  for (int p = 0; p < nr_patches; p++) {
    if (n < d_n_prts_by_patch[p]) {
      float4 xi4 = d_xi4[n + off];
      unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
      unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);
      
      int block_idx;
      if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
	block_idx = -1; // not supposed to happen here!
      } else {
	block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
      }
      d_bidx[n + off] = block_idx;
      d_ids[n + off] = n + off;
    }
    off += d_n_prts_by_patch[p];
  }
}

void
cuda_mparticles_find_block_indices_ids(struct cuda_mparticles *cmprts,
				       unsigned int *n_prts_by_patch)
{
  if (cmprts->n_patches == 0) {
    return;
  }

  int max_n_prts = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    if (n_prts_by_patch[p] > max_n_prts) {
      max_n_prts = n_prts_by_patch[p];
    }
  }

  hipError_t ierr;
  ierr = hipMemcpy(cmprts->d_n_prts_by_patch, n_prts_by_patch,
		    cmprts->n_patches * sizeof(unsigned int),
		    hipMemcpyHostToDevice); cudaCheck(ierr);

  struct cuda_params2 prm;
  cuda_params2_set(&prm, cmprts);
    
  dim3 dimGrid((max_n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  dim3 dimBlock(THREADS_PER_BLOCK);

  mprts_find_block_indices_ids<<<dimGrid, dimBlock>>>(prm,
						      cmprts->d_xi4, 
						      cmprts->d_n_prts_by_patch,
						      cmprts->d_bidx,
						      cmprts->d_id,
						      cmprts->n_patches);
  cuda_sync_if_enabled();
  cuda_params2_free(&prm);
}

// ----------------------------------------------------------------------
// cuda_mparticles_reorder_and_offsets

__global__ static void
mprts_reorder_and_offsets(int nr_prts, float4 *xi4, float4 *pxi4, float4 *alt_xi4, float4 *alt_pxi4,
			  unsigned int *d_bidx, unsigned int *d_ids, unsigned int *d_off, int last_block)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > nr_prts)
    return;

  int block, prev_block;
  if (i < nr_prts) {
    alt_xi4[i] = xi4[d_ids[i]];
    alt_pxi4[i] = pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else { // needed if there is no particle in the last block
    block = last_block;
  }

  // OPT: d_bidx[i-1] could use shmem
  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_off[b] = i;
  }
}

void
cuda_mparticles_reorder_and_offsets(struct cuda_mparticles *cmprts)
{
  if (cmprts->n_patches == 0) {
    return;
  }

  dim3 dimGrid((cmprts->n_prts + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  dim3 dimBlock(THREADS_PER_BLOCK);

  mprts_reorder_and_offsets<<<dimGrid, dimBlock>>>(cmprts->n_prts, cmprts->d_xi4, cmprts->d_pxi4,
						   cmprts->d_alt_xi4, cmprts->d_alt_pxi4,
						   cmprts->d_bidx, cmprts->d_id,
						   cmprts->d_off, cmprts->n_blocks);
  cuda_sync_if_enabled();

  cuda_mparticles_swap_alt(cmprts);
  cmprts->need_reorder = false;
}

void
cuda_mparticles_reorder_and_offsets_slow(struct cuda_mparticles *cmprts)
{
  if (cmprts->n_patches == 0) {
    return;
  }

  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<float4> d_alt_xi4(cmprts->d_alt_xi4);
  thrust::device_ptr<float4> d_alt_pxi4(cmprts->d_alt_pxi4);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  thrust::host_vector<float4> h_xi4(d_xi4, d_xi4 + cmprts->n_prts);
  thrust::host_vector<float4> h_pxi4(d_pxi4, d_pxi4 + cmprts->n_prts);
  thrust::host_vector<float4> h_alt_xi4(d_alt_xi4, d_alt_xi4 + cmprts->n_prts);
  thrust::host_vector<float4> h_alt_pxi4(d_alt_pxi4, d_alt_pxi4 + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + cmprts->n_blocks + 1);
  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_id(d_id, d_id + cmprts->n_prts);

  for (int i = 0; i <= cmprts->n_prts; i++) {
    //    unsigned int bidx;
    unsigned int block;
    if (i < cmprts->n_prts) {
      h_alt_xi4[i] = h_xi4[h_id[i]];
      h_alt_pxi4[i] = h_pxi4[h_id[i]];
      //bidx = get_block_idx(cmprts, h_alt_xi4[i], 0);
      block = h_bidx[i];
    } else {
      //bidx = cmprts->n_blocks;
      block = cmprts->n_blocks;
    }
    // if (i < 10) {
    //   printf("i %d bidx %d block %d xi4 %g %g\n", bidx, block, h_alt_xi4[i].y, h_alt_xi4[i].z);
    // }
    int prev_block = (i > 0) ? (int) h_bidx[i-1] : -1;
    for (int b = prev_block + 1; b <= block; b++) {
      h_off[b] = i;
    }
  }

  thrust::copy(h_alt_xi4.begin(), h_alt_xi4.end(), d_alt_xi4);
  thrust::copy(h_alt_pxi4.begin(), h_alt_pxi4.end(), d_alt_pxi4);
  thrust::copy(h_off.begin(), h_off.end(), d_off);
  
  cuda_mparticles_swap_alt(cmprts);
  cmprts->need_reorder = false;
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_in_patch_unordered_slow

void
cuda_mparticles_check_in_patch_unordered_slow(struct cuda_mparticles *cmprts,
					      unsigned int *nr_prts_by_patch)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int n = 0; n < nr_prts_by_patch[p]; n++) {
      int bidx = get_block_idx(cmprts, d_xi4[off + n], p);
      assert(bidx >= 0 && bidx <= cmprts->n_blocks);
    }
    off += nr_prts_by_patch[p];
  }

  assert(off == cmprts->n_prts);
  printf("PASS: cuda_mparticles_check_in_patch_unordered_slow()\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_bix_id_unordered_slow

void
cuda_mparticles_check_bidx_id_unordered_slow(struct cuda_mparticles *cmprts,
					     unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = get_block_idx(cmprts, d_xi4[off + n], p);
      assert(bidx == d_bidx[off+n]);
      assert(off+n == d_id[off+n]);
    }
    off += n_prts_by_patch[p];
  }

  assert(off == cmprts->n_prts);
  printf("PASS: cuda_mparticles_check_bidx_id_unordered_slow()\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_ordered_slow

void
cuda_mparticles_check_ordered_slow(struct cuda_mparticles *cmprts)
{
  bool need_reorder = cmprts->need_reorder;
  
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  unsigned int off = 0;
  for (int b = 0; b < cmprts->n_blocks; b++) {
    int p = b / cmprts->n_blocks_per_patch;
    unsigned int off_b = d_off[b], off_e = d_off[b+1];
    assert(off_e >= off_b);
    // printf("cuda_mparticles_check_ordered: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    assert(d_off[b] == off);
    for (int n = d_off[b]; n < d_off[b+1]; n++) {
      float4 xi4;
      if (need_reorder) {
	xi4 = d_xi4[d_id[n]];
      } else {
	xi4 = d_xi4[n];
      }
      unsigned int bidx = get_block_idx(cmprts, xi4, p);
      //printf("cuda_mparticles_check_ordered: bidx %d\n", bidx);
      if (b != bidx) {
	printf("b %d bidx %d n %d p %d xi4 %g %g %g\n",
	       b, bidx, n, p, xi4.x, xi4.y, xi4.z);
	unsigned int block_pos_y = (int) floorf(xi4.y * cmprts->b_dxi[1]);
	unsigned int block_pos_z = (int) floorf(xi4.z * cmprts->b_dxi[2]);
	printf("block_pos %d %d %g %g\n", block_pos_y, block_pos_z, xi4.y * cmprts->b_dxi[1],
	       xi4.z * cmprts->b_dxi[2]);
      }
      assert(b == bidx);
    }
    off += off_e - off_b;
  }
  assert(off == cmprts->n_prts);
  printf("cuda_mparticles_check_ordered: PASS\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_ordered

void
cuda_mparticles_check_ordered(struct cuda_mparticles *cmprts)
{
  bool need_reorder = cmprts->need_reorder;

  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::host_vector<float4> h_xi4(d_xi4, d_xi4 + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + cmprts->n_blocks + 1);
  thrust::host_vector<unsigned int> h_id(d_id, d_id + cmprts->n_prts);

  //printf("cuda_mparticles_check_ordered: need_reorder %s\n", need_reorder ? "true" : "false");

  // for (int n = 0; n < 10; n++) {
  //   unsigned int bidx = d_bidx[n];
  //   printf("n %d bidx %d xi4 %g %g\n", n, bidx, h_xi4[n].y, h_xi4[n].z);
  // }
  unsigned int off = 0;
  for (int b = 0; b < cmprts->n_blocks; b++) {
    int p = b / cmprts->n_blocks_per_patch;
    unsigned int off_b = h_off[b], off_e = h_off[b+1];
    assert(off_e >= off_b);
    //printf("cuda_mparticles_check_ordered: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    assert(off_b == off);
    for (int n = h_off[b]; n < h_off[b+1]; n++) {
      float4 xi4;
      if (need_reorder) {
	xi4 = h_xi4[h_id[n]];
      } else {
	xi4 = h_xi4[n];
      }
      unsigned int bidx = get_block_idx(cmprts, xi4, p);
      //printf("cuda_mparticles_check_ordered: bidx %d\n", bidx);
      if (b != bidx) {
	printf("b %d bidx %d n %d p %d xi4 %g %g %g\n",
	       b, bidx, n, p, xi4.x, xi4.y, xi4.z);
	unsigned int block_pos_y = (int) floorf(xi4.y * cmprts->b_dxi[1]);
	unsigned int block_pos_z = (int) floorf(xi4.z * cmprts->b_dxi[2]);
	printf("block_pos %d %d %g %g\n", block_pos_y, block_pos_z, xi4.y * cmprts->b_dxi[1],
	       xi4.z * cmprts->b_dxi[2]);
      }
      assert(b == bidx);
    }
    off += off_e - off_b;
  }
  assert(off == cmprts->n_prts);
  printf("cuda_mparticles_check_ordered: PASS\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_sort_initial

void
cuda_mparticles_sort_initial(struct cuda_mparticles *cmprts,
			     unsigned int *n_prts_by_patch)
{
  static int first_time = false;
  if (first_time) {
    cuda_mparticles_check_in_patch_unordered_slow(cmprts, n_prts_by_patch);
  }

  cuda_mparticles_find_block_indices_ids(cmprts, n_prts_by_patch);
  if (first_time) {
    cuda_mparticles_check_bidx_id_unordered_slow(cmprts, n_prts_by_patch);
  }

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::stable_sort_by_key(d_bidx, d_bidx + cmprts->n_prts, d_id);
  cuda_mparticles_reorder_and_offsets(cmprts);

  if (first_time) {
    cuda_mparticles_check_ordered(cmprts);
    first_time = false;
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_set_n_prts_by_patch

void
cuda_mparticles_set_n_prts_by_patch(struct cuda_mparticles *cmprts,
				    const unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::host_vector<unsigned int> h_off(cmprts->n_blocks + 1);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    h_off[p * cmprts->n_blocks_per_patch] = off;
    off += n_prts_by_patch[p];
    // printf("set_n_prts p%d: %d\n", p, n_prts_by_patch[p]);
  }
  h_off[cmprts->n_blocks] = off;
  cmprts->n_prts = off;

  thrust::copy(h_off.begin(), h_off.end(), d_off);
}

// ----------------------------------------------------------------------
// cuda_mparticles_get_n_prts_by_patch

void
cuda_mparticles_get_n_prts_by_patch(struct cuda_mparticles *cmprts,
				    unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + cmprts->n_blocks + 1);

  for (int p = 0; p < cmprts->n_patches; p++) {
    n_prts_by_patch[p] = h_off[(p+1) * cmprts->n_blocks_per_patch] - h_off[p * cmprts->n_blocks_per_patch];
    //printf("p %d n_prts_by_patch %d\n", p, n_prts_by_patch[p]);
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_reorder

__global__ static void
k_cuda_mparticles_reorder(int nr_prts, unsigned int *d_ids,
		 float4 *xi4, float4 *pxi4,
		 float4 *alt_xi4, float4 *alt_pxi4)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < nr_prts) {
    int j = d_ids[i];
    alt_xi4[i] = xi4[j];
    alt_pxi4[i] = pxi4[j];
  }
}

void
cuda_mparticles_reorder(struct cuda_mparticles *cmprts)
{
  if (!cmprts->need_reorder) {
    return;
  }
  
  dim3 dimGrid((cmprts->n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  
  k_cuda_mparticles_reorder<<<dimGrid, THREADS_PER_BLOCK>>>
    (cmprts->n_prts, cmprts->d_id,
     cmprts->d_xi4, cmprts->d_pxi4,
     cmprts->d_alt_xi4, cmprts->d_alt_pxi4);
  
  cuda_mparticles_swap_alt(cmprts);

  cmprts->need_reorder = false;
}

// ----------------------------------------------------------------------
// cuda_mparticles_inject

void
cuda_mparticles_inject(struct cuda_mparticles *cmprts, struct cuda_mparticles_prt *buf,
		       unsigned int *buf_n_by_patch)
{
  if (cmprts->need_reorder) {
    cuda_mparticles_reorder(cmprts);
    cmprts->need_reorder = false;
  }
  
  unsigned int buf_n = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    buf_n += buf_n_by_patch[p];
    //    printf("p %d buf_n_by_patch %d\n", p, buf_n_by_patch[p]);
  }
  //  printf("buf_n %d\n", buf_n);

  thrust::host_vector<float4> h_xi4(buf_n);
  thrust::host_vector<float4> h_pxi4(buf_n);
  thrust::host_vector<unsigned int> h_bidx(buf_n);
  thrust::host_vector<unsigned int> h_id(buf_n);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int n = 0; n < buf_n_by_patch[p]; n++) {
      float4 *xi4 = &h_xi4[off + n];
      float4 *pxi4 = &h_pxi4[off + n];
      cuda_mparticles_prt *prt = &buf[off + n];
      
      xi4->x  = prt->xi[0];
      xi4->y  = prt->xi[1];
      xi4->z  = prt->xi[2];
      xi4->w  = cuda_int_as_float(prt->kind);
      pxi4->x = prt->pxi[0];
      pxi4->y = prt->pxi[1];
      pxi4->z = prt->pxi[2];
      pxi4->w = prt->qni_wni;

      h_bidx[off + n] = get_block_idx(cmprts, *xi4, p);
      h_id[off + n] = cmprts->n_prts + off + n;
    }
    off += buf_n_by_patch[p];
  }
  assert(off == buf_n);

  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + cmprts->n_blocks + 1);
  unsigned int n_prts_by_patch[cmprts->n_patches];
  for (int p = 0; p < cmprts->n_patches; p++) {
    n_prts_by_patch[p] = h_off[(p+1) * cmprts->n_blocks_per_patch] - h_off[p * cmprts->n_blocks_per_patch];
    //printf("p %d n_prts_by_patch %d\n", p, n_prts_by_patch[p]);
  }

  //cuda_mparticles_check_in_patch_unordered_slow(cmprts, n_prts_by_patch);

  cuda_mparticles_find_block_indices_ids(cmprts, n_prts_by_patch);
  //cuda_mparticles_check_bidx_id_unordered_slow(cmprts, n_prts_by_patch);

  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  assert(cmprts->n_prts + buf_n <= cmprts->n_alloced);
  thrust::copy(h_xi4.begin(), h_xi4.end(), d_xi4 + cmprts->n_prts);
  thrust::copy(h_pxi4.begin(), h_pxi4.end(), d_pxi4 + cmprts->n_prts);
  thrust::copy(h_bidx.begin(), h_bidx.end(), d_bidx + cmprts->n_prts);
  //thrust::copy(h_id.begin(), h_id.end(), d_id + cmprts->n_prts);
  thrust::sequence(d_id, d_id + cmprts->n_prts + buf_n);

  // for (int i = -5; i <= 5; i++) {
  //   //    float4 xi4 = d_xi4[cmprts->n_prts + i];
  //   unsigned int bidx = d_bidx[cmprts->n_prts + i];
  //   unsigned int id = d_id[cmprts->n_prts + i];
  //   printf("i %d bidx %d %d\n", i, bidx, id);
  // }

  // cuda_mparticles_check_ordered(cmprts);

  cmprts->n_prts += buf_n;

  thrust::stable_sort_by_key(d_bidx, d_bidx + cmprts->n_prts, d_id);
  cuda_mparticles_reorder_and_offsets(cmprts);

  // cuda_mparticles_check_ordered(cmprts);
}

// ----------------------------------------------------------------------
// cuda_mparticles_set_particles

void
cuda_mparticles_set_particles(struct cuda_mparticles *cmprts, unsigned int n_prts, unsigned int off,
			      void (*get_particle)(struct cuda_mparticles_prt *prt, int n, void *ctx),
			      void *ctx)
{
  float4 *xi4  = new float4[n_prts];
  float4 *pxi4 = new float4[n_prts];
  
  for (int n = 0; n < n_prts; n++) {
    struct cuda_mparticles_prt prt;
    get_particle(&prt, n, ctx);

    for (int d = 0; d < 3; d++) {
      int bi = floorf(prt.xi[d] * cmprts->b_dxi[d]); // FIXME, consolidate the fint stuff
      if (bi < 0 || bi >= cmprts->b_mx[d]) {
	printf("XXX xi %g %g %g\n", prt.xi[0], prt.xi[1], prt.xi[2]);
	printf("XXX n %d d %d xi4[n] %g biy %d // %d\n",
	       n, d, prt.xi[d], bi, cmprts->b_mx[d]);
	if (bi < 0) {
	  prt.xi[d] = 0.f;
	} else {
	  prt.xi[d] *= (1. - 1e-6);
	}
      }
      bi = floorf(prt.xi[d] * cmprts->b_dxi[d]);
      assert(bi >= 0 && bi < cmprts->b_mx[d]);
    }

    xi4[n].x  = prt.xi[0];
    xi4[n].y  = prt.xi[1];
    xi4[n].z  = prt.xi[2];
    xi4[n].w  = cuda_int_as_float(prt.kind);
    pxi4[n].x = prt.pxi[0];
    pxi4[n].y = prt.pxi[1];
    pxi4[n].z = prt.pxi[2];
    pxi4[n].w = prt.qni_wni;
  }

  cuda_mparticles_to_device(cmprts, xi4, pxi4, n_prts, off);
  
  delete[] xi4;
  delete[] pxi4;
}

// ----------------------------------------------------------------------
// cuda_mparticles_get_particles

void
cuda_mparticles_get_particles(struct cuda_mparticles *cmprts, unsigned int n_prts, unsigned int off,
			      void (*put_particle)(struct cuda_mparticles_prt *, int, void *),
			      void *ctx)
{
  float4 *xi4  = new float4[n_prts];
  float4 *pxi4 = new float4[n_prts];

  cuda_mparticles_reorder(cmprts);
  cuda_mparticles_from_device(cmprts, xi4, pxi4, n_prts, off);
  
  for (int n = 0; n < n_prts; n++) {
    struct cuda_mparticles_prt prt;
    prt.xi[0]   = xi4[n].x;
    prt.xi[1]   = xi4[n].y;
    prt.xi[2]   = xi4[n].z;
    prt.kind    = cuda_float_as_int(xi4[n].w);
    prt.pxi[0]  = pxi4[n].x;
    prt.pxi[1]  = pxi4[n].y;
    prt.pxi[2]  = pxi4[n].z;
    prt.qni_wni = pxi4[n].w;

    put_particle(&prt, n, ctx);

#if 0
    for (int d = 0; d < 3; d++) {
      int bi = particle_single_real_fint(prt.xi[d] * cmprts->b_dxi[d]);
      if (bi < 0 || bi >= cmprts->b_mx[d]) {
	MHERE;
	mprintf("XXX xi %.10g %.10g %.10g\n", prt.xi[0], prt.xi[1], prt.xi[2]);
	mprintf("XXX n %d d %d xi %.10g b_dxi %.10g bi %d // %d\n",
		n, d, prt.xi[d] * cmprts->b_dxi[d], cmprts->b_dxi[d], bi, cmprts->b_mx[d]);
      }
    }
#endif
  }

  delete[] (xi4);
  delete[] (pxi4);
}

