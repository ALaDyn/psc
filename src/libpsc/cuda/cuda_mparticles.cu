#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"

#include <cstdio>
#include <cassert>

#define cudaCheck(ierr) do {						\
    if (ierr != hipSuccess)						\
      fprintf(stderr, "IERR = %d (%s)\n", ierr, hipGetErrorName(ierr)); \
    assert(ierr == hipSuccess);					\
  } while(0)

// ----------------------------------------------------------------------
// cuda_mparticles_create

struct cuda_mparticles *
cuda_mparticles_create()
{
  struct cuda_mparticles *cmprts = 
    (struct cuda_mparticles *) calloc(1, sizeof(*cmprts));

  return cmprts;
}

// ----------------------------------------------------------------------
// cuda_mparticles_destroy

void
cuda_mparticles_destroy(struct cuda_mparticles *cmprts)
{
  free(cmprts);
}

// ----------------------------------------------------------------------
// cuda_mparticles_alloc

void
cuda_mparticles_alloc(struct cuda_mparticles *cmprts, unsigned int *n_prts_by_patch)
{
  hipError_t ierr;

  cmprts->n_prts = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    cmprts->n_prts += n_prts_by_patch[p];
  }

  cmprts->n_alloced = cmprts->n_prts * 1.4;
  unsigned int n_alloced = cmprts->n_alloced;

  ierr = hipMalloc((void **) &cmprts->d_xi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_pxi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_xi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_pxi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_bidx, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_id, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_dealloc

void
cuda_mparticles_dealloc(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  ierr = hipFree(cmprts->d_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_bidx); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_id); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_swap_alt

void
cuda_mparticles_swap_alt(struct cuda_mparticles *cmprts)
{
  float4 *tmp_xi4 = cmprts->d_alt_xi4;
  float4 *tmp_pxi4 = cmprts->d_alt_pxi4;
  cmprts->d_alt_xi4 = cmprts->d_xi4;
  cmprts->d_alt_pxi4 = cmprts->d_pxi4;
  cmprts->d_xi4 = tmp_xi4;
  cmprts->d_pxi4 = tmp_pxi4;
}

