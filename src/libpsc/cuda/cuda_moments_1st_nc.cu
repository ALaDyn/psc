#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_mfields.h"
#include "cuda_mparticles_const.h"
#include "cuda_mfields_const.h"

struct DParticles : DParticleIndexer
{
  static const int MAX_N_KINDS = 4;
  
  DParticles(const cuda_mparticles& cmprts)
    : DParticleIndexer{cmprts.b_mx(), cmprts.b_dxi(), cmprts.dxi()},
      fnqs_(cmprts.grid_.fnqs)
  {
    int n_kinds = cmprts.grid_.kinds.size();
    assert(n_kinds <= MAX_N_KINDS);
    for (int k = 0; k < n_kinds; k++) {
      q_inv_[k] = 1.f / cmprts.grid_.kinds[k].q;
    }
  }

  __device__ real_t fnqs() const { return fnqs_; }
  __device__ real_t q_inv(int k) const { return q_inv_[k]; }

private:
  real_t fnqs_;
  real_t q_inv_[MAX_N_KINDS];
};

#define THREADS_PER_BLOCK (512)

// FIXME/TODO: we could do this w/o prior reordering, but currently the
// generic moment calculation code first reorders anyway (which it shouldn't)

// ======================================================================
// GCurr

class GCurr {
public:
  DFields d_flds;

  __device__ GCurr(DFields _d_flds) :
    d_flds(_d_flds)
  {
  }

  __device__ void add(int m, int jy, int jz, float val)
  {
    float *addr = &d_flds(m, 0,jy,jz);
    atomicAdd(addr, val);
  }
};

// ======================================================================

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
rho_1st_nc_cuda_run(DParticles dmprts,
		    float4 *d_xi4, float4 *d_pxi4,
		    uint *d_off, int nr_total_blocks, uint *d_ids,
		    DMFields d_flds0)
{
  int block_pos[3];
  int p = dmprts.find_block_pos_patch<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>(block_pos);
  int bid = dmprts.find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0[p]);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      uint id = d_ids[n];
      LOAD_PARTICLE_POS(prt, d_xi4, id);
      LOAD_PARTICLE_MOM(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS(prt, d_xi4, n);
      LOAD_PARTICLE_MOM(prt, d_pxi4, n);
    }

    float fnq = prt.qni_wni * dmprts.fnqs();
    
    int lf[3];
    float of[3];
    dmprts.find_idx_off_1st(prt.xi, lf, of, float(0.));

    scurr.add(0, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// n_1st_cuda_run

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
n_1st_cuda_run(DParticles dmprts,
	       float4 *d_xi4, float4 *d_pxi4,
	       uint *d_off, int nr_total_blocks, uint *d_ids,
	       DMFields d_flds0)
{
  int block_pos[3];
  int p = dmprts.find_block_pos_patch<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>(block_pos);
  int bid = dmprts.find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0[p]);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      uint id = d_ids[n];
      LOAD_PARTICLE_POS(prt, d_xi4, id);
      LOAD_PARTICLE_MOM(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS(prt, d_xi4, n);
      LOAD_PARTICLE_MOM(prt, d_pxi4, n);
    }

    int kind = __float_as_int(prt.kind_as_float);
    float wni = prt.qni_wni * dmprts.q_inv(kind);
    float fnq = wni * dmprts.fnqs();
    
    int lf[3];
    float of[3];
    dmprts.find_idx_off_1st(prt.xi, lf, of, float(-.5));

    scurr.add(kind, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches_no_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
static void
rho_1st_nc_cuda_run_patches_no_reorder(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_const_set(cmprts);
  cuda_mfields_const_set(cmres);
  
  dim3 dimGrid(cmprts->b_mx()[1], cmprts->b_mx()[2] * cmprts->n_patches);

  rho_1st_nc_cuda_run<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (*cmprts, cmprts->d_xi4.data().get(), cmprts->d_pxi4.data().get(),
     cmprts->d_off.data().get(),
     cmprts->n_blocks, cmprts->d_id.data().get(), *cmres);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// n_1st_cuda_run_patches_no_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
static void
n_1st_cuda_run_patches_no_reorder(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_const_set(cmprts);
  cuda_mfields_const_set(cmres);

  dim3 dimGrid(cmprts->b_mx()[1], cmprts->b_mx()[2] * cmprts->n_patches);

  n_1st_cuda_run<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (*cmprts, cmprts->d_xi4.data().get(), cmprts->d_pxi4.data().get(), cmprts->d_off.data().get(),
     cmprts->n_blocks, cmprts->d_id.data().get(), *cmres);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
rho_1st_nc_cuda_run_patches(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cmprts->reorder(); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    rho_1st_nc_cuda_run_patches_no_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// n_1st_cuda_run_patches

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
n_1st_cuda_run_patches(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cmprts->reorder(); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    n_1st_cuda_run_patches_no_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// cuda_moments_yz_rho_1st_nc

void
cuda_moments_yz_rho_1st_nc(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  rho_1st_nc_cuda_run_patches<1, 4, 4>(cmprts, cmres);
}

// ----------------------------------------------------------------------
// cuda_moments_yz_n_1st

void
cuda_moments_yz_n_1st(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  n_1st_cuda_run_patches<1, 4, 4>(cmprts, cmres);
}

