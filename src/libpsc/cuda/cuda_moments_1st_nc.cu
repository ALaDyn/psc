#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_mfields.h"
#include "cuda_mparticles_const.h"
#include "cuda_mfields_const.h"

#include "psc_cuda.h"

#undef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK (512)

// FIXME/TODO: we could do this w/o prior reordering, but currently the
// generic moment calculation code first reorders anyway (which it shouldn't)

// ======================================================================
// GCurr

class GCurr {
public:
  float *d_flds;

  __device__ GCurr(float *_d_flds) :
    d_flds(_d_flds)
  {
  }

  __device__ void add(int m, int jy, int jz, float val)
  {
    float *addr = &D_F3(d_flds, m, 0,jy,jz);
    atomicAdd(addr, val);
  }
};

#define LOAD_PARTICLE_POS_(pp, d_xi4, n) do {				\
    float4 _xi4 = d_xi4[n];						\
    (pp).xi[0]         = _xi4.x;					\
    (pp).xi[1]         = _xi4.y;					\
    (pp).xi[2]         = _xi4.z;					\
    (pp).kind_as_float = _xi4.w;					\
} while (0)

#define LOAD_PARTICLE_MOM_(pp, d_pxi4, n) do {				\
    float4 _pxi4 = d_pxi4[n];						\
    (pp).pxi[0]        = _pxi4.x;					\
    (pp).pxi[1]        = _pxi4.y;					\
    (pp).pxi[2]        = _pxi4.z;					\
    (pp).qni_wni       = _pxi4.w;					\
} while (0)

__device__ static void
find_idx_off_1st(const float xi[3], int j[3], float h[3], float shift)
{
  for (int d = 0; d < 3; d++) {
    float pos = xi[d] * d_cmprts_const.dxi[d] + shift;
    j[d] = __float2int_rd(pos);
    h[d] = pos - j[d];
  }
}

// ======================================================================

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static int
find_block_pos_patch(int *block_pos)
{
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % d_cmprts_const.b_mx[2];

  return blockIdx.y / d_cmprts_const.b_mx[2];
}

__device__ static int
find_bid()
{
  return blockIdx.y * d_cmprts_const.b_mx[1] + blockIdx.x;
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
rho_1st_nc_cuda_run(int block_start,
		    float4 *d_xi4, float4 *d_pxi4,
		    unsigned int *d_off, int nr_total_blocks, unsigned int *d_ids,
		    float *d_flds0, unsigned int size)
{
  int block_pos[3];
  int p = find_block_pos_patch<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>(block_pos);
  int bid = find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0 + p * size);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      unsigned int id = d_ids[n];
      LOAD_PARTICLE_POS_(prt, d_xi4, id);
      LOAD_PARTICLE_MOM_(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS_(prt, d_xi4, n);
      LOAD_PARTICLE_MOM_(prt, d_pxi4, n);
    }

    float fnq = prt.qni_wni * d_cmprts_const.fnqs;
    
    int lf[3];
    float of[3];
    find_idx_off_1st(prt.xi, lf, of, float(0.));

    scurr.add(0, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// n_1st_cuda_run

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
n_1st_cuda_run(int block_start,
	       float4 *d_xi4, float4 *d_pxi4,
	       unsigned int *d_off, int nr_total_blocks, unsigned int *d_ids,
	       float *d_flds0, unsigned int size)
{
  int block_pos[3];
  int p = find_block_pos_patch<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>(block_pos);
  int bid = find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0 + p * size);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      unsigned int id = d_ids[n];
      LOAD_PARTICLE_POS_(prt, d_xi4, id);
      LOAD_PARTICLE_MOM_(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS_(prt, d_xi4, n);
      LOAD_PARTICLE_MOM_(prt, d_pxi4, n);
    }

    int kind = __float_as_int(prt.kind_as_float);
    float wni = prt.qni_wni * d_cmprts_const.q_inv[kind];
    float fnq = wni * d_cmprts_const.fnqs;
    
    int lf[3];
    float of[3];
    find_idx_off_1st(prt.xi, lf, of, float(-.5));

    scurr.add(kind, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches_no_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
static void
rho_1st_nc_cuda_run_patches_no_reorder(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_const_set(cmprts);
  cuda_mfields_const_set(cmres);
  
  unsigned int fld_size = cmres->n_fields * cmres->im[0] * cmres->im[1] * cmres->im[2];

  dim3 dimGrid(cmprts->b_mx[1], cmprts->b_mx[2] * cmprts->n_patches);

  rho_1st_nc_cuda_run<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (0, cmprts->d_xi4, cmprts->d_pxi4,
     cmprts->d_off,
     cmprts->n_blocks, cmprts->d_id,
     cmres->d_flds, fld_size);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// n_1st_cuda_run_patches_no_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
static void
n_1st_cuda_run_patches_no_reorder(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_const_set(cmprts);
  cuda_mfields_const_set(cmres);

  unsigned int fld_size = cmres->n_fields * cmres->im[0] * cmres->im[1] * cmres->im[2];

  dim3 dimGrid(cmprts->b_mx[1], cmprts->b_mx[2] * cmprts->n_patches);

  n_1st_cuda_run<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (0, cmprts->d_xi4, cmprts->d_pxi4, cmprts->d_off,
     cmprts->n_blocks, cmprts->d_id,
     cmres->d_flds, fld_size);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
rho_1st_nc_cuda_run_patches(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_reorder(cmprts); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    rho_1st_nc_cuda_run_patches_no_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// n_1st_cuda_run_patches

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
n_1st_cuda_run_patches(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_reorder(cmprts); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    n_1st_cuda_run_patches_no_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// cuda_moments_yz_rho_1st_nc

void
cuda_moments_yz_rho_1st_nc(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  rho_1st_nc_cuda_run_patches<1, 4, 4>(cmprts, cmres);
}

// ----------------------------------------------------------------------
// cuda_moments_yz_n_1st

void
cuda_moments_yz_n_1st(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  n_1st_cuda_run_patches<1, 4, 4>(cmprts, cmres);
}

