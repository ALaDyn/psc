#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_mfields.h"

#define THREADS_PER_BLOCK (512)

// FIXME/TODO: we could do this w/o prior reordering, but currently the
// generic moment calculation code first reorders anyway (which it shouldn't)

// ======================================================================
// GCurr

class GCurr {
public:
  DFields d_flds;

  __device__ GCurr(DFields _d_flds) :
    d_flds(_d_flds)
  {
  }

  __device__ void add(int m, int jy, int jz, float val)
  {
    float *addr = &d_flds(m, 0,jy,jz);
    atomicAdd(addr, val);
  }
};

// ======================================================================

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run

template<typename BS, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
rho_1st_nc_cuda_run(DMparticlesCuda dmprts,
		    float4 *d_xi4, float4 *d_pxi4,
		    uint *d_off, int nr_total_blocks, uint *d_ids,
		    DMFields d_flds0)
{
  int block_pos[3];
  int p = dmprts.find_block_pos_patch<BS::x::value, BS::y::value, BS::z::value>(block_pos);
  int bid = dmprts.find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0[p]);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      uint id = d_ids[n];
      LOAD_PARTICLE_POS(prt, d_xi4, id);
      LOAD_PARTICLE_MOM(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS(prt, d_xi4, n);
      LOAD_PARTICLE_MOM(prt, d_pxi4, n);
    }

    float fnq = prt.qni_wni * dmprts.fnqs();
    
    int lf[3];
    float of[3];
    dmprts.find_idx_off_1st(prt.xi, lf, of, float(0.));

    scurr.add(0, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// n_1st_cuda_run

template<typename BS, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
n_1st_cuda_run(DMparticlesCuda dmprts,
	       float4 *d_xi4, float4 *d_pxi4,
	       uint *d_off, int nr_total_blocks, uint *d_ids,
	       DMFields d_flds0)
{
  int block_pos[3];
  int p = dmprts.find_block_pos_patch<BS::x::value, BS::y::value, BS::z::value>(block_pos);
  int bid = dmprts.find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0[p]);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      uint id = d_ids[n];
      LOAD_PARTICLE_POS(prt, d_xi4, id);
      LOAD_PARTICLE_MOM(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS(prt, d_xi4, n);
      LOAD_PARTICLE_MOM(prt, d_pxi4, n);
    }

    int kind = __float_as_int(prt.kind_as_float);
    float wni = prt.qni_wni * dmprts.q_inv(kind);
    float fnq = wni * dmprts.fnqs();
    
    int lf[3];
    float of[3];
    dmprts.find_idx_off_1st(prt.xi, lf, of, float(-.5));

    scurr.add(kind, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches_no_reorder

template<typename BS, bool REORDER>
static void
rho_1st_nc_cuda_run_patches_no_reorder(cuda_mparticles<BS>* cmprts, struct cuda_mfields *cmres)
{
  dim3 dimGrid(cmprts->b_mx()[1], cmprts->b_mx()[2] * cmprts->n_patches);

  rho_1st_nc_cuda_run<BS, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (*cmprts, cmprts->d_xi4.data().get(), cmprts->d_pxi4.data().get(),
     cmprts->d_off.data().get(),
     cmprts->n_blocks, cmprts->d_id.data().get(), *cmres);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// n_1st_cuda_run_patches_no_reorder

template<typename BS, bool REORDER>
static void
n_1st_cuda_run_patches_no_reorder(cuda_mparticles<BS>* cmprts, struct cuda_mfields *cmres)
{
  dim3 dimGrid(cmprts->b_mx()[1], cmprts->b_mx()[2] * cmprts->n_patches);

  n_1st_cuda_run<BS, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (*cmprts, cmprts->d_xi4.data().get(), cmprts->d_pxi4.data().get(), cmprts->d_off.data().get(),
     cmprts->n_blocks, cmprts->d_id.data().get(), *cmres);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// cuda_moments_yz_rho_1st_nc

template<typename BS>
void cuda_moments_yz_rho_1st_nc(cuda_mparticles<BS>* cmprts, struct cuda_mfields *cmres)
{
  cmprts->reorder(); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    rho_1st_nc_cuda_run_patches_no_reorder<BS, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// cuda_moments_yz_n_1st

template<typename BS>
void cuda_moments_yz_n_1st(cuda_mparticles<BS>* cmprts, struct cuda_mfields *cmres)
{
  cmprts->reorder(); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    n_1st_cuda_run_patches_no_reorder<BS, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

template void cuda_moments_yz_rho_1st_nc<BS144>(cuda_mparticles<BS144>* cmprts, struct cuda_mfields *cmres);
template void cuda_moments_yz_n_1st<BS144>(cuda_mparticles<BS144>* cmprts, struct cuda_mfields *cmres);
