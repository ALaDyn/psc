#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "psc_cuda.h"
#include "particles_cuda.h"

#undef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK (512)

#define NO_CHECKERBOARD
//#define DEBUG

#include "cuda_common.h"

// FIXME/TODO: we could do this w/o prior reordering, but currently the
// generic moment calculation code first reorders anyway (which it shouldn't)

// ======================================================================
// GCurr

class GCurr {
public:
  real *d_flds;

  __device__ GCurr(real *_d_flds) :
    d_flds(_d_flds)
  {
  }

  __device__ void add(int m, int jy, int jz, float val, struct cuda_params prm)
  {
    float *addr = &F3_DEV_YZ(m, jy,jz);
    atomicAdd(addr, val);
  }
};

#define LOAD_PARTICLE_POS_(pp, d_xi4, n) do {				\
    float4 _xi4 = d_xi4[n];						\
    (pp).xi[0]         = _xi4.x;					\
    (pp).xi[1]         = _xi4.y;					\
    (pp).xi[2]         = _xi4.z;					\
    (pp).kind_as_float = _xi4.w;					\
} while (0)

#define LOAD_PARTICLE_MOM_(pp, d_pxi4, n) do {				\
    float4 _pxi4 = d_pxi4[n];						\
    (pp).pxi[0]        = _pxi4.x;					\
    (pp).pxi[1]        = _pxi4.y;					\
    (pp).pxi[2]        = _pxi4.z;					\
    (pp).qni_wni       = _pxi4.w;					\
} while (0)

__device__ static void
find_idx_off_1st(const real xi[3], int j[3], real h[3], real shift,
		 struct cuda_params prm)
{
  for (int d = 0; d < 3; d++) {
    real pos = xi[d] * prm.dxi[d] + shift;
    j[d] = __float2int_rd(pos);
    h[d] = pos - j[d];
  }
}

// ======================================================================

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static int
find_block_pos_patch(struct cuda_params prm, int *block_pos)
{
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % prm.b_mx[2];

  return blockIdx.y / prm.b_mx[2];
}

__device__ static int
find_bid(struct cuda_params prm)
{
  return blockIdx.y * prm.b_mx[1] + blockIdx.x;
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
rho_1st_nc_cuda_run(int block_start, struct cuda_params prm, float4 *d_xi4, float4 *d_pxi4,
	      unsigned int *d_off, int nr_total_blocks, unsigned int *d_ids,
	      float *d_flds0, unsigned int size)
{
  int block_pos[3];
  int p = find_block_pos_patch<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>
    (prm, block_pos);
  int bid = find_bid(prm);
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0 + p * size);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      unsigned int id = d_ids[n];
      LOAD_PARTICLE_POS_(prt, d_xi4, id);
      LOAD_PARTICLE_MOM_(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS_(prt, d_xi4, n);
      LOAD_PARTICLE_MOM_(prt, d_pxi4, n);
    }

    real fnq = prt.qni_wni * prm.fnqs;
    
    int lf[3];
    real of[3];
    find_idx_off_1st(prt.xi, lf, of, real(0.), prm);

    scurr.add(0, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq, prm);
    scurr.add(0, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq, prm);
    scurr.add(0, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq, prm);
    scurr.add(0, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq, prm);
  }
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches_no_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
static void
rho_1st_nc_cuda_run_patches_no_reorder(struct psc_mparticles *mprts, struct psc_mfields *mres)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
  struct cuda_mparticles *cmprts = mprts_cuda->cmprts;
  assert(cmprts);
  struct psc_mfields_cuda *mres_cuda = psc_mfields_cuda(mres);

  struct cuda_params prm;
  set_params(&prm, ppsc, mprts, mres);

  unsigned int fld_size = mres->nr_fields *
    mres_cuda->im[0] * mres_cuda->im[1] * mres_cuda->im[2];

  int gx = prm.b_mx[1];
  int gy = prm.b_mx[2] * mprts->nr_patches;
  dim3 dimGrid(gx, gy);

  rho_1st_nc_cuda_run<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (0, prm, cmprts->d_xi4, cmprts->d_pxi4,
     mprts_cuda->d_off,
     cmprts->n_blocks, cmprts->d_id,
     mres_cuda->d_flds, fld_size);
  cuda_sync_if_enabled();

  free_params(&prm);
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
rho_1st_nc_cuda_run_patches(struct psc_mparticles *mprts, struct psc_mfields *mres)
{
  struct psc_mparticles_cuda *mprts_cuda = psc_mparticles_cuda(mprts);
    
  psc_mparticles_cuda_copy_to_dev(mprts);
  
  if (!mprts_cuda->need_reorder) {
    rho_1st_nc_cuda_run_patches_no_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, false>(mprts, mres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// yz_moments_rho_1st_nc_cuda_run_patches

void
yz_moments_rho_1st_nc_cuda_run_patches(struct psc_mparticles *mprts, struct psc_mfields *mres)
{
  rho_1st_nc_cuda_run_patches<1, 4, 4>(mprts, mres);
#if 0
  // FIXME, make sure is reordered -- or handle if not
  for (int p = 0; p < mres->nr_patches; p++) {
    do_rho_run(p, psc_mfields_get_patch(mres, p), psc_mparticles_get_patch(mprts, p));
  }
#endif
}

