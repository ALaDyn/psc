#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_mfields.h"
#include "cuda_mparticles_const.h"
#include "cuda_mfields_const.h"

#define THREADS_PER_BLOCK (512)

// FIXME/TODO: we could do this w/o prior reordering, but currently the
// generic moment calculation code first reorders anyway (which it shouldn't)

// ======================================================================
// GCurr

class GCurr {
public:
  float *d_flds;

  __device__ GCurr(float *_d_flds) :
    d_flds(_d_flds)
  {
  }

  __device__ void add(int m, int jy, int jz, float val)
  {
    float *addr = &D_F3(d_flds, m, 0,jy,jz);
    atomicAdd(addr, val);
  }
};

// ======================================================================

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
__device__ static int
find_block_pos_patch(int *block_pos)
{
  block_pos[1] = blockIdx.x;
  block_pos[2] = blockIdx.y % d_cmprts_const.b_mx[2];

  return blockIdx.y / d_cmprts_const.b_mx[2];
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
rho_1st_nc_cuda_run(int block_start,
		    float4 *d_xi4, float4 *d_pxi4,
		    uint *d_off, int nr_total_blocks, uint *d_ids,
		    float *d_flds0, uint size)
{
  int block_pos[3];
  int p = find_block_pos_patch<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>(block_pos);
  int bid = find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0 + p * size);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      uint id = d_ids[n];
      LOAD_PARTICLE_POS(prt, d_xi4, id);
      LOAD_PARTICLE_MOM(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS(prt, d_xi4, n);
      LOAD_PARTICLE_MOM(prt, d_pxi4, n);
    }

    float fnq = prt.qni_wni * d_cmprts_const.fnqs;
    
    int lf[3];
    float of[3];
    find_idx_off_1st(prt.xi, lf, of, float(0.));

    scurr.add(0, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(0, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(0, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// n_1st_cuda_run

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
n_1st_cuda_run(int block_start,
	       float4 *d_xi4, float4 *d_pxi4,
	       uint *d_off, int nr_total_blocks, uint *d_ids,
	       float *d_flds0, uint size)
{
  int block_pos[3];
  int p = find_block_pos_patch<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z>(block_pos);
  int bid = find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  GCurr scurr(d_flds0 + p * size);

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    struct d_particle prt;
    if (REORDER) {
      uint id = d_ids[n];
      LOAD_PARTICLE_POS(prt, d_xi4, id);
      LOAD_PARTICLE_MOM(prt, d_pxi4, id);
    } else {
      LOAD_PARTICLE_POS(prt, d_xi4, n);
      LOAD_PARTICLE_MOM(prt, d_pxi4, n);
    }

    int kind = __float_as_int(prt.kind_as_float);
    float wni = prt.qni_wni * d_cmprts_const.q_inv[kind];
    float fnq = wni * d_cmprts_const.fnqs;
    
    int lf[3];
    float of[3];
    find_idx_off_1st(prt.xi, lf, of, float(-.5));

    scurr.add(kind, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnq);
    scurr.add(kind, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnq);
    scurr.add(kind, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnq);
  }
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches_no_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
static void
rho_1st_nc_cuda_run_patches_no_reorder(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_const_set(cmprts);
  cuda_mfields_const_set(cmres);
  
  uint fld_size = cmres->n_fields * cmres->im[0] * cmres->im[1] * cmres->im[2];

  dim3 dimGrid(cmprts->b_mx[1], cmprts->b_mx[2] * cmprts->n_patches);

  rho_1st_nc_cuda_run<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (0, cmprts->d_xi4, cmprts->d_pxi4,
     cmprts->d_off.data().get(),
     cmprts->n_blocks, cmprts->d_id,
     cmres->d_flds, fld_size);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// n_1st_cuda_run_patches_no_reorder

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z, bool REORDER>
static void
n_1st_cuda_run_patches_no_reorder(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_const_set(cmprts);
  cuda_mfields_const_set(cmres);

  uint fld_size = cmres->n_fields * cmres->im[0] * cmres->im[1] * cmres->im[2];

  dim3 dimGrid(cmprts->b_mx[1], cmprts->b_mx[2] * cmprts->n_patches);

  n_1st_cuda_run<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, REORDER>
    <<<dimGrid, THREADS_PER_BLOCK>>>
    (0, cmprts->d_xi4, cmprts->d_pxi4, cmprts->d_off.data().get(),
     cmprts->n_blocks, cmprts->d_id,
     cmres->d_flds, fld_size);
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// rho_1st_nc_cuda_run_patches

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
rho_1st_nc_cuda_run_patches(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_reorder(cmprts); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    rho_1st_nc_cuda_run_patches_no_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// n_1st_cuda_run_patches

template<int BLOCKSIZE_X, int BLOCKSIZE_Y, int BLOCKSIZE_Z>
static void
n_1st_cuda_run_patches(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  cuda_mparticles_reorder(cmprts); // FIXME/OPT?

  if (!cmprts->need_reorder) {
    n_1st_cuda_run_patches_no_reorder<BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z, false>(cmprts, cmres);
  } else {
    assert(0);
  }
}

// ----------------------------------------------------------------------
// cuda_moments_yz_rho_1st_nc

void
cuda_moments_yz_rho_1st_nc(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  rho_1st_nc_cuda_run_patches<1, 4, 4>(cmprts, cmres);
}

// ----------------------------------------------------------------------
// cuda_moments_yz_n_1st

void
cuda_moments_yz_n_1st(struct cuda_mparticles *cmprts, struct cuda_mfields *cmres)
{
  n_1st_cuda_run_patches<1, 4, 4>(cmprts, cmres);
}

