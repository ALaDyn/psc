
#include "cuda_iface.h"
#include "cuda_mfields.h"
#include "cuda_bits.h"

#include "psc_fields_cuda.h"

#if 1
#define dprintf(...) mprintf(__VA_ARGS__)
#else
#define dprintf(...) do {} while (0)
#endif

MfieldsCuda::MfieldsCuda(const Grid_t& grid, int n_fields, const Int3& ibn)
  : MfieldsBase(grid, n_fields)
{
  dprintf("CMFLDS: ctor\n");
  cmflds = new cuda_mfields(grid, n_fields, ibn);
}

MfieldsCuda::~MfieldsCuda()
{
  dprintf("CMFLDS: dtor\n");
  delete cmflds;
}

fields_single_t MfieldsCuda::get_host_fields()
{
  dprintf("CMFLDS: get_host_fields\n");
  return cmflds->get_host_fields();
}

void MfieldsCuda::copy_to_device(int p, fields_single_t h_flds, int mb, int me)
{
  dprintf("CMFLDS: copy_to_device\n");
  cmflds->copy_to_device(p, h_flds, mb, me);
}

void MfieldsCuda::copy_from_device(int p, fields_single_t h_flds, int mb, int me)
{
  dprintf("CMFLDS: copy_from_device\n");
  cmflds->copy_from_device(p, h_flds, mb, me);
}

void MfieldsCuda::axpy_comp_yz(int ym, float a, PscMfieldsCuda mflds_x, int xm)
{
  dprintf("CMFLDS: axpy_comp_yz\n");
  cmflds->axpy_comp_yz(ym, a, mflds_x->cmflds, xm);
}

void MfieldsCuda::zero_comp(int m)
{
  dprintf("CMFLDS: zero_comp\n");
  assert(grid_.gdims[0] == 1);
  cmflds->zero_comp_yz(m);
}

void MfieldsCuda::zero()
{
  dprintf("CMFLDS: zero\n");
  for (int m = 0; m < cmflds->n_fields; m++) {
    zero_comp(m);
  }
}