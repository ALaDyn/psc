
#include <cstdio>
#include <cassert>
#include <cuda_bits.h>
#include <mrc_common.h>

// ----------------------------------------------------------------------
// cuda_base_init

static size_t used_base;

void
cuda_base_init(void)
{
  static bool first_time = true;
  if (!first_time)
    return;

  first_time = false;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0) {
    printf("There is no device supporting CUDA\n");
    return;
  }

  for (int dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    if (dev == 0) {
      // This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
      if (deviceProp.major == 9999 && deviceProp.minor == 9999)
	printf("There is no device supporting CUDA.\n");
      else if (deviceCount == 1)
	printf("There is 1 device supporting CUDA\n");
      else
	printf("There are %d devices supporting CUDA\n", deviceCount);
    }
    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
    printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
    printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
    printf("  Total amount of global memory:                 %lu bytes\n", deviceProp.totalGlobalMem);
#if CUDART_VERSION >= 2000
    printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
    printf("  Number of cores:                               %d\n", 8 * deviceProp.multiProcessorCount);
#endif
    printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem); 
    printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
    printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n", deviceProp.warpSize);
    printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
	   deviceProp.maxThreadsDim[0],
	   deviceProp.maxThreadsDim[1],
	   deviceProp.maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
	   deviceProp.maxGridSize[0],
	   deviceProp.maxGridSize[1],
	   deviceProp.maxGridSize[2]);
    printf("  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
    printf("  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
    printf("  Clock rate:                                    %.2f GHz\n", deviceProp.clockRate * 1e-6f);
#if CUDART_VERSION >= 2000
    printf("  Concurrent copy and execution:                 %s\n", deviceProp.deviceOverlap ? "Yes" : "No");
#endif
#if CUDART_VERSION >= 2020
    printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
    printf("  Integrated:                                    %s\n", deviceProp.integrated ? "Yes" : "No");
    printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
    printf("  Compute mode:                                  %s\n", deviceProp.computeMode == hipComputeModeDefault ?
	   "Default (multiple host threads can use this device simultaneously)" :
	   deviceProp.computeMode == hipComputeModeExclusive ?
	   "Exclusive (only one host thread at a time can use this device)" :
	   deviceProp.computeMode == hipComputeModeProhibited ?
	   "Prohibited (no host thread can use this device)" :
	   "Unknown");
#endif
  }
  size_t free_byte, total_byte;
  hipError_t ierr;
  ierr = hipMemGetInfo(&free_byte, &total_byte);
  cudaCheck(ierr);
  used_base = total_byte - free_byte;
  mprintf("MEM: used_base = %ld\n", used_base);
  cuda_mem_status("init");
}

// ----------------------------------------------------------------------
// myCudaMalloc

void* myCudaMalloc(size_t len)
{
  void* rv;
  hipError_t ierr;
  ierr = hipMalloc(&rv, len);
  cudaCheck(ierr);
  //mprintf("myCudaMalloc %ld\n", len);

  return rv;
}

// ----------------------------------------------------------------------
// myCudaFree

void myCudaFree(void *ptr)
{
  hipError_t ierr;
  ierr = hipFree(ptr);
  cudaCheck(ierr);
  //mprintf("myCudaFree\n");
}

size_t mem_cuda_mparticles;
size_t mem_cuda_mparticles_alt;
size_t mem_cuda_mfields;
size_t mem_cuda_heating_curand;
size_t mem_cuda_collision_curand;
size_t mem_cuda_bnd_maps;
size_t mem_cuda_bndp;

void cuda_mem_status(const char *msg)
{
  const double MB = 1024.*1024.;
  static size_t last_used;
  hipError_t ierr;
  size_t free_byte, total_byte, used_byte;
  ierr = hipMemGetInfo(&free_byte, &total_byte);
  cudaCheck(ierr);
  used_byte = total_byte - free_byte - used_base;
  size_t accounted = mem_cuda_mparticles + mem_cuda_mfields + mem_cuda_mparticles_alt +
    mem_cuda_heating_curand + mem_cuda_collision_curand +
    mem_cuda_bnd_maps;
  mprintf("MEM: change %ld used %.3g MB (acc: %.3g MB) free %.3g MB (%s)\n", used_byte - last_used, used_byte / MB, accounted / MB, free_byte / MB, msg);
  last_used = used_byte;

  mprintf("MEM: cuda_mparticles: %.3f MB alt %.3f MB\n", mem_cuda_mparticles / MB, mem_cuda_mparticles_alt / MB);
  mprintf("MEM: cuda_mfields: %.3f MB\n", mem_cuda_mfields / MB);
  mprintf("MEM: hiprand: heating %.3f MB collision %.3f MB\n", mem_cuda_heating_curand / MB, mem_cuda_collision_curand / MB);
  mprintf("MEM: bnd: maps %.3f MB bndp %.3f MB\n", mem_cuda_bnd_maps / MB, mem_cuda_bndp / MB);
}
