#include "hip/hip_runtime.h"

#include "psc_cuda.h"

#define PFX(x) push_fields_yz_##x
#include "constants.c"

__global__ static void
push_fields_a_E_yz(real *d_flds)
{
  // FIXME, precalc
  real cny = .5f * d_dt * d_dxi[1];
  real cnz = .5f * d_dt * d_dxi[2];
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_mx[1] - 2*3 && iz < d_mx[2] - 2*3)) // FIXME
    return;

  F3_DEV(EX, 0,iy,iz) +=
    cny * (F3_DEV(HZ, 0,iy,iz) - F3_DEV(HZ, 0,iy-1,iz)) -
    cnz * (F3_DEV(HY, 0,iy,iz) - F3_DEV(HY, 0,iy,iz-1)) -
    .5f * d_dt * F3_DEV(JXI, 0,iy,iz);
  
  F3_DEV(EY, 0,iy,iz) +=
    cnz * (F3_DEV(HX, 0,iy,iz) - F3_DEV(HX, 0,iy,iz-1)) -
    0.f -
    .5f * d_dt * F3_DEV(JYI, 0,iy,iz);
  
  F3_DEV(EZ, 0,iy,iz) +=
    0.f -
    cny * (F3_DEV(HX, 0,iy,iz) - F3_DEV(HX, 0,iy-1,iz)) -
    .5f * d_dt * F3_DEV(JZI, 0,iy,iz);
}

__global__ static void
push_fields_a_H_yz(real *d_flds)
{
  // FIXME, precalc
  real cny = .5f * d_dt * d_dxi[1];
  real cnz = .5f * d_dt * d_dxi[2];
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_mx[1] - 2*3 && iz < d_mx[2] - 2*3)) // FIXME
    return;

  F3_DEV(HX, 0,iy,iz) -=
    cny * (F3_DEV(EZ, 0,iy+1,iz) - F3_DEV(EZ, 0,iy,iz)) -
    cnz * (F3_DEV(EY, 0,iy,iz+1) - F3_DEV(EY, 0,iy,iz));
  
  F3_DEV(HY, 0,iy,iz) -=
    cnz * (F3_DEV(EX, 0,iy,iz+1) - F3_DEV(EX, 0,iy,iz)) -
    0.f;
  
  F3_DEV(HZ, 0,iy,iz) -=
    0.f -
    cny * (F3_DEV(EX, 0,iy+1,iz) - F3_DEV(EX, 0,iy,iz));
}

__global__ static void
push_fields_b_H_yz(real *d_flds)
{
  // FIXME, precalc
  real cny = .5f * d_dt * d_dxi[1];
  real cnz = .5f * d_dt * d_dxi[2];
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_mx[1] - 2*3 && iz < d_mx[2] - 2*3)) // FIXME
    return;

  F3_DEV(HX, 0,iy,iz) -=
    cny * (F3_DEV(EZ, 0,iy+1,iz) - F3_DEV(EZ, 0,iy,iz)) -
    cnz * (F3_DEV(EY, 0,iy,iz+1) - F3_DEV(EY, 0,iy,iz));
  
  F3_DEV(HY, 0,iy,iz) -=
    cnz * (F3_DEV(EX, 0,iy,iz+1) - F3_DEV(EX, 0,iy,iz)) -
    0.f;
      
  F3_DEV(HZ, 0,iy,iz) -=
    0.f -
    cny * (F3_DEV(EX, 0,iy+1,iz) - F3_DEV(EX, 0,iy,iz));
}

__global__ static void
push_fields_b_E_yz(real *d_flds)
{
  // FIXME, precalc
  real cny = .5f * d_dt * d_dxi[1];
  real cnz = .5f * d_dt * d_dxi[2];
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (!(iy < d_mx[1] - 2*3 && iz < d_mx[2] - 2*3)) // FIXME
    return;

  F3_DEV(EX, 0,iy,iz) +=
    cny * (F3_DEV(HZ, 0,iy,iz) - F3_DEV(HZ, 0,iy-1,iz)) -
    cnz * (F3_DEV(HY, 0,iy,iz) - F3_DEV(HY, 0,iy,iz-1)) -
    .5 * d_dt * F3_DEV(JXI, 0,iy,iz);
  
  F3_DEV(EY, 0,iy,iz) +=
    cnz * (F3_DEV(HX, 0,iy,iz) - F3_DEV(HX, 0,iy,iz-1)) -
    0.f -
    .5 * d_dt * F3_DEV(JYI, 0,iy,iz);
  
  F3_DEV(EZ, 0,iy,iz) +=
    0.f -
    cny * (F3_DEV(HX, 0,iy,iz) - F3_DEV(HX, 0,iy-1,iz)) -
    .5 * d_dt * F3_DEV(JZI, 0,iy,iz);
}

EXTERN_C void
cuda_push_fields_a_E_yz(int p, fields_cuda_t *pf)
{
  struct psc_patch *patch = &ppsc->patch[p];

  particles_cuda_t pp;
  push_fields_yz_set_constants(&pp, pf);

  int dimBlock[2] = { BLOCKSIZE_Y, BLOCKSIZE_Z };
  int dimGrid[2]  = { (patch->ldims[1] + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		      (patch->ldims[2] + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  RUN_KERNEL(dimGrid, dimBlock,
	     push_fields_a_E_yz, (pf->d_flds));
}

EXTERN_C void
cuda_push_fields_a_H_yz(int p, fields_cuda_t *pf)
{
  struct psc_patch *patch = &ppsc->patch[p];
  int dimBlock[2] = { BLOCKSIZE_Y, BLOCKSIZE_Z };
  int dimGrid[2]  = { (patch->ldims[1] + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		      (patch->ldims[2] + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  RUN_KERNEL(dimGrid, dimBlock,
	     push_fields_a_H_yz, (pf->d_flds));
}

EXTERN_C void
cuda_push_fields_b_H_yz(int p, fields_cuda_t *pf)
{
  struct psc_patch *patch = &ppsc->patch[p];
  int dimBlock[2] = { BLOCKSIZE_Y, BLOCKSIZE_Z };
  int dimGrid[2]  = { (patch->ldims[1] + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		      (patch->ldims[2] + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  RUN_KERNEL(dimGrid, dimBlock,
	     push_fields_b_H_yz, (pf->d_flds));
}

EXTERN_C void
cuda_push_fields_b_E_yz(int p, fields_cuda_t *pf)
{
  struct psc_patch *patch = &ppsc->patch[p];
  int dimBlock[2] = { BLOCKSIZE_Y, BLOCKSIZE_Z };
  int dimGrid[2]  = { (patch->ldims[1] + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		      (patch->ldims[2] + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  RUN_KERNEL(dimGrid, dimBlock,
	     push_fields_b_E_yz, (pf->d_flds));
}

