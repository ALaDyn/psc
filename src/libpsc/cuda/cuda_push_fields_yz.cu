#include "hip/hip_runtime.h"

#include "cuda_bits.h"
#include "cuda_mfields.h"
#include "cuda_mfields_const.h"

#include "fields.hxx"

#include <psc.h>

// the loops include 2 levels of ghost cells
// they really only need -1:2 and -1:1, respectively (for 1st order)
// but always doing 2:2 seems cheap enough

#define BND 2

// OPT: precalc offset


template<int N_COMPS>
struct DFieldsPatch
{
  using real_t = float;
  
  __device__ DFieldsPatch(real_t* d_flds)
    : d_flds_(d_flds)
  {}
  
  __device__ real_t  operator()(int m, int i, int j, int k) const { return D_F3(d_flds_, m, i,j,k); }
  __device__ real_t& operator()(int m, int i, int j, int k)       { return D_F3(d_flds_, m, i,j,k); }

  real_t *d_flds_;
  uint stride_;
};

template<int N_COMPS>
struct DFields
{
  using real_t = float;
  
  __device__ DFields(real_t* d_flds)
    : d_flds_(d_flds),
      stride_(d_cmflds_const.n_cells_per_patch * N_COMPS)
  {}
  
  DFields(const DFields&) = delete;
  
  __device__ real_t  operator()(int m, int i, int j, int k, int p) const { return D_F3(d_flds_ + p * stride_, m, i,j,k); }
  __device__ real_t& operator()(int m, int i, int j, int k, int p)       { return D_F3(d_flds_ + p * stride_, m, i,j,k); }

  __device__ DFieldsPatch<N_COMPS> operator[](int p) { return DFieldsPatch<N_COMPS>(d_flds_ + p * stride_); }
  
  real_t *d_flds_;
  uint stride_;
};

__global__ static void
push_fields_E_yz(float *d_flds0, float dt, float cny, float cnz,
		 uint size, int gridy)
{
  DFields<NR_FIELDS> MF(d_flds0);
  int bidx_y = blockIdx.y % gridy;
  int p = blockIdx.y / gridy;
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = bidx_y * blockDim.y + threadIdx.y;

  if (!(iy < d_cmflds_const.im[1] - 2 * (2-BND) &&
	iz < d_cmflds_const.im[2] - 2 * (2-BND)))
    return;
  iy -= BND;
  iz -= BND;

  DFieldsPatch<NR_FIELDS> F = MF[p];

  F(EX, 0,iy,iz) +=
    cny * (F(HZ, 0,iy,iz) - F(HZ, 0,iy-1,iz)) -
    cnz * (F(HY, 0,iy,iz) - F(HY, 0,iy,iz-1)) -
    dt * F(JXI, 0,iy,iz);
  
  F(EY, 0,iy,iz) +=
    cnz * (F(HX, 0,iy,iz) - F(HX, 0,iy,iz-1)) -
    0.f -
    dt * F(JYI, 0,iy,iz);
  
  F(EZ, 0,iy,iz) +=
    0.f -
    cny * (F(HX, 0,iy,iz) - F(HX, 0,iy-1,iz)) -
    dt * F(JZI, 0,iy,iz);
}

__global__ static void
push_fields_H_yz(float *d_flds0, float cny, float cnz,
		 uint size, int gridy)
{
  int bidx_y = blockIdx.y % gridy;
  int p = blockIdx.y / gridy;
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = bidx_y * blockDim.y + threadIdx.y;

  if (!(iy < d_cmflds_const.im[1] - 2 * (2-BND) &&
	iz < d_cmflds_const.im[2] - 2 * (2-BND)))
    return;
  iy -= BND;
  iz -= BND;

  float *d_flds = d_flds0 + p * size;

  D_F3(d_flds, HX, 0,iy,iz) -=
    cny * (D_F3(d_flds, EZ, 0,iy+1,iz) - D_F3(d_flds, EZ, 0,iy,iz)) -
    cnz * (D_F3(d_flds, EY, 0,iy,iz+1) - D_F3(d_flds, EY, 0,iy,iz));
  
  D_F3(d_flds, HY, 0,iy,iz) -=
    cnz * (D_F3(d_flds, EX, 0,iy,iz+1) - D_F3(d_flds, EX, 0,iy,iz)) -
    0.f;
  
  D_F3(d_flds, HZ, 0,iy,iz) -=
    0.f -
    cny * (D_F3(d_flds, EX, 0,iy+1,iz) - D_F3(d_flds, EX, 0,iy,iz));
}

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 16
#define BLOCKSIZE_Z 16

void
cuda_push_fields_E_yz(struct cuda_mfields *cmflds, float dt)
{
  if (cmflds->n_patches == 0) {
    return;
  }

  cuda_mfields_const_set(cmflds);
  assert(cmflds->n_fields == NR_FIELDS);

  float cny = dt / cmflds->dx[1];
  float cnz = dt / cmflds->dx[2];
  assert(cmflds->ldims[0] == 1);

  uint size = cmflds->n_fields * cmflds->n_cells_per_patch;

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1] * cmflds->n_patches);

  push_fields_E_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds.data().get(), dt, cny, cnz, size, grid[1]);
  cuda_sync_if_enabled();
}

void
cuda_push_fields_H_yz(struct cuda_mfields *cmflds, float dt)
{
  if (cmflds->n_patches == 0) {
    return;
  }

  cuda_mfields_const_set(cmflds);

  float cny = dt / cmflds->dx[1];
  float cnz = dt / cmflds->dx[2];

  uint size = cmflds->n_fields * cmflds->n_cells_per_patch;

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1] * cmflds->n_patches);

  push_fields_H_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds.data().get(), cny, cnz, size, grid[1]);
  cuda_sync_if_enabled();
}

void
cuda_marder_correct_yz_gold(struct cuda_mfields *cmflds, struct cuda_mfields *cmf,
			    int p, float fac[3],
			    int ly[3], int ry[3],
			    int lz[3], int rz[3])
{
  fields_single_t flds = cmflds->get_host_fields();
  Fields3d<fields_single_t> Flds(flds);
  fields_single_t f = cmf->get_host_fields();
  Fields3d<fields_single_t> F(f);
  
  cmflds->copy_from_device(p, flds, EX, EX + 3);
  cmf->copy_from_device(p, f, 0, 1);
  
  for (int iz = -1; iz < cmflds->ldims[2]; iz++) {
    for (int iy = -1; iy < cmflds->ldims[1]; iy++) {
      if (iy >= -ly[1] && iy < ry[1] &&
	  iz >= -ly[2] && iz < ry[2]) {
	Flds(EY, 0,iy,iz) += fac[1] * (F(0, 0,iy+1,iz) - F(0, 0,iy,iz));
	}
      
      if (iy >= -lz[1] && iy < rz[1] &&
	  iz >= -lz[2] && iz < rz[2]) {
	Flds(EZ, 0,iy,iz) += fac[2] * (F(0, 0,iy,iz+1) - F(0, 0,iy,iz));
      }
    }
  }
  
  cmflds->copy_to_device(p, flds, EX, EX + 3);

  flds.dtor();
  f.dtor();
}

__global__ static void
marder_correct_yz(float *d_flds, float *d_f, float facy, float facz,
		  int lyy, int lyz, int ryy, int ryz,
		  int lzy, int lzz, int rzy, int rzz, int my, int mz)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  iy -= BND;
  iz -= BND;

  if (iy >= -lyy && iy < ryy &&
      iz >= -lyz && iz < ryz) {
    D_F3(d_flds, EY, 0,iy,iz) += 
      facy * (D_F3(d_f, 0, 0,iy+1,iz) - D_F3(d_f, 0, 0,iy,iz));
  }
  
  if (iy >= -lzy && iy < rzy &&
      iz >= -lzz && iz < rzz) {
    D_F3(d_flds, EZ, 0,iy,iz) += 
      facz * (D_F3(d_f, 0, 0,iy,iz+1) - D_F3(d_f, 0, 0,iy,iz));
  }
}

void
cuda_marder_correct_yz(struct cuda_mfields *cmflds, struct cuda_mfields *cmf,
		       int p, float fac[3],
		       int ly[3], int ry[3],
		       int lz[3], int rz[3])
{
#if 0
  cuda_marder_correct_yz_gold(mflds, mf, p, fac, ly, ry, lz, rz);
  return;
#endif

  if (cmflds->n_patches == 0) {
    return;
  }

  uint size = cmflds->n_cells_per_patch;
  int my = cmflds->im[1];
  int mz = cmflds->im[2];

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1]);

  marder_correct_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds.data().get() + p * size * cmflds->n_fields,
					   cmf->d_flds.data().get() + p * size * cmf->n_fields, fac[1], fac[2],
					   ly[1], ly[2], ry[1], ry[2],
					   lz[1], lz[2], rz[1], rz[2], my, mz);
  cuda_sync_if_enabled();
}

// ======================================================================

__global__ static void
calc_dive_yz(float *flds, float *f, float dy, float dz,
	     int ldimsy, int ldimsz, int my, int mz)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy >= ldimsy || iz >= ldimsz) {
    return;
  }

  D_F3(f, 0, 0,iy,iz) = 
    ((D_F3(flds, EY, 0,iy,iz) - D_F3(flds, EY, 0,iy-1,iz)) / dy +
     (D_F3(flds, EZ, 0,iy,iz) - D_F3(flds, EZ, 0,iy,iz-1)) / dz);
}

void
cuda_mfields_calc_dive_yz(struct cuda_mfields *cmflds, struct cuda_mfields *cmf, int p)
{
  float dy = cmflds->dx[1];
  float dz = cmflds->dx[2];

  int my = cmflds->im[1];
  int mz = cmflds->im[2];

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1]);

  calc_dive_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds_by_patch[p], cmf->d_flds_by_patch[p], dy, dz,
				      cmflds->ldims[1], cmflds->ldims[2], my, mz);
  cuda_sync_if_enabled();
}

