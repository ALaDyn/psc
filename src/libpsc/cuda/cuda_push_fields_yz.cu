#include "hip/hip_runtime.h"

#include "cuda_bits.h"
#include "cuda_mfields.h"
#include "cuda_mfields_const.h"

#include "fields.hxx"

#include <psc.h>

// the loops include 2 levels of ghost cells
// they really only need -1:2 and -1:1, respectively (for 1st order)
// but always doing 2:2 seems cheap enough

#define BND 2

// OPT: precalc offset

__global__ static void
push_fields_E_yz(float *d_flds0, float dt, float cny, float cnz,
		 uint size, int gridy)
{
  int bidx_y = blockIdx.y % gridy;
  int p = blockIdx.y / gridy;
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = bidx_y * blockDim.y + threadIdx.y;

  if (!(iy < d_cmflds_const.im[1] - 2 * (2-BND) &&
	iz < d_cmflds_const.im[2] - 2 * (2-BND)))
    return;
  iy -= BND;
  iz -= BND;

  float *d_flds = d_flds0 + p * size;

  D_F3(d_flds, EX, 0,iy,iz) +=
    cny * (D_F3(d_flds, HZ, 0,iy,iz) - D_F3(d_flds, HZ, 0,iy-1,iz)) -
    cnz * (D_F3(d_flds, HY, 0,iy,iz) - D_F3(d_flds, HY, 0,iy,iz-1)) -
    dt * D_F3(d_flds, JXI, 0,iy,iz);
  
  D_F3(d_flds, EY, 0,iy,iz) +=
    cnz * (D_F3(d_flds, HX, 0,iy,iz) - D_F3(d_flds, HX, 0,iy,iz-1)) -
    0.f -
    dt * D_F3(d_flds, JYI, 0,iy,iz);
  
  D_F3(d_flds, EZ, 0,iy,iz) +=
    0.f -
    cny * (D_F3(d_flds, HX, 0,iy,iz) - D_F3(d_flds, HX, 0,iy-1,iz)) -
    dt * D_F3(d_flds, JZI, 0,iy,iz);
}

__global__ static void
push_fields_H_yz(float *d_flds0, float cny, float cnz,
		 uint size, int gridy)
{
  int bidx_y = blockIdx.y % gridy;
  int p = blockIdx.y / gridy;
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = bidx_y * blockDim.y + threadIdx.y;

  if (!(iy < d_cmflds_const.im[1] - 2 * (2-BND) &&
	iz < d_cmflds_const.im[2] - 2 * (2-BND)))
    return;
  iy -= BND;
  iz -= BND;

  float *d_flds = d_flds0 + p * size;

  D_F3(d_flds, HX, 0,iy,iz) -=
    cny * (D_F3(d_flds, EZ, 0,iy+1,iz) - D_F3(d_flds, EZ, 0,iy,iz)) -
    cnz * (D_F3(d_flds, EY, 0,iy,iz+1) - D_F3(d_flds, EY, 0,iy,iz));
  
  D_F3(d_flds, HY, 0,iy,iz) -=
    cnz * (D_F3(d_flds, EX, 0,iy,iz+1) - D_F3(d_flds, EX, 0,iy,iz)) -
    0.f;
  
  D_F3(d_flds, HZ, 0,iy,iz) -=
    0.f -
    cny * (D_F3(d_flds, EX, 0,iy+1,iz) - D_F3(d_flds, EX, 0,iy,iz));
}

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 16
#define BLOCKSIZE_Z 16

void
cuda_push_fields_E_yz(struct cuda_mfields *cmflds, float dt)
{
  if (cmflds->n_patches == 0) {
    return;
  }

  cuda_mfields_const_set(cmflds);

  float cny = dt / cmflds->dx[1];
  float cnz = dt / cmflds->dx[2];
  assert(cmflds->ldims[0] == 1);

  uint size = cmflds->n_fields * cmflds->n_cells_per_patch;

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1] * cmflds->n_patches);

  push_fields_E_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds, dt, cny, cnz, size, grid[1]);
  cuda_sync_if_enabled();
}

void
cuda_push_fields_H_yz(struct cuda_mfields *cmflds, float dt)
{
  if (cmflds->n_patches == 0) {
    return;
  }

  cuda_mfields_const_set(cmflds);

  float cny = dt / cmflds->dx[1];
  float cnz = dt / cmflds->dx[2];

  uint size = cmflds->n_fields * cmflds->n_cells_per_patch;

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1] * cmflds->n_patches);

  push_fields_H_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds, cny, cnz, size, grid[1]);
  cuda_sync_if_enabled();
}

void
cuda_marder_correct_yz_gold(struct cuda_mfields *cmflds, struct cuda_mfields *cmf,
			    int p, float fac[3],
			    int ly[3], int ry[3],
			    int lz[3], int rz[3])
{
  fields_single_t flds = cuda_mfields_get_host_fields(cmflds);
  Fields3d<fields_single_t> Flds(flds);
  fields_single_t f = cuda_mfields_get_host_fields(cmf);
  Fields3d<fields_single_t> F(f);
  
  cuda_mfields_copy_from_device(cmflds, p, flds, EX, EX + 3);
  cuda_mfields_copy_from_device(cmf, p, f, 0, 1);
  
  for (int iz = -1; iz < cmflds->ldims[2]; iz++) {
    for (int iy = -1; iy < cmflds->ldims[1]; iy++) {
      if (iy >= -ly[1] && iy < ry[1] &&
	  iz >= -ly[2] && iz < ry[2]) {
	Flds(EY, 0,iy,iz) += fac[1] * (F(0, 0,iy+1,iz) - F(0, 0,iy,iz));
	}
      
      if (iy >= -lz[1] && iy < rz[1] &&
	  iz >= -lz[2] && iz < rz[2]) {
	Flds(EZ, 0,iy,iz) += fac[2] * (F(0, 0,iy,iz+1) - F(0, 0,iy,iz));
      }
    }
  }
  
  cuda_mfields_copy_to_device(cmflds, p, flds, EX, EX + 3);

  flds.dtor();
  f.dtor();
}

__global__ static void
marder_correct_yz(float *d_flds, float *d_f, float facy, float facz,
		  int lyy, int lyz, int ryy, int ryz,
		  int lzy, int lzz, int rzy, int rzz, int my, int mz)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  iy -= BND;
  iz -= BND;

  if (iy >= -lyy && iy < ryy &&
      iz >= -lyz && iz < ryz) {
    D_F3(d_flds, EY, 0,iy,iz) += 
      facy * (D_F3(d_f, 0, 0,iy+1,iz) - D_F3(d_f, 0, 0,iy,iz));
  }
  
  if (iy >= -lzy && iy < rzy &&
      iz >= -lzz && iz < rzz) {
    D_F3(d_flds, EZ, 0,iy,iz) += 
      facz * (D_F3(d_f, 0, 0,iy,iz+1) - D_F3(d_f, 0, 0,iy,iz));
  }
}

void
cuda_marder_correct_yz(struct cuda_mfields *cmflds, struct cuda_mfields *cmf,
		       int p, float fac[3],
		       int ly[3], int ry[3],
		       int lz[3], int rz[3])
{
#if 0
  cuda_marder_correct_yz_gold(mflds, mf, p, fac, ly, ry, lz, rz);
  return;
#endif

  if (cmflds->n_patches == 0) {
    return;
  }

  uint size = cmflds->n_cells_per_patch;
  int my = cmflds->im[1];
  int mz = cmflds->im[2];

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1]);

  marder_correct_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds + p * size * cmflds->n_fields,
					   cmf->d_flds + p * size * cmf->n_fields, fac[1], fac[2],
					   ly[1], ly[2], ry[1], ry[2],
					   lz[1], lz[2], rz[1], rz[2], my, mz);
  cuda_sync_if_enabled();
}

// ======================================================================

__global__ static void
calc_dive_yz(float *flds, float *f, float dy, float dz,
	     int ldimsy, int ldimsz, int my, int mz)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy >= ldimsy || iz >= ldimsz) {
    return;
  }

  D_F3(f, 0, 0,iy,iz) = 
    ((D_F3(flds, EY, 0,iy,iz) - D_F3(flds, EY, 0,iy-1,iz)) / dy +
     (D_F3(flds, EZ, 0,iy,iz) - D_F3(flds, EZ, 0,iy,iz-1)) / dz);
}

void
cuda_mfields_calc_dive_yz(struct cuda_mfields *cmflds, struct cuda_mfields *cmf, int p)
{
  float dy = cmflds->dx[1];
  float dz = cmflds->dx[2];

  int my = cmflds->im[1];
  int mz = cmflds->im[2];

  int grid[2]  = { (cmflds->ldims[1] + 2*BND + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
		   (cmflds->ldims[2] + 2*BND + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z };
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1]);

  calc_dive_yz<<<dimGrid, dimBlock>>>(cmflds->d_flds_by_patch[p], cmf->d_flds_by_patch[p], dy, dz,
				      cmflds->ldims[1], cmflds->ldims[2], my, mz);
  cuda_sync_if_enabled();
}

