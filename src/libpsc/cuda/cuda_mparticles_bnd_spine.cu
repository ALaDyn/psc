#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_bits.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include <b40c/radixsort_reduction_kernel.h>
#include <b40c/radixsort_scanscatter_kernel3.h>

#include <mrc_profile.h>

using namespace b40c_thrust;

typedef unsigned int K;
typedef unsigned int V;

static const int RADIX_BITS = 4;

#define THREADS_PER_BLOCK 256

// ----------------------------------------------------------------------
// spine_reduce

void cuda_mparticles_bnd::spine_reduce(cuda_mparticles *cmprts)
{
  unsigned int n_blocks = cmprts->n_blocks;
  int *b_mx = cmprts->b_mx;

  thrust::device_ptr<unsigned int> d_spine_cnts(d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(d_bnd_spine_sums);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);

  // OPT?
  thrust::fill(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  const int threads = B40C_RADIXSORT_THREADS;
  if (b_mx[0] == 1 && b_mx[1] == 2 && b_mx[2] == 2) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 2, 2> <<<n_blocks, threads>>>
      (d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 4 && b_mx[2] == 4) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 4, 4> <<<n_blocks, threads>>>
      (d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 8, 8> <<<n_blocks, threads>>>
      (d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 16, 16> <<<n_blocks, threads>>>
      (d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 32, 32> <<<n_blocks, threads>>>
      (d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 64, 64> <<<n_blocks, threads>>>
      (d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
                      NopFunctor<K>, 128, 128> <<<n_blocks, threads>>>
      (d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else {
    printf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();

  thrust::exclusive_scan(d_spine_cnts + n_blocks * 10,
			 d_spine_cnts + n_blocks * 10 + n_blocks + 1,
			 d_spine_sums + n_blocks * 10);
}

// ----------------------------------------------------------------------
// cuda_mprts_spine_reduce_gold

void cuda_mparticles_bnd::spine_reduce_gold(cuda_mparticles *cmprts)
{
  unsigned int n_blocks = cmprts->n_blocks;
  unsigned int n_blocks_per_patch = cmprts->n_blocks_per_patch;
  int *b_mx = cmprts->b_mx;

  thrust::device_ptr<unsigned int> d_spine_cnts(d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(d_bnd_spine_sums);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);

  thrust::fill(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + n_blocks + 1);
  thrust::host_vector<unsigned int> h_spine_cnts(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (CUDA_BND_STRIDE + 1));

  
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int b = 0; b < n_blocks_per_patch; b++) {
      unsigned int bid = b + p * n_blocks_per_patch;
      for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
	unsigned int key = h_bidx[n];
	if (key < 9) {
	  int dy = key % 3;
	  int dz = key / 3;
	  int by = b % b_mx[1];
	  int bz = b / b_mx[1];
	  unsigned int bby = by + 1 - dy;
	  unsigned int bbz = bz + 1 - dz;
	  unsigned int bb = bbz * b_mx[1] + bby;
	  if (bby < b_mx[1] && bbz < b_mx[2]) {
	    h_spine_cnts[(bb + p * n_blocks_per_patch) * 10 + key]++;
	  } else {
	    assert(0);
	  }
	} else if (key == CUDA_BND_S_OOB) {
	  h_spine_cnts[b_mx[1]*b_mx[2]*cmprts->n_patches * 10 + bid]++;
	}
      }
    }
  }  

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts);
  thrust::exclusive_scan(d_spine_cnts + n_blocks * 10,
			 d_spine_cnts + n_blocks * 10 + n_blocks + 1,
			 d_spine_sums + n_blocks * 10);
}

// ----------------------------------------------------------------------
// cuda_mparticles_count_received

__global__ static void
mprts_count_received(int nr_total_blocks, unsigned int *d_alt_bidx, unsigned int *d_spine_cnts)
{
  int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (bid < nr_total_blocks) {
    d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = d_alt_bidx[bid];
  }
}

void
cuda_mparticles_count_received(struct cuda_mparticles *cmprts)
{
  unsigned int n_blocks = cmprts->n_blocks;
  
  mprts_count_received<<<n_blocks, THREADS_PER_BLOCK>>>
    (n_blocks, cmprts->bnd.d_bnd_spine_cnts + 10 * n_blocks, cmprts->bnd.d_bnd_spine_cnts);
}

void
cuda_mparticles_count_received_gold(struct cuda_mparticles *cmprts)
{
  int n_blocks = cmprts->n_blocks;

  thrust::device_ptr<unsigned int> d_spine_cnts(cmprts->bnd.d_bnd_spine_cnts);

  thrust::host_vector<unsigned int> h_spine_cnts(1 + n_blocks * (10 + 1));

  thrust::copy(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (10 + 1), h_spine_cnts.begin());

  for (int bid = 0; bid < n_blocks; bid++) {
    h_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = h_spine_cnts[10 * n_blocks + bid];
  }

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts);
}

void
cuda_mparticles_count_received_v1(struct cuda_mparticles *cmprts)
{
  int n_blocks = cmprts->n_blocks;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_spine_cnts(cmprts->bnd.d_bnd_spine_cnts);

  thrust::host_vector<unsigned int> h_bidx(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_spine_cnts(1 + n_blocks * (10 + 1));

  thrust::copy(d_bidx, d_bidx + cmprts->n_prts, h_bidx.begin());
  thrust::copy(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (10 + 1), h_spine_cnts.begin());
  for (int n = cmprts->n_prts - cmprts->bnd.n_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < n_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }
  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts);
}

// ----------------------------------------------------------------------
// cuda_mparticles_scan_scatter_received

static void __global__
mprts_scan_scatter_received(unsigned int nr_recv, unsigned int nr_prts_prev,
			    unsigned int *d_spine_sums, unsigned int *d_alt_bidx,
			    unsigned int *d_bidx, unsigned int *d_ids)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (n >= nr_recv) {
    return;
  }

  n += nr_prts_prev;

  int nn = d_spine_sums[d_bidx[n] * 10 + CUDA_BND_S_NEW] + d_alt_bidx[n];
  d_ids[nn] = n;
}

void
cuda_mparticles_scan_scatter_received(struct cuda_mparticles *cmprts)
{
  int nr_recv = cmprts->bnd.n_prts_recv;

  if (nr_recv == 0) {
    return;
  }
  
  int nr_prts_prev = cmprts->n_prts - nr_recv;

  int dimGrid = (nr_recv + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  mprts_scan_scatter_received<<<dimGrid, THREADS_PER_BLOCK>>>
    (nr_recv, nr_prts_prev, cmprts->bnd.d_bnd_spine_sums, cmprts->bnd.d_alt_bidx,
     cmprts->d_bidx, cmprts->d_id);
  cuda_sync_if_enabled();
}

void
cuda_mparticles_scan_scatter_received_gold(struct cuda_mparticles *cmprts)
{
  unsigned int n_blocks = cmprts->n_blocks;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_alt_bidx(cmprts->bnd.d_alt_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_spine_sums(cmprts->bnd.d_bnd_spine_sums);

  thrust::host_vector<unsigned int> h_bidx(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_alt_bidx(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_id(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_spine_sums(1 + n_blocks * (10 + 1));

  thrust::copy(d_spine_sums, d_spine_sums + n_blocks * 11, h_spine_sums.begin());
  thrust::copy(d_bidx, d_bidx + cmprts->n_prts, h_bidx.begin());
  thrust::copy(d_alt_bidx, d_alt_bidx + cmprts->n_prts, h_alt_bidx.begin());
  for (int n = cmprts->n_prts - cmprts->bnd.n_prts_recv; n < cmprts->n_prts; n++) {
    int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW] + h_alt_bidx[n];
    h_id[nn] = n;
  }
  thrust::copy(h_id.begin(), h_id.end(), d_id);
}

// ----------------------------------------------------------------------
// sort_pairs_device

void cuda_mparticles_bnd::sort_pairs_device(cuda_mparticles *cmprts)
{
  static int pr_A, pr_B, pr_C, pr_D;
  if (!pr_B) {
    pr_A = prof_register("xchg_cnt_recvd", 1., 0, 0);
    pr_B = prof_register("xchg_top_scan", 1., 0, 0);
    pr_C = prof_register("xchg_ss_recvd", 1., 0, 0);
    pr_D = prof_register("xchg_bottom_scan", 1., 0, 0);
  }

  unsigned int n_blocks = cmprts->n_blocks;

  prof_start(pr_A);
  cuda_mparticles_count_received(cmprts);
  prof_stop(pr_A);

  prof_start(pr_B);
  // FIXME why isn't 10 + 0 enough?
  thrust::device_ptr<unsigned int> d_spine_cnts(d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(d_bnd_spine_sums);
  thrust::exclusive_scan(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (10 + 1), d_spine_sums);
  prof_stop(pr_B);

  prof_start(pr_C);
  cuda_mparticles_scan_scatter_received(cmprts);
  prof_stop(pr_C);

  prof_start(pr_D);
  int *b_mx = cmprts->b_mx;
  if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			8, 8> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_bnd_spine_sums, cmprts->d_bidx, cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			16, 16> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_bnd_spine_sums, cmprts->d_bidx, cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			32, 32> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_bnd_spine_sums, cmprts->d_bidx,cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			64, 64> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_bnd_spine_sums, cmprts->d_bidx, cmprts->d_id, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
                        NopFunctor<K>,
                        NopFunctor<K>,
                        128, 128>
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_bnd_spine_sums, cmprts->d_bidx, cmprts->d_id, cmprts->d_off, n_blocks);
  } else {
    printf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();
  prof_stop(pr_D);

  // d_ids now contains the indices to reorder by
}

void cuda_mparticles_bnd::sort_pairs_gold(cuda_mparticles *cmprts)
{
  unsigned int n_blocks_per_patch = cmprts->n_blocks_per_patch;
  unsigned int n_blocks = cmprts->n_blocks;
  int *b_mx = cmprts->b_mx;

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_spine_cnts(d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(d_bnd_spine_sums);

  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_id(cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + n_blocks + 1);
  thrust::host_vector<unsigned int> h_spine_cnts(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (10 + 1));

  thrust::host_vector<unsigned int> h_spine_sums(1 + n_blocks * (10 + 1));

  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < n_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }

  thrust::exclusive_scan(h_spine_cnts.begin(), h_spine_cnts.end(), h_spine_sums.begin());
  thrust::copy(h_spine_sums.begin(), h_spine_sums.end(), d_spine_sums);

  for (int bid = 0; bid < n_blocks; bid++) {
    int b = bid % n_blocks_per_patch;
    int p = bid / n_blocks_per_patch;
    for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
      unsigned int key = h_bidx[n];
      if (key < 9) {
	int dy = key % 3;
	int dz = key / 3;
	int by = b % b_mx[1];
	int bz = b / b_mx[1];
	unsigned int bby = by + 1 - dy;
	unsigned int bbz = bz + 1 - dz;
	assert(bby < b_mx[1] && bbz < b_mx[2]);
	unsigned int bb = bbz * b_mx[1] + bby;
	int nn = h_spine_sums[(bb + p * n_blocks_per_patch) * 10 + key]++;
	h_id[nn] = n;
      } else { // OOB
	assert(0);
      }
    }
  }
  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
      int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
      h_id[nn] = n;
  }

  thrust::copy(h_id.begin(), h_id.end(), d_id);
  // d_ids now contains the indices to reorder by
}

