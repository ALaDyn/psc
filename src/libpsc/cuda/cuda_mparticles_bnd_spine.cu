#include "hip/hip_runtime.h"

#include "cuda_particles_bnd.h"
#include "cuda_mparticles.h"
#include "cuda_bits.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include <b40c/radixsort_reduction_kernel.h>
#include <b40c/radixsort_scanscatter_kernel3.h>

#include <mrc_profile.h>

using namespace b40c_thrust;

typedef uint K;
typedef uint V;

static const int RADIX_BITS = 4;

#define THREADS_PER_BLOCK 256

// ----------------------------------------------------------------------
// spine_reduce

void cuda_particles_bnd::spine_reduce(cuda_mparticles *cmprts)
{
  uint n_blocks = cmprts->n_blocks;
  int *b_mx = cmprts->indexer.b_mx_;

  // OPT?
  thrust::fill(d_spine_cnts.data(), d_spine_cnts.data() + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  const int threads = B40C_RADIXSORT_THREADS;
  if (b_mx[0] == 1 && b_mx[1] == 2 && b_mx[2] == 2) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 2, 2> <<<n_blocks, threads>>>
      (d_spine_cnts.data().get(), cmprts->d_bidx.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 4 && b_mx[2] == 4) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 4, 4> <<<n_blocks, threads>>>
      (d_spine_cnts.data().get(), cmprts->d_bidx.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 8, 8> <<<n_blocks, threads>>>
      (d_spine_cnts.data().get(), cmprts->d_bidx.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 16, 16> <<<n_blocks, threads>>>
      (d_spine_cnts.data().get(), cmprts->d_bidx.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 32, 32> <<<n_blocks, threads>>>
      (d_spine_cnts.data().get(), cmprts->d_bidx.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 64, 64> <<<n_blocks, threads>>>
      (d_spine_cnts.data().get(), cmprts->d_bidx.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
                      NopFunctor<K>, 128, 128> <<<n_blocks, threads>>>
      (d_spine_cnts.data().get(), cmprts->d_bidx.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else {
    printf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();

  thrust::exclusive_scan(d_spine_cnts.data() + n_blocks * 10,
			 d_spine_cnts.data() + n_blocks * 10 + n_blocks + 1,
			 d_spine_sums.data() + n_blocks * 10);
}

// ----------------------------------------------------------------------
// cuda_mprts_spine_reduce_gold

void cuda_particles_bnd::spine_reduce_gold(cuda_mparticles *cmprts)
{
  uint n_blocks = cmprts->n_blocks;
  uint n_blocks_per_patch = cmprts->n_blocks_per_patch;
  int *b_mx = cmprts->indexer.b_mx_;

  thrust::fill(d_spine_cnts.data(), d_spine_cnts.data() + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  thrust::host_vector<uint> h_bidx(cmprts->d_bidx.data(), cmprts->d_bidx.data() + cmprts->n_prts);
  thrust::host_vector<uint> h_off(cmprts->d_off);
  thrust::host_vector<uint> h_spine_cnts(d_spine_cnts.data(), d_spine_cnts.data() + 1 + n_blocks * (CUDA_BND_STRIDE + 1));

  
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int b = 0; b < n_blocks_per_patch; b++) {
      uint bid = b + p * n_blocks_per_patch;
      for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
	uint key = h_bidx[n];
	if (key < 9) {
	  int dy = key % 3;
	  int dz = key / 3;
	  int by = b % b_mx[1];
	  int bz = b / b_mx[1];
	  uint bby = by + 1 - dy;
	  uint bbz = bz + 1 - dz;
	  uint bb = bbz * b_mx[1] + bby;
	  if (bby < b_mx[1] && bbz < b_mx[2]) {
	    h_spine_cnts[(bb + p * n_blocks_per_patch) * 10 + key]++;
	  } else {
	    assert(0);
	  }
	} else if (key == CUDA_BND_S_OOB) {
	  h_spine_cnts[b_mx[1]*b_mx[2]*cmprts->n_patches * 10 + bid]++;
	}
      }
    }
  }  

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts.begin());
  thrust::exclusive_scan(d_spine_cnts.data() + n_blocks * 10,
			 d_spine_cnts.data() + n_blocks * 10 + n_blocks + 1,
			 d_spine_sums.data() + n_blocks * 10);
}

// ----------------------------------------------------------------------
// k_count_received

__global__ static void
k_count_received(int nr_total_blocks, uint *d_n_recv_by_block, uint *d_spine_cnts)
{
  int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (bid < nr_total_blocks) {
    d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = d_n_recv_by_block[bid];
  }
}

// ----------------------------------------------------------------------
// count_received

void cuda_particles_bnd::count_received(cuda_mparticles *cmprts)
{
  uint n_blocks = cmprts->n_blocks;
  
  k_count_received<<<n_blocks, THREADS_PER_BLOCK>>>
    (n_blocks, d_spine_cnts.data().get() + 10 * n_blocks, d_spine_cnts.data().get());
}

// ----------------------------------------------------------------------
// count_received_gold

void cuda_particles_bnd::count_received_gold(cuda_mparticles *cmprts)
{
  int n_blocks = cmprts->n_blocks;
  thrust::host_vector<uint> h_spine_cnts(1 + n_blocks * (10 + 1));

  thrust::copy(d_spine_cnts.data(), d_spine_cnts.data() + 1 + n_blocks * (10 + 1), h_spine_cnts.begin());

  for (int bid = 0; bid < n_blocks; bid++) {
    h_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = h_spine_cnts[10 * n_blocks + bid];
  }

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts.begin());
}

#if 0
void cuda_mparticles_bnd::count_received_v1(cuda_mparticles *cmprts)
{
  int n_blocks = cmprts->n_blocks;

  thrust::device_ptr<uint> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<uint> d_spine_cnts(d_bnd_spine_cnts);

  thrust::host_vector<uint> h_bidx(cmprts->n_prts);
  thrust::host_vector<uint> h_spine_cnts(1 + n_blocks * (10 + 1));

  thrust::copy(d_bidx, d_bidx + cmprts->n_prts, h_bidx.begin());
  thrust::copy(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (10 + 1), h_spine_cnts.begin());
  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < n_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }
  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts.begin());
}
#endif

// ----------------------------------------------------------------------
// k_scan_scatter_received

static void __global__
k_scan_scatter_received(uint nr_recv, uint nr_prts_prev,
			    uint *d_spine_sums, uint *d_alt_bidx,
			    uint *d_bidx, uint *d_ids)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (n >= nr_recv) {
    return;
  }

  n += nr_prts_prev;

  int nn = d_spine_sums[d_bidx[n] * 10 + CUDA_BND_S_NEW] + d_alt_bidx[n];
  d_ids[nn] = n;
}

// ----------------------------------------------------------------------
// scan_scatter_received

void cuda_particles_bnd::scan_scatter_received(cuda_mparticles *cmprts, uint n_prts_recv)
{
  if (n_prts_recv == 0) {
    return;
  }
  
  uint n_prts_prev = cmprts->n_prts - n_prts_recv;

  int dimGrid = (n_prts_recv + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  k_scan_scatter_received<<<dimGrid, THREADS_PER_BLOCK>>>
    (n_prts_recv, n_prts_prev, d_spine_sums.data().get(), cmprts->d_alt_bidx.data().get(),
     cmprts->d_bidx.data().get(), cmprts->d_id.data().get());
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// scan_scatter_received_gold

void cuda_particles_bnd::scan_scatter_received_gold(cuda_mparticles *cmprts, uint n_prts_recv)
{
  uint n_blocks = cmprts->n_blocks;

  thrust::host_vector<uint> h_bidx(cmprts->n_prts);
  thrust::host_vector<uint> h_alt_bidx(n_prts_recv);
  thrust::host_vector<uint> h_id(cmprts->n_prts);
  thrust::host_vector<uint> h_spine_sums(1 + n_blocks * (10 + 1));

  thrust::copy(d_spine_sums.data(), d_spine_sums.data() + n_blocks * 11, h_spine_sums.begin());
  thrust::copy(cmprts->d_bidx.data(), cmprts->d_bidx.data() + cmprts->n_prts, h_bidx.begin());
  thrust::copy(cmprts->d_alt_bidx.data() + cmprts->n_prts - n_prts_recv,
	       cmprts->d_alt_bidx.data() + cmprts->n_prts,
	       h_alt_bidx.begin());
  for (int n0 = 0; n0 < n_prts_recv; n0++) {
    int n = cmprts->n_prts - n_prts_recv + n0;
    int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW] + h_alt_bidx[n0];
    h_id[nn] = n;
  }
  thrust::copy(h_id.begin(), h_id.end(), cmprts->d_id.begin());
}

// ----------------------------------------------------------------------
// sort_pairs_device

void cuda_particles_bnd::sort_pairs_device(cuda_mparticles *cmprts, uint n_prts_recv)
{
  static int pr_A, pr_B, pr_C, pr_D;
  if (!pr_B) {
    pr_A = prof_register("xchg_cnt_recvd", 1., 0, 0);
    pr_B = prof_register("xchg_top_scan", 1., 0, 0);
    pr_C = prof_register("xchg_ss_recvd", 1., 0, 0);
    pr_D = prof_register("xchg_bottom_scan", 1., 0, 0);
  }

  uint n_blocks = cmprts->n_blocks;

  prof_start(pr_A);
  count_received(cmprts);
  prof_stop(pr_A);

  prof_start(pr_B);
  // FIXME why isn't 10 + 0 enough?
  thrust::exclusive_scan(d_spine_cnts.data(), d_spine_cnts.data() + 1 + n_blocks * (10 + 1), d_spine_sums.data());
  prof_stop(pr_B);

  prof_start(pr_C);
  scan_scatter_received(cmprts, n_prts_recv);
  prof_stop(pr_C);

  prof_start(pr_D);
  int *b_mx = cmprts->indexer.b_mx_;
  if (b_mx[0] == 1 && b_mx[1] == 4 && b_mx[2] == 4) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			4, 4> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_spine_sums.data().get(), cmprts->d_bidx.data().get(), cmprts->d_id.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			8, 8> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_spine_sums.data().get(), cmprts->d_bidx.data().get(), cmprts->d_id.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			16, 16> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_spine_sums.data().get(), cmprts->d_bidx.data().get(), cmprts->d_id.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			32, 32> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_spine_sums.data().get(), cmprts->d_bidx.data().get(), cmprts->d_id.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
			NopFunctor<K>,
			NopFunctor<K>,
			64, 64> 
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_spine_sums.data().get(), cmprts->d_bidx.data().get(), cmprts->d_id.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    ScanScatterDigits3x<K, V, 0, RADIX_BITS, 0,
                        NopFunctor<K>,
                        NopFunctor<K>,
                        128, 128>
      <<<n_blocks, B40C_RADIXSORT_THREADS>>>
      (d_spine_sums.data().get(), cmprts->d_bidx.data().get(), cmprts->d_id.data().get(), cmprts->d_off.data().get(), n_blocks);
  } else {
    printf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();
  prof_stop(pr_D);

  // d_ids now contains the indices to reorder by
}

void cuda_particles_bnd::sort_pairs_gold(cuda_mparticles *cmprts, uint n_prts_recv)
{
  uint n_blocks_per_patch = cmprts->n_blocks_per_patch;
  uint n_blocks = cmprts->n_blocks;
  int *b_mx = cmprts->indexer.b_mx_;

  thrust::host_vector<uint> h_bidx(cmprts->d_bidx.data(), cmprts->d_bidx.data() + cmprts->n_prts);
  thrust::host_vector<uint> h_id(cmprts->n_prts);
  thrust::host_vector<uint> h_off(cmprts->d_off);
  thrust::host_vector<uint> h_spine_cnts(d_spine_cnts.data(), d_spine_cnts.data() + 1 + n_blocks * (10 + 1));

  thrust::host_vector<uint> h_spine_sums(1 + n_blocks * (10 + 1));

  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
    assert(h_bidx[n] < n_blocks);
    h_spine_cnts[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
  }

  thrust::exclusive_scan(h_spine_cnts.begin(), h_spine_cnts.end(), h_spine_sums.begin());
  thrust::copy(h_spine_sums.begin(), h_spine_sums.end(), d_spine_sums.begin());

  for (int bid = 0; bid < n_blocks; bid++) {
    int b = bid % n_blocks_per_patch;
    int p = bid / n_blocks_per_patch;
    for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
      uint key = h_bidx[n];
      if (key < 9) {
	int dy = key % 3;
	int dz = key / 3;
	int by = b % b_mx[1];
	int bz = b / b_mx[1];
	uint bby = by + 1 - dy;
	uint bbz = bz + 1 - dz;
	assert(bby < b_mx[1] && bbz < b_mx[2]);
	uint bb = bbz * b_mx[1] + bby;
	int nn = h_spine_sums[(bb + p * n_blocks_per_patch) * 10 + key]++;
	h_id[nn] = n;
      } else { // OOB
	assert(0);
      }
    }
  }
  for (int n = cmprts->n_prts - n_prts_recv; n < cmprts->n_prts; n++) {
      int nn = h_spine_sums[h_bidx[n] * 10 + CUDA_BND_S_NEW]++;
      h_id[nn] = n;
  }

  thrust::copy(h_id.begin(), h_id.end(), cmprts->d_id.begin());
  // d_ids now contains the indices to reorder by
}

