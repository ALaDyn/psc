#include "hip/hip_runtime.h"

#include "cuda_mparticles.cuh"
#include "cuda_bits.h"

// ----------------------------------------------------------------------
// check_in_patch_unordered_slow

template<typename BS>
bool cuda_mparticles<BS>::check_in_patch_unordered_slow()
{
  auto n_prts_by_patch = this->sizeByPatch();

  uint off = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = this->blockIndex(this->storage.load(off + n), p);
      if (!(bidx >= 0 && bidx <= this->n_blocks)) return false;
    }
    off += n_prts_by_patch[p];
  }

  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_in_patch_unordered_slow()\n");
  return true;
}

// ----------------------------------------------------------------------
// check_bix_id_unordered_slow
//
// checks that block indices are correct,
// id is just enumerating particles

template<typename BS>
bool cuda_mparticles<BS>::check_bidx_id_unordered_slow()
{
  auto n_prts_by_patch = this->sizeByPatch();

  uint off = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = this->blockIndex(this->storage.load(off + n), p);
      if (!(bidx == this->by_block_.d_idx[off+n])) return false;
      if (!(off+n == this->by_block_.d_id[off+n])) return false;
    }
    off += n_prts_by_patch[p];
  }

  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_bidx_id_unordered_slow()\n");
  return true;
}

// ----------------------------------------------------------------------
// check_ordered

template<typename BS>
bool cuda_mparticles<BS>::check_ordered()
{
  thrust::host_vector<float4> h_xi4(this->storage.xi4);
  thrust::host_vector<uint> h_off(this->by_block_.d_off);
  thrust::host_vector<uint> h_id(this->by_block_.d_id);

  printf("check_ordered: need_reorder %s\n", need_reorder ? "true" : "false");

  cuda_mparticles_sort_by_block sort(this->n_blocks);
  if (!need_reorder) {
    MHERE;
    // doesn't have any side effects, just triggers validBlockIndex on the GPU
    sort.find_indices_ids(*this);
    MHERE;
  }

  uint off = 0;
  for (int b = 0; b < this->n_blocks; b++) {
    int p = b / this->n_blocks_per_patch;
    uint off_b = h_off[b], off_e = h_off[b+1];
    if (!(off_e >= off_b)) return false;
    //printf("check_ordered: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    if (!(off_b == off)) return false;
    for (int n = h_off[b]; n < h_off[b+1]; n++) {
      float4 xi4;
      int nn;
      if (need_reorder) {
	nn = h_id[n];
      } else {
	nn = n;
      }
      xi4 = h_xi4[nn];
      uint bidx = this->blockIndex(xi4, p);
      //printf("check_ordered: bidx %d\n", bidx);
      if (b != bidx) {
	int bidx_gpu = sort.d_idx[nn];
	printf("check_ordered: b %d bidx %d n %d nn %d p %d xi4 %g %g %g -- gpu %d\n",
	       b, bidx, n, nn, p, xi4.x, xi4.y, xi4.z, bidx_gpu);
	Int3 bpos = this->blockPosition(&xi4.x);
	printf("block_pos %d %d\n", bpos[1], bpos[2]);
      }
      if (!(b == bidx)) return false;
    }
    off += off_e - off_b;
  }
  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_ordered:\n");
  return true;
}

// ----------------------------------------------------------------------
// check_bidx_after_push

template<typename BS>
bool cuda_mparticles<BS>::check_bidx_after_push()
{
  bool ok = true;

  thrust::host_vector<uint> h_off(this->by_block_.d_off);
  thrust::host_vector<uint> h_bidx(this->by_block_.d_idx);
  thrust::host_vector<float4> h_xi4(this->storage.xi4);
  
  for (int p = 0; p < this->n_patches(); p++) {
    int begin = h_off[p * this->n_blocks_per_patch];
    int end = h_off[(p+1) * this->n_blocks_per_patch];
    for (int n = begin; n < end; n++) {
      float4 xi4 = h_xi4[n];
      int bidx = h_bidx[n];
      int bidx2 = this->blockIndex(xi4, p);
      if (bidx2 < 0) bidx2 = this->n_blocks + p;
      if (bidx != bidx2) {
	Int3 cpos = { int(xi4.x / this->pi_.dxi_[0]),
		      int(xi4.y / this->pi_.dxi_[1]),
		      int(xi4.z / this->pi_.dxi_[2])};
	mprintf("check_bidx: p%d n %d: xi4 %g %g %g bidx %d/%d cpos %d %d %d\n", p, n, xi4.x, xi4.y, xi4.z,
		bidx, bidx2, cpos[0], cpos[1], cpos[2]);
	ok = false;
      }
    }
  }
  return ok;
}

