
#include "cuda_mparticles.h"
#include "cuda_bits.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include <b40c/radixsort_reduction_kernel.h>
#include <b40c/radixsort_scanscatter_kernel3.h>

using namespace b40c_thrust;

typedef unsigned int K;
typedef unsigned int V;

static const int RADIX_BITS = 4;

// layout of the spine
//     lt             self             rb        # from left-top .. self .. right-bottom 
//     0   1   2   3   4   5   6   7   8   NEW
// b0 |   |   |   |   |   |   |   |   |   |   |
// b1 |   |   |   |   |   |   |   |   |   |   |
// b2 |   |   |   |   |   |   |   |   |   |   |
// ...
// bn |   |   |   |   |   |   |   |   |   |   |

//    |   |   |   |   |   |   |   |   |   |   |   |   | ... |   | # oob
//     b0  b1  b2  b3                                        bn

#include <cstdio>
#include <cassert>

// ----------------------------------------------------------------------
// cuda_mparticles_bnd_setup

void
cuda_mparticles_bnd_setup(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  cmprts->bnd.h_bnd_cnt = new unsigned int[cmprts->n_blocks];

  ierr = hipMalloc((void **) &cmprts->bnd.d_bnd_spine_cnts,
		    (1 + cmprts->n_blocks * (CUDA_BND_STRIDE + 1)) * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->bnd.d_bnd_spine_sums,
		    (1 + cmprts->n_blocks * (CUDA_BND_STRIDE + 1)) * sizeof(unsigned int)); cudaCheck(ierr);
}  

// ----------------------------------------------------------------------
// cuda_mparticles_bnd_free_particle_mem

void
cuda_mparticles_bnd_free_particle_mem(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  ierr = hipFree(cmprts->bnd.d_alt_bidx); cudaCheck(ierr);
  ierr = hipFree(cmprts->bnd.d_sums); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_bnd_destroy

void
cuda_mparticles_bnd_destroy(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  delete[] cmprts->bnd.h_bnd_cnt;

  ierr = hipFree(cmprts->bnd.d_bnd_spine_cnts); cudaCheck(ierr);
  ierr = hipFree(cmprts->bnd.d_bnd_spine_sums); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_bnd_reserve_all

void
cuda_mparticles_bnd_reserve_all(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  int n_alloced = cmprts->n_alloced;
  ierr = hipMalloc((void **) &cmprts->bnd.d_alt_bidx, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->bnd.d_sums, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_spine_reduce

void
cuda_mparticles_spine_reduce(struct cuda_mparticles *cmprts)
{
  unsigned int n_blocks = cmprts->n_blocks;
  int *b_mx = cmprts->b_mx;

  thrust::device_ptr<unsigned int> d_spine_cnts(cmprts->bnd.d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(cmprts->bnd.d_bnd_spine_sums);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);

  // OPT?
  thrust::fill(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  const int threads = B40C_RADIXSORT_THREADS;
  if (b_mx[0] == 1 && b_mx[1] == 2 && b_mx[2] == 2) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 2, 2> <<<n_blocks, threads>>>
      (cmprts->bnd.d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 4 && b_mx[2] == 4) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 4, 4> <<<n_blocks, threads>>>
      (cmprts->bnd.d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 8 && b_mx[2] == 8) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 8, 8> <<<n_blocks, threads>>>
      (cmprts->bnd.d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 16 && b_mx[2] == 16) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 16, 16> <<<n_blocks, threads>>>
      (cmprts->bnd.d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 32 && b_mx[2] == 32) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 32, 32> <<<n_blocks, threads>>>
      (cmprts->bnd.d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 64 && b_mx[2] == 64) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
		      NopFunctor<K>, 64, 64> <<<n_blocks, threads>>>
      (cmprts->bnd.d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else if (b_mx[0] == 1 && b_mx[1] == 128 && b_mx[2] == 128) {
    RakingReduction3x<K, V, 0, RADIX_BITS, 0,
                      NopFunctor<K>, 128, 128> <<<n_blocks, threads>>>
      (cmprts->bnd.d_bnd_spine_cnts, cmprts->d_bidx, cmprts->d_off, n_blocks);
  } else {
    printf("no support for b_mx %d x %d x %d!\n", b_mx[0], b_mx[1], b_mx[2]);
    assert(0);
  }
  cuda_sync_if_enabled();

  thrust::exclusive_scan(d_spine_cnts + n_blocks * 10,
			 d_spine_cnts + n_blocks * 10 + n_blocks + 1,
			 d_spine_sums + n_blocks * 10);
}

// ----------------------------------------------------------------------
// cuda_mprts_spine_reduce_gold

void
cuda_mparticles_spine_reduce_gold(struct cuda_mparticles *cmprts)
{
  unsigned int n_blocks = cmprts->n_blocks;
  unsigned int n_blocks_per_patch = cmprts->n_blocks_per_patch;
  int *b_mx = cmprts->b_mx;

  thrust::device_ptr<unsigned int> d_spine_cnts(cmprts->bnd.d_bnd_spine_cnts);
  thrust::device_ptr<unsigned int> d_spine_sums(cmprts->bnd.d_bnd_spine_sums);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);

  thrust::fill(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (CUDA_BND_STRIDE + 1), 0);

  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + n_blocks + 1);
  thrust::host_vector<unsigned int> h_spine_cnts(d_spine_cnts, d_spine_cnts + 1 + n_blocks * (CUDA_BND_STRIDE + 1));

  
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int b = 0; b < n_blocks_per_patch; b++) {
      unsigned int bid = b + p * n_blocks_per_patch;
      for (int n = h_off[bid]; n < h_off[bid+1]; n++) {
	unsigned int key = h_bidx[n];
	if (key < 9) {
	  int dy = key % 3;
	  int dz = key / 3;
	  int by = b % b_mx[1];
	  int bz = b / b_mx[1];
	  unsigned int bby = by + 1 - dy;
	  unsigned int bbz = bz + 1 - dz;
	  unsigned int bb = bbz * b_mx[1] + bby;
	  if (bby < b_mx[1] && bbz < b_mx[2]) {
	    h_spine_cnts[(bb + p * n_blocks_per_patch) * 10 + key]++;
	  } else {
	    assert(0);
	  }
	} else if (key == CUDA_BND_S_OOB) {
	  h_spine_cnts[b_mx[1]*b_mx[2]*cmprts->n_patches * 10 + bid]++;
	}
      }
    }
  }  

  thrust::copy(h_spine_cnts.begin(), h_spine_cnts.end(), d_spine_cnts);
  thrust::exclusive_scan(d_spine_cnts + n_blocks * 10,
			 d_spine_cnts + n_blocks * 10 + n_blocks + 1,
			 d_spine_sums + n_blocks * 10);
}

