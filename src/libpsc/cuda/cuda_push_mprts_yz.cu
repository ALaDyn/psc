#include "hip/hip_runtime.h"

#include "cuda_iface.h"
#include "cuda_mparticles.h"
#include "cuda_mfields.h"
#include "cuda_push_particles.cuh"
#include "push_particles_cuda_impl.hxx"
#include "range.hxx"

#define DIM DIM_YZ

#include "../psc_push_particles/inc_defs.h"

#include "psc.h" // FIXME

#include "dim.hxx"

using dim = dim_yz;

#include "interpolate.hxx"
#include "pushp.hxx"

#define BND (2) // FIXME

#define THREADS_PER_BLOCK (512)

#include "cuda_fld_cache.cuh"
#include "cuda_currmem.cuh"

// FIXME
#define CUDA_BND_S_OOB (10)

// ----------------------------------------------------------------------

// OPT: use more shmem?

// OPT: passing shared memory cache etc around is probably sub-optimal
// OPT: fld cache is much bigger than needed
// OPT: precalculating IP coeffs could be a gain, too

template<typename Config, bool REORDER>
struct CudaPushParticles_yz
{
  using BS = typename Config::Bs;
  using Currmem = typename Config::Currmem;
  using Curr = typename Currmem::Curr<BS>;
  using DMparticles = DMparticlesCuda<BS>;
  using real_t = typename DMparticles::real_t;
  using FldCache = FldCache<BS::x::value, BS::y::value, BS::z::value>;

  // ----------------------------------------------------------------------
  // push_part_one

  __device__ static void
  push_part_one(DMparticles& dmprts, struct d_particle& prt, int n, const FldCache& fld_cache)
    
  {
    uint id;
    if (REORDER) {
      id = dmprts.id_[n];
      LOAD_PARTICLE_POS(prt, dmprts.xi4_, id);
    } else {
      LOAD_PARTICLE_POS(prt, dmprts.xi4_, n);
  }
    // here we have x^{n+.5}, p^n
    
    // field interpolation
    real_t xm[3];
    dmprts.scalePos(xm, prt.xi);
    InterpolateEM<FldCache, typename Config::Ip, dim_yz> ip;
    AdvanceParticle<real_t, dim> advance{dmprts.dt()};
    
    ip.set_coeffs(xm);
    
    real_t E[3] = { ip.ex(fld_cache), ip.ey(fld_cache), ip.ez(fld_cache) };
    real_t H[3] = { ip.hx(fld_cache), ip.hy(fld_cache), ip.hz(fld_cache) };
    
    // x^(n+0.5), p^n -> x^(n+0.5), p^(n+1.0) 
    int kind = __float_as_int(prt.kind_as_float);
    real_t dq = dmprts.dq(kind);
    if (REORDER) {
      LOAD_PARTICLE_MOM(prt, dmprts.pxi4_, id);
      advance.push_p(prt.pxi, E, H, dq);
      STORE_PARTICLE_MOM(prt, dmprts.alt_pxi4_, n);
    } else {
      LOAD_PARTICLE_MOM(prt, dmprts.pxi4_, n);
      advance.push_p(prt.pxi, E, H, dq);
      STORE_PARTICLE_MOM(prt, dmprts.pxi4_, n);
    }
  }
  // ======================================================================
  // depositing current
  
  // ----------------------------------------------------------------------
  // calc_dx1
  
  __device__ static void
  calc_dx1(float dx1[3], float x[3], float dx[3], int off[3])
  {
    float o1, x1, dx_0, dx_1, dx_2, v0, v1, v2;
    if (off[1] == 0) {
      o1 = off[2];
      x1 = x[2];
      dx_0 = dx[0];
      dx_1 = dx[2];
      dx_2 = dx[1];
    } else {
      o1 = off[1];
      x1 = x[1];
      dx_0 = dx[0];
      dx_1 = dx[1];
      dx_2 = dx[2];
    }
    if ((off[1] == 0 && off[2] == 0) || dx_1 == 0.f) {
      v0 = 0.f;
      v1 = 0.f;
      v2 = 0.f;
    } else {
      v1 = .5f * o1 - x1;
      v2 = dx_2 / dx_1 * v1;
      v0 = dx_0 / dx_1 * v1;
    }
    if (off[1] == 0) {
      dx1[0] = v0;
      dx1[1] = v2;
      dx1[2] = v1;
    } else {
      dx1[0] = v0;
      dx1[1] = v1;
      dx1[2] = v2;
    }
  }
  
  // ----------------------------------------------------------------------
  // curr_vb_cell
  
  __device__ static void
  curr_vb_cell(DMparticles& dmprts, int i[3], float x[3], float dx[3], float qni_wni,
	       Curr &scurr, const Block& current_block)
  {
    float xa[3] = { 0.,
		    x[1] + .5f * dx[1],
		    x[2] + .5f * dx[2], };
    if (Config::Deposit::value == DEPOSIT_VB_3D) {
      if (dx[0] != 0.f) {
	float fnqx = qni_wni * dmprts.fnqxs();
	float h = (1.f / 12.f) * dx[0] * dx[1] * dx[2];
	scurr.add(0, i[1]  , i[2]  , fnqx * (dx[0] * (.5f - xa[1]) * (.5f - xa[2]) + h), current_block.ci0);
	scurr.add(0, i[1]+1, i[2]  , fnqx * (dx[0] * (.5f + xa[1]) * (.5f - xa[2]) - h), current_block.ci0);
	scurr.add(0, i[1]  , i[2]+1, fnqx * (dx[0] * (.5f - xa[1]) * (.5f + xa[2]) + h), current_block.ci0);
	scurr.add(0, i[1]+1, i[2]+1, fnqx * (dx[0] * (.5f + xa[1]) * (.5f + xa[2]) - h), current_block.ci0);
      }
    }
    if (dx[1] != 0.f) {
      float fnqy = qni_wni * dmprts.fnqys();
      scurr.add(1, i[1],i[2]  , fnqy * dx[1] * (.5f - xa[2]), current_block.ci0);
      scurr.add(1, i[1],i[2]+1, fnqy * dx[1] * (.5f + xa[2]), current_block.ci0);
    }
    if (dx[2] != 0.f) {
      float fnqz = qni_wni * dmprts.fnqzs();
      scurr.add(2, i[1]  ,i[2], fnqz * dx[2] * (.5f - xa[1]), current_block.ci0);
      scurr.add(2, i[1]+1,i[2], fnqz * dx[2] * (.5f + xa[1]), current_block.ci0);
    }
  }

  // ----------------------------------------------------------------------
  // curr_vb_cell_upd
  
  __device__ static void
  curr_vb_cell_upd(int i[3], float x[3], float dx1[3], float dx[3], int off[3])
  {
    dx[0] -= dx1[0];
    dx[1] -= dx1[1];
    dx[2] -= dx1[2];
    x[1] += dx1[1] - off[1];
    x[2] += dx1[2] - off[2];
    i[1] += off[1];
    i[2] += off[2];
  }
  
  // ----------------------------------------------------------------------
  // yz_calc_j
  
  __device__ static void
  yz_calc_j(DMparticles& dmprts, struct d_particle& prt, int n, float4 *d_xi4, float4 *d_pxi4,
	    Curr &scurr, const Block& current_block)
  {
    AdvanceParticle<real_t, dim> advance{dmprts.dt()};

    float vxi[3];
    advance.calc_v(vxi, prt.pxi);

    // position xm at x^(n+.5)
    float h0[3], h1[3];
    float xm[3], xp[3];
    int j[3], k[3];
  
    dmprts.find_idx_off_pos_1st(prt.xi, j, h0, xm, float(0.));

    if (Config::Deposit::value == DEPOSIT_VB_2D) {
      // x^(n+0.5), p^(n+1.0) -> x^(n+1.0), p^(n+1.0) 
      advance.push_x(prt.xi, vxi, .5f);

      float fnqx = vxi[0] * prt.qni_wni * dmprts.fnqs();

      // out-of-plane currents at intermediate time
      int lf[3];
      float of[3];
      dmprts.find_idx_off_1st(prt.xi, lf, of, float(0.));
      lf[1] -= current_block.ci0[1];
      lf[2] -= current_block.ci0[2];

      scurr.add(0, lf[1]  , lf[2]  , (1.f - of[1]) * (1.f - of[2]) * fnqx, current_block.ci0);
      scurr.add(0, lf[1]+1, lf[2]  , (      of[1]) * (1.f - of[2]) * fnqx, current_block.ci0);
      scurr.add(0, lf[1]  , lf[2]+1, (1.f - of[1]) * (      of[2]) * fnqx, current_block.ci0);
      scurr.add(0, lf[1]+1, lf[2]+1, (      of[1]) * (      of[2]) * fnqx, current_block.ci0);

      // x^(n+1.0), p^(n+1.0) -> x^(n+1.5), p^(n+1.0) 
      advance.push_x(prt.xi, vxi, .5f);
      STORE_PARTICLE_POS(prt, d_xi4, n);
    } else if (Config::Deposit::value == DEPOSIT_VB_3D) {
      // x^(n+0.5), p^(n+1.0) -> x^(n+1.5), p^(n+1.0) 
      advance.push_x(prt.xi, vxi);
      STORE_PARTICLE_POS(prt, d_xi4, n);
    }

    // has moved into which block? (given as relative shift)
    dmprts.bidx_[n] = dmprts.blockShift(prt.xi, current_block.p, current_block.bid);

    // position xm at x^(n+.5)
    dmprts.find_idx_off_pos_1st(prt.xi, k, h1, xp, float(0.));

    // deposit xm -> xp
    int idiff[3] = { 0, k[1] - j[1], k[2] - j[2] };
    int i[3] = { 0, j[1] - current_block.ci0[1], j[2] - current_block.ci0[2] };
    float x[3] = { 0.f, xm[1] - j[1] - float(.5), xm[2] - j[2] - float(.5) };
    //float dx[3] = { 0.f, xp[1] - xm[1], xp[2] - xm[2] };
    float dx[3] = { vxi[0] * dmprts.dt() * dmprts.dxi(0), xp[1] - xm[1], xp[2] - xm[2] };
  
    float x1 = x[1] * idiff[1];
    float x2 = x[2] * idiff[2];
    int d_first = (fabsf(dx[2]) * (.5f - x1) >= fabsf(dx[1]) * (.5f - x2));

    int off[3];
    if (d_first == 0) {
      off[1] = idiff[1];
      off[2] = 0;
    } else {
      off[1] = 0;
      off[2] = idiff[2];
    }

    float dx1[3];
    calc_dx1(dx1, x, dx, off);
    curr_vb_cell(dmprts, i, x, dx1, prt.qni_wni, scurr, current_block);
    curr_vb_cell_upd(i, x, dx1, dx, off);
  
    off[1] = idiff[1] - off[1];
    off[2] = idiff[2] - off[2];
    calc_dx1(dx1, x, dx, off);
    curr_vb_cell(dmprts, i, x, dx1, prt.qni_wni, scurr, current_block);
    curr_vb_cell_upd(i, x, dx1, dx, off);
    
    curr_vb_cell(dmprts, i, x, dx, prt.qni_wni, scurr, current_block);
  }

  // ----------------------------------------------------------------------
  // push_mprts

  __device__
  static void push_mprts(DMparticles& dmprts, DMFields& d_mflds, int block_start)
  {
    int block_pos[3];
    Block current_block;
    current_block.p = Currmem::template find_block_pos_patch<BS>(dmprts, block_pos, current_block.ci0, block_start);
    if (current_block.p < 0)
      return;
    current_block.bid = Currmem::find_bid(dmprts, current_block.p, block_pos);
    int block_begin = dmprts.off_[current_block.bid];
    int block_end = dmprts.off_[current_block.bid + 1];
    
    __shared__ FldCache fld_cache;
    fld_cache.load(d_mflds[current_block.p], current_block.ci0);

    __shared__ float _scurr[Curr::shared_size];
    Curr scurr(_scurr, d_mflds[current_block.p]);
    __syncthreads();

    for (int n : in_block_loop(block_begin, block_end)) {
      if (n < block_begin) {
	continue;
      }
      struct d_particle prt;
      push_part_one(dmprts, prt, n, fld_cache);
      
      if (REORDER) {
	yz_calc_j(dmprts, prt, n, dmprts.alt_xi4_, dmprts.alt_pxi4_, scurr, current_block);
      } else {
	yz_calc_j(dmprts, prt, n, dmprts.xi4_, dmprts.pxi4_, scurr, current_block);
      }
    }
    
    scurr.add_to_fld(current_block.ci0);
  }
};

// ----------------------------------------------------------------------
// push_mprts_ab

template<typename Config, bool REORDER>
__global__ static void
__launch_bounds__(THREADS_PER_BLOCK, 3)
push_mprts_ab(int block_start, DMparticlesCuda<typename Config::Bs> dmprts, DMFields d_mflds)
{
  CudaPushParticles_yz<Config, REORDER>::push_mprts(dmprts, d_mflds, block_start);
}

// ----------------------------------------------------------------------
// zero_currents

static void
zero_currents(struct cuda_mfields *cmflds)
{
  // OPT: j as separate field, so we can use a single memset?
  for (int p = 0; p < cmflds->n_patches; p++) {
    uint size = cmflds->n_cells_per_patch;
    hipError_t ierr = hipMemset((*cmflds)[p].data() + JXI * size, 0,
				3 * size * sizeof(fields_cuda_real_t));
    cudaCheck(ierr);
  }
}

// ----------------------------------------------------------------------
// cuda_push_mprts_ab

template<typename Config>
template<bool REORDER>
void CudaPushParticles_<Config>::push_mprts_ab(CudaMparticles* cmprts, struct cuda_mfields *cmflds)
{
  using Currmem = typename Config::Currmem;

  zero_currents(cmflds);

  dim3 dimGrid = Currmem::dimGrid(*cmprts);

  if (REORDER) {
    cmprts->d_alt_xi4.resize(cmprts->n_prts);
    cmprts->d_alt_pxi4.resize(cmprts->n_prts);
  }

  for (auto block_start : Currmem::block_starts()) {
    ::push_mprts_ab<Config, REORDER>
      <<<dimGrid, THREADS_PER_BLOCK>>>(block_start, *cmprts, *cmflds);
    cuda_sync_if_enabled();
  }

  if (REORDER) {
    cmprts->swap_alt();
    cmprts->need_reorder = false;
  }
}

// ----------------------------------------------------------------------
// push_mprts_yz

template<typename Config>
void CudaPushParticles_<Config>::push_mprts_yz(CudaMparticles* cmprts, struct cuda_mfields *cmflds)
{
  if (!cmprts->need_reorder) {
    //    printf("INFO: yz_cuda_push_mprts: need_reorder == false\n");
    push_mprts_ab<false>(cmprts, cmflds);
  } else {
    push_mprts_ab<true>(cmprts, cmflds);
  }
}

template struct CudaPushParticles_<Config1vb>;
template struct CudaPushParticles_<Config1vbec3d>;
