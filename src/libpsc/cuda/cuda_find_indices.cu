#include "hip/hip_runtime.h"

#include <psc_cuda.h>

// FIXME, do this always?
#define NO_CHECKERBOARD

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 4
#define BLOCKSIZE_Z 4

#define PFX(x) sort_##x
#include "constants.c"

// FIXME, use const mem for some params

__global__ static void find_cell_indices_by_cell(int n_part, particles_cuda_dev_t h_dev,
						 int *d_cnis, int *d_ids, int ldims_y)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      h_dev.xi4[i].x * d_consts.dxi[0],
      h_dev.xi4[i].y * d_consts.dxi[1],
      h_dev.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = cuda_fint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (ldims_y / 8) + (pos[1] / 8)) << 6);
    idx |=
      ((pos[2] & 4) << 3) |
      ((pos[1] & 4) << 2);
    idx |=
      ((pos[2] & 2) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 1) << 0);
    d_cnis[i] = idx;
    d_ids[i] = i;
  }
}

static void
sort_find_cell_indices_by_cell_device(struct psc_particles *prts, struct psc_patch *patch,
				      int *d_cnis, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_cell_indices_by_cell, (prts->n_part, *cuda->h_dev, d_cnis, d_ids,
					 patch->ldims[1]));
}

// FIXME, specific to 1x8x8, should be in ! .cu, so that cell_map works

static void __unused
sort_find_cell_indices_host(struct psc_particles *prts, struct psc_patch *patch,
			    int *d_cnis, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int n_part = prts->n_part;
  particles_cuda_dev_t *h_dev = cuda->h_dev;
  int *h_cnis = (int *) malloc(n_part * sizeof(*h_cnis));
  int *h_ids = (int *) malloc(n_part * sizeof(*h_ids));

  float4 *h_xi4 = (float4 *) malloc(n_part * sizeof(*h_xi4));
  check(hipMemcpy(h_xi4, h_dev->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));

  for (int i = 0; i < n_part; i++) {
    particle_cuda_real_t dxi = 1.f / ppsc->dx[0];
    particle_cuda_real_t dyi = 1.f / ppsc->dx[1];
    particle_cuda_real_t dzi = 1.f / ppsc->dx[2];
    particle_cuda_real_t xi[3] = {
      (h_xi4[i].x - patch->xb[0]) * dxi,
      (h_xi4[i].y - patch->xb[1]) * dyi,
      (h_xi4[i].z - patch->xb[2]) * dzi };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = particle_cuda_real_fint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (patch->ldims[1] / 8) + (pos[1] / 8)) << 6) |
      ((pos[2] & 4) << 3) |
      ((pos[2] & 2) << 2) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 4) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[1] & 1) << 0);
    h_cnis[i] = idx;
    h_ids[i] = i;
  }

  check(hipMemcpy(d_cnis, h_cnis, n_part * sizeof(*h_cnis),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_ids, h_ids, n_part * sizeof(*h_ids),
		   hipMemcpyHostToDevice));

  free(h_xi4);
  free(h_cnis);
  free(h_ids);
}

static void __unused
sort_reorder_host(struct psc_particles *prts, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int n_part = prts->n_part;
  particles_cuda_dev_t *h_dev = cuda->h_dev;
  int *h_ids = (int *) malloc(n_part);

  float4 *h_xi4 = (float4 *) malloc(n_part * sizeof(*h_xi4));
  float4 *h_pxi4 = (float4 *) malloc(n_part * sizeof(*h_pxi4));
  check(hipMemcpy(h_xi4, h_dev->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_pxi4, h_dev->pxi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_ids, d_ids, n_part * sizeof(*h_ids),
		   hipMemcpyDeviceToHost));

  // move into new position
  float4 *xi4 = (float4 *) malloc(n_part * sizeof(*xi4));
  float4 *pxi4 = (float4 *) malloc(n_part * sizeof(*pxi4));
  for (int i = 0; i < n_part; i++) {
    xi4[i] = h_xi4[h_ids[i]];
    pxi4[i] = h_pxi4[h_ids[i]];
  }
  
  check(hipMemcpy(h_dev->xi4, xi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(h_dev->pxi4, pxi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  free(xi4);
  free(pxi4);
  free(h_xi4);
  free(h_pxi4);
  free(h_ids);
}

__global__ static void
sort_reorder_by_cell(int n_part, particles_cuda_dev_t h_dev, float4 *xi4, float4 *pxi4,
		     int *d_cnis, int *d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < n_part) {
    xi4[i] = h_dev.xi4[d_ids[i]];
    pxi4[i] = h_dev.pxi4[d_ids[i]];
  }
}

static void
sort_reorder_by_cell_device(struct psc_particles *prts, int *d_cnis, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4, *pxi4;
  check(hipMalloc((void **) &xi4, cuda->n_alloced * sizeof(*xi4)));
  check(hipMalloc((void **) &pxi4, cuda->n_alloced * sizeof(*pxi4)));

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     sort_reorder_by_cell, (prts->n_part, *cuda->h_dev, xi4, pxi4, d_cnis, d_ids));

  check(hipFree(cuda->h_dev->xi4));
  check(hipFree(cuda->h_dev->pxi4));
  cuda->h_dev->xi4 = xi4;
  cuda->h_dev->pxi4 = pxi4;
}

EXTERN_C void
sort_patch_prep(int p, struct psc_particles *prts, int **d_cnis, int **d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  check(hipMalloc((void **) d_cnis, cuda->n_alloced * sizeof(*d_cnis)));
  check(hipMalloc((void **) d_ids, cuda->n_alloced * sizeof(*d_ids)));

  sort_set_constants(prts, NULL);
}

EXTERN_C void
sort_patch_done(int p, struct psc_particles *prts, int *d_cnis, int *d_ids)
{
  check(hipFree(d_cnis));
  check(hipFree(d_ids));
}

EXTERN_C void
sort_patch_by_cell(int p, struct psc_particles *prts)
{
  struct psc_patch *patch = &ppsc->patch[p];

  int *d_cnis, *d_ids;
  check(hipMalloc((void **) &d_cnis, prts->n_part * sizeof(*d_cnis)));
  check(hipMalloc((void **) &d_ids, prts->n_part * sizeof(*d_ids)));

  sort_set_constants(prts, NULL);

  sort_find_cell_indices_by_cell_device(prts, patch, d_cnis, d_ids);
  sort_pairs_device((unsigned int *) d_cnis, (unsigned int *) d_ids, prts->n_part);
  sort_reorder_by_cell_device(prts, d_cnis, d_ids);
  
  check(hipFree(d_cnis));
  check(hipFree(d_ids));
}

