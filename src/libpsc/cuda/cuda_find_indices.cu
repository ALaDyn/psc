#include "hip/hip_runtime.h"

#include <psc_cuda.h>

// FIXME, do this always?
#define NO_CHECKERBOARD

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 4
#define BLOCKSIZE_Z 4

#define PFX(x) sort_##x
#include "constants.c"

// FIXME, use const mem for some params

__global__ static void find_cell_indices_by_cell(int n_part, particles_cuda_dev_t d_part,
						 int *d_cnis, int *d_ids, int ldims_y)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      d_part.xi4[i].x * d_consts.dxi[0],
      d_part.xi4[i].y * d_consts.dxi[1],
      d_part.xi4[i].z * d_consts.dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = cuda_fint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (ldims_y / 8) + (pos[1] / 8)) << 6);
    idx |=
      ((pos[2] & 4) << 3) |
      ((pos[1] & 4) << 2);
    idx |=
      ((pos[2] & 2) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 1) << 0);
    d_cnis[i] = idx;
    d_ids[i] = i;
  }
}

static void
sort_find_cell_indices_by_cell_device(struct psc_particles *prts, struct psc_patch *patch,
				      int *d_cnis, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_cell_indices_by_cell, (prts->n_part, cuda->d_part, d_cnis, d_ids,
					 patch->ldims[1]));
}

// FIXME, specific to 1x8x8, should be in ! .cu, so that cell_map works

static void __unused
sort_find_cell_indices_host(struct psc_particles *prts, struct psc_patch *patch,
			    int *d_cnis, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int n_part = prts->n_part;
  particles_cuda_dev_t *d_part = &cuda->d_part;
  int *h_cnis = (int *) malloc(n_part * sizeof(*h_cnis));
  int *h_ids = (int *) malloc(n_part * sizeof(*h_ids));

  float4 *h_xi4 = (float4 *) malloc(n_part * sizeof(*h_xi4));
  check(hipMemcpy(h_xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));

  for (int i = 0; i < n_part; i++) {
    particle_cuda_real_t dxi = 1.f / ppsc->dx[0];
    particle_cuda_real_t dyi = 1.f / ppsc->dx[1];
    particle_cuda_real_t dzi = 1.f / ppsc->dx[2];
    particle_cuda_real_t xi[3] = {
      (h_xi4[i].x - patch->xb[0]) * dxi,
      (h_xi4[i].y - patch->xb[1]) * dyi,
      (h_xi4[i].z - patch->xb[2]) * dzi };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = particle_cuda_real_fint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (patch->ldims[1] / 8) + (pos[1] / 8)) << 6) |
      ((pos[2] & 4) << 3) |
      ((pos[2] & 2) << 2) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 4) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[1] & 1) << 0);
    h_cnis[i] = idx;
    h_ids[i] = i;
  }

  check(hipMemcpy(d_cnis, h_cnis, n_part * sizeof(*h_cnis),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_ids, h_ids, n_part * sizeof(*h_ids),
		   hipMemcpyHostToDevice));

  free(h_xi4);
  free(h_cnis);
  free(h_ids);
}

static void __unused
sort_reorder_host(struct psc_particles *prts, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  int n_part = prts->n_part;
  particles_cuda_dev_t *d_part = &cuda->d_part;
  int *h_ids = (int *) malloc(n_part);

  float4 *h_xi4 = (float4 *) malloc(n_part * sizeof(*h_xi4));
  float4 *h_pxi4 = (float4 *) malloc(n_part * sizeof(*h_pxi4));
  check(hipMemcpy(h_xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_pxi4, d_part->pxi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_ids, d_ids, n_part * sizeof(*h_ids),
		   hipMemcpyDeviceToHost));

  // move into new position
  float4 *xi4 = (float4 *) malloc(n_part * sizeof(*xi4));
  float4 *pxi4 = (float4 *) malloc(n_part * sizeof(*pxi4));
  for (int i = 0; i < n_part; i++) {
    xi4[i] = h_xi4[h_ids[i]];
    pxi4[i] = h_pxi4[h_ids[i]];
  }
  
  check(hipMemcpy(d_part->xi4, xi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_part->pxi4, pxi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  free(xi4);
  free(pxi4);
  free(h_xi4);
  free(h_pxi4);
  free(h_ids);
}

__global__ static void
sort_reorder_by_cell(int n_part, particles_cuda_dev_t d_part, float4 *xi4, float4 *pxi4,
		     int *d_cnis, int *d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  int blocksize = BLOCKSIZE_X * BLOCKSIZE_Y * BLOCKSIZE_Z;

  if (i > n_part)
    return;

  int block, prev_block;
  if (i < n_part) {
    xi4[i] = d_part.xi4[d_ids[i]];
    pxi4[i] = d_part.pxi4[d_ids[i]];
    
    block = d_cnis[i];
  } else if (i == n_part) { // needed if there is no particle in the last block
    block = d_consts.b_mx[0] * d_consts.b_mx[1] * d_consts.b_mx[2] * blocksize;
  }

  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_cnis[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_part.c_offsets[b] = i;
  }
}

static void
sort_reorder_by_cell_device(struct psc_particles *prts, int *d_cnis, int *d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  float4 *xi4, *pxi4;
  check(hipMalloc((void **) &xi4, cuda->n_alloced * sizeof(*xi4)));
  check(hipMalloc((void **) &pxi4, cuda->n_alloced * sizeof(*pxi4)));

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (prts->n_part + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     sort_reorder_by_cell, (prts->n_part, cuda->d_part, xi4, pxi4, d_cnis, d_ids));

  check(hipFree(cuda->d_part.xi4));
  check(hipFree(cuda->d_part.pxi4));
  cuda->d_part.xi4 = xi4;
  cuda->d_part.pxi4 = pxi4;
}

EXTERN_C void
sort_patch_prep(int p, struct psc_particles *prts, int **d_cnis, int **d_ids)
{
  struct psc_particles_cuda *cuda = psc_particles_cuda(prts);
  check(hipMalloc((void **) d_cnis, cuda->n_alloced * sizeof(*d_cnis)));
  check(hipMalloc((void **) d_ids, cuda->n_alloced * sizeof(*d_ids)));

  fields_cuda_t pf_dummy;
  sort_set_constants(prts, &pf_dummy);
}

EXTERN_C void
sort_patch_done(int p, struct psc_particles *prts, int *d_cnis, int *d_ids)
{
  check(hipFree(d_cnis));
  check(hipFree(d_ids));
}

EXTERN_C void
sort_patch_by_cell(int p, struct psc_particles *prts)
{
  struct psc_patch *patch = &ppsc->patch[p];

  int *d_cnis, *d_ids;
  check(hipMalloc((void **) &d_cnis, prts->n_part * sizeof(*d_cnis)));
  check(hipMalloc((void **) &d_ids, prts->n_part * sizeof(*d_ids)));

  fields_cuda_t pf_dummy;
  sort_set_constants(prts, &pf_dummy);

  sort_find_cell_indices_by_cell_device(prts, patch, d_cnis, d_ids);
  sort_pairs_device((unsigned int *) d_cnis, (unsigned int *) d_ids, prts->n_part);
  sort_reorder_by_cell_device(prts, d_cnis, d_ids);
  
  check(hipFree(d_cnis));
  check(hipFree(d_ids));
}

