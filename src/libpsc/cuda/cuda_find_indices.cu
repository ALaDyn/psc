#include "hip/hip_runtime.h"

#include <psc_cuda.h>

EXTERN_C void sort_pairs_device(unsigned int *d_keys, unsigned int *d_vals, int n);
EXTERN_C void sort_pairs_host(unsigned int *d_keys, unsigned int *d_vals, int n);

// FIXME, use const mem for some params

__global__ static void find_cell_indices(int n_part, particles_cuda_dev_t d_part,
					 unsigned int *d_cnis, unsigned int *d_ids,
					 real xb, real yb, real zb, int ldims_y,
					 real dxi, real dyi, real dzi)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      (d_part.xi4[i].x - xb) * dxi,
      (d_part.xi4[i].y - yb) * dyi,
      (d_part.xi4[i].z - zb) * dzi };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = cuda_nint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (ldims_y / 8) + (pos[1] / 8)) << 6) |
      ((pos[2] & 4) << 3) |
      ((pos[2] & 2) << 2) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 4) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[1] & 1) << 0);
    d_cnis[i] = idx;
    d_ids[i] = i;
  }
}

static void
sort_find_cell_indices_device(particles_cuda_t *pp, struct psc_patch *patch,
			      unsigned int *d_cnis, unsigned int *d_ids)
{
  particle_cuda_real_t dxi = 1.f / ppsc->dx[0];
  particle_cuda_real_t dyi = 1.f / ppsc->dx[1];
  particle_cuda_real_t dzi = 1.f / ppsc->dx[2];

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (pp->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_cell_indices, (pp->n_part, pp->d_part, d_cnis, d_ids,
				 patch->xb[0], patch->xb[1], patch->xb[2], patch->ldims[1],
				 dxi, dyi, dzi));
}

// FIXME, specific to 1x8x8, should be in ! .cu, so that cell_map works

static void
sort_find_cell_indices_host(particles_cuda_t *pp, struct psc_patch *patch,
			    unsigned int *d_cnis, unsigned int *d_ids)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *h_part = &pp->h_part;
  particles_cuda_dev_t *d_part = &pp->d_part;
  unsigned int *h_cnis = (unsigned int *) malloc(n_part * sizeof(*h_cnis));
  unsigned int *h_ids = (unsigned int *) malloc(n_part * sizeof(*h_ids));

  check(hipMemcpy(h_part->xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));

  for (int i = 0; i < n_part; i++) {
    particle_cuda_real_t dxi = 1.f / ppsc->dx[0];
    particle_cuda_real_t dyi = 1.f / ppsc->dx[1];
    particle_cuda_real_t dzi = 1.f / ppsc->dx[2];
    particle_cuda_real_t xi[3] = {
      (h_part->xi4[i].x - patch->xb[0]) * dxi,
      (h_part->xi4[i].y - patch->xb[1]) * dyi,
      (h_part->xi4[i].z - patch->xb[2]) * dzi };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = particle_base_real_nint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (patch->ldims[1] / 8) + (pos[1] / 8)) << 6) |
      ((pos[2] & 4) << 3) |
      ((pos[2] & 2) << 2) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 4) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[1] & 1) << 0);
    h_cnis[i] = idx;
    h_ids[i] = i;
  }

  check(hipMemcpy(d_cnis, h_cnis, n_part * sizeof(*h_cnis),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_ids, h_ids, n_part * sizeof(*h_ids),
		   hipMemcpyHostToDevice));

  free(h_cnis);
  free(h_ids);
}

static void
sort_reorder_host(particles_cuda_t *pp, unsigned int *d_ids)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *h_part = &pp->h_part;
  particles_cuda_dev_t *d_part = &pp->d_part;
  unsigned int *h_ids = (unsigned int *) malloc(n_part);

  check(hipMemcpy(h_part->xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_part->pxi4, d_part->pxi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_ids, d_ids, n_part * sizeof(*h_ids),
		   hipMemcpyDeviceToHost));

  // move into new position
  float4 *xi4 = (float4 *) malloc(n_part * sizeof(*xi4));
  float4 *pxi4 = (float4 *) malloc(n_part * sizeof(*pxi4));
  for (int i = 0; i < n_part; i++) {
    xi4[i] = pp->h_part.xi4[h_ids[i]];
    pxi4[i] = pp->h_part.pxi4[h_ids[i]];
  }
  // back to in-place
  memcpy(pp->h_part.xi4, xi4, n_part * sizeof(*xi4));
  memcpy(pp->h_part.pxi4, pxi4, n_part * sizeof(*pxi4));
  
  free(xi4);
  free(pxi4);

  check(hipMemcpy(d_part->xi4, h_part->xi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_part->pxi4, h_part->pxi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  free(h_ids);
}

__global__ static void
sort_reorder(int n_part, particles_cuda_dev_t d_part, float4 *xi4, float4 *pxi4,
	     unsigned int *d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    xi4[i] = d_part.xi4[d_ids[i]];
    pxi4[i] = d_part.pxi4[d_ids[i]];
  }
}

static void
sort_reorder_device(particles_cuda_t *pp, unsigned int *d_ids)
{
  float4 *xi4, *pxi4;
  check(hipMalloc((void **) &xi4, pp->n_part * sizeof(*xi4)));
  check(hipMalloc((void **) &pxi4, pp->n_part * sizeof(*pxi4)));

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (pp->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     sort_reorder, (pp->n_part, pp->d_part, xi4, pxi4, d_ids));

  check(hipFree(pp->d_part.xi4));
  check(hipFree(pp->d_part.pxi4));
  pp->d_part.xi4 = xi4;
  pp->d_part.pxi4 = pxi4;
}

EXTERN_C void
sort_patch(int p, particles_cuda_t *pp)
{
  struct psc_patch *patch = &ppsc->patch[p];

  unsigned int *d_cnis, *d_ids;
  check(hipMalloc((void **) &d_cnis, pp->n_part * sizeof(*d_cnis)));
  check(hipMalloc((void **) &d_ids, pp->n_part * sizeof(*d_ids)));

  sort_find_cell_indices_device(pp, patch, d_cnis, d_ids);
  sort_pairs_device(d_cnis, d_ids, pp->n_part);
  sort_reorder_device(pp, d_ids);

  check(hipFree(d_cnis));
  check(hipFree(d_ids));
}

