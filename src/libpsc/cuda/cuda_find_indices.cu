#include "hip/hip_runtime.h"

#include <psc_cuda.h>

EXTERN_C void sort_pairs_host(unsigned int *keys, unsigned int *vals, int n);

// FIXME, specific to 1x8x8, should be in ! .cu, so that cell_map works

static void
create_indices_host(unsigned int *cnis,
		    particles_cuda_t *pp, struct psc_patch *patch)
{
}

static void
sort_find_cell_indices_host(particles_cuda_t *pp, struct psc_patch *patch,
			    unsigned int *cnis, unsigned int *ids)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *h_part = &pp->h_part;
  particles_cuda_dev_t *d_part = &pp->d_part;

  check(hipMemcpy(h_part->xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));

  for (int i = 0; i < n_part; i++) {
    particle_cuda_real_t dxi = 1.f / ppsc->dx[0];
    particle_cuda_real_t dyi = 1.f / ppsc->dx[1];
    particle_cuda_real_t dzi = 1.f / ppsc->dx[2];
    particle_cuda_real_t xi[3] = {
      (h_part->xi4[i].x - patch->xb[0]) * dxi,
      (h_part->xi4[i].y - patch->xb[1]) * dyi,
      (h_part->xi4[i].z - patch->xb[2]) * dzi };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = particle_base_real_nint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (patch->ldims[1] / 8) + (pos[1] / 8)) << 6) |
      ((pos[2] & 4) << 3) |
      ((pos[2] & 2) << 2) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 4) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[1] & 1) << 0);
    cnis[i] = idx;
    ids[i] = i;
  }
}

static void
sort_reorder_host(particles_cuda_t *pp, unsigned int *ids)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *h_part = &pp->h_part;
  particles_cuda_dev_t *d_part = &pp->d_part;

  check(hipMemcpy(h_part->xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_part->pxi4, d_part->pxi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));

  // move into new position
  float4 *xi4 = (float4 *) malloc(n_part * sizeof(*xi4));
  float4 *pxi4 = (float4 *) malloc(n_part * sizeof(*pxi4));
  for (int i = 0; i < n_part; i++) {
    xi4[i] = pp->h_part.xi4[ids[i]];
    pxi4[i] = pp->h_part.pxi4[ids[i]];
  }
  // back to in-place
  memcpy(pp->h_part.xi4, xi4, n_part * sizeof(*xi4));
  memcpy(pp->h_part.pxi4, pxi4, n_part * sizeof(*pxi4));
  
  free(xi4);
  free(pxi4);

  check(hipMemcpy(d_part->xi4, h_part->xi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_part->pxi4, h_part->pxi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
}

EXTERN_C void
sort_patch(int p, particles_cuda_t *pp)
{
  struct psc_patch *patch = &ppsc->patch[p];

  unsigned int *cnis = (unsigned int *) malloc(pp->n_part * sizeof(*cnis));
  unsigned int *ids = (unsigned int *) malloc(pp->n_part * sizeof(*ids));

  sort_find_cell_indices_host(pp, patch, cnis, ids);
  sort_pairs_host(cnis, ids, pp->n_part);
  sort_reorder_host(pp, ids);

  free(cnis);
  free(ids);
}

