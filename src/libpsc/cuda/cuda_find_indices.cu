#include "hip/hip_runtime.h"

#include <psc_cuda.h>

#define PFX(x) sort_##x
#include "constants.c"

EXTERN_C void sort_pairs_device(unsigned int *d_keys, unsigned int *d_vals, int n);
EXTERN_C void sort_pairs_host(unsigned int *d_keys, unsigned int *d_vals, int n);

// FIXME, use const mem for some params

__global__ static void find_cell_indices(int n_part, particles_cuda_dev_t d_part,
					 unsigned int *d_cnis, unsigned int *d_ids,
					 int ldims_y)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    particle_cuda_real_t xi[3] = {
      d_part.xi4[i].x * d_dxi[0],
      d_part.xi4[i].y * d_dxi[1],
      d_part.xi4[i].z * d_dxi[2] };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = cuda_fint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (ldims_y / 8) + (pos[1] / 8)) << 6);
    idx |=
      ((pos[2] & 4) << 3) |
      ((pos[1] & 4) << 2);
#if 0
    idx |=
      ((pos[2] & 2) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 1) << 0);
#endif
    d_cnis[i] = idx;
    d_ids[i] = i;
  }
}

static void
sort_find_cell_indices_device(particles_cuda_t *pp, struct psc_patch *patch,
			      unsigned int *d_cnis, unsigned int *d_ids)
{
  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (pp->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     find_cell_indices, (pp->n_part, pp->d_part, d_cnis, d_ids,
				 patch->ldims[1]));
}

// FIXME, specific to 1x8x8, should be in ! .cu, so that cell_map works

static void __unused
sort_find_cell_indices_host(particles_cuda_t *pp, struct psc_patch *patch,
			    unsigned int *d_cnis, unsigned int *d_ids)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *d_part = &pp->d_part;
  unsigned int *h_cnis = (unsigned int *) malloc(n_part * sizeof(*h_cnis));
  unsigned int *h_ids = (unsigned int *) malloc(n_part * sizeof(*h_ids));

  float4 *h_xi4 = (float4 *) malloc(n_part * sizeof(*h_xi4));
  check(hipMemcpy(h_xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));

  for (int i = 0; i < n_part; i++) {
    particle_cuda_real_t dxi = 1.f / ppsc->dx[0];
    particle_cuda_real_t dyi = 1.f / ppsc->dx[1];
    particle_cuda_real_t dzi = 1.f / ppsc->dx[2];
    particle_cuda_real_t xi[3] = {
      (h_xi4[i].x - patch->xb[0]) * dxi,
      (h_xi4[i].y - patch->xb[1]) * dyi,
      (h_xi4[i].z - patch->xb[2]) * dzi };
    int pos[3];
    for (int d = 0; d < 3; d++) {
      pos[d] = particle_cuda_real_fint(xi[d]);
    }
    
    int idx = (((pos[2] / 8) * (patch->ldims[1] / 8) + (pos[1] / 8)) << 6) |
      ((pos[2] & 4) << 3) |
      ((pos[2] & 2) << 2) |
      ((pos[2] & 1) << 1) |
      ((pos[1] & 4) << 2) |
      ((pos[1] & 2) << 1) |
      ((pos[1] & 1) << 0);
    h_cnis[i] = idx;
    h_ids[i] = i;
  }

  check(hipMemcpy(d_cnis, h_cnis, n_part * sizeof(*h_cnis),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_ids, h_ids, n_part * sizeof(*h_ids),
		   hipMemcpyHostToDevice));

  free(h_xi4);
  free(h_cnis);
  free(h_ids);
}

static void __unused
sort_reorder_host(particles_cuda_t *pp, unsigned int *d_ids)
{
  int n_part = pp->n_part;
  particles_cuda_dev_t *d_part = &pp->d_part;
  unsigned int *h_ids = (unsigned int *) malloc(n_part);

  float4 *h_xi4 = (float4 *) malloc(n_part * sizeof(*h_xi4));
  float4 *h_pxi4 = (float4 *) malloc(n_part * sizeof(*h_pxi4));
  check(hipMemcpy(h_xi4, d_part->xi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_pxi4, d_part->pxi4, n_part * sizeof(float4),
		   hipMemcpyDeviceToHost));
  check(hipMemcpy(h_ids, d_ids, n_part * sizeof(*h_ids),
		   hipMemcpyDeviceToHost));

  // move into new position
  float4 *xi4 = (float4 *) malloc(n_part * sizeof(*xi4));
  float4 *pxi4 = (float4 *) malloc(n_part * sizeof(*pxi4));
  for (int i = 0; i < n_part; i++) {
    xi4[i] = h_xi4[h_ids[i]];
    pxi4[i] = h_pxi4[h_ids[i]];
  }
  
  check(hipMemcpy(d_part->xi4, xi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  check(hipMemcpy(d_part->pxi4, pxi4, n_part * sizeof(float4),
		   hipMemcpyHostToDevice));
  free(xi4);
  free(pxi4);
  free(h_xi4);
  free(h_pxi4);
  free(h_ids);
}

__global__ static void
sort_reorder(int n_part, particles_cuda_dev_t d_part, float4 *xi4, float4 *pxi4,
	     unsigned int *d_cnis, unsigned int *d_ids)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  if (i < n_part) {
    xi4[i] = d_part.xi4[d_ids[i]];
    pxi4[i] = d_part.pxi4[d_ids[i]];
    
    // create offsets per block into particle array
    int blocksize = BLOCKSIZE_X * BLOCKSIZE_Y * BLOCKSIZE_Z;
    int block = d_cnis[i] / blocksize;
    int prev_block = -1;
    if (i > 0) {
      prev_block = d_cnis[i-1] / blocksize;
    }
    for (int b = prev_block + 1; b <= block; b++) {
      d_part.offsets[b] = i;
    }
  }
}

static void
sort_reorder_device(particles_cuda_t *pp, unsigned int *d_cnis, unsigned int *d_ids)
{
  float4 *xi4, *pxi4;
  check(hipMalloc((void **) &xi4, pp->n_part * sizeof(*xi4)));
  check(hipMalloc((void **) &pxi4, pp->n_part * sizeof(*pxi4)));

  int dimBlock[2] = { THREADS_PER_BLOCK, 1 };
  int dimGrid[2]  = { (pp->n_part + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1 };
  RUN_KERNEL(dimGrid, dimBlock,
	     sort_reorder, (pp->n_part, pp->d_part, xi4, pxi4, d_cnis, d_ids));

  check(hipFree(pp->d_part.xi4));
  check(hipFree(pp->d_part.pxi4));
  pp->d_part.xi4 = xi4;
  pp->d_part.pxi4 = pxi4;
}

EXTERN_C void
sort_patch(int p, particles_cuda_t *pp)
{
  struct psc_patch *patch = &ppsc->patch[p];

  unsigned int *d_cnis, *d_ids;
  check(hipMalloc((void **) &d_cnis, pp->n_part * sizeof(*d_cnis)));
  check(hipMalloc((void **) &d_ids, pp->n_part * sizeof(*d_ids)));

  fields_cuda_t pf_dummy;
  sort_set_constants(pp, &pf_dummy);

  sort_find_cell_indices_device(pp, patch, d_cnis, d_ids);
  sort_pairs_device(d_cnis, d_ids, pp->n_part);
  sort_reorder_device(pp, d_cnis, d_ids);
  
#if 0
  unsigned int *h_cnis = (unsigned int *) calloc(pp->n_part, sizeof(*h_cnis));
  check(hipMemcpy(h_cnis, d_cnis, pp->n_part * sizeof(*h_cnis),
		   hipMemcpyDeviceToHost));
  int *offsets = (int *) calloc(pp->nr_blocks + 1, sizeof(*offsets));
  check(hipMemcpy(offsets, pp->d_part.offsets, (pp->nr_blocks + 1) * sizeof(*offsets),
		   hipMemcpyDeviceToHost));

  for (int b = 0; b < pp->nr_blocks; b++) {
    if (pp->h_part.offsets) {
      printf("block %d: %d:%d\n", b, pp->h_part.offsets[b], pp->h_part.offsets[b+1]);
    }
    printf("block %d: %d:%d\n", b, offsets[b], offsets[b+1]);
  }
  int last_block = -1;
  for (int i = 0; i < pp->n_part; i++) {
    if (h_cnis[i] / 16 != last_block) {
      last_block = h_cnis[i] / 16;
      printf("i %d bid %d\n", i, last_block);
    }
  }
  free(offsets);
  free(h_cnis);
#endif

  check(hipFree(d_cnis));
  check(hipFree(d_ids));
}

