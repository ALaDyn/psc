
#include "gtest/gtest.h"

#include "../libpsc/cuda/cuda_mparticles.cuh"
#include "../libpsc/cuda/cuda_mparticles_sort.cuh"

#include "../libpsc/cuda/collision_cuda_impl.hxx"
#include "../libpsc/psc_collision/psc_collision_impl.hxx"
#include "psc_fields_single.h"
#include "psc_particles_single.h"
#include "testing.hxx"


using dim = dim_yz;
using Mparticles = MparticlesCuda<BS144>;

// ======================================================================
// make_psc
//
// FIXME, duplicated in various testing environments

template <typename dim>
static Grid_t& make_psc(const Grid_t::Kinds& kinds)
{
  Int3 gdims = {16, 16, 16};
  Int3 ibn = {2, 2, 2};
  Vec3<double> length = {160., 160., 160.};
  if (dim::InvarX::value) {
    gdims[0] = 1;
    ibn[0] = 0;
  }
  if (dim::InvarY::value) {
    gdims[1] = 1;
    ibn[1] = 0;
  }
  if (dim::InvarZ::value) {
    gdims[2] = 1;
    ibn[2] = 0;
  }

  auto grid_domain = Grid_t::Domain{gdims, length};
  auto grid_bc =
    psc::grid::BC{{BND_FLD_PERIODIC, BND_FLD_PERIODIC, BND_FLD_PERIODIC},
                  {BND_FLD_PERIODIC, BND_FLD_PERIODIC, BND_FLD_PERIODIC},
                  {BND_PRT_PERIODIC, BND_PRT_PERIODIC, BND_PRT_PERIODIC},
                  {BND_PRT_PERIODIC, BND_PRT_PERIODIC, BND_PRT_PERIODIC}};

  auto norm_params = Grid_t::NormalizationParams::dimensionless();
  norm_params.nicell = 200;
  auto coeff = Grid_t::Normalization{norm_params};
  return *new Grid_t{grid_domain, grid_bc, kinds, coeff, 1.};
}

static Mparticles make_mparticles(const Grid_t& grid)
{
  Mparticles mprts{grid};
  auto inj = mprts.injector();
  auto injector = inj[0];
  injector({{5., 5., 5.}, {1., 0., 0.}, 1., 0});
  injector({{5., 5., 5.}, {0., 0., 0.}, 1., 0});
  injector({{5., 15., 15.}, {.6, 0., 0.}, 1., 0});
  injector({{5., 15., 15.}, {.7, 0., 0.}, 1., 0});
  injector({{5., 15., 15.}, {.8, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.1, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.2, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.3, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.4, 0., 0.}, 1., 0});
  return mprts;
}

TEST(cuda_mparticles_sort, sort)
{
  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto& cmprts = *mprts.cmprts();
  auto sort = cuda_mparticles_sort(cmprts.n_cells());

  sort.find_indices_ids(cmprts);
  EXPECT_EQ(sort.d_idx, (std::vector<int>{0, 0, 17, 17, 17, 1, 1, 1, 1}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8}));

  sort.stable_sort_cidx();
  EXPECT_EQ(sort.d_idx, (std::vector<int>{0, 0, 1, 1, 1, 1, 17, 17, 17}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 5, 6, 7, 8, 2, 3, 4}));

  sort.find_offsets();
  std::vector<int> off(cmprts.n_cells() + 1);
  off[0] = 0;
  off[1] = 2;
  for (int i = 2; i < 18; i++) {
    off[i] = 6;
  }
  for (int i = 18; i <= 256; i++) {
    off[i] = 9;
  }
  EXPECT_EQ(sort.d_off, off);
}

TEST(cuda_mparticles_randomize_sort, sort)
{
  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto& cmprts = *mprts.cmprts();
  auto sort = cuda_mparticles_randomize_sort(cmprts.n_cells());
  
  sort.find_indices_ids(cmprts);
  EXPECT_EQ(sort.d_idx, (std::vector<int>{0, 0, 17, 17, 17, 1, 1, 1, 1}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8}));

  sort.sort();
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 5, 8, 7, 6, 2, 3, 4}));

  float last = sort.d_random_idx[0];
  for (int i = 1; i < cmprts.size(); i++) {
    EXPECT_GE(sort.d_random_idx[i], last);
    last = sort.d_random_idx[i];
  }
  for (int i = 0; i < cmprts.size(); i++) {
    mprintf("i %d idx %d r_idx %g id %d\n", i, (int)sort.d_idx[i],
            (float)sort.d_random_idx[i], (int)sort.d_id[i]);
  }
  
  sort.find_offsets();
  std::vector<int> off(cmprts.n_cells() + 1);
  off[0] = 0;
  off[1] = 2;
  for (int i = 2; i < 18; i++) {
    off[i] = 6;
  }
  for (int i = 18; i <= 256; i++) {
    off[i] = 9;
  }
  EXPECT_EQ(sort.d_off, off);

#if 0
  // do over
  sort.find_indices_ids(cmprts);
  thrust::transform(thrust::make_counting_iterator(uint(0)),
                    thrust::make_counting_iterator(cmprts.size()),
		    sort.d_idx.begin(),
                    sort.d_random_idx.begin(), add_random);
  sort.sort();
  for (int i = 0; i < cmprts.size(); i++) {
    mprintf("i %d idx %d r_idx %g id %d\n", i, (int)sort.d_idx[i],
            (float)sort.d_random_idx[i], (int)sort.d_id[i]);
  }
#endif
}

TEST(CollisionTest, Test2)
{
  using Collision = CollisionCuda<MparticlesCuda<BS144>, RngStateFake>;
  const typename Mparticles::real_t eps = 1e-5;

  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto collision = Collision(grid, 1, 1.);

  auto& cmprts = *mprts.cmprts();
  auto sort_by_cell = cuda_mparticles_sort(cmprts.n_cells());
  sort_by_cell.find_indices_ids(cmprts);
  EXPECT_EQ(sort_by_cell.d_idx,
            (std::vector<int>{0, 0, 17, 17, 17, 1, 1, 1, 1}));
  EXPECT_EQ(sort_by_cell.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8}));
  sort_by_cell.stable_sort_cidx();
  EXPECT_EQ(sort_by_cell.d_idx,
            (std::vector<int>{0, 0, 1, 1, 1, 1, 17, 17, 17}));
  EXPECT_EQ(sort_by_cell.d_id, (std::vector<int>{0, 1, 5, 6, 7, 8, 2, 3, 4}));
  sort_by_cell.find_offsets();
  // for (int i = 0; i < cmprts.size(); i++) {
  //   mprintf("i %d idx %d id %d\n", i, (int)sort_by_cell.d_idx[i],
  //   (int)sort_by_cell.d_id[i]);

  collision(mprts);

  auto accessor = mprts.accessor();
  auto it = accessor[0].begin();
  auto prtf0 = *it++;
  auto prtf1 = *it++;
  EXPECT_NEAR(prtf0.u()[0] + prtf1.u()[0], 1., eps);
  EXPECT_NEAR(prtf0.u()[1] + prtf1.u()[1], 0., eps);
  EXPECT_NEAR(prtf0.u()[2] + prtf1.u()[2], 0., eps);

#if 0
  // depends on random numbers, but for RngFake, we know
  EXPECT_NEAR(prtf0.u()[0],  0.96226911, eps);
  EXPECT_NEAR(prtf0.u()[1],  0.        , eps);
  EXPECT_NEAR(prtf0.u()[2], -0.17342988, eps);
  EXPECT_NEAR(prtf1.u()[0],  0.03773088, eps);
  EXPECT_NEAR(prtf1.u()[1], -0.        , eps);
  EXPECT_NEAR(prtf1.u()[2],  0.17342988, eps);
#endif
}

// ======================================================================
// main

int main(int argc, char** argv)
{
  MPI_Init(&argc, &argv);
  ::testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();
  MPI_Finalize();
  return rc;
}
