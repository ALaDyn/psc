#include "hip/hip_runtime.h"

#include <gtest/gtest.h>

#include "fields3d.hxx"
#include "psc_fields_cuda.h"
#include "psc_fields_cuda.inl"
#include "setup_fields.hxx"

#include "../libpsc/cuda/setup_fields_cuda.hxx"

#include "kg/io.h"
#include "fields3d.inl"

#include "psc.h" // FIXME, just for EX etc

#include "cuda_mfields.h"
#include "cuda_mfields.inl"

static Grid_t make_grid()
{
  auto domain =
    Grid_t::Domain{{8, 4, 2}, {80., 40., 20.}, {-40., -20., 0.}, {2, 1, 1}};
  auto bc = psc::grid::BC{};
  auto kinds = Grid_t::Kinds{};
  auto norm = Grid_t::Normalization{};
  double dt = .1;
  return Grid_t{domain, bc, kinds, norm, dt};
}

template <typename T>
class CudaMfieldsTest : public ::testing::Test
{};

using CudaMfieldsTestTypes = ::testing::Types<MfieldsCuda>;

TYPED_TEST_SUITE(CudaMfieldsTest, CudaMfieldsTestTypes);

TYPED_TEST(CudaMfieldsTest, WriteRead)
{
  using Mfields = TypeParam;

  auto grid = make_grid();
  auto mflds = Mfields{grid, NR_FIELDS, {}};

  setupFields(mflds, [](int m, double crd[3]) {
    return m + crd[0] + 100 * crd[1] + 10000 * crd[2];
  });

  DMFields d_mflds = *mflds.cmflds();
  
  auto io = kg::io::IOAdios2{};

  {
    auto writer = io.open("test.bp", kg::io::Mode::Write);
    writer.put("d_mflds", d_mflds);
    writer.close();
  }

  auto box = d_mflds.box();
  auto n_comps = d_mflds.n_comps();
  auto n_patches = d_mflds.n_patches();
  thrust::device_vector<float> d_data(box.size() * n_comps * n_patches);
  DMFields d_mflds3;
  auto d_mflds2 = DMFields{box, n_comps, n_patches, d_data.data().get(), d_data.size()};

  {
    auto reader = io.open("test.bp", kg::io::Mode::Read);
    reader.get("d_mflds", d_mflds3);
    reader.close();
  }

#if 0
  for (int p = 0; p < mflds.n_patches(); ++p) {
    grid.Foreach_3d(0, 0, [&](int i, int j, int k) {
#if 0
	mprintf("[%d, %d, %d] = %06g %06g %06g\n", i, j, k,
		(double) mflds[p](EX, i, j, k), (double) mflds[p](EY, i, j, k), (double) mflds[p](EZ, i, j, k));
#endif
      for (int m = 0; m < NR_FIELDS; m++) {
        EXPECT_EQ(mflds[p](m, i, j, k), mflds2[p](m, i, j, k));
      }
    });
  }
#endif
}

TYPED_TEST(CudaMfieldsTest, WriteWithGhostsRead)
{
  using Mfields = TypeParam;

  auto grid = make_grid();
  auto mflds = Mfields{grid, NR_FIELDS, {2, 2, 2}};

  setupFields(mflds, [](int m, double crd[3]) {
    return m + crd[0] + 100 * crd[1] + 10000 * crd[2];
  });

  DMFields d_mflds = *mflds.cmflds();
  
  auto io = kg::io::IOAdios2{};

  {
    auto writer = io.open("test1.bp", kg::io::Mode::Write);
    writer.put("d_mflds", d_mflds);
    writer.close();
  }

#if 0
  auto mflds2 = Mfields{grid, NR_FIELDS, {}};
  {
    auto reader = io.open("test1.bp", kg::io::Mode::Read);
    reader.get("mflds", mflds2);
    reader.close();
  }

  for (int p = 0; p < mflds.n_patches(); ++p) {
    grid.Foreach_3d(0, 0, [&](int i, int j, int k) {
#if 0
	mprintf("[%d, %d, %d] = %06g %06g %06g\n", i, j, k,
		(double) mflds[p](EX, i, j, k), (double) mflds[p](EY, i, j, k), (double) mflds[p](EZ, i, j, k));
#endif
      for (int m = 0; m < NR_FIELDS; m++) {
        EXPECT_EQ(mflds[p](m, i, j, k), mflds2[p](m, i, j, k))
          << " i " << i << " j " << j << " k " << k << " m " << m;
      }
    });
  }
#endif
}

#if 0
TYPED_TEST(CudaMfieldsTest, WriteReadWithGhosts)
{
  using Mfields = TypeParam;

  auto grid = make_grid();
  auto mflds = Mfields{grid, NR_FIELDS, {}};

  setupFields(mflds, [](int m, double crd[3]) {
    return m + crd[0] + 100 * crd[1] + 10000 * crd[2];
  });

  auto io = kg::io::IOAdios2{};

  {
    auto writer = io.open("test.bp", kg::io::Mode::Write);
    writer.put("mflds", mflds);
    writer.close();
  }

  auto mflds2 = Mfields{grid, NR_FIELDS, {2, 2, 2}};
  {
    auto reader = io.open("test.bp", kg::io::Mode::Read);
    reader.get("mflds", mflds2);
    reader.close();
  }

  for (int p = 0; p < mflds.n_patches(); ++p) {
    grid.Foreach_3d(0, 0, [&](int i, int j, int k) {
#if 0
	mprintf("p%d [%d, %d, %d] = %06g %06g %06g\n", p, i, j, k,
		(double) mflds2[p](EX, i, j, k), (double) mflds2[p](EY, i, j, k), (double) mflds2[p](EZ, i, j, k));
#endif
#if 1
      for (int m = 0; m < NR_FIELDS; m++) {
        EXPECT_EQ(mflds[p](m, i, j, k), mflds2[p](m, i, j, k))
          << " i " << i << " j " << j << " k " << k << " m " << m;
      }
#endif
    });
  }
}
#endif

// ======================================================================
// main

int main(int argc, char** argv)
{
  MPI_Init(&argc, &argv);
  ::testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();
  MPI_Finalize();
  return rc;
}
