#include "hip/hip_runtime.h"

#include <type_traits>
#include "bs.hxx"

#include "cuda_mfields.h"
#include "cuda_mfields.inl"
#include "cuda_mparticles.cuh"
#include "cuda_mparticles.inl"

#include <kg/io.h>
#include <iostream>

int debug_patch_ = -1;

void debug_ab(DMparticlesCuda<BS144> d_mprts, DMFields d_mflds);
void debug_rho(DMparticlesCuda<BS144> d_mprts, DMFields d_mflds);

std::ostream& operator<<(std::ostream& of, const float4 &xi)
{
  of << "{" << xi.x << ", " << xi.y << ", " << xi.z << ", " << xi.w << "}";
  return of;
}

bool operator==(const float4& a, const float4& b)
{
  return a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w;
}

bool operator!=(const float4& a, const float4& b)
{
  return !(a == b);
}

class MF
{
  using T = float;

public:
  MF(const DMFields& dmf) : dmf_(dmf) {}

  T operator()(int i, int j, int k, int m, int p) const
  {
    auto data = thrust::device_pointer_cast(dmf_.storage_.data());
    i -= dmf_.box().ib()[0];
    j -= dmf_.box().ib()[1];
    k -= dmf_.box().ib()[2];
    int idx = (((p * dmf_.n_comps() + m) * dmf_.box().im()[2] + k) * dmf_.box().im()[1] + j) * dmf_.box().im()[0] + i;
    return data[idx];
  }

private:
  DMFields dmf_;
};

using DMparticles = DMparticlesCuda<BS144>;

int main(int argc, char **argv)
{
  MPI_Init(&argc, &argv);

  DMFields d_mflds_1, d_mflds_2;
  DMparticles d_mprts_1, d_mprts_2;

  std::string dir(".");
  std::string step("4");
  std::string proc("0");
  auto io = kg::io::IOAdios2{};
  auto reader = io.open(dir + "/bef-proc-" + proc + "-time-" + step + ".bp", kg::io::Mode::Read);
  reader.get("d_mflds", d_mflds_1);
  reader.get("d_mprts", d_mprts_1);
  reader.close();

  auto reader2 = io.open(dir + "/aft-proc-" + proc + "-time-" + step + ".bp", kg::io::Mode::Read);
  reader2.get("d_mflds", d_mflds_2);
  reader2.get("d_mprts", d_mprts_2);
  reader2.close();
  
  auto d_xi4_1 = thrust::device_pointer_cast<float4>(d_mprts_1.storage.xi4);
  auto d_xi4_2 = thrust::device_pointer_cast<float4>(d_mprts_2.storage.xi4);
  auto d_pxi4_1 = thrust::device_pointer_cast<float4>(d_mprts_1.storage.pxi4);
  auto d_pxi4_2 = thrust::device_pointer_cast<float4>(d_mprts_2.storage.pxi4);

  // for (int n = 0; n < N; n++) {
  //   std::cout << "0: " << d_xi4_1[n] << "\n";
  //   std::cout << "2: " << d_xi4_2[n] << "\n";
  // }
  // std::cout << "\n";

  for (int p = 0; p < d_mflds_1.n_patches_; p++) {
    uint size = d_mflds_1.box().size();
    hipError_t ierr = hipMemset(d_mflds_1.storage_.data() + (p * 9 + JXI) * size, 0,
    				3 * size * sizeof(fields_cuda_real_t));
    cudaCheck(ierr);
  }

  d_mprts_1.fnqzs_ = d_mprts_1.fnqys_; // fixing wrong data

  DMFields d_rho_p;
  d_rho_p.box_ = d_mflds_1.box();
  d_rho_p.n_patches_ = d_mflds_1.n_patches();
  d_rho_p.n_fields_ = 1;
  thrust::device_vector<float> d_rho_p_data(d_rho_p.box().size() * d_rho_p.n_comps() * d_rho_p.n_patches());
  d_rho_p.storage_.data_ = d_rho_p_data.data().get();
  d_rho_p.storage_.size_ = d_rho_p_data.size();

  DMFields d_rho_m;
  d_rho_m.box_ = d_mflds_1.box();
  d_rho_m.n_patches_ = d_mflds_1.n_patches();
  d_rho_m.n_fields_ = 1;
  thrust::device_vector<float> d_rho_m_data(d_rho_m.box().size() * d_rho_m.n_comps() * d_rho_m.n_patches());
  d_rho_m.storage_.data_ = d_rho_m_data.data().get();
  d_rho_m.storage_.size_ = d_rho_m_data.size();

  
  debug_rho(d_mprts_1, d_rho_m);
  debug_ab(d_mprts_1, d_mflds_1);
  debug_rho(d_mprts_1, d_rho_p);
  
  auto writer = io.open(dir + "/after.bp", kg::io::Mode::Write, MPI_COMM_SELF);
  writer.put("d_mflds", d_mflds_1);
  writer.put("d_mprts", d_mprts_1);
  writer.put("d_rho_m", d_rho_m);
  writer.put("d_rho_p", d_rho_p);
  writer.close();

  int n_prts = d_mprts_1.size_;
  for (int n = 0; n < n_prts; n++) {
    if (d_xi4_1[n] != d_xi4_2[n]) {
      std::cout << "1 : " << d_xi4_1[n] << "\n";
      std::cout << "2 : " << d_xi4_2[n] << "\n";
    }
    if (d_pxi4_1[n] != d_pxi4_2[n]) {
      std::cout << "1p: " << d_pxi4_1[n] << "\n";
      std::cout << "2p: " << d_pxi4_2[n] << "\n";
    }
  }

#if 0
  MF mf1(d_mflds_1), mf2(d_mflds_2);
  auto& ib = d_mflds_1.box().ib();
  auto& im = d_mflds_1.box().im();
  for (int p = 0; p < d_mflds_1.n_patches(); p++) {
    for (int m = JXI; m <= JZI; m++) {
      for (int k = ib[2]; k < ib[2] + im[2]; k++) {
	for (int j = ib[1]; j < ib[1] + im[1]; j++) {
	  for (int i = ib[0]; i < ib[0] + im[0]; i++) {
	    if (mf1(i, j, k, m, p) != mf2(i, j, k, m, p)) {
	      std:: cout << "!!! m" << m << " " << mf1(i, j, k, m, p) << " -- " << mf2(i, j, k, m, p) << "\n";
	    }
	  }
	}
      }
    }
  }
#endif
  
  MPI_Finalize();
  return 0;
}