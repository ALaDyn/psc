#include "hip/hip_runtime.h"

#include <type_traits>
#include "bs.hxx"

#include "cuda_mfields.h"
#include "cuda_mfields.inl"
#include "cuda_mparticles.cuh"
#include "cuda_mparticles.inl"

#include <kg/io.h>
#include <iostream>

void debug_ab(DMparticlesCuda<BS144> d_mprts, DMFields d_mflds);

std::ostream& operator<<(std::ostream& of, const float4 &xi)
{
  of << "{" << xi.x << ", " << xi.y << ", " << xi.z << ", " << xi.w << "}";
  return of;
}

using DMparticles = DMparticlesCuda<BS144>;

int main(int argc, char **argv)
{
  MPI_Init(&argc, &argv);

  DMFields d_mflds_1, d_mflds_2;
  DMparticles d_mprts_1, d_mprts_2;

  auto io = kg::io::IOAdios2{};
  auto reader = io.open("bef-proc-27-time-1207.bp", kg::io::Mode::Read);
  reader.get("d_mflds", d_mflds_1);
  reader.get("d_mprts", d_mprts_1);
  reader.close();

  auto reader2 = io.open("aft-proc-27-time-1207.bp", kg::io::Mode::Read);
  reader2.get("d_mflds", d_mflds_2);
  reader2.get("d_mprts", d_mprts_2);
  reader2.close();
  
  auto d_xi4_1 = thrust::device_pointer_cast<float4>(d_mprts_1.storage.xi4);
  auto d_xi4_2 = thrust::device_pointer_cast<float4>(d_mprts_2.storage.xi4);

  for (int n = 0; n < 5; n++) {
    std::cout << "0: " << d_xi4_1[n] << "\n";
    std::cout << "2: " << d_xi4_2[n] << "\n";
  }
  std::cout << "\n";

  for (int p = 0; p < d_mflds_1.n_patches_; p++) {
    uint size = d_mflds_1.box().size();
    hipError_t ierr = hipMemset(d_mflds_1.storage_.data() + (p * 9 + JXI) * size, 0,
    				3 * size * sizeof(fields_cuda_real_t));
    cudaCheck(ierr);
  }
  
  debug_ab(d_mprts_1, d_mflds_1);

  for (int n = 0; n < 5; n++) {
    std::cout << "1: " << d_xi4_1[n] << "\n";
    std::cout << "2: " << d_xi4_2[n] << "\n";
  }
  
  MPI_Finalize();
  return 0;
}