#include "hip/hip_runtime.h"

#include <type_traits>
#include "bs.hxx"

#include "cuda_mfields.h"
#include "cuda_mfields.inl"
#include "cuda_mparticles.cuh"
#include "cuda_mparticles.inl"

#include <kg/io.h>
#include <iostream>

void debug_ab(DMparticlesCuda<BS144>& d_mprts, DMFields& d_mflds);

std::ostream& operator<<(std::ostream& of, const float4 &xi)
{
  of << "{" << xi.x << ", " << xi.y << ", " << xi.z << ", " << xi.w << "}";
  return of;
}

using DMparticles = DMparticlesCuda<BS144>;

int main(int argc, char **argv)
{
  MPI_Init(&argc, &argv);

  DMFields d_mflds_1, d_mflds_2;
  DMparticles d_mprts_1, d_mprts_2;

  auto io = kg::io::IOAdios2{};
  auto reader = io.open("before-proc-68-time-1206.bp", kg::io::Mode::Read);
  reader.get("d_mflds", d_mflds_1);
  reader.get("d_mprts", d_mprts_1);
  reader.close();

  auto reader2 = io.open("after-proc-68-time-1206.bp", kg::io::Mode::Read);
  reader2.get("d_mflds", d_mflds_2);
  reader2.get("d_mprts", d_mprts_2);
  reader2.close();
  
  auto d_xi4_1 = thrust::device_pointer_cast<float4>(d_mprts_1.storage.xi4);
  auto d_xi4_2 = thrust::device_pointer_cast<float4>(d_mprts_2.storage.xi4);
  
  debug_ab(d_mprts_1, d_mflds_1);

  for (int n = 0; n < 5; n++) {
    std::cout << "1: " << d_xi4_1[n] << "\n";
    std::cout << "2: " << d_xi4_2[n] << "\n";
  }
  
  MPI_Finalize();
  return 0;
}