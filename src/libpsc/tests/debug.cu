#include "hip/hip_runtime.h"

#include <type_traits>
#include "bs.hxx"

#include "cuda_mfields.h"
#include "cuda_mfields.inl"
#include "cuda_mparticles.cuh"
#include "cuda_mparticles.inl"

#include <kg/io.h>

void debug_ab(DMparticlesCuda<BS144>& d_mprts, DMFields& d_mflds);

using DMparticles = DMparticlesCuda<BS144>;

int main(int argc, char **argv)
{
  MPI_Init(&argc, &argv);

  DMFields d_mflds;
  DMparticles d_mprts;

  auto io = kg::io::IOAdios2{};
  auto reader = io.open("before-proc-51-time-1207.bp", kg::io::Mode::Read);
  reader.get("d_mflds", d_mflds);
  reader.get("d_mprts", d_mprts);
  reader.close();

  auto d_xi4 = thrust::device_pointer_cast<float4>(d_mprts.storage.xi4);
  float4 xi4 = d_xi4[4504];
  printf("4504 %g %g %g\n", xi4.x, xi4.y, xi4.z);
  
  debug_ab(d_mprts, d_mflds);
  
  MPI_Finalize();
  return 0;
}