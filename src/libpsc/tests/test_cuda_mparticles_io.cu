
#include <gtest/gtest.h>

#include "test_common.hxx"

#include "psc_particles_double.h"
#include "psc_particles_single.h"
#include "particle_with_id.h"
#include "setup_particles.hxx"
#include "../libpsc/cuda/mparticles_cuda.hxx"
#include "../libpsc/cuda/mparticles_cuda.inl"
#include "../libpsc/cuda/cuda_mparticles.cuh"
#include "../libpsc/cuda/cuda_mparticles.inl"
#include "particles_simple.inl"

template <typename _Mparticles, typename _MakeGrid = MakeTestGrid1>
struct Config
{
  using Mparticles = _Mparticles;
  using MakeGrid = _MakeGrid;
};

using MparticlesTestTypes = ::testing::Types<
  Config<MparticlesCuda<BS144>, MakeTestGridYZ>
  >;

TYPED_TEST_SUITE(MparticlesTest, MparticlesTestTypes);

// ======================================================================
// MparticlesTest

template <typename T>
struct MparticlesTest : ::testing::Test
{
  using Mparticles = typename T::Mparticles;
  using Particle = typename Mparticles::Particle;
  using MakeGrid = typename T::MakeGrid;

  MparticlesTest() : grid_{MakeGrid{}()}
  {
    grid_.kinds.emplace_back(Grid_t::Kind(1., 1., "test_species"));
  }

  Mparticles mk_mprts()
  {
    Mparticles mprts(grid_);
    mprts.define_species("test_species", 1., 1., 100, 10, 10, 0);
    return mprts;
  }

  template <typename _Mparticles>
  void inject_test_particles(_Mparticles& mprts, int n_prts)
  {
    auto inj = mprts.injector();
    for (int p = 0; p < mprts.n_patches(); ++p) {
      auto injector = inj[p];
      auto& patch = mprts.grid().patches[p];
      for (int n = 0; n < n_prts; n++) {
        double nn = double(n) / n_prts;
        auto L = patch.xe - patch.xb;
        psc::particle::Inject prt = {
	  {patch.xb[0] + nn * L[0],
	   patch.xb[1] + nn * L[1],
	   patch.xb[2] + nn * L[2]}, {}, 1., 0};
        injector(prt);
      }
    }
  }

  const Grid_t& grid() { return grid_; }

private:
  Grid_t grid_;
};

#ifdef PSC_HAVE_ADIOS2

// ======================================================================
// MparticlesIOTest

template <typename T>
struct MparticlesIOTest : MparticlesTest<T>
{};

using MparticlesIOTestTypes =
  ::testing::Types<Config<MparticlesCuda<BS144>, MakeTestGridYZ>>;

TYPED_TEST_SUITE(MparticlesIOTest, MparticlesIOTestTypes);

TYPED_TEST(MparticlesIOTest, WriteRead)
{
  int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  auto mprts = this->mk_mprts();
  this->inject_test_particles(mprts, 4 + rank);

  auto io = kg::io::IOAdios2{};

  cuda_mparticles<BS144>& cmprts = *mprts.cmprts();
  DMparticlesCuda<BS144> d_mprts = cmprts;
  
  {
    auto writer = io.open("test.bp", kg::io::Mode::Write);
    writer.put("d_mprts", d_mprts);
    writer.close();
  }

  DMparticlesCuda<BS144> d_mprts2;
  {
    auto reader = io.open("test.bp", kg::io::Mode::Read);
    reader.get("d_mprts", d_mprts2);
    reader.close();
  }

#if 0
  auto accessor = mprts.accessor();
  auto accessor2 = mprts2.accessor();
  for (int p = 0; p < mprts.n_patches(); ++p) {
    auto prts = accessor[p];
    auto prts2 = accessor2[p];
    ASSERT_EQ(prts.size(), prts2.size());
    for (int n = 0; n < prts.size(); n++) {
      EXPECT_EQ(prts[n].x(), prts2[n].x());
    }
  }
#endif
}

#endif

int main(int argc, char** argv)
{
  MPI_Init(&argc, &argv);

  ::testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();

  MPI_Finalize();
  return rc;
}
