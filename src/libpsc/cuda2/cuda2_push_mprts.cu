#include "hip/hip_runtime.h"

#include "../psc_push_particles/inc_params.c"
#include "../psc_push_particles/inc_cache.c"
#include "../psc_push_particles/inc_interpolate.c"
#include "../psc_push_particles/inc_push.c"
#include "../psc_push_particles/inc_curr.c"
#include "../psc_push_particles/inc_step.c"

#define THREADS_PER_BLOCK (512)

// ----------------------------------------------------------------------
// find_block_pos_patch

__device__ static int
find_block_pos_patch(int *block_pos, int *ci0)
{
  block_pos[0] = blockIdx.x;
  block_pos[1] = blockIdx.y;
  block_pos[2] = blockIdx.z % prm.b_mx[2];

#if EM_CACHE == EM_CACHE_CUDA
  ci0[0] = block_pos[0] * BLOCKSIZE_X;
  ci0[1] = block_pos[1] * BLOCKSIZE_Y;
  ci0[2] = block_pos[2] * BLOCKSIZE_Z;
#endif

  return blockIdx.z / prm.b_mx[2];
}

// ----------------------------------------------------------------------
// find_bid

__device__ static int
find_bid()
{
  return (blockIdx.z * prm.b_mx[1] + blockIdx.y) * prm.b_mx[0] + blockIdx.x;
}

// ----------------------------------------------------------------------
// push_mprts_ab

__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 3)
push_mprts_ab(float4 *d_xi4, float4 *d_pxi4,
	      unsigned int *d_off,
	      float *d_flds0, unsigned int size)
{
  int block_pos[3], ci0[3];
  int p, bid;
  p = find_block_pos_patch(block_pos, ci0);
  real *d_flds = d_flds0 + p * size;

  DECLARE_EM_CACHE(flds_em, d_flds, size, ci0);
  real *flds_curr = d_flds;

  bid = find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    push_one_mprts(d_xi4, d_pxi4, n, flds_em, flds_curr, ci0);
  }
}

// ----------------------------------------------------------------------
// zero_currents

static void
zero_currents(struct psc_mfields *mflds)
{
  struct psc_mfields_cuda2 *mflds_sub = psc_mfields_cuda2(mflds);

  unsigned int size = mflds_sub->im[0] * mflds_sub->im[1] * mflds_sub->im[2];

  for (int p = 0; p < mflds->nr_patches; p++) {
    fields_cuda2_real_t *d_flds = mflds_sub->d_flds + p * size * mflds->nr_fields;
    check(hipMemset(d_flds + JXI * size, 0, 3 * size * sizeof(*d_flds)));
  }
}

// ----------------------------------------------------------------------
// cuda_push_mprts_ab

static void
cuda_push_mprts_ab(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda2 *mprts_sub = psc_mparticles_cuda2(mprts);
  struct psc_mfields_cuda2 *mflds_sub = psc_mfields_cuda2(mflds);

  params_1vb_set(ppsc, mprts, mflds);

  unsigned int fld_size = mflds->nr_fields *
    mflds_sub->im[0] * mflds_sub->im[1] * mflds_sub->im[2];

  zero_currents(mflds);

  int gx, gy, gz;
  gx = mprts_sub->b_mx[0];
  gy = mprts_sub->b_mx[1];
  gz = mprts_sub->b_mx[2] * mprts->nr_patches;
  dim3 dimGrid(gx, gy, gz);

  push_mprts_ab<<<dimGrid, THREADS_PER_BLOCK>>>
    (mprts_sub->d_xi4, mprts_sub->d_pxi4, mprts_sub->d_b_off,
     mflds_sub->d_flds, fld_size);

  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// cuda2_1vbec_push_mprts

void
SFX(cuda2_1vbec_push_mprts)(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda2 *mprts_sub = psc_mparticles_cuda2(mprts);

  int *bs = mprts_sub->bs;
  assert(bs[0] == BLOCKSIZE_X && bs[1] == BLOCKSIZE_Y && bs[2] == BLOCKSIZE_Z);
  cuda_push_mprts_ab(mprts, mflds);
}

