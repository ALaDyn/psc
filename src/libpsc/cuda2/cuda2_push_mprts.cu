#include "hip/hip_runtime.h"

#include "psc_cuda2.h"

#include "psc_fields_cuda2.h"
#include "psc_particles_as_cuda2.h"

// OPT: precalc offsets into fld_cache (including ci[])
// OPT: use more shmem?

#define EM_CACHE EM_CACHE_NONE
#define CALC_J CALC_J_1VB_VAR1
#define F3_CURR(flds, m, ix,iy,iz) ((float *) flds->data)[0]

#if DIM == DIM_YZ
#define BLOCKBND_X 0
#define BLOCKBND_Y 2
#define BLOCKBND_Z 2
#elif DIM == DIM_XYZ
#define BLOCKBND_X 2
#define BLOCKBND_Y 2
#define BLOCKBND_Z 2
#endif

#define BLOCKGSIZE_X (BLOCKSIZE_X + 2 * BLOCKBND_X)
#define BLOCKGSIZE_Y (BLOCKSIZE_Y + 2 * BLOCKBND_Y)
#define BLOCKGSIZE_Z (BLOCKSIZE_Z + 2 * BLOCKBND_Z)

#include "../psc_push_particles/inc_params.c"
#include "../psc_push_particles/inc_push.c"
#include "../psc_push_particles/inc_interpolate.c"

// ======================================================================
// field access

#if DIM == DIM_YZ

#define F3_DEV_OFF(fldnr, jx,jy,jz)					\
  ((((fldnr)								\
     *prm.mx[2] + ((jz)-prm.ilg[2]))					\
    *prm.mx[1] + ((jy)-prm.ilg[1])))

#else

#define F3_DEV_OFF(fldnr, jx,jy,jz)					\
  ((((fldnr)								\
     *prm.mx[2] + ((jz)-prm.ilg[2]))					\
    *prm.mx[1] + ((jy)-prm.ilg[1]))					\
   *prm.mx[0] + ((jx)-prm.ilg[0]))

#endif

#define F3_DEV(d_flds, fldnr, jx,jy,jz)		\
  ((d_flds)[F3_DEV_OFF(fldnr, jx,jy,jz)])

// ======================================================================
// particle access

#define LOAD_PARTICLE_POS_(pp, d_xi4, n) do {				\
    float4 _xi4 = d_xi4[n];						\
    (pp).xi[0]         = _xi4.x;					\
    (pp).xi[1]         = _xi4.y;					\
    (pp).xi[2]         = _xi4.z;					\
    (pp).kind_as_float = _xi4.w;					\
} while (0)

#define LOAD_PARTICLE_MOM_(pp, d_pxi4, n) do {				\
    float4 _pxi4 = d_pxi4[n];						\
    (pp).pxi[0]        = _pxi4.x;					\
    (pp).pxi[1]        = _pxi4.y;					\
    (pp).pxi[2]        = _pxi4.z;					\
    (pp).qni_wni       = _pxi4.w;					\
} while (0)

#define STORE_PARTICLE_POS_(pp, d_xi4, n) do {				\
    float4 xi4 = { (pp).xi[0], (pp).xi[1], (pp).xi[2], (pp).kind_as_float }; \
    d_xi4[n] = xi4;							\
} while (0)

#define STORE_PARTICLE_MOM_(pp, d_pxi4, n) do {				\
    float4 pxi4 = { (pp).pxi[0], (pp).pxi[1], (pp).pxi[2], (pp).qni_wni }; \
    d_pxi4[n] = pxi4;							\
} while (0)


// ----------------------------------------------------------------------

#include "../psc_push_particles/inc_curr.c"

// ----------------------------------------------------------------------

#undef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK (512)

// ======================================================================
// field caching

#if EM_CACHE == EM_CACHE_NONE

#define F3_CACHE(flds_em, m, jx, jy, jz)	\
  (F3_DEV(flds_em, m, jx,jy,jz))

#define DECLARE_EM_CACHE(flds_em, d_flds, size, ci0)	\
  real *flds_em = d_flds

#elif EM_CACHE == EM_CACHE_CUDA

#if DIM == DIM_YZ
#define F3_CACHE(flds_em, m, jx, jy, jz)				\
  ((flds_em)[(((m-EX)							\
	       *BLOCKGSIZE_Z + ((jz-ci0[2])-(-BLOCKBND_Z)))		\
	      *BLOCKGSIZE_Y + ((jy-ci0[1])-(-BLOCKBND_Y)))])
#elif DIM == DIM_XYZ
#define F3_CACHE(flds_em, m, jx, jy, jz)				\
  ((flds_em)[((((m-EX)							\
		*BLOCKGSIZE_Z + ((jz-ci0[2])-(-BLOCKBND_Z)))		\
	       *BLOCKGSIZE_Y + ((jy-ci0[1])-(-BLOCKBND_Y)))		\
	      *BLOCKGSIZE_X + ((jx-ci0[0])-(-BLOCKBND_X)))])
#endif

__device__ static void
cache_fields(float *flds_em, float *d_flds, int size, int *ci0)
{
  int ti = threadIdx.x;
  while (ti < n) {
    int tmp = ti;
    int jx = tmp % BLOCKGSIZE_X - BLOCKBND_X;
    tmp /= dims[0];
    int jy = tmp % BLOCKGSIZE_Y - BLOCKBND_Y;
    tmp /= dims[1];
    int jz = tmp % BLOCKGSIZE_Z - BLOCKBND_Z;
    // OPT? currently it seems faster to do the loop rather than do m by threadidx
    for (int m = EX; m <= HZ; m++) {
      F3_CACHE(flds_em, m, jx+ci0[0],jy+ci0[1] jz+ci0[2]) = 
	F3_DEV(d_flds, m, jx+ci0[0],jy+ci0[1],jz+ci0[2]);
    }
    ti += THREADS_PER_BLOCK;
  }
}

#define DECLARE_EM_CACHE(flds_em, d_flds, size, ci0)	\
  __shared__ real flds_em[6 * BLOCKGSIZE_X * BLOCKGSIZE_Y * BLOCKGSIZE_Z];\
  cache_fields(flds_em, d_flds, size, ci0)

#endif

// ----------------------------------------------------------------------
// push_one

__device__ static void
push_one(particle_t *prt, int n, float4 *d_xi4, float4 *d_pxi4,
	 real *flds_em, flds_curr_t flds_curr, int ci0[3])
{
  LOAD_PARTICLE_POS_(*prt, d_xi4, n);

  // here we have x^{n+.5}, p^n

  // field interpolation
  real exq, eyq, ezq, hxq, hyq, hzq;
  int lg[3];
  real og[3];
  find_idx_off_1st_rel(prt->xi, lg, og, real(0.));
  INTERPOLATE_1ST_EC(flds_em, exq, eyq, ezq, hxq, hyq, hzq);

  // x^(n+0.5), p^n -> x^(n+0.5), p^(n+1.0) 
  LOAD_PARTICLE_MOM_(*prt, d_pxi4, n);
  int kind = particle_kind(prt);
  real dq = prm.dq_kind[kind];
  push_pxi(prt, exq, eyq, ezq, hxq, hyq, hzq, dq);
  STORE_PARTICLE_MOM_(*prt, d_pxi4, n);

  real vxi[3];
  calc_vxi(vxi, prt);

  particle_real_t xm[3], xp[3];
  int lf[3];

  // position xm at x^(n+.5)
  real h0[3];
  find_idx_off_pos_1st_rel(prt->xi, lg, h0, xm, real(0.));

  // x^(n+0.5), p^(n+1.0) -> x^(n+1.5), p^(n+1.0) 
  push_xi(prt, vxi, prm.dt);
  STORE_PARTICLE_POS_(*prt, d_xi4, n);

  // position xp at x^(n+.5)
  real h1[3];
  find_idx_off_pos_1st_rel(prt->xi, lf, h1, xp, real(0.));

  calc_j(flds_curr, xm, xp, lf, lg, prt, vxi);
}

#include "../psc_push_particles/inc_step.c"

// ----------------------------------------------------------------------
// find_block_pos_patch

__device__ static int
find_block_pos_patch(int *block_pos, int *ci0)
{
  block_pos[0] = blockIdx.x;
  block_pos[1] = blockIdx.y;
  block_pos[2] = blockIdx.z % prm.b_mx[2];

#if EM_CACHE == EM_CACHE_CUDA
  ci0[0] = block_pos[0] * BLOCKSIZE_X;
  ci0[1] = block_pos[1] * BLOCKSIZE_Y;
  ci0[2] = block_pos[2] * BLOCKSIZE_Z;
#endif

  return blockIdx.z / prm.b_mx[2];
}

// ----------------------------------------------------------------------
// find_bid

__device__ static int
find_bid()
{
  return (blockIdx.z * prm.b_mx[1] + blockIdx.y) * prm.b_mx[0] + blockIdx.x;
}

// ======================================================================

__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 3)
push_mprts_ab(float4 *d_xi4, float4 *d_pxi4,
	      unsigned int *d_off,
	      float *d_flds0, unsigned int size)
{
  int block_pos[3], ci0[3];
  int p, bid;
  p = find_block_pos_patch(block_pos, ci0);
  real *d_flds = d_flds0 + p * size;

  DECLARE_EM_CACHE(flds_em, d_flds, size, ci0);
  real *flds_curr = d_flds;

  bid = find_bid();
  int block_begin = d_off[bid];
  int block_end = d_off[bid + 1];

  __syncthreads();
  for (int n = (block_begin & ~31) + threadIdx.x; n < block_end; n += THREADS_PER_BLOCK) {
    if (n < block_begin) {
      continue;
    }
    push_one_mprts(d_xi4, d_pxi4, n, flds_em, flds_curr, ci0);
  }

}

// ----------------------------------------------------------------------
// zero_currents

static void
zero_currents(struct psc_mfields *mflds)
{
  struct psc_mfields_cuda2 *mflds_sub = psc_mfields_cuda2(mflds);

  unsigned int size = mflds_sub->im[0] * mflds_sub->im[1] * mflds_sub->im[2];

  for (int p = 0; p < mflds->nr_patches; p++) {
    fields_cuda2_real_t *d_flds = mflds_sub->d_flds + p * size * mflds->nr_fields;
    check(hipMemset(d_flds + JXI * size, 0, 3 * size * sizeof(*d_flds)));
  }
}

// ----------------------------------------------------------------------
// cuda_push_mprts_ab

static void
cuda_push_mprts_ab(struct psc_mparticles *mprts, struct psc_mfields *mflds)
{
  struct psc_mparticles_cuda2 *mprts_sub = psc_mparticles_cuda2(mprts);
  struct psc_mfields_cuda2 *mflds_sub = psc_mfields_cuda2(mflds);

  params_1vb_set(ppsc, mprts, mflds);

  unsigned int fld_size = mflds->nr_fields *
    mflds_sub->im[0] * mflds_sub->im[1] * mflds_sub->im[2];

  zero_currents(mflds);

  int gx, gy, gz;
#if DIM == DIM_YZ
  assert(mprts_sub->b_mx[0] == 1);
#endif
  gx = mprts_sub->b_mx[0];
  gy = mprts_sub->b_mx[1];
  gz = mprts_sub->b_mx[2] * mprts->nr_patches;
  dim3 dimGrid(gx, gy, gz);

  push_mprts_ab<<<dimGrid, THREADS_PER_BLOCK>>>
    (mprts_sub->d_xi4, mprts_sub->d_pxi4, mprts_sub->d_b_off,
     mflds_sub->d_flds, fld_size);

  cuda_sync_if_enabled();
}

